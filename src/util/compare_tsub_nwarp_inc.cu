/**
 * @file compare_tsub_nwarp_inc.cu
 *
 * @brief Source code for comparing values on GPU
 *
 * @author Yohei Miki (University of Tokyo)
 * @author Masayuki Umemura (University of Tsukuba)
 *
 * @date 2018/06/01 (Fri)
 *
 * Copyright (C) 2017 Yohei Miki and Masayuki Umemura
 * All rights reserved.
 *
 * The MIT License is applied to this software, see LICENSE.txt
 *
 */

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>

#include "macro.h"
#include "cudalib.h"

#include "../util/compare_tsub_nwarp_inc.cuh"
#include "../util/comparison_inc.cu"

#   if  (GPUGEN >= 70) && !defined(_COOPERATIVE_GROUPS_H_)
#include <hip/hip_cooperative_groups.h>
using namespace cooperative_groups;
#endif//(GPUGEN >= 70) && !defined(_COOPERATIVE_GROUPS_H_)


/**
 * @fn GET_MIN_TSUB_NWARP
 *
 * @brief Get minimum value within a group of TSUB_TN_COMPARE_INC threads (NWARP_TN_COMPARE_INC continuous threads have the identical value).
 * @detail implicit synchronization within TSUB_TN_COMPARE_INC (<= 32) threads (a warp) is assumed.
 */
template <typename Type>
__device__ __forceinline__ Type GET_MIN_TSUB_NWARP
(Type val
#ifdef  USE_WARP_SHUFFLE_FUNC_COMPARE_TSUB_NWARP_INC
 , const uint mask
#else///USE_WARP_SHUFFLE_FUNC_COMPARE_TSUB_NWARP_INC
 , volatile Type * smem, const int tidx, const int head
#endif//USE_WARP_SHUFFLE_FUNC_COMPARE_TSUB_NWARP_INC
 )
{
#ifdef  USE_WARP_SHUFFLE_FUNC_COMPARE_TSUB_NWARP_INC

  Type tmp;
#   if  TSUB_TN_COMPARE_INC >= ( 2 * NWARP_TN_COMPARE_INC)
  tmp = __SHFL_XOR(mask, val,      NWARP_TN_COMPARE_INC, TSUB_TN_COMPARE_INC);  val = getMinVal(val, tmp);
#   if  TSUB_TN_COMPARE_INC >= ( 4 * NWARP_TN_COMPARE_INC)
  tmp = __SHFL_XOR(mask, val,  2 * NWARP_TN_COMPARE_INC, TSUB_TN_COMPARE_INC);  val = getMinVal(val, tmp);
#   if  TSUB_TN_COMPARE_INC >= ( 8 * NWARP_TN_COMPARE_INC)
  tmp = __SHFL_XOR(mask, val,  4 * NWARP_TN_COMPARE_INC, TSUB_TN_COMPARE_INC);  val = getMinVal(val, tmp);
#   if  TSUB_TN_COMPARE_INC >= (16 * NWARP_TN_COMPARE_INC)
  tmp = __SHFL_XOR(mask, val,  8 * NWARP_TN_COMPARE_INC, TSUB_TN_COMPARE_INC);  val = getMinVal(val, tmp);
#   if  TSUB_TN_COMPARE_INC == (32 * NWARP_TN_COMPARE_INC)
  tmp = __SHFL_XOR(mask, val, 16 * NWARP_TN_COMPARE_INC, TSUB_TN_COMPARE_INC);  val = getMinVal(val, tmp);
#endif//TSUB_TN_COMPARE_INC == (32 * NWARP_TN_COMPARE_INC)
#endif//TSUB_TN_COMPARE_INC >= (16 * NWARP_TN_COMPARE_INC)
#endif//TSUB_TN_COMPARE_INC >= ( 8 * NWARP_TN_COMPARE_INC)
#endif//TSUB_TN_COMPARE_INC >= ( 4 * NWARP_TN_COMPARE_INC)
#endif//TSUB_TN_COMPARE_INC >= ( 2 * NWARP_TN_COMPARE_INC)
  val = __SHFL(mask, val, 0, TSUB_TN_COMPARE_INC);

#else///USE_WARP_SHUFFLE_FUNC_COMPARE_TSUB_NWARP_INC

  smem[tidx] = val;
#   if  TSUB_TN_COMPARE_INC >= ( 2 * NWARP_TN_COMPARE_INC)
  val = getMinVal(val, smem[tidx ^ (     NWARP_TN_COMPARE_INC)]);  smem[tidx] = val;
#   if  TSUB_TN_COMPARE_INC >= ( 4 * NWARP_TN_COMPARE_INC)
  val = getMinVal(val, smem[tidx ^ ( 2 * NWARP_TN_COMPARE_INC)]);  smem[tidx] = val;
#   if  TSUB_TN_COMPARE_INC >= ( 8 * NWARP_TN_COMPARE_INC)
  val = getMinVal(val, smem[tidx ^ ( 4 * NWARP_TN_COMPARE_INC)]);  smem[tidx] = val;
#   if  TSUB_TN_COMPARE_INC >= (16 * NWARP_TN_COMPARE_INC)
  val = getMinVal(val, smem[tidx ^ ( 8 * NWARP_TN_COMPARE_INC)]);  smem[tidx] = val;
#   if  TSUB_TN_COMPARE_INC == (32 * NWARP_TN_COMPARE_INC)
  val = getMinVal(val, smem[tidx ^ (16 * NWARP_TN_COMPARE_INC)]);  smem[tidx] = val;
#endif//TSUB_TN_COMPARE_INC == (32 * NWARP_TN_COMPARE_INC)
#endif//TSUB_TN_COMPARE_INC >= (16 * NWARP_TN_COMPARE_INC)
#endif//TSUB_TN_COMPARE_INC >= ( 8 * NWARP_TN_COMPARE_INC)
#endif//TSUB_TN_COMPARE_INC >= ( 4 * NWARP_TN_COMPARE_INC)
#endif//TSUB_TN_COMPARE_INC >= ( 2 * NWARP_TN_COMPARE_INC)
  val = smem[head];

#endif//USE_WARP_SHUFFLE_FUNC_COMPARE_TSUB_NWARP_INC

  return (val);
}


/**
 * @fn GET_MAX_TSUB_NWARP
 *
 * @brief Get maximum value within a group of TSUB_TN_COMPARE_INC threads (NWARP_TN_COMPARE_INC continuous threads have the identical value).
 * @detail implicit synchronization within TSUB_TN_COMPARE_INC (<= 32) threads (a warp) is assumed.
 */
template <typename Type>
__device__ __forceinline__ Type GET_MAX_TSUB_NWARP
(Type val
#ifdef  USE_WARP_SHUFFLE_FUNC_COMPARE_TSUB_NWARP_INC
 , const uint mask
#else///USE_WARP_SHUFFLE_FUNC_COMPARE_TSUB_NWARP_INC
 , volatile Type * smem, const int tidx, const int head
#endif//USE_WARP_SHUFFLE_FUNC_COMPARE_TSUB_NWARP_INC
 )
{
#ifdef  USE_WARP_SHUFFLE_FUNC_COMPARE_TSUB_NWARP_INC

  Type tmp;
#   if  TSUB_TN_COMPARE_INC >= ( 2 * NWARP_TN_COMPARE_INC)
  tmp = __SHFL_XOR(mask, val,      NWARP_TN_COMPARE_INC, TSUB_TN_COMPARE_INC);  val = getMaxVal(val, tmp);
#   if  TSUB_TN_COMPARE_INC >= ( 4 * NWARP_TN_COMPARE_INC)
  tmp = __SHFL_XOR(mask, val,  2 * NWARP_TN_COMPARE_INC, TSUB_TN_COMPARE_INC);  val = getMaxVal(val, tmp);
#   if  TSUB_TN_COMPARE_INC >= ( 8 * NWARP_TN_COMPARE_INC)
  tmp = __SHFL_XOR(mask, val,  4 * NWARP_TN_COMPARE_INC, TSUB_TN_COMPARE_INC);  val = getMaxVal(val, tmp);
#   if  TSUB_TN_COMPARE_INC >= (16 * NWARP_TN_COMPARE_INC)
  tmp = __SHFL_XOR(mask, val,  8 * NWARP_TN_COMPARE_INC, TSUB_TN_COMPARE_INC);  val = getMaxVal(val, tmp);
#   if  TSUB_TN_COMPARE_INC == (32 * NWARP_TN_COMPARE_INC)
  tmp = __SHFL_XOR(mask, val, 16 * NWARP_TN_COMPARE_INC, TSUB_TN_COMPARE_INC);  val = getMaxVal(val, tmp);
#endif//TSUB_TN_COMPARE_INC == (32 * NWARP_TN_COMPARE_INC)
#endif//TSUB_TN_COMPARE_INC >= (16 * NWARP_TN_COMPARE_INC)
#endif//TSUB_TN_COMPARE_INC >= ( 8 * NWARP_TN_COMPARE_INC)
#endif//TSUB_TN_COMPARE_INC >= ( 4 * NWARP_TN_COMPARE_INC)
#endif//TSUB_TN_COMPARE_INC >= ( 2 * NWARP_TN_COMPARE_INC)
  val = __SHFL(mask, val, 0, TSUB_TN_COMPARE_INC);

#else///USE_WARP_SHUFFLE_FUNC_COMPARE_TSUB_NWARP_INC

  smem[tidx] = val;
#   if  TSUB_TN_COMPARE_INC >= ( 2 * NWARP_TN_COMPARE_INC)
  val = getMaxVal(val, smem[tidx ^ (     NWARP_TN_COMPARE_INC)]);  smem[tidx] = val;
#   if  TSUB_TN_COMPARE_INC >= ( 4 * NWARP_TN_COMPARE_INC)
  val = getMaxVal(val, smem[tidx ^ ( 2 * NWARP_TN_COMPARE_INC)]);  smem[tidx] = val;
#   if  TSUB_TN_COMPARE_INC >= ( 8 * NWARP_TN_COMPARE_INC)
  val = getMaxVal(val, smem[tidx ^ ( 4 * NWARP_TN_COMPARE_INC)]);  smem[tidx] = val;
#   if  TSUB_TN_COMPARE_INC >= (16 * NWARP_TN_COMPARE_INC)
  val = getMaxVal(val, smem[tidx ^ ( 8 * NWARP_TN_COMPARE_INC)]);  smem[tidx] = val;
#   if  TSUB_TN_COMPARE_INC == (32 * NWARP_TN_COMPARE_INC)
  val = getMaxVal(val, smem[tidx ^ (16 * NWARP_TN_COMPARE_INC)]);  smem[tidx] = val;
#endif//TSUB_TN_COMPARE_INC == (32 * NWARP_TN_COMPARE_INC)
#endif//TSUB_TN_COMPARE_INC >= (16 * NWARP_TN_COMPARE_INC)
#endif//TSUB_TN_COMPARE_INC >= ( 8 * NWARP_TN_COMPARE_INC)
#endif//TSUB_TN_COMPARE_INC >= ( 4 * NWARP_TN_COMPARE_INC)
#endif//TSUB_TN_COMPARE_INC >= ( 2 * NWARP_TN_COMPARE_INC)
  val = smem[head];

#endif//USE_WARP_SHUFFLE_FUNC_COMPARE_TSUB_NWARP_INC

  return (val);
}


/**
 * @fn GET_MINLOC_TSUB_NWARP
 *
 * @brief Get minimum value with location within a group of TSUB_TN_COMPARE_INC threads (NWARP_TN_COMPARE_INC continuous threads have the identical value).
 * @detail implicit synchronization within TSUB_TN_COMPARE_INC (<= 32) threads (a warp) is assumed.
 */
template <typename Type>
__device__ __forceinline__ Type GET_MINLOC_TSUB_NWARP
(Type val, volatile Type * smem, const int tidx, const int head)
{
  Type tmp;
  smem[tidx] = val;

#   if  (__CUDA_ARCH__ >= 700) && (TSUB_TN_COMPARE_INC < 32)
  thread_block_tile<TSUB_TN_COMPARE_INC> tile = tiled_partition<TSUB_TN_COMPARE_INC>(this_thread_block());
#endif//(__CUDA_ARCH__ >= 700) && (TSUB_TN_COMPARE_INC < 32)

#   if  TSUB_TN_COMPARE_INC >= ( 2 * NWARP_TN_COMPARE_INC)
  tmp = smem[tidx ^ (     NWARP_TN_COMPARE_INC)];  if( tmp.val < val.val )    val = tmp;
#   if  __CUDA_ARCH__ >= 700
#   if  TSUB_TN_COMPARE_INC == 32
  __syncwarp();
#else///TSUB_TN_COMPARE_INC == 32
  tile.sync();
#endif//TSUB_TN_COMPARE_INC == 32
#endif//__CUDA_ARCH__ >= 700
  smem[tidx] = val;
#   if  TSUB_TN_COMPARE_INC >= ( 4 * NWARP_TN_COMPARE_INC)
  tmp = smem[tidx ^ ( 2 * NWARP_TN_COMPARE_INC)];  if( tmp.val < val.val )    val = tmp;
#   if  __CUDA_ARCH__ >= 700
#   if  TSUB_TN_COMPARE_INC == 32
  __syncwarp();
#else///TSUB_TN_COMPARE_INC == 32
  tile.sync();
#endif//TSUB_TN_COMPARE_INC == 32
#endif//__CUDA_ARCH__ >= 700
  smem[tidx] = val;
#   if  TSUB_TN_COMPARE_INC >= ( 8 * NWARP_TN_COMPARE_INC)
  tmp = smem[tidx ^ ( 4 * NWARP_TN_COMPARE_INC)];  if( tmp.val < val.val )    val = tmp;
#   if  __CUDA_ARCH__ >= 700
#   if  TSUB_TN_COMPARE_INC == 32
  __syncwarp();
#else///TSUB_TN_COMPARE_INC == 32
  tile.sync();
#endif//TSUB_TN_COMPARE_INC == 32
#endif//__CUDA_ARCH__ >= 700
  smem[tidx] = val;
#   if  TSUB_TN_COMPARE_INC >= (16 * NWARP_TN_COMPARE_INC)
  tmp = smem[tidx ^ ( 8 * NWARP_TN_COMPARE_INC)];  if( tmp.val < val.val )    val = tmp;
#   if  __CUDA_ARCH__ >= 700
#   if  TSUB_TN_COMPARE_INC == 32
  __syncwarp();
#else///TSUB_TN_COMPARE_INC == 32
  tile.sync();
#endif//TSUB_TN_COMPARE_INC == 32
#endif//__CUDA_ARCH__ >= 700
  smem[tidx] = val;
#   if  TSUB_TN_COMPARE_INC == (32 * NWARP_TN_COMPARE_INC)
  tmp = smem[tidx ^ (16 * NWARP_TN_COMPARE_INC)];  if( tmp.val < val.val )    val = tmp;
#   if  __CUDA_ARCH__ >= 700
  __syncwarp();
#endif//__CUDA_ARCH__ >= 700
  smem[tidx] = val;
#endif//TSUB_TN_COMPARE_INC == (32 * NWARP_TN_COMPARE_INC)
#endif//TSUB_TN_COMPARE_INC >= (16 * NWARP_TN_COMPARE_INC)
#endif//TSUB_TN_COMPARE_INC >= ( 8 * NWARP_TN_COMPARE_INC)
#endif//TSUB_TN_COMPARE_INC >= ( 4 * NWARP_TN_COMPARE_INC)
#endif//TSUB_TN_COMPARE_INC >= ( 2 * NWARP_TN_COMPARE_INC)

  val = smem[head];
  return (val);
}


/**
 * @fn GET_MAXLOC_TSUB_NWARP
 *
 * @brief Get maximum value with location within a group of TSUB_TN_COMPARE_INC threads (NWARP_TN_COMPARE_INC continuous threads have the identical value).
 * @detail implicit synchronization within TSUB_TN_COMPARE_INC (<= 32) threads (a warp) is assumed.
 */
template <typename Type>
__device__ __forceinline__ Type GET_MAXLOC_TSUB_NWARP
(Type val, volatile Type * smem, const int tidx, const int head)
{
  Type tmp;
  smem[tidx] = val;

#   if  (__CUDA_ARCH__ >= 700) && (TSUB_TN_COMPARE_INC < 32)
  thread_block_tile<TSUB_TN_COMPARE_INC> tile = tiled_partition<TSUB_TN_COMPARE_INC>(this_thread_block());
#endif//(__CUDA_ARCH__ >= 700) && (TSUB_TN_COMPARE_INC < 32)

#   if  TSUB_TN_COMPARE_INC >= ( 2 * NWARP_TN_COMPARE_INC)
  tmp = smem[tidx ^ (     NWARP_TN_COMPARE_INC)];  if( tmp.val > val.val )    val = tmp;
#   if  __CUDA_ARCH__ >= 700
#   if  TSUB_TN_COMPARE_INC == 32
  __syncwarp();
#else///TSUB_TN_COMPARE_INC == 32
  tile.sync();
#endif//TSUB_TN_COMPARE_INC == 32
#endif//__CUDA_ARCH__ >= 700
  smem[tidx] = val;
#   if  TSUB_TN_COMPARE_INC >= ( 4 * NWARP_TN_COMPARE_INC)
  tmp = smem[tidx ^ ( 2 * NWARP_TN_COMPARE_INC)];  if( tmp.val > val.val )    val = tmp;
#   if  __CUDA_ARCH__ >= 700
#   if  TSUB_TN_COMPARE_INC == 32
  __syncwarp();
#else///TSUB_TN_COMPARE_INC == 32
  tile.sync();
#endif//TSUB_TN_COMPARE_INC == 32
#endif//__CUDA_ARCH__ >= 700
  smem[tidx] = val;
#   if  TSUB_TN_COMPARE_INC >= ( 8 * NWARP_TN_COMPARE_INC)
  tmp = smem[tidx ^ ( 4 * NWARP_TN_COMPARE_INC)];  if( tmp.val > val.val )    val = tmp;
#   if  __CUDA_ARCH__ >= 700
#   if  TSUB_TN_COMPARE_INC == 32
  __syncwarp();
#else///TSUB_TN_COMPARE_INC == 32
  tile.sync();
#endif//TSUB_TN_COMPARE_INC == 32
#endif//__CUDA_ARCH__ >= 700
  smem[tidx] = val;
#   if  TSUB_TN_COMPARE_INC >= (16 * NWARP_TN_COMPARE_INC)
  tmp = smem[tidx ^ ( 8 * NWARP_TN_COMPARE_INC)];  if( tmp.val > val.val )    val = tmp;
#   if  __CUDA_ARCH__ >= 700
#   if  TSUB_TN_COMPARE_INC == 32
  __syncwarp();
#else///TSUB_TN_COMPARE_INC == 32
  tile.sync();
#endif//TSUB_TN_COMPARE_INC == 32
#endif//__CUDA_ARCH__ >= 700
  smem[tidx] = val;
#   if  TSUB_TN_COMPARE_INC == (32 * NWARP_TN_COMPARE_INC)
  tmp = smem[tidx ^ (16 * NWARP_TN_COMPARE_INC)];  if( tmp.val > val.val )    val = tmp;
#   if  __CUDA_ARCH__ >= 700
  __syncwarp();
#endif//__CUDA_ARCH__ >= 700
  smem[tidx] = val;
#endif//TSUB_TN_COMPARE_INC == (32 * NWARP_TN_COMPARE_INC)
#endif//TSUB_TN_COMPARE_INC >= (16 * NWARP_TN_COMPARE_INC)
#endif//TSUB_TN_COMPARE_INC >= ( 8 * NWARP_TN_COMPARE_INC)
#endif//TSUB_TN_COMPARE_INC >= ( 4 * NWARP_TN_COMPARE_INC)
#endif//TSUB_TN_COMPARE_INC >= ( 2 * NWARP_TN_COMPARE_INC)

  val = smem[head];
  return (val);
}
