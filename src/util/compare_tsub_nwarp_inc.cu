#include "hip/hip_runtime.h"
/**
 * @file compare_tsub_nwarp_inc.cu
 *
 * @brief Source code for comparing values on GPU
 *
 * @author Yohei Miki (University of Tokyo)
 * @author Masayuki Umemura (University of Tsukuba)
 *
 * @date 2020/11/04 (Wed)
 *
 * Copyright (C) 2017 Yohei Miki and Masayuki Umemura
 * All rights reserved.
 *
 * The MIT License is applied to this software, see LICENSE.txt
 *
 */

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>

#include "macro.h"
#include "cudalib.h"

#include "../util/compare_tsub_nwarp_inc.cuh"
#include "../util/comparison_inc.cu"

#   if  (GPUGEN >= 70) && !defined(_COOPERATIVE_GROUPS_H_)
#include <hip/hip_cooperative_groups.h>
using namespace cooperative_groups;
#endif//(GPUGEN >= 70) && !defined(_COOPERATIVE_GROUPS_H_)


#ifdef  USE_WARP_REDUCE_FUNCTIONS_COMPARE_TSUB_NWARP_INC
__device__ __forceinline__ uint flipFP32(const uint src){  uint mask = -int(src >> 31)   | 0x80000000;  return (src ^ mask);}
__device__ __forceinline__ uint undoFP32(const uint src){  uint mask = ((src >> 31) - 1) | 0x80000000;  return (src ^ mask);}
#endif//USE_WARP_REDUCE_FUNCTIONS_COMPARE_TSUB_NWARP_INC


/**
 * @fn GET_MIN_TSUB_NWARP
 *
 * @brief Get minimum value within a group of TSUB_TN_COMPARE_INC threads (NWARP_TN_COMPARE_INC continuous threads have the identical value).
 * @detail implicit synchronization within TSUB_TN_COMPARE_INC (<= 32) threads (a warp) is assumed.
 */
#ifdef  USE_WARP_REDUCE_FUNCTIONS_COMPARE_TSUB_NWARP_INC
__device__ __forceinline__      int GET_MIN_TSUB_NWARP(     int val, const uint mask){  return (__reduce_min_sync(mask, val));}
__device__ __forceinline__ unsigned GET_MIN_TSUB_NWARP(unsigned val, const uint mask){  return (__reduce_min_sync(mask, val));}
__device__ __forceinline__    float GET_MIN_TSUB_NWARP(   float val, const uint mask){
  union {uint u; float f;} tmp;
  tmp.f = val;
  tmp.u = undoFP32(GET_MIN_TSUB_NWARP(flipFP32(tmp.u), mask));
  return (tmp.f);
}
#endif//USE_WARP_REDUCE_FUNCTIONS_COMPARE_TSUB_NWARP_INC
template <typename Type>
__device__ __forceinline__ Type GET_MIN_TSUB_NWARP
(Type val
#ifdef  USE_WARP_SHUFFLE_FUNC_COMPARE_TSUB_NWARP_INC
 , const uint mask
#else///USE_WARP_SHUFFLE_FUNC_COMPARE_TSUB_NWARP_INC
 , volatile Type * smem, const int tidx, const int head
#endif//USE_WARP_SHUFFLE_FUNC_COMPARE_TSUB_NWARP_INC
 )
{
#ifdef  USE_WARP_SHUFFLE_FUNC_COMPARE_TSUB_NWARP_INC

  Type tmp;
#   if  TSUB_TN_COMPARE_INC >= ( 2 * NWARP_TN_COMPARE_INC)
  tmp = __SHFL_XOR(mask, val,      NWARP_TN_COMPARE_INC, TSUB_TN_COMPARE_INC);  val = getMinVal(val, tmp);
#   if  TSUB_TN_COMPARE_INC >= ( 4 * NWARP_TN_COMPARE_INC)
  tmp = __SHFL_XOR(mask, val,  2 * NWARP_TN_COMPARE_INC, TSUB_TN_COMPARE_INC);  val = getMinVal(val, tmp);
#   if  TSUB_TN_COMPARE_INC >= ( 8 * NWARP_TN_COMPARE_INC)
  tmp = __SHFL_XOR(mask, val,  4 * NWARP_TN_COMPARE_INC, TSUB_TN_COMPARE_INC);  val = getMinVal(val, tmp);
#   if  TSUB_TN_COMPARE_INC >= (16 * NWARP_TN_COMPARE_INC)
  tmp = __SHFL_XOR(mask, val,  8 * NWARP_TN_COMPARE_INC, TSUB_TN_COMPARE_INC);  val = getMinVal(val, tmp);
#   if  TSUB_TN_COMPARE_INC == (32 * NWARP_TN_COMPARE_INC)
  tmp = __SHFL_XOR(mask, val, 16 * NWARP_TN_COMPARE_INC, TSUB_TN_COMPARE_INC);  val = getMinVal(val, tmp);
#endif//TSUB_TN_COMPARE_INC == (32 * NWARP_TN_COMPARE_INC)
#endif//TSUB_TN_COMPARE_INC >= (16 * NWARP_TN_COMPARE_INC)
#endif//TSUB_TN_COMPARE_INC >= ( 8 * NWARP_TN_COMPARE_INC)
#endif//TSUB_TN_COMPARE_INC >= ( 4 * NWARP_TN_COMPARE_INC)
#endif//TSUB_TN_COMPARE_INC >= ( 2 * NWARP_TN_COMPARE_INC)
  val = __SHFL(mask, val, 0, TSUB_TN_COMPARE_INC);

#else///USE_WARP_SHUFFLE_FUNC_COMPARE_TSUB_NWARP_INC

  smem[tidx] = val;
#   if  TSUB_TN_COMPARE_INC >= ( 2 * NWARP_TN_COMPARE_INC)
  val = getMinVal(val, smem[tidx ^ (     NWARP_TN_COMPARE_INC)]);  smem[tidx] = val;
#   if  TSUB_TN_COMPARE_INC >= ( 4 * NWARP_TN_COMPARE_INC)
  val = getMinVal(val, smem[tidx ^ ( 2 * NWARP_TN_COMPARE_INC)]);  smem[tidx] = val;
#   if  TSUB_TN_COMPARE_INC >= ( 8 * NWARP_TN_COMPARE_INC)
  val = getMinVal(val, smem[tidx ^ ( 4 * NWARP_TN_COMPARE_INC)]);  smem[tidx] = val;
#   if  TSUB_TN_COMPARE_INC >= (16 * NWARP_TN_COMPARE_INC)
  val = getMinVal(val, smem[tidx ^ ( 8 * NWARP_TN_COMPARE_INC)]);  smem[tidx] = val;
#   if  TSUB_TN_COMPARE_INC == (32 * NWARP_TN_COMPARE_INC)
  val = getMinVal(val, smem[tidx ^ (16 * NWARP_TN_COMPARE_INC)]);  smem[tidx] = val;
#endif//TSUB_TN_COMPARE_INC == (32 * NWARP_TN_COMPARE_INC)
#endif//TSUB_TN_COMPARE_INC >= (16 * NWARP_TN_COMPARE_INC)
#endif//TSUB_TN_COMPARE_INC >= ( 8 * NWARP_TN_COMPARE_INC)
#endif//TSUB_TN_COMPARE_INC >= ( 4 * NWARP_TN_COMPARE_INC)
#endif//TSUB_TN_COMPARE_INC >= ( 2 * NWARP_TN_COMPARE_INC)
  val = smem[head];

#endif//USE_WARP_SHUFFLE_FUNC_COMPARE_TSUB_NWARP_INC

  return (val);
}


/**
 * @fn GET_MAX_TSUB_NWARP
 *
 * @brief Get maximum value within a group of TSUB_TN_COMPARE_INC threads (NWARP_TN_COMPARE_INC continuous threads have the identical value).
 * @detail implicit synchronization within TSUB_TN_COMPARE_INC (<= 32) threads (a warp) is assumed.
 */
#ifdef  USE_WARP_REDUCE_FUNCTIONS_COMPARE_TSUB_NWARP_INC
__device__ __forceinline__      int GET_MAX_TSUB_NWARP(     int val, const uint mask){  return (__reduce_max_sync(mask, val));}
__device__ __forceinline__ unsigned GET_MAX_TSUB_NWARP(unsigned val, const uint mask){  return (__reduce_max_sync(mask, val));}
__device__ __forceinline__    float GET_MAX_TSUB_NWARP(   float val, const uint mask){
  union {uint u; float f;} tmp;
  tmp.f = val;
  tmp.u = undoFP32(GET_MAX_TSUB_NWARP(flipFP32(tmp.u), mask));
  return (tmp.f);
}
#endif//USE_WARP_REDUCE_FUNCTIONS_COMPARE_TSUB_NWARP_INC
template <typename Type>
__device__ __forceinline__ Type GET_MAX_TSUB_NWARP
(Type val
#ifdef  USE_WARP_SHUFFLE_FUNC_COMPARE_TSUB_NWARP_INC
 , const uint mask
#else///USE_WARP_SHUFFLE_FUNC_COMPARE_TSUB_NWARP_INC
 , volatile Type * smem, const int tidx, const int head
#endif//USE_WARP_SHUFFLE_FUNC_COMPARE_TSUB_NWARP_INC
 )
{
#ifdef  USE_WARP_SHUFFLE_FUNC_COMPARE_TSUB_NWARP_INC

  Type tmp;
#   if  TSUB_TN_COMPARE_INC >= ( 2 * NWARP_TN_COMPARE_INC)
  tmp = __SHFL_XOR(mask, val,      NWARP_TN_COMPARE_INC, TSUB_TN_COMPARE_INC);  val = getMaxVal(val, tmp);
#   if  TSUB_TN_COMPARE_INC >= ( 4 * NWARP_TN_COMPARE_INC)
  tmp = __SHFL_XOR(mask, val,  2 * NWARP_TN_COMPARE_INC, TSUB_TN_COMPARE_INC);  val = getMaxVal(val, tmp);
#   if  TSUB_TN_COMPARE_INC >= ( 8 * NWARP_TN_COMPARE_INC)
  tmp = __SHFL_XOR(mask, val,  4 * NWARP_TN_COMPARE_INC, TSUB_TN_COMPARE_INC);  val = getMaxVal(val, tmp);
#   if  TSUB_TN_COMPARE_INC >= (16 * NWARP_TN_COMPARE_INC)
  tmp = __SHFL_XOR(mask, val,  8 * NWARP_TN_COMPARE_INC, TSUB_TN_COMPARE_INC);  val = getMaxVal(val, tmp);
#   if  TSUB_TN_COMPARE_INC == (32 * NWARP_TN_COMPARE_INC)
  tmp = __SHFL_XOR(mask, val, 16 * NWARP_TN_COMPARE_INC, TSUB_TN_COMPARE_INC);  val = getMaxVal(val, tmp);
#endif//TSUB_TN_COMPARE_INC == (32 * NWARP_TN_COMPARE_INC)
#endif//TSUB_TN_COMPARE_INC >= (16 * NWARP_TN_COMPARE_INC)
#endif//TSUB_TN_COMPARE_INC >= ( 8 * NWARP_TN_COMPARE_INC)
#endif//TSUB_TN_COMPARE_INC >= ( 4 * NWARP_TN_COMPARE_INC)
#endif//TSUB_TN_COMPARE_INC >= ( 2 * NWARP_TN_COMPARE_INC)
  val = __SHFL(mask, val, 0, TSUB_TN_COMPARE_INC);

#else///USE_WARP_SHUFFLE_FUNC_COMPARE_TSUB_NWARP_INC

  smem[tidx] = val;
#   if  TSUB_TN_COMPARE_INC >= ( 2 * NWARP_TN_COMPARE_INC)
  val = getMaxVal(val, smem[tidx ^ (     NWARP_TN_COMPARE_INC)]);  smem[tidx] = val;
#   if  TSUB_TN_COMPARE_INC >= ( 4 * NWARP_TN_COMPARE_INC)
  val = getMaxVal(val, smem[tidx ^ ( 2 * NWARP_TN_COMPARE_INC)]);  smem[tidx] = val;
#   if  TSUB_TN_COMPARE_INC >= ( 8 * NWARP_TN_COMPARE_INC)
  val = getMaxVal(val, smem[tidx ^ ( 4 * NWARP_TN_COMPARE_INC)]);  smem[tidx] = val;
#   if  TSUB_TN_COMPARE_INC >= (16 * NWARP_TN_COMPARE_INC)
  val = getMaxVal(val, smem[tidx ^ ( 8 * NWARP_TN_COMPARE_INC)]);  smem[tidx] = val;
#   if  TSUB_TN_COMPARE_INC == (32 * NWARP_TN_COMPARE_INC)
  val = getMaxVal(val, smem[tidx ^ (16 * NWARP_TN_COMPARE_INC)]);  smem[tidx] = val;
#endif//TSUB_TN_COMPARE_INC == (32 * NWARP_TN_COMPARE_INC)
#endif//TSUB_TN_COMPARE_INC >= (16 * NWARP_TN_COMPARE_INC)
#endif//TSUB_TN_COMPARE_INC >= ( 8 * NWARP_TN_COMPARE_INC)
#endif//TSUB_TN_COMPARE_INC >= ( 4 * NWARP_TN_COMPARE_INC)
#endif//TSUB_TN_COMPARE_INC >= ( 2 * NWARP_TN_COMPARE_INC)
  val = smem[head];

#endif//USE_WARP_SHUFFLE_FUNC_COMPARE_TSUB_NWARP_INC

  return (val);
}


/**
 * @fn GET_MINLOC_TSUB_NWARP
 *
 * @brief Get minimum value with location within a group of TSUB_TN_COMPARE_INC threads (NWARP_TN_COMPARE_INC continuous threads have the identical value).
 * @detail implicit synchronization within TSUB_TN_COMPARE_INC (<= 32) threads (a warp) is assumed.
 */
template <typename Type>
__device__ __forceinline__ Type GET_MINLOC_TSUB_NWARP
(Type val, volatile Type * smem, const int tidx, const int head)
{
  Type tmp;
  smem[tidx] = val;

#   if  !defined(ENABLE_IMPLICIT_SYNC_WITHIN_WARP) && (TSUB_TN_COMPARE_INC < 32)
  thread_block_tile<TSUB_TN_COMPARE_INC> tile = tiled_partition<TSUB_TN_COMPARE_INC>(this_thread_block());
#endif//!defined(ENABLE_IMPLICIT_SYNC_WITHIN_WARP) && (TSUB_TN_COMPARE_INC < 32)

#   if  TSUB_TN_COMPARE_INC >= ( 2 * NWARP_TN_COMPARE_INC)
  tmp = smem[tidx ^ (     NWARP_TN_COMPARE_INC)];  if( tmp.val < val.val )    val = tmp;
#ifndef ENABLE_IMPLICIT_SYNC_WITHIN_WARP
#   if  TSUB_TN_COMPARE_INC == 32
  __syncwarp();
#else///TSUB_TN_COMPARE_INC == 32
  tile.sync();
#endif//TSUB_TN_COMPARE_INC == 32
#endif//ENABLE_IMPLICIT_SYNC_WITHIN_WARP
  smem[tidx] = val;
#   if  TSUB_TN_COMPARE_INC >= ( 4 * NWARP_TN_COMPARE_INC)
  tmp = smem[tidx ^ ( 2 * NWARP_TN_COMPARE_INC)];  if( tmp.val < val.val )    val = tmp;
#ifndef ENABLE_IMPLICIT_SYNC_WITHIN_WARP
#   if  TSUB_TN_COMPARE_INC == 32
  __syncwarp();
#else///TSUB_TN_COMPARE_INC == 32
  tile.sync();
#endif//TSUB_TN_COMPARE_INC == 32
#endif//ENABLE_IMPLICIT_SYNC_WITHIN_WARP
  smem[tidx] = val;
#   if  TSUB_TN_COMPARE_INC >= ( 8 * NWARP_TN_COMPARE_INC)
  tmp = smem[tidx ^ ( 4 * NWARP_TN_COMPARE_INC)];  if( tmp.val < val.val )    val = tmp;
#ifndef ENABLE_IMPLICIT_SYNC_WITHIN_WARP
#   if  TSUB_TN_COMPARE_INC == 32
  __syncwarp();
#else///TSUB_TN_COMPARE_INC == 32
  tile.sync();
#endif//TSUB_TN_COMPARE_INC == 32
#endif//ENABLE_IMPLICIT_SYNC_WITHIN_WARP
  smem[tidx] = val;
#   if  TSUB_TN_COMPARE_INC >= (16 * NWARP_TN_COMPARE_INC)
  tmp = smem[tidx ^ ( 8 * NWARP_TN_COMPARE_INC)];  if( tmp.val < val.val )    val = tmp;
#ifndef ENABLE_IMPLICIT_SYNC_WITHIN_WARP
#   if  TSUB_TN_COMPARE_INC == 32
  __syncwarp();
#else///TSUB_TN_COMPARE_INC == 32
  tile.sync();
#endif//TSUB_TN_COMPARE_INC == 32
#endif//ENABLE_IMPLICIT_SYNC_WITHIN_WARP
  smem[tidx] = val;
#   if  TSUB_TN_COMPARE_INC == (32 * NWARP_TN_COMPARE_INC)
  tmp = smem[tidx ^ (16 * NWARP_TN_COMPARE_INC)];  if( tmp.val < val.val )    val = tmp;
#ifndef ENABLE_IMPLICIT_SYNC_WITHIN_WARP
  __syncwarp();
#endif//ENABLE_IMPLICIT_SYNC_WITHIN_WARP
  smem[tidx] = val;
#endif//TSUB_TN_COMPARE_INC == (32 * NWARP_TN_COMPARE_INC)
#endif//TSUB_TN_COMPARE_INC >= (16 * NWARP_TN_COMPARE_INC)
#endif//TSUB_TN_COMPARE_INC >= ( 8 * NWARP_TN_COMPARE_INC)
#endif//TSUB_TN_COMPARE_INC >= ( 4 * NWARP_TN_COMPARE_INC)
#endif//TSUB_TN_COMPARE_INC >= ( 2 * NWARP_TN_COMPARE_INC)

  val = smem[head];
  return (val);
}


/**
 * @fn GET_MAXLOC_TSUB_NWARP
 *
 * @brief Get maximum value with location within a group of TSUB_TN_COMPARE_INC threads (NWARP_TN_COMPARE_INC continuous threads have the identical value).
 * @detail implicit synchronization within TSUB_TN_COMPARE_INC (<= 32) threads (a warp) is assumed.
 */
template <typename Type>
__device__ __forceinline__ Type GET_MAXLOC_TSUB_NWARP
(Type val, volatile Type * smem, const int tidx, const int head)
{
  Type tmp;
  smem[tidx] = val;

#   if  !defined(ENABLE_IMPLICIT_SYNC_WITHIN_WARP) && (TSUB_TN_COMPARE_INC < 32)
  thread_block_tile<TSUB_TN_COMPARE_INC> tile = tiled_partition<TSUB_TN_COMPARE_INC>(this_thread_block());
#endif//!defined(ENABLE_IMPLICIT_SYNC_WITHIN_WARP) && (TSUB_TN_COMPARE_INC < 32)

#   if  TSUB_TN_COMPARE_INC >= ( 2 * NWARP_TN_COMPARE_INC)
  tmp = smem[tidx ^ (     NWARP_TN_COMPARE_INC)];  if( tmp.val > val.val )    val = tmp;
#ifndef ENABLE_IMPLICIT_SYNC_WITHIN_WARP
#   if  TSUB_TN_COMPARE_INC == 32
  __syncwarp();
#else///TSUB_TN_COMPARE_INC == 32
  tile.sync();
#endif//TSUB_TN_COMPARE_INC == 32
#endif//ENABLE_IMPLICIT_SYNC_WITHIN_WARP
  smem[tidx] = val;
#   if  TSUB_TN_COMPARE_INC >= ( 4 * NWARP_TN_COMPARE_INC)
  tmp = smem[tidx ^ ( 2 * NWARP_TN_COMPARE_INC)];  if( tmp.val > val.val )    val = tmp;
#ifndef ENABLE_IMPLICIT_SYNC_WITHIN_WARP
#   if  TSUB_TN_COMPARE_INC == 32
  __syncwarp();
#else///TSUB_TN_COMPARE_INC == 32
  tile.sync();
#endif//TSUB_TN_COMPARE_INC == 32
#endif//ENABLE_IMPLICIT_SYNC_WITHIN_WARP
  smem[tidx] = val;
#   if  TSUB_TN_COMPARE_INC >= ( 8 * NWARP_TN_COMPARE_INC)
  tmp = smem[tidx ^ ( 4 * NWARP_TN_COMPARE_INC)];  if( tmp.val > val.val )    val = tmp;
#ifndef ENABLE_IMPLICIT_SYNC_WITHIN_WARP
#   if  TSUB_TN_COMPARE_INC == 32
  __syncwarp();
#else///TSUB_TN_COMPARE_INC == 32
  tile.sync();
#endif//TSUB_TN_COMPARE_INC == 32
#endif//ENABLE_IMPLICIT_SYNC_WITHIN_WARP
  smem[tidx] = val;
#   if  TSUB_TN_COMPARE_INC >= (16 * NWARP_TN_COMPARE_INC)
  tmp = smem[tidx ^ ( 8 * NWARP_TN_COMPARE_INC)];  if( tmp.val > val.val )    val = tmp;
#ifndef ENABLE_IMPLICIT_SYNC_WITHIN_WARP
#   if  TSUB_TN_COMPARE_INC == 32
  __syncwarp();
#else///TSUB_TN_COMPARE_INC == 32
  tile.sync();
#endif//TSUB_TN_COMPARE_INC == 32
#endif//ENABLE_IMPLICIT_SYNC_WITHIN_WARP
  smem[tidx] = val;
#   if  TSUB_TN_COMPARE_INC == (32 * NWARP_TN_COMPARE_INC)
  tmp = smem[tidx ^ (16 * NWARP_TN_COMPARE_INC)];  if( tmp.val > val.val )    val = tmp;
#ifndef ENABLE_IMPLICIT_SYNC_WITHIN_WARP
  __syncwarp();
#endif//ENABLE_IMPLICIT_SYNC_WITHIN_WARP
  smem[tidx] = val;
#endif//TSUB_TN_COMPARE_INC == (32 * NWARP_TN_COMPARE_INC)
#endif//TSUB_TN_COMPARE_INC >= (16 * NWARP_TN_COMPARE_INC)
#endif//TSUB_TN_COMPARE_INC >= ( 8 * NWARP_TN_COMPARE_INC)
#endif//TSUB_TN_COMPARE_INC >= ( 4 * NWARP_TN_COMPARE_INC)
#endif//TSUB_TN_COMPARE_INC >= ( 2 * NWARP_TN_COMPARE_INC)

  val = smem[head];
  return (val);
}
