/**
 * @file compare_tsub_inc.cu
 *
 * @brief Source code for comparing values on GPU
 *
 * @author Yohei Miki (University of Tokyo)
 * @author Masayuki Umemura (University of Tsukuba)
 *
 * @date 2020/09/14 (Mon)
 *
 * Copyright (C) 2017 Yohei Miki and Masayuki Umemura
 * All rights reserved.
 *
 * The MIT License is applied to this software, see LICENSE.txt
 *
 */

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>

#include "macro.h"
#include "cudalib.h"

#include "../util/compare_tsub_inc.cuh"
#include "../util/comparison_inc.cu"

#   if  (GPUGEN >= 70) && !defined(_COOPERATIVE_GROUPS_H_)
#include <hip/hip_cooperative_groups.h>
using namespace cooperative_groups;
#endif//(GPUGEN >= 70) && !defined(_COOPERATIVE_GROUPS_H_)


/**
 * @fn GET_MIN_TSUB
 *
 * @brief Get minimum value within a group of TSUB_COMPARE_INC threads.
 * @detail implicit synchronization within TSUB_COMPARE_INC (<= 32) threads (a warp) is assumed.
 */
template <typename Type>
__device__ __forceinline__ Type GET_MIN_TSUB
(Type val
#ifdef  USE_WARP_SHUFFLE_FUNC_COMPARE_TSUB_INC
 , const uint mask
#else///USE_WARP_SHUFFLE_FUNC_COMPARE_TSUB_INC
 , volatile Type * smem, const int tidx, const int head
#endif//USE_WARP_SHUFFLE_FUNC_COMPARE_TSUB_INC
 )
{
#ifdef  USE_WARP_SHUFFLE_FUNC_COMPARE_TSUB_INC

  Type tmp;
#   if  TSUB_COMPARE_INC >=  2
  tmp = __SHFL_XOR(mask, val,  1, TSUB_COMPARE_INC);  val = getMinVal(val, tmp);
#   if  TSUB_COMPARE_INC >=  4
  tmp = __SHFL_XOR(mask, val,  2, TSUB_COMPARE_INC);  val = getMinVal(val, tmp);
#   if  TSUB_COMPARE_INC >=  8
  tmp = __SHFL_XOR(mask, val,  4, TSUB_COMPARE_INC);  val = getMinVal(val, tmp);
#   if  TSUB_COMPARE_INC >= 16
  tmp = __SHFL_XOR(mask, val,  8, TSUB_COMPARE_INC);  val = getMinVal(val, tmp);
#   if  TSUB_COMPARE_INC == 32
  tmp = __SHFL_XOR(mask, val, 16, TSUB_COMPARE_INC);  val = getMinVal(val, tmp);
#endif//TSUB_COMPARE_INC == 32
#endif//TSUB_COMPARE_INC >= 16
#endif//TSUB_COMPARE_INC >=  8
#endif//TSUB_COMPARE_INC >=  4
#endif//TSUB_COMPARE_INC >=  2
  val = __SHFL(mask, val, 0, TSUB_COMPARE_INC);

#else///USE_WARP_SHUFFLE_FUNC_COMPARE_TSUB_INC

  smem[tidx] = val;
#   if  TSUB_COMPARE_INC >=  2
  val = getMinVal(val, smem[tidx ^  1]);  smem[tidx] = val;
#   if  TSUB_COMPARE_INC >=  4
  val = getMinVal(val, smem[tidx ^  2]);  smem[tidx] = val;
#   if  TSUB_COMPARE_INC >=  8
  val = getMinVal(val, smem[tidx ^  4]);  smem[tidx] = val;
#   if  TSUB_COMPARE_INC >= 16
  val = getMinVal(val, smem[tidx ^  8]);  smem[tidx] = val;
#   if  TSUB_COMPARE_INC == 32
  val = getMinVal(val, smem[tidx ^ 16]);  smem[tidx] = val;
#endif//TSUB_COMPARE_INC == 32
#endif//TSUB_COMPARE_INC >= 16
#endif//TSUB_COMPARE_INC >=  8
#endif//TSUB_COMPARE_INC >=  4
#endif//TSUB_COMPARE_INC >=  2
  val = smem[head];

#endif//USE_WARP_SHUFFLE_FUNC_COMPARE_TSUB_INC

  return (val);
}


/**
 * @fn GET_MAX_TSUB
 *
 * @brief Get maximum value within a group of TSUB_COMPARE_INC threads.
 * @detail implicit synchronization within TSUB_COMPARE_INC (<= 32) threads (a warp) is assumed.
 */
template <typename Type>
__device__ __forceinline__ Type GET_MAX_TSUB
(Type val
#ifdef  USE_WARP_SHUFFLE_FUNC_COMPARE_TSUB_INC
 , const uint mask
#else///USE_WARP_SHUFFLE_FUNC_COMPARE_TSUB_INC
 , volatile Type * smem, const int tidx, const int head
#endif//USE_WARP_SHUFFLE_FUNC_COMPARE_TSUB_INC
 )
{
#ifdef  USE_WARP_SHUFFLE_FUNC_COMPARE_TSUB_INC

  Type tmp;
#   if  TSUB_COMPARE_INC >=  2
  tmp = __SHFL_XOR(mask, val,  1, TSUB_COMPARE_INC);  val = getMaxVal(val, tmp);
#   if  TSUB_COMPARE_INC >=  4
  tmp = __SHFL_XOR(mask, val,  2, TSUB_COMPARE_INC);  val = getMaxVal(val, tmp);
#   if  TSUB_COMPARE_INC >=  8
  tmp = __SHFL_XOR(mask, val,  4, TSUB_COMPARE_INC);  val = getMaxVal(val, tmp);
#   if  TSUB_COMPARE_INC >= 16
  tmp = __SHFL_XOR(mask, val,  8, TSUB_COMPARE_INC);  val = getMaxVal(val, tmp);
#   if  TSUB_COMPARE_INC == 32
  tmp = __SHFL_XOR(mask, val, 16, TSUB_COMPARE_INC);  val = getMaxVal(val, tmp);
#endif//TSUB_COMPARE_INC == 32
#endif//TSUB_COMPARE_INC >= 16
#endif//TSUB_COMPARE_INC >=  8
#endif//TSUB_COMPARE_INC >=  4
#endif//TSUB_COMPARE_INC >=  2
  val = __SHFL(mask, val, 0, TSUB_COMPARE_INC);

#else///USE_WARP_SHUFFLE_FUNC_COMPARE_TSUB_INC

  smem[tidx] = val;
#   if  TSUB_COMPARE_INC >=  2
  val = getMaxVal(val, smem[tidx ^  1]);  smem[tidx] = val;
#   if  TSUB_COMPARE_INC >=  4
  val = getMaxVal(val, smem[tidx ^  2]);  smem[tidx] = val;
#   if  TSUB_COMPARE_INC >=  8
  val = getMaxVal(val, smem[tidx ^  4]);  smem[tidx] = val;
#   if  TSUB_COMPARE_INC >= 16
  val = getMaxVal(val, smem[tidx ^  8]);  smem[tidx] = val;
#   if  TSUB_COMPARE_INC == 32
  val = getMaxVal(val, smem[tidx ^ 16]);  smem[tidx] = val;
#endif//TSUB_COMPARE_INC == 32
#endif//TSUB_COMPARE_INC >= 16
#endif//TSUB_COMPARE_INC >=  8
#endif//TSUB_COMPARE_INC >=  4
#endif//TSUB_COMPARE_INC >=  2
  val = smem[head];

#endif//USE_WARP_SHUFFLE_FUNC_COMPARE_TSUB_INC

  return (val);
}


/**
 * @fn GET_MINLOC_TSUB
 *
 * @brief Get minimum value with location within a group of TSUB_COMPARE_INC threads.
 * @detail implicit synchronization within TSUB_COMPARE_INC (<= 32) threads (a warp) is assumed.
 */
template <typename Type>
__device__ __forceinline__ Type GET_MINLOC_TSUB
(Type val, volatile Type * smem, const int tidx, const int head)
{
  Type tmp;
  smem[tidx] = val;

#   if  !defined(ENABLE_IMPLICIT_SYNC_WITHIN_WARP) && (TSUB_COMPARE_INC < 32)
  thread_block_tile<TSUB_COMPARE_INC> tile = tiled_partition<TSUB_COMPARE_INC>(this_thread_block());
#endif//!defined(ENABLE_IMPLICIT_SYNC_WITHIN_WARP) && (TSUB_COMPARE_INC < 32)

#   if  TSUB_COMPARE_INC >=  2
  tmp = smem[tidx ^  1];  if( tmp.val < val.val )    val = tmp;
#ifndef ENABLE_IMPLICIT_SYNC_WITHIN_WARP
#   if  TSUB_COMPARE_INC == 32
  __syncwarp();
#else///TSUB_COMPARE_INC == 32
  tile.sync();
#endif//TSUB_COMPARE_INC == 32
#endif//ENABLE_IMPLICIT_SYNC_WITHIN_WARP
  smem[tidx] = val;
#   if  TSUB_COMPARE_INC >=  4
  tmp = smem[tidx ^  2];  if( tmp.val < val.val )    val = tmp;
#ifndef ENABLE_IMPLICIT_SYNC_WITHIN_WARP
#   if  TSUB_COMPARE_INC == 32
  __syncwarp();
#else///TSUB_COMPARE_INC == 32
  tile.sync();
#endif//TSUB_COMPARE_INC == 32
#endif//ENABLE_IMPLICIT_SYNC_WITHIN_WARP
  smem[tidx] = val;
#   if  TSUB_COMPARE_INC >=  8
  tmp = smem[tidx ^  4];  if( tmp.val < val.val )    val = tmp;
#ifndef ENABLE_IMPLICIT_SYNC_WITHIN_WARP
#   if  TSUB_COMPARE_INC == 32
  __syncwarp();
#else///TSUB_COMPARE_INC == 32
  tile.sync();
#endif//TSUB_COMPARE_INC == 32
#endif//ENABLE_IMPLICIT_SYNC_WITHIN_WARP
  smem[tidx] = val;
#   if  TSUB_COMPARE_INC >= 16
  tmp = smem[tidx ^  8];  if( tmp.val < val.val )    val = tmp;
#ifndef ENABLE_IMPLICIT_SYNC_WITHIN_WARP
#   if  TSUB_COMPARE_INC == 32
  __syncwarp();
#else///TSUB_COMPARE_INC == 32
  tile.sync();
#endif//TSUB_COMPARE_INC == 32
#endif//ENABLE_IMPLICIT_SYNC_WITHIN_WARP
  smem[tidx] = val;
#   if  TSUB_COMPARE_INC == 32
  tmp = smem[tidx ^ 16];  if( tmp.val < val.val )    val = tmp;
#ifndef ENABLE_IMPLICIT_SYNC_WITHIN_WARP
  __syncwarp();
#endif//ENABLE_IMPLICIT_SYNC_WITHIN_WARP
  smem[tidx] = val;
#endif//TSUB_COMPARE_INC == 32
#endif//TSUB_COMPARE_INC >= 16
#endif//TSUB_COMPARE_INC >=  8
#endif//TSUB_COMPARE_INC >=  4
#endif//TSUB_COMPARE_INC >=  2

  val = smem[head];
  return (val);
}


/**
 * @fn GET_MAXLOC_TSUB
 *
 * @brief Get maximum value with location within a group of TSUB_COMPARE_INC threads.
 * @detail implicit synchronization within TSUB_COMPARE_INC (<= 32) threads (a warp) is assumed.
 */
template <typename Type>
__device__ __forceinline__ Type GET_MAXLOC_TSUB
(Type val, volatile Type * smem, const int tidx, const int head)
{
  Type tmp;
  smem[tidx] = val;

#   if  !defined(ENABLE_IMPLICIT_SYNC_WITHIN_WARP) && (TSUB_COMPARE_INC < 32)
  thread_block_tile<TSUB_COMPARE_INC> tile = tiled_partition<TSUB_COMPARE_INC>(this_thread_block());
#endif//!defined(ENABLE_IMPLICIT_SYNC_WITHIN_WARP) && (TSUB_COMPARE_INC < 32)

#   if  TSUB_COMPARE_INC >=  2
  tmp = smem[tidx ^  1];  if( tmp.val > val.val )    val = tmp;
#ifndef ENABLE_IMPLICIT_SYNC_WITHIN_WARP
#   if  TSUB_COMPARE_INC == 32
  __syncwarp();
#else///TSUB_COMPARE_INC == 32
  tile.sync();
#endif//TSUB_COMPARE_INC == 32
#endif//ENABLE_IMPLICIT_SYNC_WITHIN_WARP
  smem[tidx] = val;
#   if  TSUB_COMPARE_INC >=  4
  tmp = smem[tidx ^  2];  if( tmp.val > val.val )    val = tmp;
#ifndef ENABLE_IMPLICIT_SYNC_WITHIN_WARP
#   if  TSUB_COMPARE_INC == 32
  __syncwarp();
#else///TSUB_COMPARE_INC == 32
  tile.sync();
#endif//TSUB_COMPARE_INC == 32
#endif//ENABLE_IMPLICIT_SYNC_WITHIN_WARP
  smem[tidx] = val;
#   if  TSUB_COMPARE_INC >=  8
  tmp = smem[tidx ^  4];  if( tmp.val > val.val )    val = tmp;
#ifndef ENABLE_IMPLICIT_SYNC_WITHIN_WARP
#   if  TSUB_COMPARE_INC == 32
  __syncwarp();
#else///TSUB_COMPARE_INC == 32
  tile.sync();
#endif//TSUB_COMPARE_INC == 32
#endif//ENABLE_IMPLICIT_SYNC_WITHIN_WARP
  smem[tidx] = val;
#   if  TSUB_COMPARE_INC >= 16
  tmp = smem[tidx ^  8];  if( tmp.val > val.val )    val = tmp;
#ifndef ENABLE_IMPLICIT_SYNC_WITHIN_WARP
#   if  TSUB_COMPARE_INC == 32
  __syncwarp();
#else///TSUB_COMPARE_INC == 32
  tile.sync();
#endif//TSUB_COMPARE_INC == 32
#endif//ENABLE_IMPLICIT_SYNC_WITHIN_WARP
  smem[tidx] = val;
#   if  TSUB_COMPARE_INC == 32
  tmp = smem[tidx ^ 16];  if( tmp.val > val.val )    val = tmp;
#ifndef ENABLE_IMPLICIT_SYNC_WITHIN_WARP
  __syncwarp();
#endif//ENABLE_IMPLICIT_SYNC_WITHIN_WARP
  smem[tidx] = val;
#endif//TSUB_COMPARE_INC == 32
#endif//TSUB_COMPARE_INC >= 16
#endif//TSUB_COMPARE_INC >=  8
#endif//TSUB_COMPARE_INC >=  4
#endif//TSUB_COMPARE_INC >=  2

  val = smem[head];
  return (val);
}
