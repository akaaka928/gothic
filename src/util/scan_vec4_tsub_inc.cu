/**
 * @file scan_vec4_tsub_inc.cu
 *
 * @brief Source code for parallel prefix sum library for 4-components vector on GPU
 *
 * @author Yohei Miki (University of Tsukuba)
 * @author Masayuki Umemura (University of Tsukuba)
 *
 * @date 2017/04/05 (Wed)
 *
 * Copyright (C) 2017 Yohei Miki and Masayuki Umemura
 * All rights reserved.
 *
 * The MIT License is applied to this software, see LICENSE.txt
 *
 */

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>

#include "macro.h"
#include "cudalib.h"

#include "../util/vector_inc.cu"
#include "../util/scan_vec4_tsub_inc.cuh"


/**
 * @fn PREFIX_SUM_VEC4_TSUB
 *
 * @brief Get parallel (inclusive) prefix sum within a group of TSUB_SCAN_VEC4_INC threads.
 * @detail implicit synchronization within TSUB_SCAN_VEC4_INC (<= 32) threads (a warp) is assumed.
 */
template <typename Type>
__device__ __forceinline__ Type PREFIX_SUM_VEC4_TSUB(Type val, const int lane, volatile Type * smem, const int tidx)
{
  Type tmp;

  stvec((Type *)smem, tidx, val);
#   if  TSUB_SCAN_VEC4_INC >=  2
  if( lane >=  1 ){    tmp = ldvec(smem[tidx -  1]);    val.x += tmp.x;    val.y += tmp.y;    val.z += tmp.z;    val.w += tmp.w;    stvec((Type *)smem, tidx, val);  }
#   if  TSUB_SCAN_VEC4_INC >=  4
  if( lane >=  2 ){    tmp = ldvec(smem[tidx -  2]);    val.x += tmp.x;    val.y += tmp.y;    val.z += tmp.z;    val.w += tmp.w;    stvec((Type *)smem, tidx, val);  }
#   if  TSUB_SCAN_VEC4_INC >=  8
  if( lane >=  4 ){    tmp = ldvec(smem[tidx -  4]);    val.x += tmp.x;    val.y += tmp.y;    val.z += tmp.z;    val.w += tmp.w;    stvec((Type *)smem, tidx, val);  }
#   if  TSUB_SCAN_VEC4_INC >= 16
  if( lane >=  8 ){    tmp = ldvec(smem[tidx -  8]);    val.x += tmp.x;    val.y += tmp.y;    val.z += tmp.z;    val.w += tmp.w;    stvec((Type *)smem, tidx, val);  }
#   if  TSUB_SCAN_VEC4_INC == 32
  if( lane >= 16 ){    tmp = ldvec(smem[tidx - 16]);    val.x += tmp.x;    val.y += tmp.y;    val.z += tmp.z;    val.w += tmp.w;    stvec((Type *)smem, tidx, val);  }
#endif//TSUB_SCAN_VEC4_INC == 32
#endif//TSUB_SCAN_VEC4_INC >= 16
#endif//TSUB_SCAN_VEC4_INC >=  8
#endif//TSUB_SCAN_VEC4_INC >=  4
#endif//TSUB_SCAN_VEC4_INC >=  2

  return (val);
}


/**
 * @fn TOTAL_SUM_VEC4_TSUB
 *
 * @brief Get total sum within a group of TSUB_SCAN_VEC4_INC threads.
 * @detail implicit synchronization within TSUB_SCAN_VEC4_INC (<= 32) threads (a warp) is assumed.
 */
template <typename Type>
__device__ __forceinline__ Type TOTAL_SUM_VEC4_TSUB(Type val, volatile Type * smem, const int tidx, const int head)
{
  Type tmp;

  stvec((Type *)smem, tidx, val);
#   if  TSUB_SCAN_VEC4_INC >=  2
  tmp = ldvec(smem[tidx ^  1]);  val.x += tmp.x;  val.y += tmp.y;  val.z += tmp.z;  val.w += tmp.w;  stvec((Type *)smem, tidx, val);
#   if  TSUB_SCAN_VEC4_INC >=  4
  tmp = ldvec(smem[tidx ^  2]);  val.x += tmp.x;  val.y += tmp.y;  val.z += tmp.z;  val.w += tmp.w;  stvec((Type *)smem, tidx, val);
#   if  TSUB_SCAN_VEC4_INC >=  8
  tmp = ldvec(smem[tidx ^  4]);  val.x += tmp.x;  val.y += tmp.y;  val.z += tmp.z;  val.w += tmp.w;  stvec((Type *)smem, tidx, val);
#   if  TSUB_SCAN_VEC4_INC >= 16
  tmp = ldvec(smem[tidx ^  8]);  val.x += tmp.x;  val.y += tmp.y;  val.z += tmp.z;  val.w += tmp.w;  stvec((Type *)smem, tidx, val);
#   if  TSUB_SCAN_VEC4_INC == 32
  tmp = ldvec(smem[tidx ^ 16]);  val.x += tmp.x;  val.y += tmp.y;  val.z += tmp.z;  val.w += tmp.w;  stvec((Type *)smem, tidx, val);
#endif//TSUB_SCAN_VEC4_INC == 32
#endif//TSUB_SCAN_VEC4_INC >= 16
#endif//TSUB_SCAN_VEC4_INC >=  8
#endif//TSUB_SCAN_VEC4_INC >=  4
#endif//TSUB_SCAN_VEC4_INC >=  2

  val = ldvec(smem[head]);

  return (val);
}
