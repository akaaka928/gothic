#include "hip/hip_runtime.h"
/**
 * @file gsync_dev.cu
 *
 * @brief Utility tool for inter-block GPU synchronization
 *        (based on GPU Lock-Free Synchronization by Xiao & Feng 2009)
 *
 * @author Yohei Miki (University of Tsukuba)
 * @author Masayuki Umemura (University of Tsukuba)
 *
 * @date 2017/02/28 (Tue)
 *
 * Copyright (C) 2017 Yohei Miki and Masayuki Umemura
 * All rights reserved.
 *
 * The MIT License is applied to this software, see LICENSE.txt
 *
 */

#ifndef GSYNC_DEV_CU
#define GSYNC_DEV_CU


#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>

#include "macro.h"
#include "cudalib.h"


/**
 * @fn globalSync
 *
 * @brief Function to execute global synchronization.
 *
 * @param (tidx) thread index within a block
 * @param (bidx) block index
 * @param (bnum) number of blocks
 * @param (gsync0) temporary array on the global memory
 * @param (gsync1) temporary array on the global memory
 */
__device__ __forceinline__ void globalSync(const int tidx, const int bidx, const int bnum, volatile int * gsync0, volatile int * gsync1)
{
  /** Phase 0. tell */
  __syncthreads();
  if( tidx == 0 )
    gsync0[bidx] = 1;


  /** Phase 1. watch */
  if( bidx == 0 ){
    for(int ii = tidx; ii < bnum; ii += BLOCKDIM_X1D)
      while( true )
  	if( gsync0[ii] ){
  	  gsync0[ii] = 0;
  	  break;
  	}/* if( gsync0[ii] ){ */

    __syncthreads();

    for(int ii = tidx; ii < bnum; ii += BLOCKDIM_X1D)
      gsync1[ii] = 1;
  }/* if( bidx == 0 ){ */


  /** Phase 2. check */
  if( tidx == 0 )
    while( true )
      if( gsync1[bidx] ){
  	gsync1[bidx] = 0;
  	break;
      }/* if( gsync1[bidx] ){ */


  __syncthreads();
}


/**
 * @fn initGsync_kernel
 *
 * @brief Initialize arrays for inter-block GPU synchronization.
 *
 * @param (num) number of blocks
 * @return (gsync0) temporary array on the global memory
 * @return (gsync1) temporary array on the global memory
 */
__global__ static void initGsync_kernel(const int num, int * RESTRICT gsync0, int * RESTRICT gsync1)
{
  const int gidx = GLOBALIDX_X1D;

  if( gidx < num ){
    gsync0[gidx] = 0;
    gsync1[gidx] = 0;
  }/* if( gidx < num ){ */
}


#endif//GSYNC_DEV_CU
