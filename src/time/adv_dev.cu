#include "hip/hip_runtime.h"
/*************************************************************************\
 *                                                                       *
                  last updated on 2016/06/21(Tue) 17:06:05
 *                                                                       *
 *    Orbit integration of N-body particles in collisionless systems     *
 *                                                                       *
 *                                                                       *
 *                                                                       *
 *                                             written by Yohei MIKI     *
 *                                                                       *
\*************************************************************************/
//-------------------------------------------------------------------------
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <math.h>
#include <hip/hip_runtime_api.h>
//-------------------------------------------------------------------------
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/find.h>
/* #ifdef  CUB_AVAILABLE */
/* #include <cub/device/device_radix_sort.cuh> */
/* #else///CUB_AVAILABLE */
#include <thrust/sort.h>
/* #endif//CUB_AVAILABLE */
//-------------------------------------------------------------------------
#include <macro.h>
#include <cudalib.h>
#include <sys/time.h>
#include <timer.h>
//-------------------------------------------------------------------------
#include "../misc/benchmark.h"
#include "../misc/structure.h"
#include "../misc/device.h"
//-------------------------------------------------------------------------
#ifndef SERIALIZED_EXECUTION
#       include <mpi.h>
#       include <mpilib.h>
#       include "../para/mpicfg.h"
#endif//SERIALIZED_EXECUTION
//-------------------------------------------------------------------------
#include "../tree/walk_dev.h"
//-------------------------------------------------------------------------
#include "adv_dev.h"
//-------------------------------------------------------------------------


//-------------------------------------------------------------------------
/* memory allocation on global memory of GPU(s) */
//-------------------------------------------------------------------------
#ifndef BLOCK_TIME_STEP
//-------------------------------------------------------------------------
extern "C"
muse allocTimeStep_dev(real **dt_dev)
{
  //-----------------------------------------------------------------------
  __NOTE__("%s\n", "start");
  //-----------------------------------------------------------------------
  muse alloc = {0, 0};
  mycudaMalloc((void **)dt_dev, 1 * sizeof(real));
  alloc.device +=               1 * sizeof(real);
  //-----------------------------------------------------------------------
  __NOTE__("%s\n", "end");
  //-----------------------------------------------------------------------
  return (alloc);
  //-----------------------------------------------------------------------
}
//-------------------------------------------------------------------------
extern "C"
void  freeTimeStep_dev(real  *dt_dev)
{
  //-----------------------------------------------------------------------
  __NOTE__("%s\n", "start");
  //-----------------------------------------------------------------------
  mycudaFree(dt_dev);
  //-----------------------------------------------------------------------
  __NOTE__("%s\n", "end");
  //-----------------------------------------------------------------------
}
//-------------------------------------------------------------------------
#endif//BLOCK_TIME_STEP
//-------------------------------------------------------------------------


//-------------------------------------------------------------------------
#ifdef  BLOCK_TIME_STEP
//-------------------------------------------------------------------------
/* #ifdef  CUB_AVAILABLE */
/* //------------------------------------------------------------------------- */
/* extern "C" */
/* muse allocTimeStep_dev(void **temp_storage, laneinfo **info, double **time, soaCUBtime *buf, laneinfo *laneInfo_dev, double *laneTime_dev, const int Ngrp) */
/* { */
/*   //----------------------------------------------------------------------- */
/*   __NOTE__("%s\n", "start"); */
/*   //----------------------------------------------------------------------- */
/*   // */
/*   //----------------------------------------------------------------------- */
/*   muse alloc = {0, 0}; */
/*   //----------------------------------------------------------------------- */
/*   mycudaMalloc((void **)info, (size_t)Ngrp * sizeof(laneinfo));  alloc.device += (size_t)Ngrp * sizeof(laneinfo); */
/*   mycudaMalloc((void **)time, (size_t)Ngrp * sizeof(  double));  alloc.device += (size_t)Ngrp * sizeof(  double); */
/*   //----------------------------------------------------------------------- */
/*   size_t temp_storage_size = 0; */
/*   *temp_storage = NULL; */
/*   hipcub::DeviceRadixSort::SortPairs(*temp_storage, temp_storage_size, laneTime_dev, *time, laneInfo_dev, *info, Ngrp); */
/*   mycudaMalloc(temp_storage, temp_storage_size);  alloc.device += temp_storage_size; */
/*   //----------------------------------------------------------------------- */
/*   buf->temp_storage = *temp_storage; */
/*   buf->temp_storage_size = temp_storage_size; */
/*   buf->time = *time; */
/*   buf->info = *info; */
/*   //----------------------------------------------------------------------- */
/*   // */
/*   //----------------------------------------------------------------------- */
/*   __NOTE__("%s\n", "end"); */
/*   //----------------------------------------------------------------------- */
/*   return (alloc); */
/*   //----------------------------------------------------------------------- */
/* } */
/* //------------------------------------------------------------------------- */
/* extern "C" */
/* void  freeTimeStep_dev(void  *temp_storage, laneinfo  *info, double  *time) */
/* { */
/*   //----------------------------------------------------------------------- */
/*   __NOTE__("%s\n", "start"); */
/*   //----------------------------------------------------------------------- */
/*   mycudaFree(temp_storage); */
/*   mycudaFree(info); */
/*   mycudaFree(time); */
/*   //----------------------------------------------------------------------- */
/*   __NOTE__("%s\n", "end"); */
/*   //----------------------------------------------------------------------- */
/* } */
/* //------------------------------------------------------------------------- */
/* #endif//CUB_AVAILABLE */
//-------------------------------------------------------------------------
__global__ void adjustTimeStep_kernel
(const double tnew, const int laneNum, READ_ONLY laneinfo * RESTRICT laneInfo, velocity * RESTRICT ivel, ibody_time * RESTRICT time)
{
  //-----------------------------------------------------------------------
#if 0
  const int tidx = THREADIDX_X1D;
  const int lane = tidx & (TSUB - 1);/* index of the thread within a thread group */
  /* const int laneIdx = GLOBALIDX_X1D / TSUB; */
  const int laneIdx = DIV_TSUB(GLOBALIDX_X1D);
#else
  /* const int lane    = THREADIDX_X1D & ((TSUB / NWARP) - 1); */
  /* const int laneIdx = GLOBALIDX_X1D /  (TSUB / NWARP); */
  const int lane    = THREADIDX_X1D & (DIV_NWARP(TSUB) - 1);
  const int laneIdx = GLOBALIDX_X1D /  DIV_NWARP(TSUB);
#endif
  //-----------------------------------------------------------------------
#if 0
  const laneinfo info = laneInfo[laneIdx];
#else
  laneinfo info = {NUM_BODY_MAX, 0};
  if( laneIdx < laneNum )
    info = laneInfo[laneIdx];
#endif
  //-----------------------------------------------------------------------

  //-----------------------------------------------------------------------
  if( lane < info.num ){
    //---------------------------------------------------------------------
    const int idx = info.head + lane;
    //---------------------------------------------------------------------
    ibody_time ti = time[idx];
    ti.t1 = tnew;
    time[idx] = ti;
    //---------------------------------------------------------------------
    velocity vi = ivel[idx];
    vi.dt = (real)(ti.t1 - ti.t0);
    ivel[idx] = vi;
    //---------------------------------------------------------------------
  }
  //-----------------------------------------------------------------------
}
//-------------------------------------------------------------------------
struct greater_than
{
  double val;
greater_than(double val) : val(val) {}
  __host__ __device__
  int operator()(const double &x) const {
    return (x > val);
  }
};
//-------------------------------------------------------------------------
extern "C"
void setTimeStep_dev
(const int Ngrp, laneinfo * RESTRICT laneInfo_dev, double * RESTRICT laneTime_dev, int *grpNum, const iparticle pi,
 const double told, double *tnew, double *dt, bool adjustAllTimeStep, const double invSnapshotInterval, const uint previous, uint *present
/* #ifdef  CUB_AVAILABLE */
/*  , soaCUBtime buf */
/* #endif//CUB_AVAILABLE */
#ifndef SERIALIZED_EXECUTION
 , MPIcfg_tree mpi
#endif//SERIALIZED_EXECUTION
#ifdef  EXEC_BENCHMARK
 , wall_clock_time *elapsed
#endif//EXEC_BENCHMARK
)
{
  //-----------------------------------------------------------------------
  __NOTE__("%s\n", "start");
  //-----------------------------------------------------------------------

  //-----------------------------------------------------------------------
#ifdef  EXEC_BENCHMARK
  initStopwatch();
#endif//EXEC_BENCHMARK
  //-----------------------------------------------------------------------
/* #ifdef  CUB_AVAILABLE */
/*   checkCudaErrors(hipMemcpy(buf.time, laneTime_dev, sizeof(  double) * Ngrp, hipMemcpyDeviceToDevice)); */
/*   checkCudaErrors(hipMemcpy(buf.info, laneInfo_dev, sizeof(laneinfo) * Ngrp, hipMemcpyDeviceToDevice)); */
/*   hipcub::DeviceRadixSort::SortPairs(buf.temp_storage, buf.temp_storage_size, buf.time, laneTime_dev, buf.info, laneInfo_dev, Ngrp); */
/* #else///CUB_AVAILABLE */
  thrust::stable_sort_by_key((thrust::device_ptr<double>)laneTime_dev, (thrust::device_ptr<double>)(laneTime_dev + Ngrp), (thrust::device_ptr<laneinfo>)laneInfo_dev);
/* #endif//CUB_AVAILABLE */
  checkCudaErrors(hipMemcpy(tnew, laneTime_dev, sizeof(double), hipMemcpyDeviceToHost));
#ifndef SERIALIZED_EXECUTION
  chkMPIerr(MPI_Allreduce(MPI_IN_PLACE, tnew, 1, MPI_DOUBLE, MPI_MIN, mpi.comm));
#endif//SERIALIZED_EXECUTION
  *dt = (*tnew) - told;
  //-----------------------------------------------------------------------
  *present = (uint)((*tnew) * invSnapshotInterval);
  if( *present != previous )
    adjustAllTimeStep = true;
  if( !adjustAllTimeStep ){
    thrust::device_vector<double>::iterator iter1 =                 (thrust::device_ptr<double>)laneTime_dev;
    thrust::device_vector<double>::iterator iter2 = thrust::find_if((thrust::device_ptr<double>)laneTime_dev, (thrust::device_ptr<double>)(laneTime_dev + Ngrp), greater_than(*tnew));
    *grpNum = thrust::distance(iter1, iter2);
  }/* if( !adjustAllTimeStep ){ */
  else
    *grpNum = Ngrp;
  //-----------------------------------------------------------------------
#ifndef SERIALIZED_EXECUTION
  if( *grpNum != 0 )
#endif//SERIALIZED_EXECUTION
    {
      //-------------------------------------------------------------------
      int Nrem = BLOCKSIZE(*grpNum, NWARP * NGROUPS);
      if( Nrem <= MAX_BLOCKS_PER_GRID )
	adjustTimeStep_kernel<<<Nrem, NTHREADS>>>(*tnew, BLOCKSIZE(*grpNum, NGROUPS) * NGROUPS, laneInfo_dev, pi.vel, pi.time);
      //-------------------------------------------------------------------
      else{
	//-----------------------------------------------------------------
	const int Niter = BLOCKSIZE(Nrem, MAX_BLOCKS_PER_GRID);
	int hidx = 0;
	//-----------------------------------------------------------------
	for(int iter = 0; iter < Niter; iter++){
	  //---------------------------------------------------------------
	  int Nblck = MAX_BLOCKS_PER_GRID;
	  if( Nblck > Nrem )	    Nblck = Nrem;
	  //---------------------------------------------------------------
	  int Nsub = Nblck * NWARP * NGROUPS;
	  adjustTimeStep_kernel<<<Nblck, NTHREADS>>>(*tnew, BLOCKSIZE(Nsub, NGROUPS) * NGROUPS, &laneInfo_dev[hidx], pi.vel, pi.time);
	  //-------------------------------------------------------------------
	  hidx += Nsub;
	  Nrem -= Nblck;
	  //-------------------------------------------------------------------
	}/* for(int iter = 0; iter < Niter; iter++){ */
	//-----------------------------------------------------------------
      }/* else{ */
      //-------------------------------------------------------------------
      getLastCudaError("adjustTimeStep_kernel");
      //-------------------------------------------------------------------
      /* /\* adjustTimeStep_kernel<<<BLOCKSIZE(*grpNum,         NGROUPS), NTHREADS>>>(*tnew,                                        laneInfo_dev, pi.vel, pi.time); *\/ */
      /* adjustTimeStep_kernel<<<BLOCKSIZE(*grpNum, NWARP * NGROUPS), NTHREADS>>>(*tnew, BLOCKSIZE(*grpNum, NGROUPS) * NGROUPS, laneInfo_dev, pi.vel, pi.time); */
      /* getLastCudaError("adjustTimeStep_kernel"); */
      //-------------------------------------------------------------------
    }
  //-----------------------------------------------------------------------
#ifdef  EXEC_BENCHMARK
  stopStopwatch(&(elapsed->setTimeStep_dev));
#endif//EXEC_BENCHMARK
  //-----------------------------------------------------------------------
#ifdef  SHOW_NI_DEPENDENCE
  laneinfo *laneInfo_hst;  mycudaMallocHost((void **)&laneInfo_hst, sizeof(laneinfo) * Ngrp);
  double   *laneTime_hst;  mycudaMallocHost((void **)&laneTime_hst, sizeof(double  ) * Ngrp);
  checkCudaErrors(hipMemcpy(laneInfo_hst, laneInfo_dev, sizeof(laneinfo) * Ngrp, hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(laneTime_hst, laneTime_dev, sizeof(double  ) * Ngrp, hipMemcpyDeviceToHost));
  printf("#group ID\tnew time\n");
  int Ni_active = 0;
  for(int ii = 0; ii < Ngrp; ii++){
    Ni_active += laneInfo_hst[ii].num;
    printf("%d\t%d\t%e\n", ii, Ni_active, laneTime_hst[ii]);
  }
  printf("#*grpNum is %d\n\n", *grpNum);
  fflush(stdout);
  mycudaFreeHost(laneInfo_hst);
  mycudaFreeHost(laneTime_hst);
#endif//SHOW_NI_DEPENDENCE
  //-----------------------------------------------------------------------

  //-----------------------------------------------------------------------
  __NOTE__("%s\n", "end");
  //-----------------------------------------------------------------------
}
//-------------------------------------------------------------------------
#else///BLOCK_TIME_STEP
//-------------------------------------------------------------------------
__global__ void setTimeStep_kernel
(const int Ni,
 READ_ONLY real * RESTRICT vix, READ_ONLY real * RESTRICT viy, READ_ONLY real * RESTRICT viz,
 READ_ONLY acceleration * RESTRICT iacc,
 const real eta, const real eps,
 real * RESTRICT dt)
{
  //-----------------------------------------------------------------------
  /* identify thread properties */
  //-----------------------------------------------------------------------
  const int tidx = THREADIDX_X1D;
  //-----------------------------------------------------------------------

  //-----------------------------------------------------------------------
  /* calculate time step of individual N-body particle */
  real dtloc = REAL_MAX;
  //-----------------------------------------------------------------------
  for(int ih = 0; ih < Ni; ih += NTHREADS_TIME){
    //---------------------------------------------------------------------
    /* const uint ii = ih + lane; */
    const int ii = ih + tidx;
    //---------------------------------------------------------------------
    if( ii < Ni ){
      //-------------------------------------------------------------------
      const real         vx = vix [ii];
      const real         vy = viy [ii];
      const real         vz = viz [ii];
      const acceleration ai = iacc[ii];
      //-------------------------------------------------------------------
      const real v2 = EPSILON +   vx *   vx +   vy *   vy +   vz *   vz;
      const real a2 = EPSILON + ai.x * ai.x + ai.y * ai.y + ai.z * ai.z;
      //-------------------------------------------------------------------
      const real vdt =      eps * RSQRT(v2);
      const real adt = SQRT(eps * RSQRT(a2));
      //-------------------------------------------------------------------
      real dttmp = (vdt < adt) ? (vdt) : (adt);
      if( dttmp < dtloc )
	dtloc = dttmp;
      //-------------------------------------------------------------------
    }
    //---------------------------------------------------------------------
  }
  //-----------------------------------------------------------------------


  //-----------------------------------------------------------------------
  /* find the minimum time step */
  //-----------------------------------------------------------------------
#ifdef  USE_WARP_SHUFFLE_FUNC_TIME
  __shared__ real dtmin[32];
#else///USE_WARP_SHUFFLE_FUNC_TIME
  __shared__ real dtmin[NTHREADS_TIME];
#endif//USE_WARP_SHUFFLE_FUNC_TIME
  //-----------------------------------------------------------------------
  /* find minimum time step within a warp */
  real dttmp;
#ifdef  USE_WARP_SHUFFLE_FUNC_TIME
  dttmp = __shfl_xor(dtloc,  1, warpSize);  if( dttmp < dtloc )    dtloc = dttmp;
  dttmp = __shfl_xor(dtloc,  2, warpSize);  if( dttmp < dtloc )    dtloc = dttmp;
  dttmp = __shfl_xor(dtloc,  4, warpSize);  if( dttmp < dtloc )    dtloc = dttmp;
  dttmp = __shfl_xor(dtloc,  8, warpSize);  if( dttmp < dtloc )    dtloc = dttmp;
  dttmp = __shfl_xor(dtloc, 16, warpSize);  if( dttmp < dtloc )    dtloc = dttmp;
  if( (tidx & (warpSize - 1)) == 0 )
    dtmin[tidx / warpSize] = dtloc;
#else///USE_WARP_SHUFFLE_FUNC_TIME
  dtmin[tidx] = dtloc;
  dttmp = dtmin[tidx ^  1];  if( dttmp < dtloc ){    dtloc = dttmp;  }  dtmin[tidx] = dtloc;/* w/ \pm  1 */
  dttmp = dtmin[tidx ^  2];  if( dttmp < dtloc ){    dtloc = dttmp;  }  dtmin[tidx] = dtloc;/* w/ \pm  2 */
  dttmp = dtmin[tidx ^  4];  if( dttmp < dtloc ){    dtloc = dttmp;  }  dtmin[tidx] = dtloc;/* w/ \pm  4 */
  dttmp = dtmin[tidx ^  8];  if( dttmp < dtloc ){    dtloc = dttmp;  }  dtmin[tidx] = dtloc;/* w/ \pm  8 */
  dttmp = dtmin[tidx ^ 16];  if( dttmp < dtloc ){    dtloc = dttmp;  }  dtmin[tidx] = dtloc;/* w/ \pm 16 */
#endif//USE_WARP_SHUFFLE_FUNC_TIME
  //-----------------------------------------------------------------------
  /* warpSize^2 = 32^2 = 1024 is the maximum of the number of threads */
  __syncthreads();
  /* if( tidx < warpSize ){ */
  if( tidx < (NTHREADS_TIME / warpSize) ){
    //---------------------------------------------------------------------
    /* share the minimum time step in each warp */
#ifdef  USE_WARP_SHUFFLE_FUNC_TIME
    dtloc = dtmin[tidx];
#else///USE_WARP_SHUFFLE_FUNC_TIME
    /* if( tidx * warpSize < NTHREADS_TIME ) */
    dttmp = dtmin[tidx * warpSize];
#endif//USE_WARP_SHUFFLE_FUNC_TIME
    //---------------------------------------------------------------------
    /* find the minimum time step within the whole threads */
#ifdef  USE_WARP_SHUFFLE_FUNC_TIME
#   if  NTHREADS_TIME >=   64
  dttmp = __shfl_xor(dtloc,  1, NTHREADS_TIME / warpSize);  if( dttmp < dtloc )    dtloc = dttmp;
#   if  NTHREADS_TIME >=  128
  dttmp = __shfl_xor(dtloc,  2, NTHREADS_TIME / warpSize);  if( dttmp < dtloc )    dtloc = dttmp;
#   if  NTHREADS_TIME >=  256
  dttmp = __shfl_xor(dtloc,  4, NTHREADS_TIME / warpSize);  if( dttmp < dtloc )    dtloc = dttmp;
#   if  NTHREADS_TIME >=  512
  dttmp = __shfl_xor(dtloc,  8, NTHREADS_TIME / warpSize);  if( dttmp < dtloc )    dtloc = dttmp;
#   if  NTHREADS_TIME == 1024
  dttmp = __shfl_xor(dtloc, 16, NTHREADS_TIME / warpSize);  if( dttmp < dtloc )    dtloc = dttmp;
#endif//NTHREADS_TIME == 1024
#endif//NTHREADS_TIME >=  512
#endif//NTHREADS_TIME >=  256
#endif//NTHREADS_TIME >=  128
#endif//NTHREADS_TIME >=   64
#else///USE_WARP_SHUFFLE_FUNC_TIME
    dtmin[tidx] = dttmp;
#   if  NTHREADS_TIME >=   64
    dttmp = dtmin[tidx ^  1];    if( dttmp < dtloc ){      dtloc = dttmp;    }    dtmin[tidx] = dtloc;
#   if  NTHREADS_TIME >=  128
    dttmp = dtmin[tidx ^  2];    if( dttmp < dtloc ){      dtloc = dttmp;    }    dtmin[tidx] = dtloc;
#   if  NTHREADS_TIME >=  256
    dttmp = dtmin[tidx ^  4];    if( dttmp < dtloc ){      dtloc = dttmp;    }    dtmin[tidx] = dtloc;
#   if  NTHREADS_TIME >=  512
    dttmp = dtmin[tidx ^  8];    if( dttmp < dtloc ){      dtloc = dttmp;    }    dtmin[tidx] = dtloc;
#   if  NTHREADS_TIME == 1024
    dttmp = dtmin[tidx ^ 16];    if( dttmp < dtloc ){      dtloc = dttmp;    }    dtmin[tidx] = dtloc;
#endif//NTHREADS_TIME == 1024
#endif//NTHREADS_TIME >=  512
#endif//NTHREADS_TIME >=  256
#endif//NTHREADS_TIME >=  128
#endif//NTHREADS_TIME >=   64
    dtloc = dtmin[0];
#endif//USE_WARP_SHUFFLE_FUNC_TIME
    //---------------------------------------------------------------------
  }
  //-----------------------------------------------------------------------
  if( tidx == 0 )
    *dt = LDEXP(UNITY, (int)FLOOR(LOG2(eta * dtloc)));
  //-----------------------------------------------------------------------
}
//-------------------------------------------------------------------------
extern "C"
void setTimeStep_dev(const int Ni, iparticle ibody, const real eta, const real eps, real *dt_dev, double *dt_hst
#ifndef SERIALIZED_EXECUTION
		     , MPIcfg_tree mpi
#endif//SERIALIZED_EXECUTION
#ifdef  EXEC_BENCHMARK
		     , wall_clock_time *elapsed
#endif//EXEC_BENCHMARK
		     )
{
  //-----------------------------------------------------------------------
  __NOTE__("%s\n", "start");
  //-----------------------------------------------------------------------

  //-----------------------------------------------------------------------
#ifdef  EXEC_BENCHMARK
  initStopwatch();
#endif//EXEC_BENCHMARK
  //-----------------------------------------------------------------------
  setTimeStep_kernel<<<1, NTHREADS_TIME>>>(Ni, ibody.vx, ibody.vy, ibody.vz, ibody.acc, eta, eps, dt_dev);
  getLastCudaError("setTimeStep_kernel");
  //-----------------------------------------------------------------------
  real dt_tmp;
  checkCudaErrors(hipMemcpy(&dt_tmp, dt_dev, sizeof(real), hipMemcpyDeviceToHost));
  *dt_hst = (double)dt_tmp;
  //-----------------------------------------------------------------------
#ifndef SERIALIZED_EXECUTION
  chkMPIerr(MPI_Allreduce(MPI_IN_PLACE, dt_hst, 1, MPI_DOUBLE, MPI_MIN, mpi.comm));
#endif//SERIALIZED_EXECUTION
  //-----------------------------------------------------------------------
#ifdef  EXEC_BENCHMARK
  stopStopwatch(&(elapsed->setTimeStep_dev));
#endif//EXEC_BENCHMARK
  //-----------------------------------------------------------------------

  //-----------------------------------------------------------------------
  __NOTE__("%s\n", "end");
  //-----------------------------------------------------------------------
}
//-------------------------------------------------------------------------
#endif//BLOCK_TIME_STEP
//-------------------------------------------------------------------------


//-------------------------------------------------------------------------
/* time integration */
//-------------------------------------------------------------------------
#ifdef  BLOCK_TIME_STEP
//-------------------------------------------------------------------------
/* pi, vi, ai --> pj, vj */
__global__ void prediction_kernel
(const int Nj, const double tnew,
 READ_ONLY position * RESTRICT ipos, READ_ONLY velocity * RESTRICT ivel, READ_ONLY ibody_time * RESTRICT time, READ_ONLY acceleration * RESTRICT iacc,
 position * RESTRICT jpos, velocity * RESTRICT jvel)
{
  //-----------------------------------------------------------------------
  const int jj = GLOBALIDX_X1D;
  position pj = {ZERO, ZERO, ZERO, ZERO};
  velocity vj = {ZERO, ZERO, ZERO, ZERO};
  //-----------------------------------------------------------------------
  if( jj < Nj ){
    //---------------------------------------------------------------------
    /* load information of all i-particles */
    pj = ipos[jj];
    vj = ivel[jj];
    const acceleration aj = iacc[jj];
    const real dt = (real)(tnew - time[jj].t0);
    const real dt_2 = HALF * dt;
    //---------------------------------------------------------------------
    /* predict position and velocity of j-particle based on 2nd-order Runge-Kutta scheme */
    vj.x += dt_2 * aj.x;    pj.x += dt * vj.x;
    vj.y += dt_2 * aj.y;    pj.y += dt * vj.y;
    vj.z += dt_2 * aj.z;    pj.z += dt * vj.z;
    //---------------------------------------------------------------------
  }
  //-----------------------------------------------------------------------
  jpos[jj] = pj;
  jvel[jj] = vj;
  //-----------------------------------------------------------------------
}
//-------------------------------------------------------------------------
#ifdef  USE_WARP_SHUFFLE_FUNC_TIME
__device__ __forceinline__ double getMinimumDblTsub(const double  min)
#else///USE_WARP_SHUFFLE_FUNC_TIME
__device__ __forceinline__ void   getMinimumDblTsub(      double *min, volatile double * smem, const int tidx, const int head)
#endif//USE_WARP_SHUFFLE_FUNC_TIME
{
  //-----------------------------------------------------------------------
#ifdef  USE_WARP_SHUFFLE_FUNC_TIME
  //-----------------------------------------------------------------------
  union {int2 i; double d;} val, tmp;  val.d = min;
/* #   if  (TSUB / NWARP) >=  2 */
/*   tmp.i.x = __shfl_xor(val.i.x,  1, TSUB / NWARP);  tmp.i.y = __shfl_xor(val.i.y,  1, TSUB / NWARP);  if( tmp.d < val.d )    val.d = tmp.d; */
/* #   if  (TSUB / NWARP) >=  4 */
/*   tmp.i.x = __shfl_xor(val.i.x,  2, TSUB / NWARP);  tmp.i.y = __shfl_xor(val.i.y,  2, TSUB / NWARP);  if( tmp.d < val.d )    val.d = tmp.d; */
/* #   if  (TSUB / NWARP) >=  8 */
/*   tmp.i.x = __shfl_xor(val.i.x,  4, TSUB / NWARP);  tmp.i.y = __shfl_xor(val.i.y,  4, TSUB / NWARP);  if( tmp.d < val.d )    val.d = tmp.d; */
/* #   if  (TSUB / NWARP) >= 16 */
/*   tmp.i.x = __shfl_xor(val.i.x,  8, TSUB / NWARP);  tmp.i.y = __shfl_xor(val.i.y,  8, TSUB / NWARP);  if( tmp.d < val.d )    val.d = tmp.d; */
/* #   if  (TSUB / NWARP) == 32 */
/*   tmp.i.x = __shfl_xor(val.i.x, 16, TSUB / NWARP);  tmp.i.y = __shfl_xor(val.i.y, 16, TSUB / NWARP);  if( tmp.d < val.d )    val.d = tmp.d; */
/* #endif//(TSUB / NWARP) == 32 */
/* #endif//(TSUB / NWARP) >= 16 */
/* #endif//(TSUB / NWARP) >=  8 */
/* #endif//(TSUB / NWARP) >=  4 */
/* #endif//(TSUB / NWARP) >=  2 */
/*   tmp.i.x = __shfl(val.i.x, 0, TSUB / NWARP); */
/*   tmp.i.y = __shfl(val.i.y, 0, TSUB / NWARP); */
#   if  DIV_NWARP(TSUB) >=  2
  tmp.i.x = __shfl_xor(val.i.x,  1, DIV_NWARP(TSUB));  tmp.i.y = __shfl_xor(val.i.y,  1, DIV_NWARP(TSUB));  if( tmp.d < val.d )    val.d = tmp.d;
#   if  DIV_NWARP(TSUB) >=  4
  tmp.i.x = __shfl_xor(val.i.x,  2, DIV_NWARP(TSUB));  tmp.i.y = __shfl_xor(val.i.y,  2, DIV_NWARP(TSUB));  if( tmp.d < val.d )    val.d = tmp.d;
#   if  DIV_NWARP(TSUB) >=  8
  tmp.i.x = __shfl_xor(val.i.x,  4, DIV_NWARP(TSUB));  tmp.i.y = __shfl_xor(val.i.y,  4, DIV_NWARP(TSUB));  if( tmp.d < val.d )    val.d = tmp.d;
#   if  DIV_NWARP(TSUB) >= 16
  tmp.i.x = __shfl_xor(val.i.x,  8, DIV_NWARP(TSUB));  tmp.i.y = __shfl_xor(val.i.y,  8, DIV_NWARP(TSUB));  if( tmp.d < val.d )    val.d = tmp.d;
#   if  DIV_NWARP(TSUB) == 32
  tmp.i.x = __shfl_xor(val.i.x, 16, DIV_NWARP(TSUB));  tmp.i.y = __shfl_xor(val.i.y, 16, DIV_NWARP(TSUB));  if( tmp.d < val.d )    val.d = tmp.d;
#endif//DIV_NWARP(TSUB) == 32
#endif//DIV_NWARP(TSUB) >= 16
#endif//DIV_NWARP(TSUB) >=  8
#endif//DIV_NWARP(TSUB) >=  4
#endif//DIV_NWARP(TSUB) >=  2
  tmp.i.x = __shfl(val.i.x, 0, DIV_NWARP(TSUB));
  tmp.i.y = __shfl(val.i.y, 0, DIV_NWARP(TSUB));
  return (tmp.d);
  //-----------------------------------------------------------------------
#else///USE_WARP_SHUFFLE_FUNC_TIME
  //-----------------------------------------------------------------------
  smem[tidx] = *min;
  //-----------------------------------------------------------------------
/* #   if  (TSUB / NWARP) >=  2 */
/*   double tmp; */
/*   tmp = smem[tidx ^  1];  if( tmp < *min ){    *min = tmp;  }  smem[tidx] = *min; */
/* #   if  (TSUB / NWARP) >=  4 */
/*   tmp = smem[tidx ^  2];  if( tmp < *min ){    *min = tmp;  }  smem[tidx] = *min; */
/* #   if  (TSUB / NWARP) >=  8 */
/*   tmp = smem[tidx ^  4];  if( tmp < *min ){    *min = tmp;  }  smem[tidx] = *min; */
/* #   if  (TSUB / NWARP) >= 16 */
/*   tmp = smem[tidx ^  8];  if( tmp < *min ){    *min = tmp;  }  smem[tidx] = *min; */
/* #   if  (TSUB / NWARP) >= 32 */
/*   tmp = smem[tidx ^ 16];  if( tmp < *min ){    *min = tmp;  }  smem[tidx] = *min; */
/* #endif//(TSUB / NWARP) >= 32 */
/* #endif//(TSUB / NWARP) >= 16 */
/* #endif//(TSUB / NWARP) >=  8 */
/* #endif//(TSUB / NWARP) >=  4 */
/* #endif//(TSUB / NWARP) >=  2 */
#   if  DIV_NWARP(TSUB) >=  2
  double tmp;
  tmp = smem[tidx ^  1];  if( tmp < *min ){    *min = tmp;  }  smem[tidx] = *min;
#   if  DIV_NWARP(TSUB) >=  4
  tmp = smem[tidx ^  2];  if( tmp < *min ){    *min = tmp;  }  smem[tidx] = *min;
#   if  DIV_NWARP(TSUB) >=  8
  tmp = smem[tidx ^  4];  if( tmp < *min ){    *min = tmp;  }  smem[tidx] = *min;
#   if  DIV_NWARP(TSUB) >= 16
  tmp = smem[tidx ^  8];  if( tmp < *min ){    *min = tmp;  }  smem[tidx] = *min;
#   if  DIV_NWARP(TSUB) >= 32
  tmp = smem[tidx ^ 16];  if( tmp < *min ){    *min = tmp;  }  smem[tidx] = *min;
#endif//DIV_NWARP(TSUB) >= 32
#endif//DIV_NWARP(TSUB) >= 16
#endif//DIV_NWARP(TSUB) >=  8
#endif//DIV_NWARP(TSUB) >=  4
#endif//DIV_NWARP(TSUB) >=  2
  //-----------------------------------------------------------------------
  *min = smem[head];
  //-----------------------------------------------------------------------
#endif//USE_WARP_SHUFFLE_FUNC_TIME
  //-----------------------------------------------------------------------
}
//-------------------------------------------------------------------------
__device__ __forceinline__ real setParticleTime(const velocity vi, const acceleration ai, const real eps, const real eta)
{
  //-----------------------------------------------------------------------
  /* estimate the required time step to resolve eps */
  const real v2 = EPSILON + vi.x * vi.x + vi.y * vi.y + vi.z * vi.z;  const real vdt =      eps * RSQRT(v2);
  const real a2 = EPSILON + ai.x * ai.x + ai.y * ai.y + ai.z * ai.z;  const real adt = SQRT(eps * RSQRT(a2));
  //-----------------------------------------------------------------------
  /* set new time step */
  return (LDEXP(UNITY, (int)FLOOR(LOG2(eta * ((vdt < adt) ? (vdt) : (adt))))));
  //-----------------------------------------------------------------------
}
//-------------------------------------------------------------------------
/* pj, vj, ai --> pi, vi */
__global__ void correction_kernel
(const int laneNum, READ_ONLY laneinfo * RESTRICT laneInfo, double * RESTRICT laneTime, const real eps, const real eta,
 position * RESTRICT ipos, velocity * RESTRICT ivel, ibody_time * RESTRICT time, READ_ONLY acceleration * RESTRICT iacc,
 READ_ONLY position * RESTRICT jpos, READ_ONLY velocity * RESTRICT jvel,
 const int reuseTree)
{
  //-----------------------------------------------------------------------
  const int tidx = THREADIDX_X1D;
#if 0
  const int lane = tidx & (TSUB - 1);/* index of the thread within a thread group */
  /* const int laneIdx = GLOBALIDX_X1D / TSUB; */
  const int laneIdx = DIV_TSUB(GLOBALIDX_X1D);
#else
  /* const int lane    = tidx          & ((TSUB / NWARP) - 1); */
  /* const int laneIdx = GLOBALIDX_X1D /  (TSUB / NWARP); */
  const int lane    = tidx          & (DIV_NWARP(TSUB) - 1);
  const int laneIdx = GLOBALIDX_X1D /  DIV_NWARP(TSUB);
#endif
  //-----------------------------------------------------------------------
#ifndef USE_WARP_SHUFFLE_FUNC_TIME
  const int head = tidx - lane;
  __shared__ double smem[NTHREADS];
#endif//USE_WARP_SHUFFLE_FUNC_TIME
  //-----------------------------------------------------------------------
#if 0
  const laneinfo info = laneInfo[laneIdx];
#else
  laneinfo info = {NUM_BODY_MAX, 0};
  if( laneIdx < laneNum )
    info = laneInfo[laneIdx];
#endif
  //-----------------------------------------------------------------------
  ibody_time ti = {0.0, DBL_MAX};
  //-----------------------------------------------------------------------
  if( lane < info.num ){
    //---------------------------------------------------------------------
    const int idx = info.head + lane;
    //---------------------------------------------------------------------
    /* load pj, vj, ti, and ai */
    const acceleration ai = iacc[idx];
    velocity vi = jvel[idx];
    ti = time[idx];
    /* store pi */
    ipos[idx] = jpos[idx];
    //---------------------------------------------------------------------
    /* update vi */
    vi.dt *= HALF;
    vi.x += vi.dt * ai.x;
    vi.y += vi.dt * ai.y;
    vi.z += vi.dt * ai.z;
    //---------------------------------------------------------------------
    /* set new time step */
    vi.dt = setParticleTime(vi, ai, eps, eta);
    /* store vi */
    ivel[idx] = vi;
    //---------------------------------------------------------------------
    /* store ti */
    ti.t0 = ti.t1;
    ti.t1 += (double)vi.dt;
    time[idx] = ti;
    //---------------------------------------------------------------------
  }
  //-----------------------------------------------------------------------
  /* get minimum ti.t1 of this group (TSUB threads) */
  if( reuseTree ){
#ifdef  USE_WARP_SHUFFLE_FUNC_TIME
    double tmin = getMinimumDblTsub(ti.t1);
#else///USE_WARP_SHUFFLE_FUNC_TIME
    double tmin = ti.t1;
    getMinimumDblTsub(&tmin, smem, tidx, head);
#endif//USE_WARP_SHUFFLE_FUNC_TIME
    if( lane == 0 )
      laneTime[laneIdx] = tmin;
  }
  //-----------------------------------------------------------------------
}
//-------------------------------------------------------------------------
__global__ void adjustParticleTime_kernel
(const int laneNum, READ_ONLY laneinfo * RESTRICT laneInfo, double * RESTRICT laneTime, const real eps, const real eta,
 velocity * RESTRICT ivel, ibody_time * RESTRICT time, READ_ONLY acceleration * RESTRICT iacc)
{
  //-----------------------------------------------------------------------
  const int tidx = THREADIDX_X1D;
#if 0
  const int lane = tidx & (TSUB - 1);/* index of the thread within a thread group */
  /* const int laneIdx = GLOBALIDX_X1D / TSUB; */
  const int laneIdx = DIV_TSUB(GLOBALIDX_X1D);
#else
  /* const int lane    = tidx          & ((TSUB / NWARP) - 1); */
  /* const int laneIdx = GLOBALIDX_X1D /  (TSUB / NWARP); */
  const int lane    = tidx          & (DIV_NWARP(TSUB) - 1);
  const int laneIdx = GLOBALIDX_X1D /  DIV_NWARP(TSUB);
#endif
  //-----------------------------------------------------------------------
#ifndef USE_WARP_SHUFFLE_FUNC_TIME
  const int head = tidx - lane;
  __shared__ double smem[NTHREADS];
#endif//USE_WARP_SHUFFLE_FUNC_TIME
  //-----------------------------------------------------------------------
#if 0
  const laneinfo info = laneInfo[laneIdx];
#else
  laneinfo info = {NUM_BODY_MAX, 0};
  if( laneIdx < laneNum )
    info = laneInfo[laneIdx];
#endif
  //-----------------------------------------------------------------------
  /* ibody_time ti = {ZERO, REAL_MAX}; */
  ibody_time ti = {0.0, DBL_MAX};
  //-----------------------------------------------------------------------
  if( lane < info.num ){
    //---------------------------------------------------------------------
    const int idx = info.head + lane;
    //---------------------------------------------------------------------
    /* load pj, vj, ti, and ai */
    const acceleration ai = iacc[idx];
    velocity vi = ivel[idx];
    ti = time[idx];
    //---------------------------------------------------------------------
    /* set new time step */
    vi.dt = setParticleTime(vi, ai, eps, eta);
    /* store vi */
    ivel[idx] = vi;
    //---------------------------------------------------------------------
    /* store ti */
    ti.t1 = ti.t0 + (double)vi.dt;
    time[idx] = ti;
    //---------------------------------------------------------------------
  }
  //-----------------------------------------------------------------------
  /* get minimum ti.t1 of this group (TSUB threads) */
#ifdef  USE_WARP_SHUFFLE_FUNC_TIME
  double tmin = getMinimumDblTsub(ti.t1);
#else//USE_WARP_SHUFFLE_FUNC_TIME
  double tmin = ti.t1;
  getMinimumDblTsub(&tmin, smem, tidx, head);
#endif//USE_WARP_SHUFFLE_FUNC_TIME
  if( lane == 0 )
    laneTime[laneIdx] = tmin;
  //-----------------------------------------------------------------------
}
//-------------------------------------------------------------------------
__global__ void setLaneTime_kernel(const int laneNum, READ_ONLY laneinfo * RESTRICT laneInfo, double * RESTRICT laneTime, READ_ONLY ibody_time * RESTRICT time)
{
  //-----------------------------------------------------------------------
  const int tidx = THREADIDX_X1D;
#if 0
  const int lane = tidx & (TSUB - 1);/* index of the thread within a thread group */
  /* const int laneIdx = GLOBALIDX_X1D / TSUB; */
  const int laneIdx = DIV_TSUB(GLOBALIDX_X1D);
#else
  /* const int lane    = tidx          & ((TSUB / NWARP) - 1); */
  /* const int laneIdx = GLOBALIDX_X1D /  (TSUB / NWARP); */
  const int lane    = tidx          & (DIV_NWARP(TSUB) - 1);
  const int laneIdx = GLOBALIDX_X1D /  DIV_NWARP(TSUB);
#endif
  //-----------------------------------------------------------------------
#ifndef USE_WARP_SHUFFLE_FUNC_TIME
  const int head = tidx - lane;
  __shared__ double smem[NTHREADS];
#endif//USE_WARP_SHUFFLE_FUNC_TIME
  //-----------------------------------------------------------------------
#if 0
  const laneinfo info = laneInfo[laneIdx];
#else
  laneinfo info = {NUM_BODY_MAX, 0};
  if( laneIdx < laneNum )
    info = laneInfo[laneIdx];
#endif
  //-----------------------------------------------------------------------
  /* ibody_time ti = {ZERO, REAL_MAX}; */
  ibody_time ti = {0.0, DBL_MAX};
  //-----------------------------------------------------------------------
  if( lane < info.num )
    ti = time[info.head + lane];
  //-----------------------------------------------------------------------
  /* get minimum ti.t1 of this group (TSUB threads) */
#ifdef  USE_WARP_SHUFFLE_FUNC_TIME
  double tmin = getMinimumDblTsub(ti.t1);
#else///USE_WARP_SHUFFLE_FUNC_TIME
  double tmin = ti.t1;
  getMinimumDblTsub(&tmin, smem, tidx, head);
#endif//USE_WARP_SHUFFLE_FUNC_TIME
  if( lane == 0 )
    laneTime[laneIdx] = tmin;
  //-----------------------------------------------------------------------
}
//-------------------------------------------------------------------------
extern "C"
void prediction_dev(const int Nj, const double tnew, const iparticle pi
#ifndef CALC_MULTIPOLE_ON_DEVICE
		    , const iparticle pi_hst
#endif//CALC_MULTIPOLE_ON_DEVICE
#ifdef  EXEC_BENCHMARK
		    , wall_clock_time *elapsed
#endif//EXEC_BENCHMARK
		    )
{
  //-----------------------------------------------------------------------
  __NOTE__("%s\n", "start");
  //-----------------------------------------------------------------------

  //-----------------------------------------------------------------------
#ifdef  EXEC_BENCHMARK
  initStopwatch();
#endif//EXEC_BENCHMARK
  //-----------------------------------------------------------------------

  //-----------------------------------------------------------------------
  int Nrem = BLOCKSIZE(Nj, NTHREADS_TIME);
  //-----------------------------------------------------------------------
  if( Nrem <= MAX_BLOCKS_PER_GRID )
    prediction_kernel<<<Nrem, NTHREADS_TIME>>>(Nj, tnew, pi.pos, pi.vel, pi.time, pi.acc, pi.jpos, pi.jvel);
  //-----------------------------------------------------------------------
  else{
    //---------------------------------------------------------------------
    const int Niter = BLOCKSIZE(Nrem, MAX_BLOCKS_PER_GRID);
    int hidx = 0;
    //---------------------------------------------------------------------
    for(int iter = 0; iter < Niter; iter++){
      //-------------------------------------------------------------------
      int Nblck = MAX_BLOCKS_PER_GRID;
      if( Nblck > Nrem )	Nblck = Nrem;
      //-------------------------------------------------------------------
      int Nsub = Nblck * NTHREADS_TIME;
      prediction_kernel<<<Nblck, NTHREADS_TIME>>>(Nsub, tnew, &pi.pos[hidx], &pi.vel[hidx], &pi.time[hidx], &pi.acc[hidx], &pi.jpos[hidx], &pi.jvel[hidx]);
      //-------------------------------------------------------------------
      hidx += Nsub;
      Nrem -= Nblck;
      //-------------------------------------------------------------------
    }/* for(int iter = 0; iter < Niter; iter++){ */
    //---------------------------------------------------------------------
  }/* else{ */
  //-----------------------------------------------------------------------
  getLastCudaError("prediction_kernel");
  //-----------------------------------------------------------------------
  /* prediction_kernel<<<BLOCKSIZE(Nj, NTHREADS_TIME), NTHREADS_TIME>>>(Nj, tnew, pi.pos, pi.vel, pi.time, pi.acc, pi.jpos, pi.jvel); */
  /* getLastCudaError("prediction_kernel"); */
  //-----------------------------------------------------------------------

  //-----------------------------------------------------------------------
#ifndef CALC_MULTIPOLE_ON_DEVICE
  checkCudaErrors(hipMemcpy(pi_hst.jpos, pi.jpos, sizeof(position) * Nj, hipMemcpyDeviceToHost));
#endif//CALC_MULTIPOLE_ON_DEVICE
  //-----------------------------------------------------------------------
#ifdef  EXEC_BENCHMARK
  stopStopwatch(&(elapsed->prediction_dev));
#endif//EXEC_BENCHMARK
  //-----------------------------------------------------------------------

  //-----------------------------------------------------------------------
  __NOTE__("%s\n", "end");
  //-----------------------------------------------------------------------
}
//-------------------------------------------------------------------------
extern "C"
void correction_dev(const int Ngrp, laneinfo * RESTRICT laneInfo, double * RESTRICT laneTime, const real eps, const real eta, const iparticle pi, const int reuseTree
#ifdef  EXEC_BENCHMARK
		    , wall_clock_time *elapsed
#endif//EXEC_BENCHMARK
		    )
{
  //-----------------------------------------------------------------------
  __NOTE__("%s\n", "start");
  //-----------------------------------------------------------------------

  //-----------------------------------------------------------------------
#ifdef  EXEC_BENCHMARK
  initStopwatch();
#endif//EXEC_BENCHMARK
  //-----------------------------------------------------------------------
  /* thread-block structure must be identical to tree traversal */
#ifndef SERIALIZED_EXECUTION
  if( Ngrp != 0 )
#endif//SERIALIZED_EXECUTION
    {
      //-------------------------------------------------------------------
      int Nrem = BLOCKSIZE(Ngrp, NWARP * NGROUPS);
      //-------------------------------------------------------------------
      if( Nrem <= MAX_BLOCKS_PER_GRID )
	correction_kernel<<<Nrem, NTHREADS>>>
	  (BLOCKSIZE(Ngrp, NGROUPS) * NGROUPS, laneInfo, laneTime, eps, eta, pi.pos, pi.vel, pi.time, pi.acc, pi.jpos, pi.jvel, reuseTree);
      //-------------------------------------------------------------------
      else{
	//-----------------------------------------------------------------
	const int Niter = BLOCKSIZE(Nrem, MAX_BLOCKS_PER_GRID);
	int hidx = 0;
	//-----------------------------------------------------------------
	for(int iter = 0; iter < Niter; iter++){
	  //---------------------------------------------------------------
	  int Nblck = MAX_BLOCKS_PER_GRID;
	  if( Nblck > Nrem )	    Nblck = Nrem;
	  //---------------------------------------------------------------
	  int Nsub = Nblck * NWARP * NGROUPS;
	  correction_kernel<<<Nblck, NTHREADS>>>
	    (BLOCKSIZE(Nsub, NGROUPS) * NGROUPS, &laneInfo[hidx], &laneTime[hidx], eps, eta, pi.pos, pi.vel, pi.time, pi.acc, pi.jpos, pi.jvel, reuseTree);
	  //-------------------------------------------------------------------
	  hidx += Nsub;
	  Nrem -= Nblck;
	//-----------------------------------------------------------------
	}/* for(int iter = 0; iter < Niter; iter++){ */
	//-----------------------------------------------------------------
      }/* else{ */
      //-------------------------------------------------------------------
      getLastCudaError("correction_kernel");
      //-------------------------------------------------------------------
      /* /\* correction_kernel<<<BLOCKSIZE(Ngrp,         NGROUPS), NTHREADS>>> *\/ */
      /* /\* 	(                                    laneInfo, laneTime, eps, eta, pi.pos, pi.vel, pi.time, pi.acc, pi.jpos, pi.jvel, reuseTree); *\/ */
      /* correction_kernel<<<BLOCKSIZE(Ngrp, NWARP * NGROUPS), NTHREADS>>> */
      /* 	(BLOCKSIZE(Ngrp, NGROUPS) * NGROUPS, laneInfo, laneTime, eps, eta, pi.pos, pi.vel, pi.time, pi.acc, pi.jpos, pi.jvel, reuseTree); */
      /* getLastCudaError("correction_kernel"); */
      //-------------------------------------------------------------------
    }
  //-----------------------------------------------------------------------
#ifdef  EXEC_BENCHMARK
  stopStopwatch(&(elapsed->correction_dev));
#endif//EXEC_BENCHMARK
  //-----------------------------------------------------------------------

  //-----------------------------------------------------------------------
  __NOTE__("%s\n", "end");
  //-----------------------------------------------------------------------
}
//-------------------------------------------------------------------------
extern "C"
void adjustParticleTime_dev(const int Ngrp, laneinfo * RESTRICT laneInfo, double * RESTRICT laneTime, const real eps, const real eta, const iparticle pi
#ifdef  EXEC_BENCHMARK
			    , wall_clock_time *elapsed
#endif//EXEC_BENCHMARK
			    )
{
  //-----------------------------------------------------------------------
  __NOTE__("%s\n", "start");
  //-----------------------------------------------------------------------
#ifdef  EXEC_BENCHMARK
  initStopwatch();
#endif//EXEC_BENCHMARK
  //-----------------------------------------------------------------------

  //-----------------------------------------------------------------------
  int Nrem = BLOCKSIZE(Ngrp, NWARP * NGROUPS);
  //-----------------------------------------------------------------------
  if( Nrem <= MAX_BLOCKS_PER_GRID )
    adjustParticleTime_kernel<<<Nrem, NTHREADS>>>
      (BLOCKSIZE(Ngrp, NGROUPS) * NGROUPS, laneInfo, laneTime, eps, eta, pi.vel, pi.time, pi.acc);
  //-----------------------------------------------------------------------
  else{
    //---------------------------------------------------------------------
    const int Niter = BLOCKSIZE(Nrem, MAX_BLOCKS_PER_GRID);
    int hidx = 0;
    //---------------------------------------------------------------------
    for(int iter = 0; iter < Niter; iter++){
      //-------------------------------------------------------------------
      int Nblck = MAX_BLOCKS_PER_GRID;
      if( Nblck > Nrem )	Nblck = Nrem;
      //-------------------------------------------------------------------
      int Nsub = Nblck * NWARP * NGROUPS;
      adjustParticleTime_kernel<<<Nblck, NTHREADS>>>
	(BLOCKSIZE(Nsub, NGROUPS) * NGROUPS, &laneInfo[hidx], &laneTime[hidx], eps, eta, pi.vel, pi.time, pi.acc);
      //-------------------------------------------------------------------
      hidx += Nsub;
      Nrem -= Nblck;
      //-------------------------------------------------------------------
    }/* for(int iter = 0; iter < Niter; iter++){ */
    //---------------------------------------------------------------------
  }/* else{ */
  //-----------------------------------------------------------------------
  getLastCudaError("adjustParticleTime_kernel");
  //-----------------------------------------------------------------------
  /* /\* adjustParticleTime_kernel<<<BLOCKSIZE(Ngrp,         NGROUPS), NTHREADS>>> *\/ */
  /* /\*   (                                    laneInfo, laneTime, eps, eta, pi.vel, pi.time, pi.acc); *\/ */
  /* adjustParticleTime_kernel<<<BLOCKSIZE(Ngrp, NWARP * NGROUPS), NTHREADS>>> */
  /*   (BLOCKSIZE(Ngrp, NGROUPS) * NGROUPS, laneInfo, laneTime, eps, eta, pi.vel, pi.time, pi.acc); */
  /* getLastCudaError("adjustParticleTime_kernel"); */
  //-----------------------------------------------------------------------

  //-----------------------------------------------------------------------
#ifdef  EXEC_BENCHMARK
  stopStopwatch(&(elapsed->adjustParticleTime_dev));
#endif//EXEC_BENCHMARK
  //-----------------------------------------------------------------------
  __NOTE__("%s\n", "end");
  //-----------------------------------------------------------------------
}
//-------------------------------------------------------------------------
extern "C"
void setLaneTime_dev(const int Ngrp, laneinfo * RESTRICT laneInfo, double * RESTRICT laneTime, const iparticle pi
#ifdef  EXEC_BENCHMARK
		     , wall_clock_time *elapsed
#endif//EXEC_BENCHMARK
		     )
{
  //-----------------------------------------------------------------------
  __NOTE__("%s\n", "start");
  //-----------------------------------------------------------------------
#ifdef  EXEC_BENCHMARK
  initStopwatch();
#endif//EXEC_BENCHMARK
  //-----------------------------------------------------------------------

  //-----------------------------------------------------------------------
  int Nrem = BLOCKSIZE(Ngrp, NWARP * NGROUPS);
  //-----------------------------------------------------------------------
  if( Nrem <= MAX_BLOCKS_PER_GRID )
    setLaneTime_kernel<<<Nrem, NTHREADS>>>(BLOCKSIZE(Ngrp, NGROUPS) * NGROUPS, laneInfo, laneTime, pi.time);
  //-----------------------------------------------------------------------
  else{
    //---------------------------------------------------------------------
    const int Niter = BLOCKSIZE(Nrem, MAX_BLOCKS_PER_GRID);
    int hidx = 0;
    //---------------------------------------------------------------------
    for(int iter = 0; iter < Niter; iter++){
      //-------------------------------------------------------------------
      int Nblck = MAX_BLOCKS_PER_GRID;
      if( Nblck > Nrem )	Nblck = Nrem;
      //-------------------------------------------------------------------
      int Nsub = Nblck * NWARP * NGROUPS;
      setLaneTime_kernel<<<Nblck, NTHREADS>>>(BLOCKSIZE(Nsub, NGROUPS) * NGROUPS, &laneInfo[hidx], &laneTime[hidx], pi.time);
      //-------------------------------------------------------------------
      hidx += Nsub;
      Nrem -= Nblck;
      //-------------------------------------------------------------------
    }/* for(int iter = 0; iter < Niter; iter++){ */
    //---------------------------------------------------------------------
  }/* else{ */
  //-----------------------------------------------------------------------
  getLastCudaError("setLaneTime_kernel");
  //-----------------------------------------------------------------------
  /* /\* setLaneTime_kernel<<<BLOCKSIZE(Ngrp,         NGROUPS), NTHREADS>>>(                                    laneInfo, laneTime, pi.time); *\/ */
  /* setLaneTime_kernel<<<BLOCKSIZE(Ngrp, NWARP * NGROUPS), NTHREADS>>>(BLOCKSIZE(Ngrp, NGROUPS) * NGROUPS, laneInfo, laneTime, pi.time); */
  /* getLastCudaError("setLaneTime_kernel"); */
  //-----------------------------------------------------------------------

  //-----------------------------------------------------------------------
#ifdef  EXEC_BENCHMARK
  stopStopwatch(&(elapsed->setLaneTime_dev));
#else///EXEC_BENCHMARK
  hipDeviceSynchronize();
#endif//EXEC_BENCHMARK
  //-----------------------------------------------------------------------
  __NOTE__("%s\n", "end");
  //-----------------------------------------------------------------------
}
//-------------------------------------------------------------------------
#else///BLOCK_TIME_STEP
//-------------------------------------------------------------------------
__global__ void advPos_kernel
(const int Ni, position * RESTRICT ipos, READ_ONLY real * RESTRICT vx, READ_ONLY real * RESTRICT vy, READ_ONLY real * RESTRICT vz, const real dt)
{
  //-----------------------------------------------------------------------
  const int ii = GLOBALIDX_X1D;
  position pi = {ZERO, ZERO, ZERO, ZERO};
  if( ii < Ni ){
    //---------------------------------------------------------------------
    /* load an i-particle */
    pi = ipos[ii];
    //---------------------------------------------------------------------
    pi.x += dt * vx[ii];
    pi.y += dt * vy[ii];
    pi.z += dt * vz[ii];
    //---------------------------------------------------------------------
  }
  //-----------------------------------------------------------------------
  ipos[ii] = pi;
  //-----------------------------------------------------------------------
}
//-------------------------------------------------------------------------
__global__ void advVel_kernel
(const int Ni, READ_ONLY acceleration * RESTRICT iacc, real * RESTRICT vx, real * RESTRICT vy, real * RESTRICT vz, const real dt)
{
  //-----------------------------------------------------------------------
  const int ii = GLOBALIDX_X1D;
  if( ii < Ni ){
    //---------------------------------------------------------------------
    /* load acceleration of i-particle */
    acceleration ai = iacc[ii];
    //---------------------------------------------------------------------
    /* update velocity */
    vx[ii] += dt * ai.x;
    vy[ii] += dt * ai.y;
    vz[ii] += dt * ai.z;
    //---------------------------------------------------------------------
  }
  //-----------------------------------------------------------------------
}
//-------------------------------------------------------------------------
extern "C"
void advPos_dev(const int Ni, iparticle ibody, const real dt
#ifdef  EXEC_BENCHMARK
		, wall_clock_time *elapsed
#endif//EXEC_BENCHMARK
		)
{
  //-----------------------------------------------------------------------
  __NOTE__("%s\n", "start");
  //-----------------------------------------------------------------------

  //-----------------------------------------------------------------------
#ifdef  EXEC_BENCHMARK
  initStopwatch();
#endif//EXEC_BENCHMARK
  //-----------------------------------------------------------------------

  //-----------------------------------------------------------------------
  int Nrem = BLOCKSIZE(Ni, NTHREADS_TIME);
  //-----------------------------------------------------------------------
  if( Nrem <= MAX_BLOCKS_PER_GRID )
    advPos_kernel<<<Nrem, NTHREADS_TIME>>>(Ni, ibody.pos, ibody.vx, ibody.vy, ibody.vz, dt);
  //-----------------------------------------------------------------------
  else{
    //---------------------------------------------------------------------
    const int Niter = BLOCKSIZE(Nrem, MAX_BLOCKS_PER_GRID);
    int hidx = 0;
    //---------------------------------------------------------------------
    for(int iter = 0; iter < Niter; iter++){
      //-------------------------------------------------------------------
      int Nblck = MAX_BLOCKS_PER_GRID;
      if( Nblck > Nrem )	Nblck = Nrem;
      //-------------------------------------------------------------------
      int Nsub = Nblck * NTHREADS_TIME;
      advPos_kernel<<<Nblck, NTHREADS_TIME>>>(Nsub, &ibody.pos[hidx], &ibody.vx[hidx], &ibody.vy[hidx], &ibody.vz[hidx], dt);
      //-------------------------------------------------------------------
      hidx += Nsub;
      Nrem -= Nblck;
      //-------------------------------------------------------------------
    }/* for(int iter = 0; iter < Niter; iter++){ */
    //---------------------------------------------------------------------
  }/* else{ */
  //-----------------------------------------------------------------------
  getLastCudaError("advPos_kernel");
  //-----------------------------------------------------------------------
  /* advPos_kernel<<<BLOCKSIZE(Ni, NTHREADS_TIME), NTHREADS_TIME>>>(Ni, ibody.pos, ibody.vx, ibody.vy, ibody.vz, dt); */
  /* getLastCudaError("advPos_kernel"); */
  //-----------------------------------------------------------------------

  //-----------------------------------------------------------------------
#ifdef  EXEC_BENCHMARK
  stopStopwatch(&(elapsed->advPos_dev));
#endif//EXEC_BENCHMARK
  //-----------------------------------------------------------------------

  //-----------------------------------------------------------------------
  __NOTE__("%s\n", "end");
  //-----------------------------------------------------------------------
}
//-------------------------------------------------------------------------
extern "C"
void advVel_dev(const int Ni, iparticle ibody, const real dt
#ifdef  EXEC_BENCHMARK
		, wall_clock_time *elapsed
#endif//EXEC_BENCHMARK
		)
{
  //-----------------------------------------------------------------------
  __NOTE__("%s\n", "start");
  //-----------------------------------------------------------------------

  //-----------------------------------------------------------------------
#ifdef  EXEC_BENCHMARK
  initStopwatch();
#endif//EXEC_BENCHMARK
  //-----------------------------------------------------------------------

  //-----------------------------------------------------------------------
  int Nrem = BLOCKSIZE(Ni, NTHREADS_TIME);
  //-----------------------------------------------------------------------
  if( Nrem <= MAX_BLOCKS_PER_GRID )
    advVel_kernel<<<Nrem, NTHREADS_TIME>>>(Ni, ibody.acc, ibody.vx, ibody.vy, ibody.vz, dt);
  //-----------------------------------------------------------------------
  else{
    //---------------------------------------------------------------------
    const int Niter = BLOCKSIZE(Nrem, MAX_BLOCKS_PER_GRID);
    int hidx = 0;
    //---------------------------------------------------------------------
    for(int iter = 0; iter < Niter; iter++){
      //-------------------------------------------------------------------
      int Nblck = MAX_BLOCKS_PER_GRID;
      if( Nblck > Nrem )	Nblck = Nrem;
      //-------------------------------------------------------------------
      int Nsub = Nblck * NTHREADS_TIME;
      advVel_kernel<<<Nblck, NTHREADS_TIME>>>(Nsub, &ibody.acc[hidx], &ibody.vx[hidx], &ibody.vy[hidx], &ibody.vz[hidx], dt);
      //-------------------------------------------------------------------
      hidx += Nsub;
      Nrem -= Nblck;
      //-------------------------------------------------------------------
    }/* for(int iter = 0; iter < Niter; iter++){ */
    //---------------------------------------------------------------------
  }/* else{ */
  //-----------------------------------------------------------------------
  getLastCudaError("advVel_kernel");
  //-----------------------------------------------------------------------
  /* advVel_kernel<<<BLOCKSIZE(Ni, NTHREADS_TIME), NTHREADS_TIME>>>(Ni, ibody.acc, ibody.vx, ibody.vy, ibody.vz, dt); */
  /* getLastCudaError("advVel_kernel"); */
  //-----------------------------------------------------------------------

  //-----------------------------------------------------------------------
#ifdef  EXEC_BENCHMARK
  stopStopwatch(&(elapsed->advVel_dev));
#endif//EXEC_BENCHMARK
  //-----------------------------------------------------------------------

  //-----------------------------------------------------------------------
  __NOTE__("%s\n", "end");
  //-----------------------------------------------------------------------
}
//-------------------------------------------------------------------------
#endif//BLOCK_TIME_STEP
//-------------------------------------------------------------------------


//-------------------------------------------------------------------------
extern "C"
void copyParticle_hst2dev(const int Ni, iparticle hst, iparticle dev
#ifdef  EXEC_BENCHMARK
			  , wall_clock_time *elapsed
#endif//EXEC_BENCHMARK
			  )
{
  //-----------------------------------------------------------------------
  __NOTE__("%s\n", "start");
  //-----------------------------------------------------------------------

  //-----------------------------------------------------------------------
#ifdef  EXEC_BENCHMARK
  initStopwatch();
#endif//EXEC_BENCHMARK
  //-----------------------------------------------------------------------
  /* send i-particles from the host to the device using the default CUDA stream */
#ifdef  GENERATE_PHKEY_ON_DEVICE
  checkCudaErrors(hipMemcpy(dev. idx, hst. idx, sizeof(       ulong) * Ni, hipMemcpyHostToDevice));
#endif//GENERATE_PHKEY_ON_DEVICE
  checkCudaErrors(hipMemcpy(dev. pos, hst. pos, sizeof(    position) * Ni, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(dev. acc, hst. acc, sizeof(acceleration) * Ni, hipMemcpyHostToDevice));
#ifdef  BLOCK_TIME_STEP
  checkCudaErrors(hipMemcpy(dev. vel, hst. vel, sizeof(    velocity) * Ni, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(dev.time, hst.time, sizeof(  ibody_time) * Ni, hipMemcpyHostToDevice));
#else///BLOCK_TIME_STEP
  checkCudaErrors(hipMemcpy(dev.  vx, hst.  vx, sizeof(        real) * Ni, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(dev.  vy, hst.  vy, sizeof(        real) * Ni, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(dev.  vz, hst.  vz, sizeof(        real) * Ni, hipMemcpyHostToDevice));
#endif//BLOCK_TIME_STEP
  //-----------------------------------------------------------------------
#ifndef EXEC_BENCHMARK
#       ifndef GENERATE_PHKEY_ON_DEVICE
  checkCudaErrors(hipDeviceSynchronize());
#       endif//GENERATE_PHKEY_ON_DEVICE
#else///EXEC_BENCHMARK
  stopStopwatch(&(elapsed->copyParticle_hst2dev));
#endif//EXEC_BENCHMARK
  //-----------------------------------------------------------------------

  //-----------------------------------------------------------------------
  __NOTE__("%s\n", "end");
  //-----------------------------------------------------------------------
}
//-------------------------------------------------------------------------
extern "C"
void copyParticle_dev2hst(const int Ni, iparticle dev, iparticle hst
#ifdef  EXEC_BENCHMARK
			  , wall_clock_time *elapsed
#endif//EXEC_BENCHMARK
			  )
{
  //-----------------------------------------------------------------------
  __NOTE__("%s\n", "start");
  //-----------------------------------------------------------------------

  //-----------------------------------------------------------------------
#ifdef  EXEC_BENCHMARK
  initStopwatch();
#endif//EXEC_BENCHMARK
  //-----------------------------------------------------------------------
  /* send i-particles from the device to the host using the default CUDA stream */
#ifdef  GENERATE_PHKEY_ON_DEVICE
  checkCudaErrors(hipMemcpy(hst. idx, dev. idx, sizeof(       ulong) * Ni, hipMemcpyDeviceToHost));
#endif//GENERATE_PHKEY_ON_DEVICE
  checkCudaErrors(hipMemcpy(hst. pos, dev. pos, sizeof(    position) * Ni, hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(hst. acc, dev. acc, sizeof(acceleration) * Ni, hipMemcpyDeviceToHost));
#ifdef  BLOCK_TIME_STEP
  checkCudaErrors(hipMemcpy(hst. vel, dev. vel, sizeof(    velocity) * Ni, hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(hst.time, dev.time, sizeof(  ibody_time) * Ni, hipMemcpyDeviceToHost));
#else///BLOCK_TIME_STEP
  checkCudaErrors(hipMemcpy(hst.  vx, dev.  vx, sizeof(        real) * Ni, hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(hst.  vy, dev.  vy, sizeof(        real) * Ni, hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(hst.  vz, dev.  vz, sizeof(        real) * Ni, hipMemcpyDeviceToHost));
#endif//BLOCK_TIME_STEP
  //-----------------------------------------------------------------------
#ifdef  EXEC_BENCHMARK
  stopStopwatch(&(elapsed->copyParticle_dev2hst));
#endif//EXEC_BENCHMARK
  //-----------------------------------------------------------------------

  //-----------------------------------------------------------------------
  __NOTE__("%s\n", "end");
  //-----------------------------------------------------------------------
}
//-------------------------------------------------------------------------


//-------------------------------------------------------------------------
#ifdef  COMPARE_WITH_DIRECT_SOLVER
//-------------------------------------------------------------------------
extern "C"
void copyAccel_dev2hst(const int Ni, acceleration * RESTRICT dev, acceleration * RESTRICT hst)
{
  //-----------------------------------------------------------------------
  __NOTE__("%s\n", "start");
  //-----------------------------------------------------------------------
  checkCudaErrors(hipMemcpy(hst, dev, sizeof(acceleration) * Ni, hipMemcpyDeviceToHost));
  //-----------------------------------------------------------------------
  __NOTE__("%s\n", "end");
  //-----------------------------------------------------------------------
}
//-------------------------------------------------------------------------
#endif//COMPARE_WITH_DIRECT_SOLVER
//-------------------------------------------------------------------------


//-------------------------------------------------------------------------
#ifdef  COUNT_INTERACTIONS
//-------------------------------------------------------------------------
extern "C"
void copyCounters_dev2hst(const int Ni, iparticle_treeinfo dev, iparticle_treeinfo hst)
{
  //-----------------------------------------------------------------------
  __NOTE__("%s\n", "start");
  //-----------------------------------------------------------------------
  checkCudaErrors(hipMemcpy(hst.  Nj, dev.  Nj, sizeof(int) * Ni, hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(hst.Nbuf, dev.Nbuf, sizeof(int) * Ni, hipMemcpyDeviceToHost));
  //-----------------------------------------------------------------------
  __NOTE__("%s\n", "end");
  //-----------------------------------------------------------------------
}
//-------------------------------------------------------------------------
#endif//COUNT_INTERACTIONS
//-------------------------------------------------------------------------
