#include "hip/hip_runtime.h"
/*************************************************************************\
 *                                                                       *
                  last updated on 2017/01/17(Tue) 20:11:35
 *                                                                       *
 *    Octree N-body calculation for collisionless systems on NVIDIA GPUs *
 *                                                                       *
 *                                                                       *
 *                                                                       *
 *                                             written by Yohei MIKI     *
 *                                                                       *
\*************************************************************************/
//-------------------------------------------------------------------------
/* #define DEBUG_PRINT_FOR_PARTICLE_ACCELERATION */
#define DOUBLE_BUFFER_FOR_LET
//-------------------------------------------------------------------------
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <math.h>
#include <hip/hip_runtime_api.h>
#include <sys/time.h>
#ifndef SERIALIZED_EXECUTION
#include <mpi.h>
#endif//SERIALIZED_EXECUTION
#ifdef  PRINT_PSEUDO_PARTICLE_INFO
#include <unistd.h>
#endif//PRINT_PSEUDO_PARTICLE_INFO
//-------------------------------------------------------------------------
#include "macro.h"
#include "cudalib.h"
#include "timer.h"
#ifndef SERIALIZED_EXECUTION
#include "mpilib.h"
#endif//SERIALIZED_EXECUTION
#ifdef  PRINT_PSEUDO_PARTICLE_INFO
#include "name.h"
#endif//PRINT_PSEUDO_PARTICLE_INFO
//-------------------------------------------------------------------------
#include "../misc/benchmark.h"
#include "../misc/structure.h"
#include "../misc/device.h"
//-------------------------------------------------------------------------
#include "macutil.h"
#include "make.h"
#include "buf_inc.h"
//-------------------------------------------------------------------------
#ifndef SERIALIZED_EXECUTION
#include "../misc/tune.h"
#include "../para/mpicfg.h"
#include "let.h"
#include "let_dev.h"
#endif//SERIALIZED_EXECUTION
//-------------------------------------------------------------------------
#include "walk_dev.h"
//-------------------------------------------------------------------------
#   if  !defined(USE_CUDA_EVENT) && (!defined(SERIALIZED_EXECUTION) || defined(PRINT_PSEUDO_PARTICLE_INFO))
#define USE_GPU_BASE_CLOCK_FREQ
#if 1
#   if  (__CUDACC_VER_MINOR__ + 10 * __CUDACC_VER_MAJOR__) >= 80
#include <nvml.h>
#undef  USE_GPU_BASE_CLOCK_FREQ
#define USE_MEASURED_CLOCK_FREQ
nvmlDevice_t deviceHandler;
#endif//(__CUDACC_VER_MINOR__ + 10 * __CUDACC_VER_MAJOR__) >= 80
#endif
#endif//!defined(USE_CUDA_EVENT) && (!defined(SERIALIZED_EXECUTION) || defined(PRINT_PSEUDO_PARTICLE_INFO))
//-------------------------------------------------------------------------
__constant__  real newton;
__constant__  real epsinv;
#ifndef INDIVIDUAL_GRAVITATIONAL_SOFTENING
__constant__  real eps2;
#endif//INDIVIDUAL_GRAVITATIONAL_SOFTENING
#ifndef WS93_MAC
__constant__  real theta2;
#endif//WS93_MAC
__constant__ jnode jnode0;
//-------------------------------------------------------------------------


//-------------------------------------------------------------------------
/* set CUDA streams */
//-------------------------------------------------------------------------
extern "C"
muse setCUDAstreams_dev(hipStream_t **stream, kernelStream *sinfo, deviceInfo *info, deviceProp *prop
/* #   if  defined(USE_CUDA_EVENT) && (!defined(SERIALIZED_EXECUTION) || defined(PRINT_PSEUDO_PARTICLE_INFO)) */
/* 			, hipEvent_t **iniEvent, hipEvent_t **finEvent */
/* #endif//defined(USE_CUDA_EVENT) && (!defined(SERIALIZED_EXECUTION) || defined(PRINT_PSEUDO_PARTICLE_INFO)) */
			)
{
  //-----------------------------------------------------------------------
  __NOTE__("%s\n", "start");
  //-----------------------------------------------------------------------
  muse alloc = {0, 0};
  //-----------------------------------------------------------------------

  //-----------------------------------------------------------------------
  /* determine # of CUDA streams */
  sinfo->idx = 0;
  sinfo->num = 2;
  //-----------------------------------------------------------------------
  /* allocate array for CUDA streams */
  *stream = (hipStream_t *)malloc((size_t)(sinfo->num) * sizeof(hipStream_t));  if( *stream == NULL ){    __KILL__(stderr, "ERROR: failure to allocate stream\n");  }
  alloc.host +=                    (size_t)(sinfo->num) * sizeof(hipStream_t) ;
  sinfo->stream = *stream;
  //-----------------------------------------------------------------------
  /* set CUDA streams */
  for(int ii = 0; ii < 2; ii++)
    sinfo->stream[ii] = info->stream[ii];
  for(int ii = 2; ii < sinfo->num; ii++)
    checkCudaErrors(hipStreamCreate(&(sinfo->stream[ii])));
  //-----------------------------------------------------------------------
#if 0
  int priority;
  for(int ii = 0; ii < sinfo->num; ii++){
    checkCudaErrors(hipStreamGetPriority(sinfo->stream[ii], &priority));
    fprintf(stdout, "priority of stream[%d] is %d\n", ii, priority);
  }/* for(int ii = 0; ii < sinfo->num; ii++){ */
  fflush(stdout);
#endif
  //-----------------------------------------------------------------------
/* #if 1 */
/*   int highest, lowest; */
/*   checkCudaErrors(hipDeviceGetStreamPriorityRange(&lowest, &highest)); */
/*   for(int ii = 0; ii < *Nstream; ii++) */
/*     checkCudaErrors(hipStreamCreateWithPriority(&((*stream)[ii]), hipStreamDefault, highest)); */
/*     /\* checkCudaErrors(hipStreamCreateWithPriority(&((*stream)[ii]), hipStreamNonBlocking, highest)); *\/ */
/* #else */
/* #pragma unroll */
/*   for(int ii = 0; ii < *Nstream; ii++) */
/*     checkCudaErrors(hipStreamCreate(&((*stream)[ii]))); */
/* #endif */
  //-----------------------------------------------------------------------
/*   /\* allocate and set CUDA events *\/ */
/* #   if  defined(USE_CUDA_EVENT) && (!defined(SERIALIZED_EXECUTION) || defined(PRINT_PSEUDO_PARTICLE_INFO)) */
/*   *iniEvent = (hipEvent_t *)malloc((size_t)(sinfo->num) * sizeof(hipEvent_t));  if( *iniEvent == NULL ){    __KILL__(stderr, "ERROR: failure to allocate iniEvent\n");  } */
/*   *finEvent = (hipEvent_t *)malloc((size_t)(sinfo->num) * sizeof(hipEvent_t));  if( *finEvent == NULL ){    __KILL__(stderr, "ERROR: failure to allocate finEvent\n");  } */
/*   alloc.host +=                     (size_t)(sinfo->num) * sizeof(hipEvent_t); */
/*   alloc.host +=                     (size_t)(sinfo->num) * sizeof(hipEvent_t); */
/*   for(int ii = 0; ii < sinfo->num; ii++){ */
/*     checkCudaErrors(hipEventCreate(&((*iniEvent)[ii]))); */
/*     checkCudaErrors(hipEventCreate(&((*finEvent)[ii]))); */
/*   }/\* for(int ii = 0; ii < sinfo->num; ii++){ *\/ */
/* #endif//defined(USE_CUDA_EVENT) && (!defined(SERIALIZED_EXECUTION) || defined(PRINT_PSEUDO_PARTICLE_INFO)) */
  //-----------------------------------------------------------------------

  //-----------------------------------------------------------------------
  __NOTE__("%s\n", "end");
  //-----------------------------------------------------------------------
  return (alloc);
  //-----------------------------------------------------------------------
}
//-------------------------------------------------------------------------


//-------------------------------------------------------------------------
/* set CUDA streams */
//-------------------------------------------------------------------------
#   if  defined(USE_CUDA_EVENT) && (!defined(SERIALIZED_EXECUTION) || defined(PRINT_PSEUDO_PARTICLE_INFO))
//-------------------------------------------------------------------------
extern "C"
muse allocateCUDAevents_dev
(hipEvent_t **iniWalk, hipEvent_t **finWalk
#ifdef  MONITOR_LETGEN_TIME
 , hipEvent_t **iniMake, hipEvent_t **finMake
#endif//MONITOR_LETGEN_TIME
 , const int Ngpu)
{
  //-----------------------------------------------------------------------
  __NOTE__("%s\n", "start");
  //-----------------------------------------------------------------------
  muse alloc = {0, 0};
  //-----------------------------------------------------------------------

  //-----------------------------------------------------------------------
  /* allocate array for CUDA events */
  *iniWalk = (hipEvent_t *)malloc((size_t)Ngpu * sizeof(hipEvent_t));  if( *iniWalk == NULL ){    __KILL__(stderr, "ERROR: failure to allocate iniWalk\n");  }
  *finWalk = (hipEvent_t *)malloc((size_t)Ngpu * sizeof(hipEvent_t));  if( *finWalk == NULL ){    __KILL__(stderr, "ERROR: failure to allocate finWalk\n");  }
  alloc.host +=                    (size_t)Ngpu * sizeof(hipEvent_t) ;
  alloc.host +=                    (size_t)Ngpu * sizeof(hipEvent_t) ;
#ifdef  MONITOR_LETGEN_TIME
  *iniMake = (hipEvent_t *)malloc((size_t)(Ngpu - 1) * sizeof(hipEvent_t));	 if( *iniMake == NULL ){    __KILL__(stderr, "ERROR: failure to allocate iniMake\n");  }
  *finMake = (hipEvent_t *)malloc((size_t)(Ngpu - 1) * sizeof(hipEvent_t));	 if( *finMake == NULL ){    __KILL__(stderr, "ERROR: failure to allocate finMake\n");  }
  alloc.host +=                    (size_t)(Ngpu - 1) * sizeof(hipEvent_t) ;
  alloc.host +=                    (size_t)(Ngpu - 1) * sizeof(hipEvent_t) ;
#endif//MONITOR_LETGEN_TIME
  //-----------------------------------------------------------------------
  /* set CUDA events */
  for(int ii = 0; ii < Ngpu; ii++){
    checkCudaErrors(hipEventCreate(&((*iniWalk)[ii])));
    checkCudaErrors(hipEventCreate(&((*finWalk)[ii])));
  }/* for(int ii = 0; ii < Ngpu; ii++){ */
#ifdef  MONITOR_LETGEN_TIME
  for(int ii = 0; ii < Ngpu - 1; ii++){
    checkCudaErrors(hipEventCreate(&((*iniMake)[ii])));
    checkCudaErrors(hipEventCreate(&((*finMake)[ii])));
  }/* for(int ii = 0; ii < Ngpu; ii++){ */
#endif//MONITOR_LETGEN_TIME
  //-----------------------------------------------------------------------

  //-----------------------------------------------------------------------
  __NOTE__("%s\n", "end");
  //-----------------------------------------------------------------------
  return (alloc);
  //-----------------------------------------------------------------------
}
//-------------------------------------------------------------------------
extern "C"
void  releaseCUDAevents_dev
(hipEvent_t  *iniWalk, hipEvent_t  *finWalk
#ifdef  MONITOR_LETGEN_TIME
 , hipEvent_t  *iniMake, hipEvent_t  *finMake
#endif//MONITOR_LETGEN_TIME
 , const int Ngpu)
{
  //-----------------------------------------------------------------------
  __NOTE__("%s\n", "start");
  //-----------------------------------------------------------------------

  //-----------------------------------------------------------------------
  /* destroy CUDA events */
  for(int ii = 0; ii < Ngpu; ii++){
    mycudaEventDestroy(iniWalk[ii]);
    mycudaEventDestroy(finWalk[ii]);
  }/* for(int ii = 0; ii < Ngpu; ii++){ */
#ifdef  MONITOR_LETGEN_TIME
  for(int ii = 0; ii < Ngpu - 1; ii++){
    mycudaEventDestroy(iniMake[ii]);
    mycudaEventDestroy(finMake[ii]);
  }/* for(int ii = 0; ii < Ngpu - 1; ii++){ */
#endif//MONITOR_LETGEN_TIME
  //-----------------------------------------------------------------------
  /* deallocate CUDA events */
  free(iniWalk);
  free(finWalk);
#ifdef  MONITOR_LETGEN_TIME
  free(iniMake);
  free(finMake);
#endif//MONITOR_LETGEN_TIME
  //-----------------------------------------------------------------------

  //-----------------------------------------------------------------------
  __NOTE__("%s\n", "end");
  //-----------------------------------------------------------------------
}
//-------------------------------------------------------------------------
#endif//defined(USE_CUDA_EVENT) && (!defined(SERIALIZED_EXECUTION) || defined(PRINT_PSEUDO_PARTICLE_INFO))
//-------------------------------------------------------------------------


//-------------------------------------------------------------------------
/* arrays to store properties of tree cells (allocated on the global memory) */
//-------------------------------------------------------------------------
#ifdef  USE_SMID_TO_GET_BUFID
//-------------------------------------------------------------------------
/* complicated treatments is a remedy for ``not contiguous'' case of smid */
//-------------------------------------------------------------------------
__global__ void initFreeLst(const int numLanes, uint * RESTRICT freeLst, const int numFul, READ_ONLY int * RESTRICT smid)
{
  //-----------------------------------------------------------------------
  const int tidx = THREADIDX_X1D;
  //-----------------------------------------------------------------------
  if( tidx < numFul )
    freeLst[tidx] = INT_MAX;
  //-----------------------------------------------------------------------
  if( tidx < numLanes ){
    //---------------------------------------------------------------------
    const int target = (tidx % NBLOCKS_PER_SM) + smid[tidx / NBLOCKS_PER_SM] * NBLOCKS_PER_SM;
    //---------------------------------------------------------------------
    freeLst[target] = (uint)tidx;
    //---------------------------------------------------------------------
  }/* if( tidx < numLanes ){ */
  //-----------------------------------------------------------------------
}
//-------------------------------------------------------------------------
#else///USE_SMID_TO_GET_BUFID
//-------------------------------------------------------------------------
__global__ void initFreeLst
(const int numLanes, uint * RESTRICT freeLst
#ifndef TRY_MODE_ABOUT_BUFFER
 , uint * RESTRICT freeNum, int * RESTRICT active
#endif//TRY_MODE_ABOUT_BUFFER
 )
{
  //-----------------------------------------------------------------------
  const int tidx = THREADIDX_X1D;
  //-----------------------------------------------------------------------
  if( tidx < numLanes ){
    //---------------------------------------------------------------------
#ifdef  TRY_MODE_ABOUT_BUFFER
    freeLst[tidx] = (uint)tidx;
#else///TRY_MODE_ABOUT_BUFFER
    freeLst[tidx] = (uint)(numLanes - (tidx + 1));
#endif//TRY_MODE_ABOUT_BUFFER
    //---------------------------------------------------------------------
#ifndef TRY_MODE_ABOUT_BUFFER
    if( tidx == 0 ){
      *freeNum = (uint)numLanes;
      *active  = 1;
    }
#endif//TRY_MODE_ABOUT_BUFFER
    //---------------------------------------------------------------------
  }
  //-----------------------------------------------------------------------
}
//-------------------------------------------------------------------------
#endif//USE_SMID_TO_GET_BUFID
//-------------------------------------------------------------------------
extern "C"
void  freeTreeBuffer_dev
(int  *failure, uint  *buffer, uint  *freeLst
#   if  !defined(USE_SMID_TO_GET_BUFID) && !defined(TRY_MODE_ABOUT_BUFFER)
 , uint  *freeNum, int  *active
#endif//!defined(USE_SMID_TO_GET_BUFID) && !defined(TRY_MODE_ABOUT_BUFFER)
#ifndef USE_CUDA_EVENT
#   if  !defined(SERIALIZED_EXECUTION) || defined(PRINT_PSEUDO_PARTICLE_INFO)
 , unsigned long long int  *cycles_hst, unsigned long long int  *cycles_dev
#endif//!defined(SERIALIZED_EXECUTION) || defined(PRINT_PSEUDO_PARTICLE_INFO)
#   if  !defined(SERIALIZED_EXECUTION) && defined(MONITOR_LETGEN_TIME)
 , unsigned long long int  *cycles_let_hst, unsigned long long int  *cycles_let_dev
#endif//!defined(SERIALIZED_EXECUTION) && defined(MONITOR_LETGEN_TIME)
#endif//USE_CUDA_EVENT
 )
{
  //-----------------------------------------------------------------------
  __NOTE__("%s\n", "start");
  //-----------------------------------------------------------------------
  mycudaFree(failure);
  mycudaFree(buffer);
  mycudaFree(freeLst);
#   if  !defined(USE_SMID_TO_GET_BUFID) && !defined(TRY_MODE_ABOUT_BUFFER)
  mycudaFree(freeNum);
  mycudaFree(active);
#endif//!defined(USE_SMID_TO_GET_BUFID) && !defined(TRY_MODE_ABOUT_BUFFER)
#ifndef USE_CUDA_EVENT
#   if  !defined(SERIALIZED_EXECUTION) || defined(PRINT_PSEUDO_PARTICLE_INFO)
  mycudaFree    (cycles_dev);
  mycudaFreeHost(cycles_hst);
#endif//!defined(SERIALIZED_EXECUTION) || defined(PRINT_PSEUDO_PARTICLE_INFO)
#   if  !defined(SERIALIZED_EXECUTION) && defined(MONITOR_LETGEN_TIME)
  mycudaFree    (cycles_let_dev);
  mycudaFreeHost(cycles_let_hst);
#endif//!defined(SERIALIZED_EXECUTION) && defined(MONITOR_LETGEN_TIME)
#endif//USE_CUDA_EVENT
  //-----------------------------------------------------------------------
#ifdef  USE_MEASURED_CLOCK_FREQ
  nvmlShutdown();
#endif//USE_MEASURED_CLOCK_FREQ
  //-----------------------------------------------------------------------
  __NOTE__("%s\n", "end");
  //-----------------------------------------------------------------------
}
//-------------------------------------------------------------------------
extern "C"
muse allocTreeBuffer_dev
(int **failure, uint **buffer, uint **freeLst,
#   if  !defined(USE_SMID_TO_GET_BUFID) && !defined(TRY_MODE_ABOUT_BUFFER)
 uint **freeNum, int **active,
#endif//!defined(USE_SMID_TO_GET_BUFID) && !defined(TRY_MODE_ABOUT_BUFFER)
#ifndef USE_CUDA_EVENT
#   if  !defined(SERIALIZED_EXECUTION) || defined(PRINT_PSEUDO_PARTICLE_INFO)
 unsigned long long int **cycles_hst, unsigned long long int **cycles_dev,
#endif//!defined(SERIALIZED_EXECUTION) || defined(PRINT_PSEUDO_PARTICLE_INFO)
#   if  !defined(SERIALIZED_EXECUTION) && defined(MONITOR_LETGEN_TIME)
 unsigned long long int **cycles_let_hst, unsigned long long int **cycles_let_dev,
#endif//!defined(SERIALIZED_EXECUTION) && defined(MONITOR_LETGEN_TIME)
#endif//USE_CUDA_EVENT
 soaTreeWalkBuf *buf, const int num_max, const muse used, const deviceProp gpu)
{
  //-----------------------------------------------------------------------
  __NOTE__("%s\n", "start");
  //-----------------------------------------------------------------------
  muse alloc = {0, 0};
  //-----------------------------------------------------------------------

  //-----------------------------------------------------------------------
  mycudaMalloc((void **)failure, 1 * sizeof(int));
  alloc.device +=                1 * sizeof(int);
  const int fail_hst = 0;
  checkCudaErrors(hipMemcpy(*failure, &fail_hst, sizeof(int), hipMemcpyHostToDevice));
  //-----------------------------------------------------------------------
  const int nblocks = NBLOCKS_PER_SM * gpu.numSM;
  //-----------------------------------------------------------------------
#ifdef  USE_SMID_TO_GET_BUFID
  int last = 0;
  int num = 0;
  int *smid_dev;  mycudaMalloc    ((void **)&smid_dev, sizeof(int) * gpu.numSM);
  int *smid_hst;  mycudaMallocHost((void **)&smid_hst, sizeof(int) * gpu.numSM);
  for(int ii = 0; ii < 64; ii++)
    if( gpu.smid[ii] != -1 ){
      smid_hst[num] = gpu.smid[ii];      num++;
      last = ii;
    }
  last++;
  mycudaMalloc((void **)freeLst, (NBLOCKS_PER_SM * last) * sizeof(uint));  alloc.device += (NBLOCKS_PER_SM * last) * sizeof(uint);
#else///USE_SMID_TO_GET_BUFID
  mycudaMalloc((void **)freeLst, nblocks * sizeof(uint));  alloc.device += nblocks * sizeof(uint);
#endif//USE_SMID_TO_GET_BUFID
  //-----------------------------------------------------------------------
#   if  !defined(USE_SMID_TO_GET_BUFID) && !defined(TRY_MODE_ABOUT_BUFFER)
  mycudaMalloc((void **)freeNum,           sizeof(uint));  alloc.device +=           sizeof(uint);
  mycudaMalloc((void **) active,           sizeof( int));  alloc.device +=           sizeof( int);
#endif//!defined(USE_SMID_TO_GET_BUFID) && !defined(TRY_MODE_ABOUT_BUFFER)
  //-----------------------------------------------------------------------
#ifdef  USE_SMID_TO_GET_BUFID
  checkCudaErrors(hipMemcpy(smid_dev, smid_hst, sizeof(int) * gpu.numSM, hipMemcpyHostToDevice));
  initFreeLst<<<1, NBLOCKS_PER_SM * last>>>(nblocks, *freeLst, NBLOCKS_PER_SM * last, smid_dev);
  mycudaFree    (smid_dev);
  mycudaFreeHost(smid_hst);
#else///USE_SMID_TO_GET_BUFID
  initFreeLst<<<1, nblocks>>>(nblocks, *freeLst
#ifndef TRY_MODE_ABOUT_BUFFER
			      , *freeNum, *active
#endif//TRY_MODE_ABOUT_BUFFER
			      );
#endif//USE_SMID_TO_GET_BUFID
  //-----------------------------------------------------------------------
#ifndef USE_CUDA_EVENT
#   if  !defined(SERIALIZED_EXECUTION) || defined(PRINT_PSEUDO_PARTICLE_INFO)
  mycudaMalloc    ((void **)cycles_dev, sizeof(unsigned long long int));  alloc.device += sizeof(unsigned long long int);
  mycudaMallocHost((void **)cycles_hst, sizeof(unsigned long long int));  alloc.host   += sizeof(unsigned long long int);
#endif//!defined(SERIALIZED_EXECUTION) || defined(PRINT_PSEUDO_PARTICLE_INFO)
  //-----------------------------------------------------------------------
#   if  !defined(SERIALIZED_EXECUTION) && defined(MONITOR_LETGEN_TIME)
  mycudaMalloc    ((void **)cycles_let_dev, sizeof(unsigned long long int));  alloc.device += sizeof(unsigned long long int);
  mycudaMallocHost((void **)cycles_let_hst, sizeof(unsigned long long int));  alloc.host   += sizeof(unsigned long long int);
#endif//!defined(SERIALIZED_EXECUTION) && defined(MONITOR_LETGEN_TIME)
#endif//USE_CUDA_EVENT
  //-----------------------------------------------------------------------
#ifdef  USE_MEASURED_CLOCK_FREQ
  nvmlInit();
#if 1
  nvmlDeviceGetHandleByIndex(gpu.idx, &deviceHandler);
#else
  nvmlReturn_t nvmlMsg = nvmlDeviceGetHandleByIndex(gpu.idx, &deviceHandler);
  printf("nvmlMsg = %d\n", nvmlMsg);
  MPI_Finalize();
  exit(0);
#endif
#endif//USE_MEASURED_CLOCK_FREQ
  //-----------------------------------------------------------------------


  //-----------------------------------------------------------------------
#if 1
  size_t unused, total;
  queryFreeDeviceMemory(&unused, &total);
#else
  const size_t unused = gpu.gmemSize - used.device - alloc.device;
#endif
#ifdef  CUB_AVAILABLE
  const size_t safety = GLOBAL_MEMORY_SYSBUF;
#else///CUB_AVAILABLE
  /* latters are pessimistic guess about device memory for CUDA thrust (PH-key sort, time step sort) */
  const size_t safety = GLOBAL_MEMORY_SYSBUF + (size_t)num_max * (sizeof(PHint) + sizeof(real));
#endif//CUB_AVAILABLE
  const size_t booked = (unused > safety) ? (unused - safety) : (unused >> 1);
  if( (booked / ((size_t)(NGROUPS * nblocks) * (sizeof(uint)))) > INT_MAX ){
    __KILL__(stderr, "ERROR: expected size for bufUnit (%zu) exceeds INT_MAX\n\trewrite \"calcAcc_kernel()\" in \"src/tree/walk_dev.cu\"\n", (booked / ((size_t)(NGROUPS * nblocks) * (sizeof(uint)))));
  }/* if( (booked / ((size_t)(NGROUPS * nblocks) * (sizeof(uint)))) > INT_MAX ){ */
  int bufUnit = (int)(booked / ((size_t)(NGROUPS * nblocks) * (sizeof(uint))));
  /* *bufUnit should be aligned in 32 bytes order (= 128 bits) --> 8 or 4 elements for single or double precision, respectively */
  bufUnit -= (bufUnit & 7);
#ifndef SERIALIZED_EXECUTION
  if( ((size_t)bufUnit * (size_t)NGROUPS) > INT_MAX ){
    __KILL__(stderr, "ERROR: expected size for bufUnit for LET (%zu) exceeds INT_MAX\n\trewrite \"makeLET_kernel()\" in \"src/tree/let_dev.cu\"\n", ((size_t)bufUnit * (size_t)NGROUPS));
  }/* if( ((size_t)bufUnit * (size_t)NGROUPS) > INT_MAX ){ */
#endif//SERIALIZED_EXECUTION
  //-----------------------------------------------------------------------
  const size_t walkBufSize = (size_t)(NGROUPS * nblocks) * (size_t)bufUnit * sizeof(uint);
  mycudaMalloc((void **)buffer, walkBufSize);
  alloc.device +=               walkBufSize ;
  //-----------------------------------------------------------------------
  /* alert if the size for the walk buffer is smaller than 64 Ni B (512MiB @ N = 8M) */
  if( walkBufSize < ((size_t)num_max << 6) ){
    fprintf(stderr, "%s(%d): %s\n", __FILE__, __LINE__, __func__);
    fprintf(stderr, "warning:\tthe size for the walk buffer is %zu B (= %zu KiB = %zu MiB = %zu GiB), might be too small\n", walkBufSize, walkBufSize >> 10, walkBufSize >> 20, walkBufSize >> 30);
    fprintf(stderr, "suggestion:\tconsider decreasing \"TREE_SAFETY_VAL\" defined in src/tree/make.h (current value is %f)\n", TREE_SAFETY_VAL);
    fflush(stderr);
  }/* if( walkBufSize < ((size_t)num_max << 6) ){ */
  //-----------------------------------------------------------------------
  buf->fail    = *failure;
  buf->freeLst = *freeLst;
  buf->buffer  = *buffer;
#   if  !defined(USE_SMID_TO_GET_BUFID) && !defined(TRY_MODE_ABOUT_BUFFER)
  buf->freeNum = *freeNum;
  buf->active  = *active;
#endif//!defined(USE_SMID_TO_GET_BUFID) && !defined(TRY_MODE_ABOUT_BUFFER)
#   if  !defined(USE_SMID_TO_GET_BUFID) &&  defined(TRY_MODE_ABOUT_BUFFER)
  buf->freeNum = NBLOCKS_PER_SM * gpu.numSM;
#endif//!defined(USE_SMID_TO_GET_BUFID) &&  defined(TRY_MODE_ABOUT_BUFFER)
  buf->bufSize = bufUnit;
  //-----------------------------------------------------------------------
#if 0
  fprintf(stdout, "bufUnit = %d, bufTot = %zu, bufSize = %zu\n", bufUnit, walkBufSize / sizeof(uint), walkBufSize);
  fflush(stdout);
#endif
  //-----------------------------------------------------------------------

  //-----------------------------------------------------------------------
  __NOTE__("%s\n", "end");
  //-----------------------------------------------------------------------
  return (alloc);
  //-----------------------------------------------------------------------
}
//-------------------------------------------------------------------------


//-------------------------------------------------------------------------
#ifdef  COUNT_INTERACTIONS
//-------------------------------------------------------------------------
/* initialize count of Nj and Nbuf */
//-------------------------------------------------------------------------
__global__ void initCounter_kernel(int * RESTRICT Nj, int * RESTRICT Nb)
{
  //-----------------------------------------------------------------------
  Nj[GLOBALIDX_X1D] = 0;
  Nb[GLOBALIDX_X1D] = 0;
  //-----------------------------------------------------------------------
}
//-------------------------------------------------------------------------
#endif//COUNT_INTERACTIONS
//-------------------------------------------------------------------------


//-------------------------------------------------------------------------
/* initialize acceleration and potential */
//-------------------------------------------------------------------------
/* acc ::         output :: acceleration and potential of N-body particles */
//-------------------------------------------------------------------------
#ifdef  BLOCK_TIME_STEP
//-------------------------------------------------------------------------
__global__ void initAcc_kernel
(acceleration * RESTRICT acc, const int laneNum, const laneinfo * RESTRICT laneInfo
#ifdef  GADGET_MAC
 , acceleration * RESTRICT old
#endif//GADGET_MAC
#ifdef  DPADD_FOR_ACC
 , DPacc * RESTRICT tmp
#endif//DPADD_FOR_ACC
#   if  defined(KAHAN_SUM_CORRECTION) && defined(ACCURATE_ACCUMULATION) && (!defined(SERIALIZED_EXECUTION) || (NWARP > 1))
 , acceleration * RESTRICT res
#endif//defined(KAHAN_SUM_CORRECTION) && defined(ACCURATE_ACCUMULATION) && (!defined(SERIALIZED_EXECUTION) || (NWARP > 1))
 )
{
  //-----------------------------------------------------------------------
#if 0
  const int tidx = THREADIDX_X1D;
  const int lane = tidx & (TSUB - 1);/* index of the thread within a thread group */
  /* const int laneIdx = GLOBALIDX_X1D / TSUB; */
  const int laneIdx = DIV_TSUB(GLOBALIDX_X1D);
#else
  /* const int lane    = THREADIDX_X1D & ((TSUB / NWARP) - 1); */
  /* const int laneIdx = GLOBALIDX_X1D /  (TSUB / NWARP); */
  const int lane    = THREADIDX_X1D & (DIV_NWARP(TSUB) - 1);
  const int laneIdx = GLOBALIDX_X1D /  DIV_NWARP(TSUB);
#endif
#if 0
  const laneinfo info = laneInfo[laneIdx];
#else
  laneinfo info = {NUM_BODY_MAX, 0};
  if( laneIdx < laneNum )
    info = laneInfo[laneIdx];
#endif
  //-----------------------------------------------------------------------
  if( lane < info.num ){
    //---------------------------------------------------------------------
#ifdef  GADGET_MAC
    old[info.head + lane] = acc[info.head + lane];
#endif//GADGET_MAC
    //---------------------------------------------------------------------
    const acceleration ai = {ZERO, ZERO, ZERO, ZERO};
    acc[info.head + lane] = ai;
#ifdef  DPADD_FOR_ACC
    const DPacc dac = {0.0, 0.0, 0.0, 0.0};
    tmp[info.head + lane] = dac;
#endif//DPADD_FOR_ACC
#   if  defined(KAHAN_SUM_CORRECTION) && defined(ACCURATE_ACCUMULATION) && (!defined(SERIALIZED_EXECUTION) || (NWARP > 1))
    res[info.head + lane] = ai;
#endif//defined(KAHAN_SUM_CORRECTION) && defined(ACCURATE_ACCUMULATION) && (!defined(SERIALIZED_EXECUTION) || (NWARP > 1))
    //---------------------------------------------------------------------
  }/* if( lane < info.num ){ */
  //-----------------------------------------------------------------------
}
//-------------------------------------------------------------------------
#else///BLOCK_TIME_STEP
//-------------------------------------------------------------------------
__global__ void initAcc_kernel
(acceleration *acc
#ifdef  GADGET_MAC
 , acceleration * RESTRICT old
#endif//GADGET_MAC
#ifdef  DPADD_FOR_ACC
 , DPacc * RESTRICT tmp
#endif//DPADD_FOR_ACC
#   if  defined(KAHAN_SUM_CORRECTION) && defined(ACCURATE_ACCUMULATION) && (!defined(SERIALIZED_EXECUTION) || (NWARP > 1))
 , acceleration * RESTRICT res
#endif//defined(KAHAN_SUM_CORRECTION) && defined(ACCURATE_ACCUMULATION) && (!defined(SERIALIZED_EXECUTION) || (NWARP > 1))
 )
{
  //-----------------------------------------------------------------------
  const acceleration ai = {ZERO, ZERO, ZERO, ZERO};
  //-----------------------------------------------------------------------
#ifdef  GADGET_MAC
  old[GLOBALIDX_X1D] = acc[GLOBALIDX_X1D];
#endif//GADGET_MAC
  //-----------------------------------------------------------------------
  acc[GLOBALIDX_X1D] = ai;
#ifdef  DPADD_FOR_ACC
    const DPacc dac = {0.0, 0.0, 0.0, 0.0};
    tmp[GLOBALIDX_X1D] = dac;
#endif//DPADD_FOR_ACC
#   if  defined(KAHAN_SUM_CORRECTION) && defined(ACCURATE_ACCUMULATION) && (!defined(SERIALIZED_EXECUTION) || (NWARP > 1))
  res[GLOBALIDX_X1D] = ai;
#endif//defined(KAHAN_SUM_CORRECTION) && defined(ACCURATE_ACCUMULATION) && (!defined(SERIALIZED_EXECUTION) || (NWARP > 1))
  //-----------------------------------------------------------------------
}
//-------------------------------------------------------------------------
#endif//BLOCK_TIME_STEP
//-------------------------------------------------------------------------


//-------------------------------------------------------------------------
/* multiply Gravitational constant and subtract self-interaction */
//-------------------------------------------------------------------------
/* acc :: input / output :: acceleration and potential of N-body particles */
/* pos :: input          :: position and mass of N-body particles */
//-------------------------------------------------------------------------
#ifdef  BLOCK_TIME_STEP
//-------------------------------------------------------------------------
__global__ void trimAcc_kernel(acceleration * RESTRICT acc, READ_ONLY position * RESTRICT pos, const int laneNum, READ_ONLY laneinfo * RESTRICT laneInfo
#ifdef  DPADD_FOR_ACC
 , READ_ONLY DPacc * RESTRICT tmp
#endif//DPADD_FOR_ACC
#   if  defined(KAHAN_SUM_CORRECTION) && defined(ACCURATE_ACCUMULATION) && (!defined(SERIALIZED_EXECUTION) || (NWARP > 1))
			       , READ_ONLY acceleration * RESTRICT res
#endif//defined(KAHAN_SUM_CORRECTION) && defined(ACCURATE_ACCUMULATION) && (!defined(SERIALIZED_EXECUTION) || (NWARP > 1))
			       )
{
  //-----------------------------------------------------------------------
#if 0
  const int tidx = THREADIDX_X1D;
  const int lane = tidx & (TSUB - 1);/* index of the thread within a thread group */
  /* const int laneIdx = GLOBALIDX_X1D / TSUB; */
  const int laneIdx = DIV_TSUB(GLOBALIDX_X1D);
#else
  /* const int lane    = THREADIDX_X1D & ((TSUB / NWARP) - 1); */
  /* const int laneIdx = GLOBALIDX_X1D /  (TSUB / NWARP); */
  const int lane    = THREADIDX_X1D & (DIV_NWARP(TSUB) - 1);
  const int laneIdx = GLOBALIDX_X1D /  DIV_NWARP(TSUB);
#endif
#if 0
  const laneinfo info = laneInfo[laneIdx];
#else
  laneinfo info = {NUM_BODY_MAX, 0};
  if( laneIdx < laneNum )
    info = laneInfo[laneIdx];
#endif
  //-----------------------------------------------------------------------
  if( lane < info.num ){
    //---------------------------------------------------------------------
    const int ii = info.head + lane;
    //---------------------------------------------------------------------
#ifndef DPADD_FOR_ACC
    /* load acceleration */
    acceleration ai = acc[ii];
    /* eliminate self-interaction */
    ai.pot -= epsinv * pos[ii].m;
#endif//DPADD_FOR_ACC
    //---------------------------------------------------------------------
#ifdef  DPADD_FOR_ACC
    DPacc dacc = tmp[ii];
    acceleration ai;
    ai.x   = CAST_D2R(dacc.x);
    ai.y   = CAST_D2R(dacc.y);
    ai.z   = CAST_D2R(dacc.z);
    ai.pot = CAST_D2R(dacc.pot - CAST_R2D(epsinv * pos[ii].m));
#endif//DPADD_FOR_ACC
#   if  defined(KAHAN_SUM_CORRECTION) && defined(ACCURATE_ACCUMULATION) && (!defined(SERIALIZED_EXECUTION) || (NWARP > 1))
    acceleration corr = res[ii];
    ai.x   = CAST_D2R(CAST_R2D(ai.x  ) + CAST_R2D(corr.x  ));
    ai.y   = CAST_D2R(CAST_R2D(ai.y  ) + CAST_R2D(corr.y  ));
    ai.z   = CAST_D2R(CAST_R2D(ai.z  ) + CAST_R2D(corr.z  ));
    ai.pot = CAST_D2R(CAST_R2D(ai.pot) + CAST_R2D(corr.pot));
#if 0
    printf("res(%e) = %e, %e, %e, %e\n", pos[ii].x, corr.x, corr.y, corr.z, corr.pot);
#endif
#endif//defined(KAHAN_SUM_CORRECTION) && defined(ACCURATE_ACCUMULATION) && (!defined(SERIALIZED_EXECUTION) || (NWARP > 1))
    //---------------------------------------------------------------------
    /* multiply Gravitational constant */
    ai.x   *=  newton;
    ai.y   *=  newton;
    ai.z   *=  newton;
    ai.pot *= -newton;
    //---------------------------------------------------------------------
    /* store acceleration */
    acc[ii] = ai;
    //---------------------------------------------------------------------
  }/* if( lane < info.num ){ */
  //-----------------------------------------------------------------------
}
//-------------------------------------------------------------------------
#else///BLOCK_TIME_STEP
//-------------------------------------------------------------------------
__global__ void trimAcc_kernel(acceleration * RESTRICT acc, READ_ONLY position * RESTRICT pos
#ifdef  DPADD_FOR_ACC
			       , READ_ONLY DPacc * RESTRICT tmp
#endif//DPADD_FOR_ACC
#   if  defined(KAHAN_SUM_CORRECTION) && defined(ACCURATE_ACCUMULATION) && (!defined(SERIALIZED_EXECUTION) || (NWARP > 1))
			       , READ_ONLY acceleration * RESTRICT res
#endif//defined(KAHAN_SUM_CORRECTION) && defined(ACCURATE_ACCUMULATION) && (!defined(SERIALIZED_EXECUTION) || (NWARP > 1))
			       )
{
  //-----------------------------------------------------------------------
  const int ii = GLOBALIDX_X1D;
  //-----------------------------------------------------------------------
#ifndef DPADD_FOR_ACC
  /* load acceleration and mass */
  acceleration ai = acc[ii];
  /* eliminate self-interaction */
  ai.pot -= epsinv * pos[ii].m;
#endif//DPADD_FOR_ACC
  //-----------------------------------------------------------------------
#ifdef  DPADD_FOR_ACC
  DPacc dacc = tmp[ii];
  acceleration ai;
  ai.x   = CAST_D2R(dacc.x);
  ai.y   = CAST_D2R(dacc.y);
  ai.z   = CAST_D2R(dacc.z);
  ai.pot = CAST_D2R(dacc.pot - CAST_R2D(epsinv * pos[ii].m));
#endif//DPADD_FOR_ACC
#   if  defined(KAHAN_SUM_CORRECTION) && defined(ACCURATE_ACCUMULATION) && (!defined(SERIALIZED_EXECUTION) || (NWARP > 1))
  acceleration corr = res[ii];
  ai.x   = CAST_D2R(CAST_R2D(ai.x  ) + CAST_R2D(corr.x  ));
  ai.y   = CAST_D2R(CAST_R2D(ai.y  ) + CAST_R2D(corr.y  ));
  ai.z   = CAST_D2R(CAST_R2D(ai.z  ) + CAST_R2D(corr.z  ));
  ai.pot = CAST_D2R(CAST_R2D(ai.pot) + CAST_R2D(corr.pot));
#endif//defined(KAHAN_SUM_CORRECTION) && defined(ACCURATE_ACCUMULATION) && (!defined(SERIALIZED_EXECUTION) || (NWARP > 1))
  //-----------------------------------------------------------------------
  /* multiply Gravitational constant */
  ai.x   *=  newton;
  ai.y   *=  newton;
  ai.z   *=  newton;
  ai.pot *= -newton;
  //-----------------------------------------------------------------------
  /* store acceleration */
  acc[ii] = ai;
  //-----------------------------------------------------------------------
}
//-------------------------------------------------------------------------
#endif//BLOCK_TIME_STEP
//-------------------------------------------------------------------------


//-------------------------------------------------------------------------
/* parallel prefix sum within a group of TSUB threads (TSUB <= 32 to use implicit synchronization) */
/* type of prefix sum is inclusive */
/* NOTE: implicit synchronization within 32 threads (a warp) is assumed */
//-------------------------------------------------------------------------
#ifdef  USE_WARP_SHUFFLE_FUNC
__device__ __forceinline__  int prefixSumTsub(const int psum,                                            const int lane)
#else///USE_WARP_SHUFFLE_FUNC
__device__ __forceinline__ void prefixSumTsub(const int psum, volatile uint_real * smem, const int tidx, const int lane)
#endif//USE_WARP_SHUFFLE_FUNC
{
  //-----------------------------------------------------------------------
#ifdef  USE_WARP_SHUFFLE_FUNC
  //-----------------------------------------------------------------------
  int val = psum;
  int tmp;
#   if  TSUB >=  2
  tmp = __shfl_up(val,  1, TSUB);  if( lane >=  1 )    val += tmp;
#   if  TSUB >=  4
  tmp = __shfl_up(val,  2, TSUB);  if( lane >=  2 )    val += tmp;
#   if  TSUB >=  8
  tmp = __shfl_up(val,  4, TSUB);  if( lane >=  4 )    val += tmp;
#   if  TSUB >= 16
  tmp = __shfl_up(val,  8, TSUB);  if( lane >=  8 )    val += tmp;
#   if  TSUB == 32
  tmp = __shfl_up(val, 16, TSUB);  if( lane >= 16 )    val += tmp;
#endif//TSUB == 32
#endif//TSUB >= 16
#endif//TSUB >=  8
#endif//TSUB >=  4
#endif//TSUB >=  2
  return (val);
  //-----------------------------------------------------------------------
#else///USE_WARP_SHUFFLE_FUNC
  //-----------------------------------------------------------------------
  smem[tidx].i = psum;
#   if  TSUB >=  2
  if( lane >=  1 )    smem[tidx].i += smem[tidx -  1].i;
#   if  TSUB >=  4
  if( lane >=  2 )    smem[tidx].i += smem[tidx -  2].i;
#   if  TSUB >=  8
  if( lane >=  4 )    smem[tidx].i += smem[tidx -  4].i;
#   if  TSUB >= 16
  if( lane >=  8 )    smem[tidx].i += smem[tidx -  8].i;
#   if  TSUB == 32
  if( lane >= 16 )    smem[tidx].i += smem[tidx - 16].i;
#endif//TSUB == 32
#endif//TSUB >= 16
#endif//TSUB >=  8
#endif//TSUB >=  4
#endif//TSUB >=  2
  //-----------------------------------------------------------------------
#endif///USE_WARP_SHUFFLE_FUNC
  //-----------------------------------------------------------------------
}
//-------------------------------------------------------------------------
/* NOTE: implicit synchronization within 32 threads (a warp) is assumed */
#ifdef  USE_WARP_SHUFFLE_FUNC
__device__ __forceinline__  int prefixSumTsubMultiple(int psum,                                            const int lane, const int Niter)
#else///USE_WARP_SHUFFLE_FUNC
__device__ __forceinline__ void prefixSumTsubMultiple(int psum, volatile uint_real * smem, const int tidx, const int lane, const int Niter, const int tail)
#endif//USE_WARP_SHUFFLE_FUNC
{
  //-----------------------------------------------------------------------
#ifdef  USE_WARP_SHUFFLE_FUNC
  int smem = prefixSumTsub(psum, lane);
#else///USE_WARP_SHUFFLE_FUNC
  prefixSumTsub(psum, smem, tidx, lane);
#endif//USE_WARP_SHUFFLE_FUNC
  //-----------------------------------------------------------------------
  for(int iter = 1; iter < Niter; iter++){
#ifdef  USE_WARP_SHUFFLE_FUNC
    const uint inc = (__shfl(smem, TSUB - 1, TSUB) >> (IDX_SHIFT_BITS * (iter - 1))) & IDX_SHIFT_MASK;
    smem         += (inc << (IDX_SHIFT_BITS * iter));
#else///USE_WARP_SHUFFLE_FUNC
    const uint inc = (smem[tail].i                 >> (IDX_SHIFT_BITS * (iter - 1))) & IDX_SHIFT_MASK;
    smem[tidx].i += (inc << (IDX_SHIFT_BITS * iter));
#endif//USE_WARP_SHUFFLE_FUNC
  }
  //-----------------------------------------------------------------------
#ifdef  USE_WARP_SHUFFLE_FUNC
  return (smem);
#endif//USE_WARP_SHUFFLE_FUNC
  //-----------------------------------------------------------------------
}
//-------------------------------------------------------------------------


//-------------------------------------------------------------------------
/* maximum value within a group of TSUB threads (TSUB <= 32 to use implicit synchronization) */
/* NOTE: implicit synchronization within 32 threads (a warp) is assumed */
//-------------------------------------------------------------------------
/* continuous NWARP threads have the same value as input */
//-------------------------------------------------------------------------
__device__ __forceinline__ real getMaximumRealTsub
(
#ifdef  USE_WARP_SHUFFLE_FUNC
 const real max
#else///USE_WARP_SHUFFLE_FUNC
 real max, volatile uint_real * smem, const int tidx, const int head
#endif//USE_WARP_SHUFFLE_FUNC
 )
{
  //-----------------------------------------------------------------------
#ifdef  USE_WARP_SHUFFLE_FUNC
  //-----------------------------------------------------------------------
  real val = max;
/* #   if  TSUB >= ( 2 * NWARP) */
/*   real tmp; */
/*   tmp = __shfl_xor(val,      NWARP, TSUB);  if( tmp > val )    val = tmp; */
/* #   if  TSUB >= ( 4 * NWARP) */
/*   tmp = __shfl_xor(val,  2 * NWARP, TSUB);  if( tmp > val )    val = tmp; */
/* #   if  TSUB >= ( 8 * NWARP) */
/*   tmp = __shfl_xor(val,  4 * NWARP, TSUB);  if( tmp > val )    val = tmp; */
/* #   if  TSUB >= (16 * NWARP) */
/*   tmp = __shfl_xor(val,  8 * NWARP, TSUB);  if( tmp > val )    val = tmp; */
/* #   if  TSUB == (32 * NWARP) */
/*   tmp = __shfl_xor(val, 16 * NWARP, TSUB);  if( tmp > val )    val = tmp; */
/* #endif//TSUB == (32 * NWARP) */
/* #endif//TSUB >= (16 * NWARP) */
/* #endif//TSUB >= ( 8 * NWARP) */
/* #endif//TSUB >= ( 4 * NWARP) */
/* #endif//TSUB >= ( 2 * NWARP) */
#   if  TSUB >= ( 2 * NWARP)
  real tmp;
  tmp = __shfl_xor(val,      NWARP, TSUB);  val = FMAX(val, tmp);
#   if  TSUB >= ( 4 * NWARP)
  tmp = __shfl_xor(val,  2 * NWARP, TSUB);  val = FMAX(val, tmp);
#   if  TSUB >= ( 8 * NWARP)
  tmp = __shfl_xor(val,  4 * NWARP, TSUB);  val = FMAX(val, tmp);
#   if  TSUB >= (16 * NWARP)
  tmp = __shfl_xor(val,  8 * NWARP, TSUB);  val = FMAX(val, tmp);
#   if  TSUB == (32 * NWARP)
  tmp = __shfl_xor(val, 16 * NWARP, TSUB);  val = FMAX(val, tmp);
#endif//TSUB == (32 * NWARP)
#endif//TSUB >= (16 * NWARP)
#endif//TSUB >= ( 8 * NWARP)
#endif//TSUB >= ( 4 * NWARP)
#endif//TSUB >= ( 2 * NWARP)
  return (__shfl(val, 0, TSUB));
  //-----------------------------------------------------------------------
#else///USE_WARP_SHUFFLE_FUNC
  //-----------------------------------------------------------------------
  smem[tidx].r = max;
  //-----------------------------------------------------------------------
/* #   if  TSUB >= ( 2 * NWARP) */
/*   real tmp; */
/*   tmp = smem[tidx ^ (     NWARP)].r;  if( tmp > max ){    max = tmp;  }  smem[tidx].r = max; */
/* #   if  TSUB >= ( 4 * NWARP) */
/*   tmp = smem[tidx ^ ( 2 * NWARP)].r;  if( tmp > max ){    max = tmp;  }  smem[tidx].r = max; */
/* #   if  TSUB >= ( 8 * NWARP) */
/*   tmp = smem[tidx ^ ( 4 * NWARP)].r;  if( tmp > max ){    max = tmp;  }  smem[tidx].r = max; */
/* #   if  TSUB >= (16 * NWARP) */
/*   tmp = smem[tidx ^ ( 8 * NWARP)].r;  if( tmp > max ){    max = tmp;  }  smem[tidx].r = max; */
/* #   if  TSUB == (32 * NWARP) */
/*   tmp = smem[tidx ^ (16 * NWARP)].r;  if( tmp > max ){    max = tmp;  }  smem[tidx].r = max; */
/* #endif//TSUB == (32 * NWARP) */
/* #endif//TSUB >= (16 * NWARP) */
/* #endif//TSUB >= ( 8 * NWARP) */
/* #endif//TSUB >= ( 4 * NWARP) */
/* #endif//TSUB >= ( 2 * NWARP) */
#   if  TSUB >= ( 2 * NWARP)
  real tmp;
  tmp = smem[tidx ^ (     NWARP)].r;  max = FMAX(max, tmp);  smem[tidx].r = max;
#   if  TSUB >= ( 4 * NWARP)
  tmp = smem[tidx ^ ( 2 * NWARP)].r;  max = FMAX(max, tmp);  smem[tidx].r = max;
#   if  TSUB >= ( 8 * NWARP)
  tmp = smem[tidx ^ ( 4 * NWARP)].r;  max = FMAX(max, tmp);  smem[tidx].r = max;
#   if  TSUB >= (16 * NWARP)
  tmp = smem[tidx ^ ( 8 * NWARP)].r;  max = FMAX(max, tmp);  smem[tidx].r = max;
#   if  TSUB == (32 * NWARP)
  tmp = smem[tidx ^ (16 * NWARP)].r;  max = FMAX(max, tmp);  smem[tidx].r = max;
#endif//TSUB == (32 * NWARP)
#endif//TSUB >= (16 * NWARP)
#endif//TSUB >= ( 8 * NWARP)
#endif//TSUB >= ( 4 * NWARP)
#endif//TSUB >= ( 2 * NWARP)
  //-----------------------------------------------------------------------
  return (smem[head].r);
  //-----------------------------------------------------------------------
#endif///USE_WARP_SHUFFLE_FUNC
  //-----------------------------------------------------------------------
}
//-------------------------------------------------------------------------


//-------------------------------------------------------------------------
/* minimum value within a group of TSUB threads (TSUB <= 32 to use implicit synchronization) */
/* NOTE: implicit synchronization within 32 threads (a warp) is assumed */
//-------------------------------------------------------------------------
/* continuous NWARP threads have the same value as input */
//-------------------------------------------------------------------------
__device__ __forceinline__ real getMinimumRealTsub
(
#ifdef  USE_WARP_SHUFFLE_FUNC
 const real min
#else///USE_WARP_SHUFFLE_FUNC
 real min, volatile uint_real * smem, const int tidx, const int head
#endif//USE_WARP_SHUFFLE_FUNC
 )
{
  //-----------------------------------------------------------------------
#ifdef  USE_WARP_SHUFFLE_FUNC
  //-----------------------------------------------------------------------
  real val = min;
/* #   if  TSUB >= ( 2 * NWARP) */
/*   real tmp; */
/*   tmp = __shfl_xor(val,      NWARP, TSUB);  if( tmp < val )    val = tmp; */
/* #   if  TSUB >= ( 4 * NWARP) */
/*   tmp = __shfl_xor(val,  2 * NWARP, TSUB);  if( tmp < val )    val = tmp; */
/* #   if  TSUB >= ( 8 * NWARP) */
/*   tmp = __shfl_xor(val,  4 * NWARP, TSUB);  if( tmp < val )    val = tmp; */
/* #   if  TSUB >= (16 * NWARP) */
/*   tmp = __shfl_xor(val,  8 * NWARP, TSUB);  if( tmp < val )    val = tmp; */
/* #   if  TSUB == (32 * NWARP) */
/*   tmp = __shfl_xor(val, 16 * NWARP, TSUB);  if( tmp < val )    val = tmp; */
/* #endif//TSUB == (32 * NWARP) */
/* #endif//TSUB >= (16 * NWARP) */
/* #endif//TSUB >= ( 8 * NWARP) */
/* #endif//TSUB >= ( 4 * NWARP) */
/* #endif//TSUB >= ( 2 * NWARP) */
#   if  TSUB >= ( 2 * NWARP)
  real tmp;
  tmp = __shfl_xor(val,      NWARP, TSUB);  val = FMIN(val, tmp);
#   if  TSUB >= ( 4 * NWARP)
  tmp = __shfl_xor(val,  2 * NWARP, TSUB);  val = FMIN(val, tmp);
#   if  TSUB >= ( 8 * NWARP)
  tmp = __shfl_xor(val,  4 * NWARP, TSUB);  val = FMIN(val, tmp);
#   if  TSUB >= (16 * NWARP)
  tmp = __shfl_xor(val,  8 * NWARP, TSUB);  val = FMIN(val, tmp);
#   if  TSUB == (32 * NWARP)
  tmp = __shfl_xor(val, 16 * NWARP, TSUB);  val = FMIN(val, tmp);
#endif//TSUB == (32 * NWARP)
#endif//TSUB >= (16 * NWARP)
#endif//TSUB >= ( 8 * NWARP)
#endif//TSUB >= ( 4 * NWARP)
#endif//TSUB >= ( 2 * NWARP)
  return (__shfl(val, 0, TSUB));
  //-----------------------------------------------------------------------
#else///USE_WARP_SHUFFLE_FUNC
  //-----------------------------------------------------------------------
  smem[tidx].r = min;
  //-----------------------------------------------------------------------
/* #   if  TSUB >= ( 2 * NWARP) */
/*   real tmp; */
/*   tmp = smem[tidx ^ (     NWARP)].r;  if( tmp < min ){    min = tmp;  }  smem[tidx].r = min; */
/* #   if  TSUB >= ( 4 * NWARP) */
/*   tmp = smem[tidx ^ ( 2 * NWARP)].r;  if( tmp < min ){    min = tmp;  }  smem[tidx].r = min; */
/* #   if  TSUB >= ( 8 * NWARP) */
/*   tmp = smem[tidx ^ ( 4 * NWARP)].r;  if( tmp < min ){    min = tmp;  }  smem[tidx].r = min; */
/* #   if  TSUB >= (16 * NWARP) */
/*   tmp = smem[tidx ^ ( 8 * NWARP)].r;  if( tmp < min ){    min = tmp;  }  smem[tidx].r = min; */
/* #   if  TSUB == (32 * NWARP) */
/*   tmp = smem[tidx ^ (16 * NWARP)].r;  if( tmp < min ){    min = tmp;  }  smem[tidx].r = min; */
/* #endif//TSUB == (32 * NWARP) */
/* #endif//TSUB >= (16 * NWARP) */
/* #endif//TSUB >= ( 8 * NWARP) */
/* #endif//TSUB >= ( 4 * NWARP) */
/* #endif//TSUB >= ( 2 * NWARP) */
#   if  TSUB >= ( 2 * NWARP)
  real tmp;
  tmp = smem[tidx ^ (     NWARP)].r;  min = FMIN(min, tmp);  smem[tidx].r = min;
#   if  TSUB >= ( 4 * NWARP)
  tmp = smem[tidx ^ ( 2 * NWARP)].r;  min = FMIN(min, tmp);  smem[tidx].r = min;
#   if  TSUB >= ( 8 * NWARP)
  tmp = smem[tidx ^ ( 4 * NWARP)].r;  min = FMIN(min, tmp);  smem[tidx].r = min;
#   if  TSUB >= (16 * NWARP)
  tmp = smem[tidx ^ ( 8 * NWARP)].r;  min = FMIN(min, tmp);  smem[tidx].r = min;
#   if  TSUB == (32 * NWARP)
  tmp = smem[tidx ^ (16 * NWARP)].r;  min = FMIN(min, tmp);  smem[tidx].r = min;
#endif//TSUB == (32 * NWARP)
#endif//TSUB >= (16 * NWARP)
#endif//TSUB >= ( 8 * NWARP)
#endif//TSUB >= ( 4 * NWARP)
#endif//TSUB >= ( 2 * NWARP)
  //-----------------------------------------------------------------------
  return (smem[head].r);
  //-----------------------------------------------------------------------
#endif///USE_WARP_SHUFFLE_FUNC
  //-----------------------------------------------------------------------
}
//-------------------------------------------------------------------------


//-------------------------------------------------------------------------
__device__ __forceinline__ void copyData_s2s(uint *src, int sidx, uint *dst, int didx, const int num, const int lane)
{
  //-----------------------------------------------------------------------
  const int iter = DIV_TSUB(num);
  const int frac = num & (TSUB - 1);/* := Nload % TSUB */
  //-----------------------------------------------------------------------
  /* NOTE: implicit synchronization within 32 threads (a warp) is assumed */
  for(int kk = 0; kk < iter; kk++){
    dst[didx] = src[sidx];
    sidx += TSUB;
    didx += TSUB;
  }
  //-----------------------------------------------------------------------
  if( lane < frac )
    dst[didx] = src[sidx];
  //-----------------------------------------------------------------------
}
//-------------------------------------------------------------------------
__device__ __forceinline__ void copyData_g2s(uint * RESTRICT gbuf, size_t srcHead, uint * RESTRICT sbuf, int dstHead, int numCopy, const int lane)
{
  //-----------------------------------------------------------------------
  /* fraction processing at loading from the head of destination array */
  //-----------------------------------------------------------------------
  const int numTemp = TSUB - (int)(srcHead & (TSUB - 1));/* := TSUB - (srcHead % TSUB) */
  const int numHead = (numTemp < numCopy) ? numTemp : numCopy;
  if( lane < numHead )
    sbuf[dstHead + lane] = gbuf[srcHead + lane];
  dstHead += numHead;
  srcHead += numHead;
  numCopy -= numHead;
  //-----------------------------------------------------------------------

  //-----------------------------------------------------------------------
  /* sequential load from source on the global memory and store to destination on the shared memory */
  //-----------------------------------------------------------------------
  for(int ii = lane; ii < numCopy; ii += TSUB)
    sbuf[dstHead + ii] = gbuf[srcHead + ii];
  //-----------------------------------------------------------------------
}
//-------------------------------------------------------------------------
__device__ __forceinline__ void copyData_s2g(uint * RESTRICT sbuf, int srcHead, uint * RESTRICT gbuf, size_t dstHead, int numCopy, const int lane)
{
  //-----------------------------------------------------------------------
  /* fraction processing at storing to the head of destination array */
  //-----------------------------------------------------------------------
  const int numTemp = TSUB - (int)(dstHead & (TSUB - 1));/* := TSUB - (dstHead % TSUB) */
  const int numHead = (numTemp < numCopy) ? numTemp : numCopy;
  if( lane < numHead )
    gbuf[dstHead + lane] = sbuf[srcHead + lane];
  dstHead += numHead;
  srcHead += numHead;
  numCopy -= numHead;
  //-----------------------------------------------------------------------

  //-----------------------------------------------------------------------
  /* sequential load from source on the shared memory and store to destination on the global memory */
  //-----------------------------------------------------------------------
  for(int ii = lane; ii < numCopy; ii += TSUB)
    gbuf[dstHead + ii] = sbuf[srcHead + ii];
  //-----------------------------------------------------------------------
}
//-------------------------------------------------------------------------
__device__ __forceinline__ void copyData_g2g(uint * RESTRICT gbuf, size_t srcHead, size_t dstHead, int Ncopy, const int Ndisp, const int lane)
{
  //-----------------------------------------------------------------------
  /* configure the settings */
  //-----------------------------------------------------------------------
  const int Nfirst = Ndisp & (TSUB - 1);/* := Ndisp % TSUB */
  /* ldIdx is Nfirst, Nfirst + 1, ..., TSUB - 1, 0, 1, ..., Nfirst - 1 for lane of 0, 1, 2, ..., TSUB - 1 */
  const int  ldIdx = (lane + Nfirst) & (TSUB - 1);/* := (lane + Nfirst) % TSUB */
  const int grpIdx = (ldIdx < Nfirst) ? 0 : 1;
  //-----------------------------------------------------------------------
  srcHead += Ndisp - Nfirst;/* hereafter, srcHead is TSUB elements aligned */
  //-----------------------------------------------------------------------


  //-----------------------------------------------------------------------
  /* fraction processing at loading from the head of source array */
  //-----------------------------------------------------------------------
  uint temp = gbuf[srcHead + ldIdx];
  srcHead += TSUB;
  //-----------------------------------------------------------------------

  //-----------------------------------------------------------------------
  /* sequential load and store from source to destination on the global memory */
  //-----------------------------------------------------------------------
  const int Niter = BLOCKSIZE(Ncopy, TSUB);
  for(int iter = 0; iter < Niter; iter++){
    //---------------------------------------------------------------------
    const int Nmove = (Ncopy > TSUB) ? (TSUB) : (Ncopy);
    //---------------------------------------------------------------------
    //
    //---------------------------------------------------------------------
    /* load from the source array on the global memory */
    //---------------------------------------------------------------------
    /* load from temp (fraction processing) as initialization */
    uint local = temp;
    //---------------------------------------------------------------------
    /* load from global memory, store to shared memory or temp (fraction processing) */
    temp = gbuf[srcHead + ldIdx];
    if( !grpIdx )
      local = temp;
    //---------------------------------------------------------------------
    //
    //---------------------------------------------------------------------
    /* store to the destination array on the global memory */
    //---------------------------------------------------------------------
    gbuf[dstHead + lane] = local;
    //---------------------------------------------------------------------
    Ncopy   -= Nmove;
    srcHead += Nmove;
    dstHead += Nmove;
    //---------------------------------------------------------------------
  }/* for(int iter = 0; iter < Niter; iter++){ */
  //-----------------------------------------------------------------------
}
//-------------------------------------------------------------------------


//-------------------------------------------------------------------------
/* merge continuous tree nodes */
//-------------------------------------------------------------------------
/* uint smem[TSUB]; */
/* uint node[TSUB * NSTOCK]; */
//-------------------------------------------------------------------------
#ifdef  USE_WARP_SHUFFLE_FUNC
__device__ __forceinline__ void cpChildNodes
(                           uint * RESTRICT node, jnode jidx,
 uint leaf,                 const int lane,
 uint * RESTRICT smbuf, const    int hq, int *rem_sm, int *num_sm,
 uint * RESTRICT gmbuf, const size_t hb, int *rem_gm, int *num_gm, int *head_gm, int *tail_gm
			    )
#else///USE_WARP_SHUFFLE_FUNC
__device__ __forceinline__ void cpChildNodes
(uint_real * RESTRICT smem, uint * RESTRICT node, jnode jidx,
 uint leaf, const int tidx, const int lane, const int tail,
 uint * RESTRICT smbuf, const    int hq, int *rem_sm, int *num_sm,
 uint * RESTRICT gmbuf, const size_t hb, int *rem_gm, int *num_gm, int *head_gm, int *tail_gm
 )
#endif//USE_WARP_SHUFFLE_FUNC
{
  //-----------------------------------------------------------------------
  int iter;
  //-----------------------------------------------------------------------
  /* 1. compact the given sparse tree nodes */
  //-----------------------------------------------------------------------
#ifdef  USE_WARP_SHUFFLE_FUNC
  int smem = prefixSumTsubMultiple(leaf, lane, NSTOCK);
  uint nadd = smem         - leaf;/* exclusive prefix sum of leaf */
#else///USE_WARP_SHUFFLE_FUNC
  prefixSumTsubMultiple(leaf, smem, tidx, lane, NSTOCK, tail);
  uint nadd = smem[tidx].i - leaf;/* exclusive prefix sum of leaf */
#endif//USE_WARP_SHUFFLE_FUNC
#pragma unroll
  for(iter = 0; iter < NSTOCK; iter++){
    if( (leaf >> (IDX_SHIFT_BITS * iter)) & 1 ){
      const uint hidx = (nadd >> (IDX_SHIFT_BITS * iter)) & IDX_SHIFT_MASK;
      node[hidx] = jidx.idx[iter];
    }/* if( (leaf >> (IDX_SHIFT_BITS * iter)) & 1 ){ */
    jidx.idx[iter] = NULL_NODE;
  }/* for(iter = 0; iter < NSTOCK; iter++){ */
  //-----------------------------------------------------------------------
#ifdef  USE_WARP_SHUFFLE_FUNC
  const int nold = (int)((__shfl(smem, TSUB - 1, TSUB) >> (IDX_SHIFT_BITS * (NSTOCK - 1))) & IDX_SHIFT_MASK);
#else///USE_WARP_SHUFFLE_FUNC
  const int nold = (int)((       smem[tail].i          >> (IDX_SHIFT_BITS * (NSTOCK - 1))) & IDX_SHIFT_MASK);
#endif//USE_WARP_SHUFFLE_FUNC
  for(int ii = nold + lane; ii < NSTOCK * TSUB; ii += TSUB)
    node[ii] = NULL_NODE;
  //-----------------------------------------------------------------------


  //-----------------------------------------------------------------------
  int Ntot;
  //-----------------------------------------------------------------------
#ifdef  MERGE_QUEUED_TREE_NODES
  const int Niter = BLOCKSIZE(nold, TSUB);
  if( Niter != NSTOCK )
#endif//MERGE_QUEUED_TREE_NODES
#ifdef  USE_WARP_SHUFFLE_FUNC
    Ntot = (int)((__shfl(smem, TSUB - 1, TSUB) >> (IDX_SHIFT_BITS * (NSTOCK - 1))) & IDX_SHIFT_MASK);
#else///USE_WARP_SHUFFLE_FUNC
    Ntot = (int)((       smem[tail].i          >> (IDX_SHIFT_BITS * (NSTOCK - 1))) & IDX_SHIFT_MASK);
#endif//USE_WARP_SHUFFLE_FUNC
#ifdef  MERGE_QUEUED_TREE_NODES
  else{
    //---------------------------------------------------------------------
#if 1
    /* partial, faster version */
    //---------------------------------------------------------------------
    /* 2. examine continuity of the given tree nodes */
    /* 3. construct merged tree nodes */
    //---------------------------------------------------------------------
    iter = 0;
    leaf = 0;
#pragma unroll
    for(int ii = 2 * lane; ii < nold; ii += 2 * TSUB){
      //-------------------------------------------------------------------
      jidx.idx[2 * iter    ] = node[ii    ];
      jidx.idx[2 * iter + 1] = node[ii + 1];
      //-------------------------------------------------------------------
      const uint  numFormer = (jidx.idx[2 * iter    ] >> IDXBITS) + 1;
      const uint  numLatter = (jidx.idx[2 * iter + 1] >> IDXBITS) + 1;
      const uint tailFormer = (jidx.idx[2 * iter    ] &  IDXMASK) + numFormer;/* := tail index + 1 */
      const uint headLatter =  jidx.idx[2 * iter + 1] &  IDXMASK;
      //-------------------------------------------------------------------
      if( (tailFormer == headLatter) && ((numFormer + numLatter) <= NLEAF) ){
	jidx.idx[2 * iter    ] += (numLatter << IDXBITS);
	jidx.idx[2 * iter + 1]  = NULL_NODE;
      }/* if( (tailFormer == headLatter) && ((numFormer + numLatter) <= NLEAF) ){ */
      //-------------------------------------------------------------------
      uint numNodes = 0;
#pragma unroll
      for(int jj = 0; jj < 2; jj++)
	numNodes += (jidx.idx[2 * iter + jj] != NULL_NODE);
      leaf += (numNodes << (IDX_SHIFT_BITS * iter));
      //-------------------------------------------------------------------
      iter++;
      //-------------------------------------------------------------------
    }/* for(int ii = 2 * lane; ii < nold; ii += 2 * TSUB){ */
    //---------------------------------------------------------------------

    //---------------------------------------------------------------------
    /* 4. count up number of reconstructed tree nodes */
    //---------------------------------------------------------------------
    const int Nloop = BLOCKSIZE(nold, 2 * TSUB);
#ifdef  USE_WARP_SHUFFLE_FUNC
    smem = prefixSumTsubMultiple(leaf, lane, Nloop);
#else///USE_WARP_SHUFFLE_FUNC
    prefixSumTsubMultiple(leaf, smem, tidx, lane, Nloop, tail);
#endif//USE_WARP_SHUFFLE_FUNC
    //---------------------------------------------------------------------
#ifdef  USE_WARP_SHUFFLE_FUNC
    Ntot = (int)((__shfl(smem, TSUB - 1, TSUB) >> (IDX_SHIFT_BITS * (Nloop - 1))) & IDX_SHIFT_MASK);
#else///USE_WARP_SHUFFLE_FUNC
    Ntot = (int)((       smem[tail].i          >> (IDX_SHIFT_BITS * (Nloop - 1))) & IDX_SHIFT_MASK);
#endif//USE_WARP_SHUFFLE_FUNC
    //---------------------------------------------------------------------
    for(int ii = Ntot + lane; ii < nold; ii += TSUB)
      node[ii] = NULL_NODE;
    //---------------------------------------------------------------------

    //---------------------------------------------------------------------
    /* 5. set the reconstructed tree nodes on the shared memory */
    //---------------------------------------------------------------------
#ifdef  USE_WARP_SHUFFLE_FUNC
    smem         -= leaf;/* exclusive prefix sum */
#else///USE_WARP_SHUFFLE_FUNC
    smem[tidx].i -= leaf;/* exclusive prefix sum */
#endif//USE_WARP_SHUFFLE_FUNC
#pragma unroll
    for(int ii = 0; ii < Nloop; ii++){
      //-------------------------------------------------------------------
      const int  numNodes = (int)((leaf         >> (IDX_SHIFT_BITS * ii)) & IDX_SHIFT_MASK);
#ifdef  USE_WARP_SHUFFLE_FUNC
      const int headNodes = (int)((smem         >> (IDX_SHIFT_BITS * ii)) & IDX_SHIFT_MASK);
#else///USE_WARP_SHUFFLE_FUNC
      const int headNodes = (int)((smem[tidx].i >> (IDX_SHIFT_BITS * ii)) & IDX_SHIFT_MASK);
#endif//USE_WARP_SHUFFLE_FUNC
      //-------------------------------------------------------------------
#pragma unroll
      for(int jj = 0; jj < numNodes; jj++)
	node[headNodes + jj] = jidx.idx[2 * ii + jj];
      //-------------------------------------------------------------------
    }/* for(int ii = 0; ii < Nloop; ii++){ */
    //---------------------------------------------------------------------
#else
    /* complete, slower version */
    //---------------------------------------------------------------------
    /* 2. examine continuity of the given tree nodes */
    //---------------------------------------------------------------------
    iter = 0;
    leaf = 0;
#pragma unroll
    for(int ii = lane; ii < nold; ii += TSUB){
      //-------------------------------------------------------------------
      jidx.idx[iter] = node[ii];
      //-------------------------------------------------------------------
      if( ii != 0 ){
	//-----------------------------------------------------------------
	uint tail_id = node[ii - 1];
	const uint num  = 1 + (tail_id >> IDXBITS);
	tail_id = (tail_id & IDXMASK) + num;/* := tail index + 1 */
	//-----------------------------------------------------------------
	leaf += ((tail_id == (jidx.idx[iter] & IDXMASK)) << (IDX_SHIFT_BITS * iter));
	//-----------------------------------------------------------------
      }/* if( ii != 0 ){ */
      //-------------------------------------------------------------------
      iter++;
      //-------------------------------------------------------------------
    }/* for(int ii = lane; ii < nold; ii += TSUB){ */
    //---------------------------------------------------------------------
#ifdef  USE_WARP_SHUFFLE_FUNC
    smem = prefixSumTsubMultiple(leaf, lane, Niter);
#else///USE_WARP_SHUFFLE_FUNC
    prefixSumTsubMultiple(leaf, smem, tidx, lane, Niter, tail);
#endif//USE_WARP_SHUFFLE_FUNC
    //---------------------------------------------------------------------
#ifdef  USE_WARP_SHUFFLE_FUNC
    const int nnew = nold - (int)((__shfl(smem, TSUB - 1, TSUB) >> (IDX_SHIFT_BITS * (Niter - 1))) & IDX_SHIFT_MASK);
#else///USE_WARP_SHUFFLE_FUNC
    const int nnew = nold - (int)((       smem[tail].i          >> (IDX_SHIFT_BITS * (Niter - 1))) & IDX_SHIFT_MASK);
#endif//USE_WARP_SHUFFLE_FUNC
    const int Nloop = BLOCKSIZE(nnew, TSUB);
    //---------------------------------------------------------------------
    for(int ii = nnew + lane; ii < nold; ii += TSUB)
      node[ii] = NULL_NODE;
    //---------------------------------------------------------------------

    //---------------------------------------------------------------------
    /* 3. construct merged tree nodes */
    //---------------------------------------------------------------------
    /* store head index to the shared memory */
    iter = 0;
#pragma unroll
    for(int ii = lane; ii < nold; ii += TSUB){
      if( !((leaf >> (IDX_SHIFT_BITS * iter)) & 1) )
#ifdef  USE_WARP_SHUFFLE_FUNC
	node[ii - (int)((smem         >> (IDX_SHIFT_BITS * iter)) & IDX_SHIFT_MASK)] = jidx.idx[iter] & IDXMASK;
#else///USE_WARP_SHUFFLE_FUNC
	node[ii - (int)((smem[tidx].i >> (IDX_SHIFT_BITS * iter)) & IDX_SHIFT_MASK)] = jidx.idx[iter] & IDXMASK;
#endif//USE_WARP_SHUFFLE_FUNC
      iter++;
    }/* for(int ii = lane; ii < nold; ii += TSUB){ */
    //---------------------------------------------------------------------
    /* load head index from the shared memory */
    jnode head;
    iter = 0;
#pragma unroll
    for(int ii = lane; ii < Nloop * TSUB; ii += TSUB){
      head.idx[iter] = node[ii];
      node[ii] = 0;
      iter++;
    }/* for(int ii = lane; ii < Nloop * TSUB; ii += TSUB){ */
    //---------------------------------------------------------------------
    /* sum up number of child nodes */
    /* TENTATIVE IMPLEMENTATION: in future update, atomic operation should be removed */
    iter = 0;
#pragma unroll
    for(int ii = lane; ii < nold; ii += TSUB){
      if( jidx.idx[iter] != NULL_NODE )
#ifdef  USE_WARP_SHUFFLE_FUNC
	atomicAdd(&node[ii - (int)((smem         >> (IDX_SHIFT_BITS * iter)) & IDX_SHIFT_MASK)], 1 + (jidx.idx[iter] >> IDXBITS));
#else///USE_WARP_SHUFFLE_FUNC
	atomicAdd(&node[ii - (int)((smem[tidx].i >> (IDX_SHIFT_BITS * iter)) & IDX_SHIFT_MASK)], 1 + (jidx.idx[iter] >> IDXBITS));
#endif//USE_WARP_SHUFFLE_FUNC
      iter++;
    }/* for(int ii = lane; ii < nold; ii += TSUB){ */
    //---------------------------------------------------------------------
    /* load number of child nodes from the shared memory */
    jnode num;
    iter = 0;
#pragma unroll
    for(int ii = lane; ii < Nloop * TSUB; ii += TSUB){
      num.idx[iter] = node[ii];
      iter++;
    }/* for(int ii = lane; ii < Nloop * TSUB; ii += TSUB){ */
    //---------------------------------------------------------------------

    //---------------------------------------------------------------------
    /* 4. count up number of reconstructed tree nodes */
    //---------------------------------------------------------------------
    nadd = 0;
#pragma unroll
    for(iter = 0; iter < Nloop; iter++)
      nadd += ((uint)(BLOCKSIZE(num.idx[iter], NLEAF)) << (IDX_SHIFT_BITS * iter));
#ifdef  USE_WARP_SHUFFLE_FUNC
    smem = prefixSumTsubMultiple(nadd, lane, Nloop);
#else///USE_WARP_SHUFFLE_FUNC
    prefixSumTsubMultiple(nadd, smem, tidx, lane, Nloop, tail);
#endif//USE_WARP_SHUFFLE_FUNC
    //---------------------------------------------------------------------
#ifdef  USE_WARP_SHUFFLE_FUNC
    Ntot = (int)((__shfl(smem, TSUB - 1, TSUB) >> (IDX_SHIFT_BITS * (Nloop - 1))) & IDX_SHIFT_MASK);
#else///USE_WARP_SHUFFLE_FUNC
    Ntot = (int)((       smem[tail].i          >> (IDX_SHIFT_BITS * (Nloop - 1))) & IDX_SHIFT_MASK);
#endif//USE_WARP_SHUFFLE_FUNC
    //---------------------------------------------------------------------
#pragma unroll
    for(int ii = Ntot + lane; ii < Nloop * TSUB; ii += TSUB)
      node[ii] = NULL_NODE;
    //---------------------------------------------------------------------

    //---------------------------------------------------------------------
    /* 5. set the reconstructed tree nodes on the shared memory */
    //---------------------------------------------------------------------
#ifdef  USE_WARP_SHUFFLE_FUNC
    smem         -= nadd;/* exclusive prefix sum */
#else///USE_WARP_SHUFFLE_FUNC
    smem[tidx].i -= nadd;/* exclusive prefix sum */
#endif//USE_WARP_SHUFFLE_FUNC
    iter = 0;
    for(int ii = lane; ii < nnew; ii += TSUB){
      //-------------------------------------------------------------------
      const int nsplit = (int)((nadd         >> (IDX_SHIFT_BITS * iter)) & IDX_SHIFT_MASK);
#ifdef  USE_WARP_SHUFFLE_FUNC
      const int hidx   = (int)((smem         >> (IDX_SHIFT_BITS * iter)) & IDX_SHIFT_MASK);
#else///USE_WARP_SHUFFLE_FUNC
      const int hidx   = (int)((smem[tidx].i >> (IDX_SHIFT_BITS * iter)) & IDX_SHIFT_MASK);
#endif//USE_WARP_SHUFFLE_FUNC
      //-------------------------------------------------------------------
      uint idx_node = head.idx[iter];
      uint rem_node =  num.idx[iter];
      //-------------------------------------------------------------------
      for(int jj = 0; jj < nsplit; jj++){
	//-----------------------------------------------------------------
	const uint num_node = (rem_node < NLEAF) ? rem_node : NLEAF;
	node[hidx + jj] = ((num_node - 1) << IDXBITS) + idx_node;
	//-----------------------------------------------------------------
	idx_node += num_node;
	rem_node -= num_node;
	//-----------------------------------------------------------------
      }/* for(int jj = 0; jj < nsplit; jj++){ */
      //-------------------------------------------------------------------
      iter++;
      //-------------------------------------------------------------------
    }/* for(int ii = lane; ii < nnew; ii += TSUB){ */
    //---------------------------------------------------------------------
#endif
    //---------------------------------------------------------------------
  }/* else{ */
  //-----------------------------------------------------------------------
#endif//MERGE_QUEUED_TREE_NODES
  //-----------------------------------------------------------------------


  //-----------------------------------------------------------------------
  /* 6. copy merged tree nodes to the shared memory */
  //-----------------------------------------------------------------------
  const int Nsm = (Ntot < *rem_sm) ? (Ntot) : (*rem_sm);
  copyData_s2s(node, lane, smbuf, hq + (*num_sm), Nsm, lane);
  //-----------------------------------------------------------------------
  *num_sm += Nsm;
  *rem_sm -= Nsm;
  Ntot    -= Nsm;
  //-----------------------------------------------------------------------

  //-----------------------------------------------------------------------
  /* 7. move tree nodes on the global memory, if necessary */
  //-----------------------------------------------------------------------
  if( Ntot > *rem_gm ){
    //---------------------------------------------------------------------
    copyData_g2g(gmbuf, hb, hb, *num_gm, *head_gm, lane);
    //---------------------------------------------------------------------
    * rem_gm += *head_gm;
    *tail_gm -= *head_gm;
    *head_gm  = 0;
    //---------------------------------------------------------------------
  }/* if( Ntot > *rem_gm ){ */
  //-----------------------------------------------------------------------

  //-----------------------------------------------------------------------
  /* 8. copy merged tree nodes to the global memory */
  //-----------------------------------------------------------------------
  copyData_s2g(node, Nsm, gmbuf, hb + (*tail_gm), Ntot, lane);
  //-----------------------------------------------------------------------
  * rem_gm -= Ntot;
  * num_gm += Ntot;
  *tail_gm += Ntot;
  //-----------------------------------------------------------------------
}
//-------------------------------------------------------------------------


//-------------------------------------------------------------------------
/* calculate body--body interaction based on direct summation */
//-------------------------------------------------------------------------
/* pi   :: input          :: position and eps2 of N-body particles */
/* ai   :: input / output :: acceleration and potential of N-body particles */
/* jpos :: input          :: position and mass of N-body particles */
//-------------------------------------------------------------------------
__device__ __forceinline__ void calc_interaction
(const position pi, acceleration * RESTRICT ai, jparticle * RESTRICT jpos
#ifdef  INDIVIDUAL_GRAVITATIONAL_SOFTENING
 , real * RESTRICT eps2
#endif//INDIVIDUAL_GRAVITATIONAL_SOFTENING
#ifdef  ACCURATE_ACCUMULATION
 , acceleration * RESTRICT res
#endif//ACCURATE_ACCUMULATION
#ifdef  IJ_PARALLELIZATION
 , const int lane
#endif//IJ_PARALLELIZATION
)
{
  //-----------------------------------------------------------------------
#ifdef  PARTIAL_SUM_ACCELERATION
#       ifdef  ACCURATE_PARTIAL_SUM
  acceleration res_loc = {ZERO, ZERO, ZERO, ZERO};
#       endif//ACCURATE_PARTIAL_SUM
  acceleration acc     = {ZERO, ZERO, ZERO, ZERO};
#endif//PARTIAL_SUM_ACCELERATION
  //-----------------------------------------------------------------------
#if 0
#       if DIV_NWARP(NLOOP * TSUB) == 128
#pragma unroll 64
#       endif
#       if DIV_NWARP(NLOOP * TSUB) == 96
#pragma unroll 48
#       endif
#       if DIV_NWARP(NLOOP * TSUB) == 64
#pragma unroll 32
#       endif
#       if DIV_NWARP(NLOOP * TSUB) == 48
#pragma unroll 24
#       endif
#       if DIV_NWARP(NLOOP * TSUB) == 32
#pragma unroll 16
#       endif
#       if DIV_NWARP(NLOOP * TSUB) == 24
#pragma unroll 12
#       endif
#       if DIV_NWARP(NLOOP * TSUB) == 16
#pragma unroll  8
#       endif
#       if DIV_NWARP(NLOOP * TSUB) == 12
#pragma unroll  6
#       endif
#       if DIV_NWARP(NLOOP * TSUB) ==  8
#pragma unroll  4
#       endif
#       if DIV_NWARP(NLOOP * TSUB) ==  6
#pragma unroll  3
#       endif
#       if DIV_NWARP(NLOOP * TSUB) ==  4
#pragma unroll  2
#       endif
#else
#pragma unroll
#endif
#ifdef  IJ_PARALLELIZATION
  for(int jj = lane; jj < NLOOP * TSUB; jj += NWARP)
#else///IJ_PARALLELIZATION
  for(int jj = 0; jj < NLOOP * TSUB; jj++)
#endif//IJ_PARALLELIZATION
    {
      //-------------------------------------------------------------------
      /* load j-particle from shared memory */
      jparticle pj = jpos[jj];
      //-------------------------------------------------------------------
      /* calculate distance between j-particel and i-particle */
      const real rx = pj.x - pi.x;
      const real ry = pj.y - pi.y;
      const real rz = pj.z - pi.z;
#ifdef  INDIVIDUAL_GRAVITATIONAL_SOFTENING
      const real r2 = eps2[jj] + rx * rx + ry * ry + rz * rz;
#else///INDIVIDUAL_GRAVITATIONAL_SOFTENING
      const real r2 = pi.m     + rx * rx + ry * ry + rz * rz;
#endif//INDIVIDUAL_GRAVITATIONAL_SOFTENING
      real rinv = RSQRT(r2);
      //-------------------------------------------------------------------
      /* calculate common factor for all direction */
      pj.w *= rinv;/* mj / r */
      rinv *= rinv;/* 1  / r^2 */
      rinv *= pj.w;/* mj / r^3 */
      //-------------------------------------------------------------------
      /* calculate gravitational acceleration of i-particle */
#ifdef  PARTIAL_SUM_ACCELERATION
#       ifdef  ACCURATE_PARTIAL_SUM
      /* R := R + x_i */
      res_loc.x   += rx * rinv;
      res_loc.y   += ry * rinv;
      res_loc.z   += rz * rinv;
      res_loc.pot += r2 * rinv;
      /* T := S */
      acceleration tmp_loc = acc;
      /* S := S + R */
      acc.x   += res_loc.x;
      acc.y   += res_loc.y;
      acc.z   += res_loc.z;
      acc.pot += res_loc.pot;
      /* T := S - T */
      tmp_loc.x   = acc.x   - tmp_loc.x;
      tmp_loc.y   = acc.y   - tmp_loc.y;
      tmp_loc.z   = acc.z   - tmp_loc.z;
      tmp_loc.pot = acc.pot - tmp_loc.pot;
      /* R := R - T */
      res_loc.x   -= tmp_loc.x;
      res_loc.y   -= tmp_loc.y;
      res_loc.z   -= tmp_loc.z;
      res_loc.pot -= tmp_loc.pot;
#       else///ACCURATE_PARTIAL_SUM
      acc.x   += rx * rinv;
      acc.y   += ry * rinv;
      acc.z   += rz * rinv;
      acc.pot += r2 * rinv;/* if necessary */
#       endif//ACCURATE_PARTIAL_SUM
#else///PARTIAL_SUM_ACCELERATION
      ai->x   += rx * rinv;
      ai->y   += ry * rinv;
      ai->z   += rz * rinv;
      ai->pot += r2 * rinv;/* if necessary */
#endif//PARTIAL_SUM_ACCELERATION
      //-------------------------------------------------------------------
    }
  //-----------------------------------------------------------------------

  //-----------------------------------------------------------------------
#ifdef  PARTIAL_SUM_ACCELERATION
  //-----------------------------------------------------------------------
#ifdef  ACCURATE_ACCUMULATION
  //-----------------------------------------------------------------------
  /* R := R + x_i */
  res->x   += acc.x;
  res->y   += acc.y;
  res->z   += acc.z;
  res->pot += acc.pot;
  /* T := S */
  acceleration tmp = *ai;
  /* S := S + R */
  ai->x   += res->x;
  ai->y   += res->y;
  ai->z   += res->z;
  ai->pot += res->pot;
  /* T := S - T */
  tmp.x   = ai->x   - tmp.x;
  tmp.y   = ai->y   - tmp.y;
  tmp.z   = ai->z   - tmp.z;
  tmp.pot = ai->pot - tmp.pot;
  /* R := R - T */
  res->x   -= tmp.x;
  res->y   -= tmp.y;
  res->z   -= tmp.z;
  res->pot -= tmp.pot;
  //-----------------------------------------------------------------------
#else///ACCURATE_ACCUMULATION
  //-----------------------------------------------------------------------
#ifdef  ACCURATE_PARTIAL_SUM
  acc.x   += res_loc.x;
  acc.y   += res_loc.y;
  acc.z   += res_loc.z;
  acc.pot += res_loc.pot;
#endif//ACCURATE_PARTIAL_SUM
  ai->x   += acc.x;
  ai->y   += acc.y;
  ai->z   += acc.z;
  ai->pot += acc.pot;
  //-----------------------------------------------------------------------
#endif//ACCURATE_ACCUMULATION
  //-----------------------------------------------------------------------
#endif//PARTIAL_SUM_ACCELERATION
  //-----------------------------------------------------------------------
}
//-------------------------------------------------------------------------


//-------------------------------------------------------------------------
#ifdef  COMPARE_WITH_DIRECT_SOLVER
__global__ void calcAccDirect_kernel
(position *ipos, acceleration * RESTRICT iacc, position *jpos, const int Nj
#ifdef  INDIVIDUAL_GRAVITATIONAL_SOFTENING
 , const real eps2_val
#endif//INDIVIDUAL_GRAVITATIONAL_SOFTENING
)
{
  //-----------------------------------------------------------------------
  /* identify thread properties */
  //-----------------------------------------------------------------------
  const int tidx = THREADIDX_X1D;
  //-----------------------------------------------------------------------

  //-----------------------------------------------------------------------
  /* load poisition of an i-particle */
  //-----------------------------------------------------------------------
  const int idx = GLOBALIDX_X1D;
  position     pi = ipos[idx];
#ifndef INDIVIDUAL_GRAVITATIONAL_SOFTENING
  pi.m = eps2;
#endif//INDIVIDUAL_GRAVITATIONAL_SOFTENING
  acceleration ai = {ZERO, ZERO, ZERO, ZERO};
  //-----------------------------------------------------------------------
#ifdef  ACCURATE_ACCUMULATION
  acceleration res = {ZERO, ZERO, ZERO, ZERO};
#endif//ACCURATE_ACCUMULATION
  //-----------------------------------------------------------------------

  //-----------------------------------------------------------------------
  const position massless = {ZERO, ZERO, ZERO, ZERO};
  //-----------------------------------------------------------------------
  __shared__ jparticle pj[NTHREADS * NLOOP];
#ifdef  INDIVIDUAL_GRAVITATIONAL_SOFTENING
  __shared__      real eps2[NTHREADS * NLOOP];
#endif//INDIVIDUAL_GRAVITATIONAL_SOFTENING
  //-----------------------------------------------------------------------
  for(int jj = 0; jj < Nj; jj += NTHREADS * NLOOP){
    //---------------------------------------------------------------------
    __syncthreads();
    for(int ll = 0; ll < NLOOP; ll++){
      //-------------------------------------------------------------------
      position pj_loc = (jj + NTHREADS * ll + tidx < Nj) ? jpos[jj + NTHREADS * ll + tidx] : massless;
      //-------------------------------------------------------------------
      jparticle pj_tmp;
      pj_tmp.x = pj_loc.x;
      pj_tmp.y = pj_loc.y;
      pj_tmp.z = pj_loc.z;
      pj_tmp.w = pj_loc.m;
      //-------------------------------------------------------------------
      pj  [NTHREADS * ll + tidx] = pj_tmp;
#ifdef  INDIVIDUAL_GRAVITATIONAL_SOFTENING
      eps2[NTHREADS * ll + tidx] = eps2_val;
#endif//INDIVIDUAL_GRAVITATIONAL_SOFTENING
      //-------------------------------------------------------------------
    }
    __syncthreads();
    //---------------------------------------------------------------------
#pragma unroll
    for(int kk = 0; kk < NTHREADS * NLOOP; kk += TSUB * NLOOP)
#ifdef  IJ_PARALLELIZATION
#pragma unroll
      for(int ll = 0; ll < NWARP; ll++)
#endif//IJ_PARALLELIZATION
      calc_interaction
	(pi, &ai, &pj[kk]
#ifdef  INDIVIDUAL_GRAVITATIONAL_SOFTENING
	 , &eps2[kk]
#endif//INDIVIDUAL_GRAVITATIONAL_SOFTENING
#ifdef  ACCURATE_ACCUMULATION
	 , &res
#endif//ACCURATE_ACCUMULATION
#ifdef  IJ_PARALLELIZATION
	 , ll
#endif//IJ_PARALLELIZATION
	 );
    //---------------------------------------------------------------------
  }
  //-----------------------------------------------------------------------
#ifdef  ACCURATE_ACCUMULATION
  ai.x   += res.x;
  ai.y   += res.y;
  ai.z   += res.z;
  ai.pot += res.pot;
#endif//ACCURATE_ACCUMULATION
  //-----------------------------------------------------------------------

  //-----------------------------------------------------------------------
  /* store acceleration of an i-particle from each thread */
  //-----------------------------------------------------------------------
#if 1
  iacc[idx] = ai;
#else
  atomicAdd(&(iacc[idx].x  ), ai.x  );
  atomicAdd(&(iacc[idx].y  ), ai.y  );
  atomicAdd(&(iacc[idx].z  ), ai.z  );
  atomicAdd(&(iacc[idx].pot), ai.pot);
#endif
  //-----------------------------------------------------------------------
}
#endif//COMPARE_WITH_DIRECT_SOLVER
//-------------------------------------------------------------------------


//-------------------------------------------------------------------------
#if 1
//-------------------------------------------------------------------------
#include "buf_inc.cu"
//-------------------------------------------------------------------------
#endif
//-------------------------------------------------------------------------


//-------------------------------------------------------------------------
/* calculate gravitational acceleration based on the width-first tree traversal */
//-------------------------------------------------------------------------
/* laneInfo :: input          :: head index and number of ``active'' i-particles */
/* ipos     :: input          :: position and mass of N-body particles */
/* iacc     :: input / output :: acceleration and potential of N-body particles */
/* more     :: input          :: head index and number of child particles of the corresponding j-particle */
/* root     :: input          :: index of the root tree node */
/* jpos     :: input          :: position and squared radius of pseudo N-body particle as j-particles */
/* mj       :: input          :: mass of pseudo N-body particle as j-particles */
/* active   ::                :: a shared value to lock the shared quantities (freeNum, freeLst) to control usage of buffer */
/* freeNum  ::                :: an unsigned integer represents # of unused bufferes */
/* freeLst  ::                :: a list of unused bufferes */
/* buffer   ::                :: tentative memory space to store tree cells which does not fit within the limited space of the shared memory */
/* bufSize  :: input          :: size of the buffer */
/* overflow ::         output :: a variable to detect buffer overflow */
//-------------------------------------------------------------------------
#   if  defined(ADOPT_SMALLEST_ENCLOSING_BALL) || defined(ADOPT_APPROXIMATED_ENCLOSING_BALL)
#include "../tree/seb_dev.cu"
#endif//defined(ADOPT_SMALLEST_ENCLOSING_BALL) || defined(ADOPT_APPROXIMATED_ENCLOSING_BALL)
//-------------------------------------------------------------------------
#   if  defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 600)
/* after Pascal generation, native atomicAdd for FP64 is provided */
__device__ __forceinline__ double atomicAdd(double* address, double val)
{
  unsigned long long int* address_as_ull = (unsigned long long int*)address;
  unsigned long long int old = *address_as_ull, assumed;
  do {
    assumed = old;
    old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val + __longlong_as_double(assumed)));
  } while (assumed != old);
  return __longlong_as_double(old);
}
#endif//defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 600)
/* __device__ __forceinline__ float atomicAdd(float* addr, double val) */
/* { */
/*   unsigned int* addr_as_ui = (unsigned int*)addr; */
/*   unsigned int old = *addr_as_ui, assumed; */
/*   do { */
/*     assumed = old; */
/*     old = atomicCAS(addr_as_ui, assumed, __float_as_int((float)(val + (double)__int_as_float(assumed)))); */
/*   } while (assumed != old); */
/*   return __int_as_float(old); */
/* } */
/* __device__ __forceinline__ float atomicPrecAdd(float* addr, double val) */
/* { */
/*   int* addr_as_i = (int*)addr; */
/*   int old = *addr_as_i, assumed; */
/*   do { */
/*     assumed = old; */
/*     old = atomicCAS(addr_as_i, assumed, __float_as_int((float)(val + (double)__int_as_float(assumed)))); */
/*   } while (assumed != old); */
/*   return __int_as_float(old); */
/* } */
//-------------------------------------------------------------------------
__global__ void __launch_bounds__(NTHREADS, NBLOCKS_PER_SM) calcAcc_kernel
     (READ_ONLY laneinfo * RESTRICT laneInfo, READ_ONLY position * RESTRICT ipos, jnode * RESTRICT iacc,
#ifdef  GADGET_MAC
      READ_ONLY acceleration * RESTRICT iacc_old,
#endif//GADGET_MAC
      const int root, READ_ONLY uint * RESTRICT more, READ_ONLY jparticle * RESTRICT jpos, READ_ONLY jmass * RESTRICT mj,
#ifdef  DPADD_FOR_ACC
      DPacc * RESTRICT dacc,
#endif//DPADD_FOR_ACC
#   if  defined(KAHAN_SUM_CORRECTION) && defined(ACCURATE_ACCUMULATION) && (!defined(SERIALIZED_EXECUTION) || (NWARP > 1))
      jnode * RESTRICT ires,
#endif//defined(KAHAN_SUM_CORRECTION) && defined(ACCURATE_ACCUMULATION) && (!defined(SERIALIZED_EXECUTION) || (NWARP > 1))
#   if  !defined(USE_SMID_TO_GET_BUFID) && !defined(TRY_MODE_ABOUT_BUFFER)
      int * RESTRICT active, uint * RESTRICT freeNum,
#endif//!defined(USE_SMID_TO_GET_BUFID) && !defined(TRY_MODE_ABOUT_BUFFER)
#   if  !defined(USE_SMID_TO_GET_BUFID) &&  defined(TRY_MODE_ABOUT_BUFFER)
      const int freeNum,
#endif//!defined(USE_SMID_TO_GET_BUFID) &&  defined(TRY_MODE_ABOUT_BUFFER)
      uint * RESTRICT freeLst, uint * RESTRICT buffer, const int bufSize, int * RESTRICT overflow
#   if  !defined(USE_CUDA_EVENT) && (!defined(SERIALIZED_EXECUTION) || defined(PRINT_PSEUDO_PARTICLE_INFO))
      , unsigned long long int * RESTRICT cycles
#endif//!defined(USE_CUDA_EVENT) && (!defined(SERIALIZED_EXECUTION) || defined(PRINT_PSEUDO_PARTICLE_INFO))
#ifdef  COUNT_INTERACTIONS
      , int * RESTRICT stockNj, int * RESTRICT stockNbuf
#endif//COUNT_INTERACTIONS
      )
{
  //-----------------------------------------------------------------------
  /* start stop watch */
  //-----------------------------------------------------------------------
#   if  !defined(USE_CUDA_EVENT) && !defined(SERIALIZED_EXECUTION) && !defined(PRINT_PSEUDO_PARTICLE_INFO)
  const long long int initCycle = clock64();
#endif//!defined(USE_CUDA_EVENT) && !defined(SERIALIZED_EXECUTION) && !defined(PRINT_PSEUDO_PARTICLE_INFO)
  //-----------------------------------------------------------------------

  //-----------------------------------------------------------------------
  /* identify thread properties */
  //-----------------------------------------------------------------------
  const int tidx = THREADIDX_X1D;
  const int lane = tidx & (TSUB - 1);/* index of the thread within a thread group */
  //-----------------------------------------------------------------------
  const int head = tidx - lane;
  const int tail = head + (TSUB - 1);
  //-----------------------------------------------------------------------


  //-----------------------------------------------------------------------
  /* shared quantities in the thread parallelized version */
  //-----------------------------------------------------------------------
  __shared__ jnode   pj[NTHREADS * (NLOOP + 1)];
#ifdef  INDIVIDUAL_GRAVITATIONAL_SOFTENING
  __shared__ real  eps2[NTHREADS * (NLOOP + 1)];
#endif//INDIVIDUAL_GRAVITATIONAL_SOFTENING
  __shared__ uint queue[NTHREADS * NQUEUE];
  //-----------------------------------------------------------------------
#if 0
  for(int ii = tidx; ii < NTHREADS * (NLOOP + 1); ii += NTHREADS){
    const position massless = {ZERO, ZERO, ZERO, ZERO};
    pj[ii].pi = massless;
#ifdef  INDIVIDUAL_GRAVITATIONAL_SOFTENING
    eps2[ii] = ZERO;
#endif//INDIVIDUAL_GRAVITATIONAL_SOFTENING
  }
  for(int ii = tidx; ii < NTHREADS * NQUEUE; ii += NTHREADS)
    queue[ii] = NULL_NODE;
  __syncthreads();
#endif
  //-----------------------------------------------------------------------
  /* const int hq = lane + (head / TSUB) * TSUB * NQUEUE;/\* head index of the shared array close and queue within a thread group *\/ */
  /* const int hp =        (head / TSUB) * TSUB * (NLOOP + 1);/\* head index of the shared array pj within a thread group *\/ */
  const int hq = lane + DIV_TSUB(head) * TSUB * NQUEUE;/* head index of the shared array close and queue within a thread group */
  const int hp =        DIV_TSUB(head) * TSUB * (NLOOP + 1);/* head index of the shared array pj within a thread group */
  //-----------------------------------------------------------------------

  //-----------------------------------------------------------------------
  /* shared values within the threads */
  //-----------------------------------------------------------------------
  /* to store prefix sum */
#ifdef  USE_WARP_SHUFFLE_FUNC
  int smem;
#else///USE_WARP_SHUFFLE_FUNC
  __shared__ uint_real smem[NTHREADS];
#endif//USE_WARP_SHUFFLE_FUNC
  //-----------------------------------------------------------------------
#ifdef  USE_SMID_TO_GET_BUFID
  const int bufTarget = occupyBuffer(tidx, freeLst, queue);
#else///USE_SMID_TO_GET_BUFID
#ifdef  TRY_MODE_ABOUT_BUFFER
  const int bufTarget = occupyBuffer(tidx, BLOCKIDX_X1D, freeNum, freeLst, queue);
#else///TRY_MODE_ABOUT_BUFFER
  occupyBuffer(tidx, freeNum, freeLst, queue, active);
#endif//TRY_MODE_ABOUT_BUFFER
#endif//USE_SMID_TO_GET_BUFID
  const int bufIdx = (int)queue[0];
  __syncthreads();
  size_t buf0Head = (size_t)(bufIdx * NGROUPS + DIV_TSUB(head)) * (size_t)bufSize;
#ifdef  DBG_TREE_WALK
  if( lane == 0 )
    printf("buffer: %ld--%ld\n", buf0Head, buf0Head + bufSize - 1);
#endif//DBG_TREE_WALK
  //-----------------------------------------------------------------------
#if 0
  if( tidx == 0 )
    printf("walk: SM %d, tag %d\n", bufTarget / NBLOCKS_PER_SM, bufTarget);
#endif
  //-----------------------------------------------------------------------
#ifdef  DBG_TREE_WALK
  real mjtot = ZERO;
#endif//DBG_TREE_WALK
  //-----------------------------------------------------------------------


  //-----------------------------------------------------------------------
  /* calculate gravitational force using hierarchical tree-structure */
  //-----------------------------------------------------------------------
  /* const int laneIdx = GLOBALIDX_X1D / TSUB; */
  const int laneIdx = DIV_TSUB(GLOBALIDX_X1D);
  const laneinfo info = laneInfo[laneIdx];
  //-----------------------------------------------------------------------

  //-----------------------------------------------------------------------
  /* load poisition of an i-particle */
  //-----------------------------------------------------------------------
#ifdef  IJ_PARALLELIZATION
  /* const bool skip = ((lane / NWARP) < info.num) ? (false) : (true); */
  const bool skip = (DIV_NWARP(lane) < info.num) ? (false) : (true);
  int      jtag =              lane & (NWARP - 1);
  /* const int idx = info.head + (lane /  NWARP); */
  const int idx = info.head + DIV_NWARP(lane);
#ifdef  DBG_TREE_WALK
  if( lane == 0 )
    printf("%d\t%d\t%d\t%d\t%d\n", GLOBALIDX_X1D, tidx, laneIdx, info.head, info.num);
#endif//DBG_TREE_WALK
#else///IJ_PARALLELIZATION
  const bool skip = (lane < info.num) ? (false) : (true);
  const int idx = info.head + lane;
#endif//IJ_PARALLELIZATION
  //-----------------------------------------------------------------------
  position   pi = {ZERO, ZERO, ZERO, UNITY};/* x, y, z, m */
  position icom = {ZERO, ZERO, ZERO, UNITY};/* x, y, z, m; m contains r2max */
  if( !skip ){
    /* load position and mass of i-particle from global memory */
    pi = ipos[idx];
#ifndef ADOPT_ENCLOSING_BALL
    icom = pi;
#endif//ADOPT_ENCLOSING_BALL
  }/* if( !skip ){ */
  //-----------------------------------------------------------------------
#ifdef  DEBUG_PRINT_FOR_PARTICLE_ACCELERATION
  bool writeWholeData = false;
  if( !skip )
    if(     (-10.270200f < pi.x) && (pi.x < -10.270000f) )
      if(   (  0.646161f < pi.y) && (pi.y <   0.646163f) )
	if( (- 0.884657f < pi.z) && (pi.z < - 0.884655f) )
	  writeWholeData = true;
#if 1
#   if  TSUB >=  2
  writeWholeData |= __shfl_xor(writeWholeData,  1, TSUB);
#   if  TSUB >=  4
  writeWholeData |= __shfl_xor(writeWholeData,  2, TSUB);
#   if  TSUB >=  8
  writeWholeData |= __shfl_xor(writeWholeData,  4, TSUB);
#   if  TSUB >= 16
  writeWholeData |= __shfl_xor(writeWholeData,  8, TSUB);
#   if  TSUB == 32
  writeWholeData |= __shfl_xor(writeWholeData, 16, TSUB);
#endif//TSUB == 32
#endif//TSUB >= 16
#endif//TSUB >=  8
#endif//TSUB >=  4
#endif//TSUB >=  2
#else
  if( jtag != 0 )
    writeWholeData = false;
#endif
#if 0
  if( lane != 0 )
    writeWholeData = false;
#endif
#endif//DEBUG_PRINT_FOR_PARTICLE_ACCELERATION
  //-----------------------------------------------------------------------
#if 0
  if( (idx == 0) && (tidx == 0) )
    printf("pi_x = %e; pj_x = %e\n", pi.x, jpos[0].x);
  /* if( (idx == 0) && (tidx == 0) ) */
  /*   printf("G = %e, eps2 = %e\n", newton, eps2); */
#endif
  //-----------------------------------------------------------------------
#if 0
  if( (idx == 0) && (tidx == 0) )
    printf("(%e @ %e, %e, %e)\n", mj[0], jpos[0].x, jpos[0].y, jpos[0].z);
#endif
  //-----------------------------------------------------------------------
  int fail = hp + lane;
  jnode jidx;
  //-----------------------------------------------------------------------


  //-----------------------------------------------------------------------
#   if  !defined(USE_CUDA_EVENT) && defined(PRINT_PSEUDO_PARTICLE_INFO)
  const long long int initCycle = clock64();
#endif//!defined(USE_CUDA_EVENT) && defined(PRINT_PSEUDO_PARTICLE_INFO)
  //-----------------------------------------------------------------------
  /* set an enclosing sphere contains whole i-particles within TSUB threads */
  //-----------------------------------------------------------------------
#ifdef  ADOPT_ENCLOSING_BALL
  //-----------------------------------------------------------------------
  pj[hp + lane].pi = pi;
  icom = pj[hp].pi;
  if( skip )    pi = icom;
  //-----------------------------------------------------------------------
#ifdef  ADOPT_SMALLEST_ENCLOSING_BALL
  //-----------------------------------------------------------------------
  /* adopt the smallest enclosing ball */
  {
    pos4seb sebPos = {pi.x, pi.y, pi.z, false};
    real4 sebCen;
    findSEB(lane, &pj[hp], &sebPos, &sebCen, (real *)&pj[hp + TSUB], (real *)&pj[hp + TSUB + NDIM_SEB], (int *)&pj[hp + TSUB + 2 * NDIM_SEB], (real *)&pj[hp + TSUB + 2 * NDIM_SEB + 1]
#ifndef USE_WARP_SHUFFLE_FUNC
	    /* , smem, (int *)&queue[hq - tidx], tidx, head */
	    , smem, (int *)&queue[hq - lane], tidx, head
#endif//USE_WARP_SHUFFLE_FUNC
	    );
    icom.x = sebCen.x;
    icom.y = sebCen.y;
    icom.z = sebCen.z;
  }
  //-----------------------------------------------------------------------
#endif//ADOPT_SMALLEST_ENCLOSING_BALL
#ifdef  ADOPT_APPROXIMATED_ENCLOSING_BALL
  //-----------------------------------------------------------------------
  /* adopt the approximated enclosing ball proposed by Ritter (1990) */
  approxSEB(lane, &pj[hp], pi, &icom
#ifndef USE_WARP_SHUFFLE_FUNC
	    /* , smem, (int *)&queue[hq - tidx], tidx, head */
	    , smem, (int *)&queue[hq - lane], tidx, head
#endif//USE_WARP_SHUFFLE_FUNC
	    );
  //-----------------------------------------------------------------------
#endif//ADOPT_APPROXIMATED_ENCLOSING_BALL
#   if  !defined(ADOPT_SMALLEST_ENCLOSING_BALL) && !defined(ADOPT_APPROXIMATED_ENCLOSING_BALL)
  //-----------------------------------------------------------------------
  /* adopt a simple estimation of enclosing ball using the minimum bounding box in Cartesian coordinates */
  {
#ifdef  USE_WARP_SHUFFLE_FUNC
    const real xmin = getMinimumRealTsub(pi.x                  );    const real xmax = getMaximumRealTsub(pi.x                  );
    const real ymin = getMinimumRealTsub(pi.y                  );    const real ymax = getMaximumRealTsub(pi.y                  );
    const real zmin = getMinimumRealTsub(pi.z                  );    const real zmax = getMaximumRealTsub(pi.z                  );
#else///USE_WARP_SHUFFLE_FUNC
    const real xmin = getMinimumRealTsub(pi.x, smem, tidx, head);    const real xmax = getMaximumRealTsub(pi.x, smem, tidx, head);
    const real ymin = getMinimumRealTsub(pi.y, smem, tidx, head);    const real ymax = getMaximumRealTsub(pi.y, smem, tidx, head);
    const real zmin = getMinimumRealTsub(pi.z, smem, tidx, head);    const real zmax = getMaximumRealTsub(pi.z, smem, tidx, head);
#endif//USE_WARP_SHUFFLE_FUNC
    icom.x = HALF * (xmin + xmax);
    icom.y = HALF * (ymin + ymax);
    icom.z = HALF * (zmin + zmax);
  }
#ifdef  COMPARE_ENCLOSING_BALLS
  position ball = icom;
  icom = pi;
#endif//COMPARE_ENCLOSING_BALLS
  //-----------------------------------------------------------------------
#endif//!defined(ADOPT_SMALLEST_ENCLOSING_BALL) && !defined(ADOPT_APPROXIMATED_ENCLOSING_BALL)
  //-----------------------------------------------------------------------
#endif//ADOPT_ENCLOSING_BALL
#   if  !defined(ADOPT_ENCLOSING_BALL) || defined(COMPARE_ENCLOSING_BALLS)
  //-----------------------------------------------------------------------
  /* calculate center-of-mass of a group of i-particles as an enclosing sphere */
  icom.x *= icom.m;
  icom.y *= icom.m;
  icom.z *= icom.m;
  pj[fail].pi = icom;
  /* NOTE: implicit synchronization within 32 threads (a warp) is assumed */
#   if  TSUB >=  2
#                    if  NWARP <  2
  jidx = pj[fail ^  1];  icom.x += jidx.pi.x;  icom.y += jidx.pi.y;  icom.z += jidx.pi.z;  icom.m += jidx.pi.m;  pj[fail].pi = icom;
#                 endif//NWARP <  2
#   if  TSUB >=  4
#                    if  NWARP <  4
  jidx = pj[fail ^  2];  icom.x += jidx.pi.x;  icom.y += jidx.pi.y;  icom.z += jidx.pi.z;  icom.m += jidx.pi.m;  pj[fail].pi = icom;
#                 endif//NWARP <  4
#   if  TSUB >=  8
#                    if  NWARP <  8
  jidx = pj[fail ^  4];  icom.x += jidx.pi.x;  icom.y += jidx.pi.y;  icom.z += jidx.pi.z;  icom.m += jidx.pi.m;  pj[fail].pi = icom;
#                 endif//NWARP <  8
#   if  TSUB >= 16
#                    if  NWARP < 16
  jidx = pj[fail ^  8];  icom.x += jidx.pi.x;  icom.y += jidx.pi.y;  icom.z += jidx.pi.z;  icom.m += jidx.pi.m;  pj[fail].pi = icom;
#                 endif//NWARP < 16
#   if  TSUB == 32
#                    if  NWARP < 32
  jidx = pj[fail ^ 16];  icom.x += jidx.pi.x;  icom.y += jidx.pi.y;  icom.z += jidx.pi.z;  icom.m += jidx.pi.m;  pj[fail].pi = icom;
#                 endif//NWARP < 32
#endif//TSUB == 32
#endif//TSUB >= 16
#endif//TSUB >=  8
#endif//TSUB >=  4
#endif//TSUB >=  2
  icom = pj[hp].pi;/* icom.m = Mtot */
  icom.m = UNITY / icom.m;/* tentative use as minv */
  icom.x *= icom.m;
  icom.y *= icom.m;
  icom.z *= icom.m;
  //-----------------------------------------------------------------------
#ifdef  COMPARE_ENCLOSING_BALLS
  {
    real dx, dy, dz;
    /* calculate size of geometrical estimated enclosing ball */
    dx = pi.x - ball.x;
    dy = pi.y - ball.y;
    dz = pi.z - ball.z;
    ball.m = getMaximumRealTsub(FLT_MIN + dx * dx + dy * dy + dz * dz
#ifndef USE_WARP_SHUFFLE_FUNC
				, smem, tidx, head
#endif//USE_WARP_SHUFFLE_FUNC
				);
    /* calculate size of mass-weighted estimated enclosing ball */
    dx = pi.x - icom.x;
    dy = pi.y - icom.y;
    dz = pi.z - icom.z;
    icom.m = getMaximumRealTsub(FLT_MIN + dx * dx + dy * dy + dz * dz
#ifndef USE_WARP_SHUFFLE_FUNC
				, smem, tidx, head
#endif//USE_WARP_SHUFFLE_FUNC
				);
    /* adopt smaller one */
    if( ball.m < icom.m )
      icom = ball;
  }
#endif//COMPARE_ENCLOSING_BALLS
  //-----------------------------------------------------------------------
#endif//!defined(ADOPT_ENCLOSING_BALL) || defined(COMPARE_ENCLOSING_BALLS)
  //-----------------------------------------------------------------------


  //-----------------------------------------------------------------------
  /* calculate radius of the sphere which include whole i-particles within the group centered on (icom.x, icom.y, icom.z) */
  //-----------------------------------------------------------------------
#ifdef  GADGET_MAC
#ifdef  YMIKI_MAC
  acceleration amin;
#else///YMIKI_MAC
  real amin;
#endif//YMIKI_MAC
#endif//GADGET_MAC
  {
    //---------------------------------------------------------------------
    /* calculate displacement of i-particle and center-of-mass */
    const real rx = pi.x - icom.x;
    const real ry = pi.y - icom.y;
    const real rz = pi.z - icom.z;
    //---------------------------------------------------------------------
    /* calculate maximum of r squared */
    icom.m = getMaximumRealTsub(FLT_MIN + rx * rx + ry * ry + rz * rz
#ifndef USE_WARP_SHUFFLE_FUNC
				, smem, tidx, head
#endif//USE_WARP_SHUFFLE_FUNC
				);
    //---------------------------------------------------------------------
#ifdef  GADGET_MAC
    //---------------------------------------------------------------------
    acceleration ai_old = {ZERO, ZERO, ZERO, ZERO};
    if( !skip ){
      ai_old = iacc_old[idx];
/* #ifdef  YMIKI_MAC */
/*       ai_old.pot = UNITY; */
/* #endif//YMIKI_MAC */
    }/* if( !skip ){ */
    //---------------------------------------------------------------------
    /* calculate minimum of a squared */
    const real tmp = getMinimumRealTsub(FLT_MIN + ai_old.x * ai_old.x + ai_old.y * ai_old.y + ai_old.z * ai_old.z
#ifndef USE_WARP_SHUFFLE_FUNC
					, smem, tidx, head
#endif//USE_WARP_SHUFFLE_FUNC
					);
#ifndef YMIKI_MAC
    amin = tmp * RSQRT(tmp);
#endif//YMIKI_MAC
    //---------------------------------------------------------------------
#ifdef  YMIKI_MAC
    /* calculate bulk acceleration of a group of i-particles */
    pj[fail].ai = ai_old;
    /* NOTE: implicit synchronization within 32 threads (a warp) is assumed */
#   if  TSUB >=  2
#                    if  NWARP <  2
    jidx = pj[fail ^  1];    ai_old.x += jidx.ai.x;    ai_old.y += jidx.ai.y;    ai_old.z += jidx.ai.z;    /* ai_old.pot += jidx.ai.pot; */    pj[fail].ai = ai_old;
#                 endif//NWARP <  2
#   if  TSUB >=  4
#                    if  NWARP <  4
    jidx = pj[fail ^  2];    ai_old.x += jidx.ai.x;    ai_old.y += jidx.ai.y;    ai_old.z += jidx.ai.z;    /* ai_old.pot += jidx.ai.pot; */    pj[fail].ai = ai_old;
#                 endif//NWARP <  4
#   if  TSUB >=  8
#                    if  NWARP <  8
    jidx = pj[fail ^  4];    ai_old.x += jidx.ai.x;    ai_old.y += jidx.ai.y;    ai_old.z += jidx.ai.z;    /* ai_old.pot += jidx.ai.pot; */    pj[fail].ai = ai_old;
#                 endif//NWARP <  8
#   if  TSUB >= 16
#                    if  NWARP < 16
    jidx = pj[fail ^  8];    ai_old.x += jidx.ai.x;    ai_old.y += jidx.ai.y;    ai_old.z += jidx.ai.z;    /* ai_old.pot += jidx.ai.pot; */    pj[fail].ai = ai_old;
#                 endif//NWARP < 16
#   if  TSUB == 32
#                    if  NWARP < 32
    jidx = pj[fail ^ 16];    ai_old.x += jidx.ai.x;    ai_old.y += jidx.ai.y;    ai_old.z += jidx.ai.z;    /* ai_old.pot += jidx.ai.pot; */    pj[fail].ai = ai_old;
#                 endif//NWARP < 32
#endif//TSUB == 32
#endif//TSUB >= 16
#endif//TSUB >=  8
#endif//TSUB >=  4
#endif//TSUB >=  2
    amin = pj[hp].ai;
#if 1
    /* amin.pot = UNITY / amin.pot; */
    /* amin.x *= amin.pot; */
    /* amin.y *= amin.pot; */
    /* amin.z *= amin.pot; */
    amin.pot = SQRTRATIO(tmp, FLT_MIN + amin.x * amin.x + amin.y * amin.y + amin.z * amin.z);
    amin.x *= amin.pot;
    amin.y *= amin.pot;
    amin.z *= amin.pot;
#else
    amin.pot = SQRTRATIO(tmp * amin.pot * amin.pot, FLT_MIN + amin.x * amin.x + amin.y * amin.y + amin.z * amin.z);
    amin.x *= amin.pot;
    amin.y *= amin.pot;
    amin.z *= amin.pot;
#endif
#endif//YMIKI_MAC
    //---------------------------------------------------------------------
#endif//GADGET_MAC
    //---------------------------------------------------------------------
  }
  //-----------------------------------------------------------------------
#ifndef INDIVIDUAL_GRAVITATIONAL_SOFTENING
  /* set square of softening length at pi.m */
  pi.m = eps2;
#endif//INDIVIDUAL_GRAVITATIONAL_SOFTENING
  //-----------------------------------------------------------------------
#ifdef  DBG_TREE_WALK
  if( lane == 0 )
    printf("%d: %f, %f, %f w/ r2max = %e\n", GLOBALIDX_X1D, icom.x, icom.y, icom.z, icom.m);
#endif//DBG_TREE_WALK
  //-----------------------------------------------------------------------
#   if  !defined(USE_CUDA_EVENT) && defined(PRINT_PSEUDO_PARTICLE_INFO)
  long long int exitCycle = clock64();
  if( lane == 0 ){
    unsigned long long int elapsed = (unsigned long long int)(exitCycle - initCycle);
    atomicAdd(cycles, elapsed);
    iacc[DIV_TSUB(GLOBALIDX_X1D)].pi = icom;
  }/* if( tidx == 0 ){ */
#endif//!defined(USE_CUDA_EVENT) && defined(PRINT_PSEUDO_PARTICLE_INFO)
  //-----------------------------------------------------------------------
/* #if 0 */
/*   amin = ZERO; */
/* #endif */
/* #ifdef  DEBUG_PRINT_FOR_PARTICLE_ACCELERATION */
/*   if( writeWholeData && (jtag == 0) ) */
/*     printf("amin = %e, root = %d, Mj = %e\n", amin, root, mj[root]); */
/* #endif//DEBUG_PRINT_FOR_PARTICLE_ACCELERATION */
  //-----------------------------------------------------------------------



  //-----------------------------------------------------------------------
#ifndef PRINT_PSEUDO_PARTICLE_INFO
  //-----------------------------------------------------------------------
  /* sweep all j-cells by executing tree-traversal */
  //-----------------------------------------------------------------------
  /* initialize queue for j-cells */
#pragma unroll
  for(int jj = 0; jj < NQUEUE; jj++)
    queue[hq + TSUB * jj] = NULL_NODE;
  //-----------------------------------------------------------------------
  /* initialize queue for j-cells and interaction list by a representative thread */
  int Nj = 0;
  int bufHead = 0;
  int bufTail = 0;
  int bufOpen = bufSize;
  int bufUsed = 0;
#if 0
  int bufTailMax = bufTail;
#endif
  /* set child j-cells in queue on the shared memory */
#if 1
  uint jcell = more[root];
  int rem = 1 + (jcell >> IDXBITS);
  jcell &= IDXMASK;
  //-----------------------------------------------------------------------
  if( rem > TSUB ){
    //---------------------------------------------------------------------
    /* if rem exceeds TSUB, then number of child j-cells must be shrunk */
    //---------------------------------------------------------------------
    queue[hq] = jcell + lane;
    //---------------------------------------------------------------------
    if( tidx == tail )
      queue[hq] += ((rem - TSUB) << IDXBITS);
    //---------------------------------------------------------------------
    rem = TSUB;
    //---------------------------------------------------------------------
  }/* if( rem > TSUB ){ */
  else{
    //---------------------------------------------------------------------
    /* upload rem (<= TSUB) child j-cells to the shared memory */
    //---------------------------------------------------------------------
    if( lane < rem )
      queue[hq] = more[jcell + lane];
    //---------------------------------------------------------------------
  }/* else{ */
#else
  uint jcell;
  int rem = 1;
  if( lane == 0 )
    queue[hq] = 0;
#endif
  //-----------------------------------------------------------------------
  if( info.num == 0 )
    rem = 0;
  //-----------------------------------------------------------------------


  //-----------------------------------------------------------------------
  /* tree traversal in a width-first manner */
  //-----------------------------------------------------------------------
  acceleration  ai = {ZERO, ZERO, ZERO, ZERO};/* ax, ay, az, pot */
#ifdef  ACCURATE_ACCUMULATION
  acceleration res = {ZERO, ZERO, ZERO, ZERO};
#endif//ACCURATE_ACCUMULATION
  //-----------------------------------------------------------------------
  fail = 0;
  //-----------------------------------------------------------------------
#ifdef  COUNT_INTERACTIONS
  int Nj_tot = 0;
  int Nb_max = 0;
#endif//COUNT_INTERACTIONS
  //-----------------------------------------------------------------------
  while( true ){
    //---------------------------------------------------------------------
    /* if the queue becomes empty, then exit the while loop */
    //---------------------------------------------------------------------
    if( rem == 0 )
      break;
    //---------------------------------------------------------------------
/* #ifdef  DEBUG_PRINT_FOR_PARTICLE_ACCELERATION */
/*     if( writeWholeData ){ */
/*       printf("%u+%u", queue[hq] & IDXMASK, 1 + (queue[hq] >> IDXBITS)); */
/*       for(int ii = 1; ii < rem; ii++) */
/* 	printf("\t%u+%u", queue[hq + ii] & IDXMASK, 1 + (queue[hq + ii] >> IDXBITS)); */
/*       printf("\n"); */
/*     } */
/* #endif//DEBUG_PRINT_FOR_PARTICLE_ACCELERATION */
    //---------------------------------------------------------------------

    //---------------------------------------------------------------------
    /* pick up a queue from stack */
    //---------------------------------------------------------------------
    /* initialize the shared memory */
    uint leaf = 0;
#pragma unroll
    for(int iter = 0; iter < NSTOCK; iter++)
      jidx.idx[iter] = NULL_NODE;
    pj[hp + lane + NLOOP * TSUB] = jidx;
    //---------------------------------------------------------------------
    /* tentative load from the stack */
    int cnum = 0;
    jcell = NULL_NODE;
    if( lane < rem ){
      jcell = queue[hq];
      cnum = 1 + (jcell >> IDXBITS);
    }/* if( lane < rem ){ */
    //---------------------------------------------------------------------
    /* predict the head index on the shared memory by parallel prefix sum */
#ifdef  USE_WARP_SHUFFLE_FUNC
    smem = prefixSumTsub(cnum, lane);
    int hidx = smem         - cnum;/* exclusive prefix sum of cnum */
#else///USE_WARP_SHUFFLE_FUNC
    prefixSumTsub(cnum, smem, tidx, lane);
    int hidx = smem[tidx].i - cnum;/* exclusive prefix sum of cnum */
#endif//USE_WARP_SHUFFLE_FUNC
    //---------------------------------------------------------------------
    int remove = 0;
    if( (cnum != 0) && (hidx < TSUB * NSTOCK) ){
      //-------------------------------------------------------------------
      /* local data can be uploaded to the shared memory */
      int unum = TSUB * NSTOCK - hidx;
      if( cnum < unum )	  unum = cnum;
      //-------------------------------------------------------------------
      /* upload local data */
      jcell &= IDXMASK;
      for(int jj = 0; jj < unum; jj++){
	pj[hp + NLOOP * TSUB + (hidx & (TSUB - 1))].idx[DIV_TSUB(hidx)] = jcell;/* assumes TSUB is a power of 2 */
	hidx++;
	jcell++;
      }/* for(int jj = 0; jj < unum; jj++){ */
      //-------------------------------------------------------------------
      /* eliminate stocked j-cells from the queue */
      if( unum == cnum )
	remove = 1;
      else{
	jcell += ((cnum - unum - 1) << IDXBITS);
	queue[hq] = jcell;
      }/* else{ */
      //-------------------------------------------------------------------
    }/* if( (cnum != 0) && (hidx < TSUB * NSTOCK) ){ */
    //---------------------------------------------------------------------
    /* remove scanned j-cells if possible */
#ifdef  USE_WARP_SHUFFLE_FUNC
    smem = prefixSumTsub(remove, lane);
    remove = __shfl(smem, TSUB - 1, TSUB);
#else///USE_WARP_SHUFFLE_FUNC
    prefixSumTsub(remove, smem, tidx, lane);
    remove = smem[tail].i;
#endif//USE_WARP_SHUFFLE_FUNC
    //---------------------------------------------------------------------
    if( remove != 0 ){
      rem -= remove;
      copyData_s2s(queue, hq + remove, queue, hq, rem, lane);
    }/* if( remove != 0 ){ */
    //---------------------------------------------------------------------


    //---------------------------------------------------------------------
    /* pick up pseudo particles from NSTOCK buffers */
    //---------------------------------------------------------------------
    jidx = pj[hp + lane + NLOOP * TSUB];
    //---------------------------------------------------------------------
#pragma unroll
    for(int iter = 0; iter < NSTOCK; iter++){
      //-------------------------------------------------------------------
      /* set an index of j-cell */
      const uint target = jidx.idx[iter];
      //-------------------------------------------------------------------

      //-------------------------------------------------------------------
      /* only the active threads pick up a j-cell from the global memory */
      //-------------------------------------------------------------------
      jparticle jcnd;
#ifdef  INDIVIDUAL_GRAVITATIONAL_SOFTENING
      real      jeps2;
#endif//INDIVIDUAL_GRAVITATIONAL_SOFTENING
      int calc = 0;
      //-------------------------------------------------------------------
      if( target != NULL_NODE ){
	//-----------------------------------------------------------------
	jcnd = jpos[target];
	//-----------------------------------------------------------------
	/* set a pseudo i-particle */
	const real rx = jcnd.x - icom.x;
	const real ry = jcnd.y - icom.y;
	const real rz = jcnd.z - icom.z;
	const real r2 = FLT_MIN + rx * rx + ry * ry + rz * rz;
	real lambda = FMAX(UNITY - SQRTRATIO(icom.m, r2), ZERO);
	/* calculate distance between the pseudo i-particle and the candidate j-particle */
	//-----------------------------------------------------------------
#ifndef YMIKI_MAC
	lambda *= lambda * r2;
#endif//YMIKI_MAC
#ifdef  GADGET_MAC
#ifndef YMIKI_MAC
	/* alpha * |a| * r^4 > G * M * l^2 */
	if( jcnd.w < lambda * lambda * amin )
#else///YMIKI_MAC
	/* alpha * |(a, r)| * r^3 > G * M * l^2 */
	lambda *= lambda;	lambda *= lambda;	          /* lambda := lambda^4 */
	lambda *= (amin.x * rx + amin.y * ry + amin.z * rz);      /* lambda := lambda^4 * (ai, rij) */
	lambda *= r2;                                             /* lambda := lambda^4 * (ai, rij) * d^2 */
	if( jcnd.w < lambda * lambda * r2 )
#endif//YMIKI_MAC
#else///GADGET_MAC
#ifdef  WS93_MAC
	  if(   jcnd.w < lambda )
#else///WS93_MAC
	    /* (l / r) < theta */
	    if( jcnd.w < lambda * theta2 )
#endif//WS93_MAC
#endif//GADGET_MAC
	      {
		/* add the candidate j-particle to the interaction list */
#ifdef  DEBUG_PRINT_FOR_PARTICLE_ACCELERATION
		if( writeWholeData )
		  printf("%e\t%e\t%e\t%e\t%e\n", mj[target], rx, ry, rz, r2);
		  /* printf("%e\t%e\t%e\t%e\t%e\t%d\n", mj[target], jcnd.x, jcnd.y, jcnd.z, jcnd.w, target); */
#endif//DEBUG_PRINT_FOR_PARTICLE_ACCELERATION
		const jmass mj_tmp = mj[target];
#ifdef  INDIVIDUAL_GRAVITATIONAL_SOFTENING
		jcnd.w = mj_tmp.mass;
		jeps2  = mj_tmp.eps2;
#else///INDIVIDUAL_GRAVITATIONAL_SOFTENING
		jcnd.w = mj_tmp;
#endif//INDIVIDUAL_GRAVITATIONAL_SOFTENING
		calc = 1;
	      }
	    else
	      {
/* #ifdef  DEBUG_PRINT_FOR_PARTICLE_ACCELERATION */
/* 		if( writeWholeData ) */
/* 		  printf("%e\t%e\t%e\t%e\t%e\n", mj[target], jcnd.x, jcnd.y, jcnd.z, jcnd.w); */
/* #endif//DEBUG_PRINT_FOR_PARTICLE_ACCELERATION */
		/* add child-cells of near tree-cells to the tentative stack */
		leaf += (1 << (IDX_SHIFT_BITS * iter));
		jidx.idx[iter] = more[target];
	      }
	//-----------------------------------------------------------------
      }/* if( target != NULL_NODE ){ */
      //-------------------------------------------------------------------

      //-------------------------------------------------------------------
      /* prefixSum to build a local interaction list */
      //-------------------------------------------------------------------
#ifdef  USE_WARP_SHUFFLE_FUNC
      smem = prefixSumTsub(calc, lane);
      hidx = smem         - calc;/* exclusive prefix sum of calc */
#else///USE_WARP_SHUFFLE_FUNC
      prefixSumTsub(calc, smem, tidx, lane);
      hidx = smem[tidx].i - calc;/* exclusive prefix sum of calc */
#endif//USE_WARP_SHUFFLE_FUNC
      //-------------------------------------------------------------------

      //-------------------------------------------------------------------
      /* add distant tree-cells to the interaction list */
      if( calc ){
	pj  [hp + Nj + hidx].pos = jcnd;
#ifdef  INDIVIDUAL_GRAVITATIONAL_SOFTENING
	eps2[hp + Nj + hidx] = jeps2;
#endif//INDIVIDUAL_GRAVITATIONAL_SOFTENING
      }
#ifdef  USE_WARP_SHUFFLE_FUNC
      Nj += __shfl(smem, TSUB - 1, TSUB);/* inclusive prefix sum of calc */
#else///USE_WARP_SHUFFLE_FUNC
      Nj += smem[tail].i;/* inclusive prefix sum of calc */
#endif//USE_WARP_SHUFFLE_FUNC
      //-------------------------------------------------------------------

      //-------------------------------------------------------------------
      /* calculate body--body interaction if sufficient size of interaction list is available */
      //-------------------------------------------------------------------
      if( Nj >= NLOOP * TSUB ){
	//-----------------------------------------------------------------
	calc_interaction
	  (pi, &ai, (jparticle *)&pj[hp]
#ifdef  INDIVIDUAL_GRAVITATIONAL_SOFTENING
	   , &eps2[hp]
#endif//INDIVIDUAL_GRAVITATIONAL_SOFTENING
#ifdef  ACCURATE_ACCUMULATION
	   , &res
#endif//ACCURATE_ACCUMULATION
#ifdef  IJ_PARALLELIZATION
	   , jtag
#endif//IJ_PARALLELIZATION
	   );
	//-----------------------------------------------------------------
/* #ifdef  DEBUG_PRINT_FOR_PARTICLE_ACCELERATION */
/* 	if( writeWholeData ) */
/* 	  for(int ll = 0; ll < NLOOP * TSUB; ll++) */
/* 	    printf("%e\t%e\t%e\t%e\n", pj[hp + ll].pos.w, pj[hp + ll].pos.x, pj[hp + ll].pos.y, pj[hp + ll].pos.z); */
/* #endif//DEBUG_PRINT_FOR_PARTICLE_ACCELERATION */
	//-----------------------------------------------------------------
#ifdef  DBG_TREE_WALK
	for(int ll = 0; ll < NLOOP * TSUB; ll++)
	  mjtot += pj[hp + ll].pos.w;
#endif//DBG_TREE_WALK
	//-----------------------------------------------------------------
	pj  [hp + lane] = pj  [hp + lane + NLOOP * TSUB];
#ifdef  INDIVIDUAL_GRAVITATIONAL_SOFTENING
	eps2[hp + lane] = eps2[hp + lane + NLOOP * TSUB];
#endif//INDIVIDUAL_GRAVITATIONAL_SOFTENING
	Nj -= NLOOP * TSUB;
	//-----------------------------------------------------------------
#ifdef  COUNT_INTERACTIONS
	Nj_tot += NLOOP * TSUB;
#endif//COUNT_INTERACTIONS
	//-----------------------------------------------------------------
      }/* if( Nj >= NLOOP * TSUB ){ */
      //-------------------------------------------------------------------
    }/* for(int iter = 0; iter < NSTOCK; iter++){ */
    //---------------------------------------------------------------------


    //---------------------------------------------------------------------
    /* if the shared memory has open space and some tree cells are stored on the global memory, then load tree-cells from the global memory to the shared memory */
    //---------------------------------------------------------------------
    /* evaluate available size of the queue on the shared memory */
    int Nsm_rem = NQUEUE * TSUB - rem;
    //---------------------------------------------------------------------
    if(  (bufUsed != 0) && (Nsm_rem > 0) ){
      //-------------------------------------------------------------------
      const int Nload = (Nsm_rem < bufUsed) ? (Nsm_rem) : (bufUsed);
      copyData_g2s(buffer, buf0Head + bufHead, queue, hq - lane + rem, Nload, lane);
      //-------------------------------------------------------------------
      rem     += Nload;
      Nsm_rem -= Nload;
      bufUsed -= Nload;
      bufHead += Nload;
      //-------------------------------------------------------------------
      if( bufUsed == 0 ){
	bufHead = 0;
	bufTail = 0;
	bufOpen = bufSize;
      }/* if( bufUsed == 0 ){ */
      //-------------------------------------------------------------------
    }/* if( (bufUsed != 0) && (Nsm_rem > 0) ){ */
    //---------------------------------------------------------------------


    //---------------------------------------------------------------------
    /* copy child-cells of near tree-cells stored in the tentative stack to the stack on the shared memory and/or the global memory */
    //---------------------------------------------------------------------
#ifdef  USE_WARP_SHUFFLE_FUNC
    cpChildNodes(      (uint *)(&pj[hp + NLOOP * TSUB]), jidx, leaf,       lane,       queue, hq, &Nsm_rem, &rem, buffer, buf0Head, &bufOpen, &bufUsed, &bufHead, &bufTail);
#else///USE_WARP_SHUFFLE_FUNC
    cpChildNodes(smem, (uint *)(&pj[hp + NLOOP * TSUB]), jidx, leaf, tidx, lane, tail, queue, hq, &Nsm_rem, &rem, buffer, buf0Head, &bufOpen, &bufUsed, &bufHead, &bufTail);
#endif//USE_WARP_SHUFFLE_FUNC
    //---------------------------------------------------------------------
    /* fail += (bufOpen < 0); */
    fail += (bufTail > bufSize);
#if 0
    if( fail != 0 )
      buffer[ULONG_MAX] = NULL_NODE;
#endif
#ifdef  COUNT_INTERACTIONS
    if( bufUsed > Nb_max )
      Nb_max = bufUsed;
#endif//COUNT_INTERACTIONS
    //---------------------------------------------------------------------
#if 0
    if( fail > 0 )
      break;
#endif
    //---------------------------------------------------------------------
#if 0
    if( bufTail > bufTailMax )
      bufTailMax = bufTail;
#endif
    //---------------------------------------------------------------------
  }/* while( true ){ */
  //-----------------------------------------------------------------------
#if 0
  if( lane == 0 )
    printf("bufTailMax = %d\n", bufTailMax);
#endif
  //-----------------------------------------------------------------------


  //-----------------------------------------------------------------------
  /* calculate body--body interaction for remained j-particles */
  //-----------------------------------------------------------------------
  if( Nj != 0 ){
    //---------------------------------------------------------------------
    /* add massless particles at the tail of the interaction list */
    const int Ndummy = NLOOP * TSUB - Nj;
    const jparticle massless = {ZERO, ZERO, ZERO, ZERO};
    for(int jj = 0; jj < NLOOP; jj++){
      const int addr = lane + jj * TSUB;
      if( addr < Ndummy ){
	pj  [hp + Nj + addr].pos = massless;
#ifdef  INDIVIDUAL_GRAVITATIONAL_SOFTENING
	eps2[hp + Nj + addr]     = UNITY;
#endif//INDIVIDUAL_GRAVITATIONAL_SOFTENING
      }/* if( addr < Ndummy ){ */
    }/* for(int jj = 0; jj < NLOOP; jj++){ */
    //---------------------------------------------------------------------
    calc_interaction
      (pi, &ai, (jparticle *)&pj[hp]
#ifdef  INDIVIDUAL_GRAVITATIONAL_SOFTENING
       , &eps2[hp]
#endif//INDIVIDUAL_GRAVITATIONAL_SOFTENING
#ifdef  ACCURATE_ACCUMULATION
       , &res
#endif//ACCURATE_ACCUMULATION
#ifdef  IJ_PARALLELIZATION
       , jtag
#endif//IJ_PARALLELIZATION
       );
    //---------------------------------------------------------------------
/* #ifdef  DEBUG_PRINT_FOR_PARTICLE_ACCELERATION */
/*     if( writeWholeData ) */
/*       for(int ll = 0; ll < NLOOP * TSUB; ll++) */
/* 	printf("%e\t%e\t%e\t%e\n", pj[hp + ll].pos.w, pj[hp + ll].pos.x, pj[hp + ll].pos.y, pj[hp + ll].pos.z); */
/* #endif//DEBUG_PRINT_FOR_PARTICLE_ACCELERATION */
    //---------------------------------------------------------------------
#ifdef  DBG_TREE_WALK
    for(int ll = 0; ll < NLOOP * TSUB; ll++)
      mjtot += pj[hp + ll].pos.w;
#endif//DBG_TREE_WALK
#ifdef  COUNT_INTERACTIONS
    Nj_tot += NLOOP * TSUB;
#endif//COUNT_INTERACTIONS
    //---------------------------------------------------------------------
  }/* if( Nj != 0 ){ */
  //-----------------------------------------------------------------------
  /* accumulation of residuals in Kahan summation */
#   if  defined(ACCURATE_ACCUMULATION) && (NWARP > 1)
  /* set index to accumulate acceleration without atomic operations */
#   if  NWARP > 4
  const int gtag = jtag >> 2;
  jtag &= 3;
#endif//NWARP > 4
  const int itag = hp + lane - jtag;
  pj[hp + lane].ai = res;
#   if  NWARP == 2
  pj[itag].val[jtag] += pj[itag + 1].val[jtag];  jtag ^= 2;
  pj[itag].val[jtag] += pj[itag + 1].val[jtag];
#endif//NWARP == 2
#   if  NWARP == 4
  pj[itag].val[jtag] += pj[itag + 1].val[jtag] + pj[itag + 2].val[jtag] + pj[itag + 3].val[jtag];
#endif//NWARP == 4
#   if  NWARP >= 8
  pj[itag].val[jtag] += pj[itag + 1].val[jtag] + pj[itag + 2].val[jtag] + pj[itag + 3].val[jtag];
#   if  NWARP == 32
  if( gtag < 4 )
    pj[itag].val[jtag] += pj[itag + 16].val[jtag];
#endif//NWARP == 32
#   if  NWARP >= 16
  if( gtag < 2 )
    pj[itag].val[jtag] += pj[itag + 8].val[jtag];
#endif//NWARP >= 16
  if( gtag == 0 )
    pj[itag].val[jtag] += pj[itag + 4].val[jtag];
#endif//NWARP >= 8
  res = pj[itag].ai;
#endif//defined(ACCURATE_ACCUMULATION) && (NWARP > 1)
  //-----------------------------------------------------------------------


  //-----------------------------------------------------------------------
  /* store acceleration of an i-particle from each thread */
  //-----------------------------------------------------------------------
  if( !skip ){
    //---------------------------------------------------------------------
    /* NOTE: implicit synchronization within 32 threads (a warp) is assumed for NWARP = 8, 16, 32 */
#   if  defined(SERIALIZED_EXECUTION) && (NWARP == 1)
    iacc[idx].ai = ai;
#else///defined(SERIALIZED_EXECUTION) && (NWARP == 1)
#   if  NWARP > 1
#ifndef ACCURATE_ACCUMULATION
    /* set index to accumulate acceleration without atomic operations */
#   if  NWARP > 4
    const int gtag = jtag >> 2;
    jtag &= 3;
#endif//NWARP > 4
    const int itag = hp + lane - jtag;
#endif//ACCURATE_ACCUMULATION
    pj[hp + lane].ai = ai;
#   if  NWARP == 2
#   if  defined(KAHAN_SUM_CORRECTION) && defined(ACCURATE_ACCUMULATION)
    /* T := S, S := S + R; T := S - T; R := R - T */
    real sum0 = pj[itag].val[jtag] + pj[itag + 1].val[jtag];    real tmp0 = atomicAdd(&(iacc[idx].val[jtag]), sum0);    tmp0 = (tmp0 + sum0) - tmp0;    sum0 -= tmp0;    jtag ^= 2;
    real sum1 = pj[itag].val[jtag] + pj[itag + 1].val[jtag];    real tmp1 = atomicAdd(&(iacc[idx].val[jtag]), sum1);    tmp1 = (tmp1 + sum1) - tmp1;    sum1 -= tmp1;
    pj[hp + lane].ai = res;
    atomicAdd(&(ires[idx].val[jtag]), sum1 + pj[itag].val[jtag]);    jtag ^= 2;
    atomicAdd(&(ires[idx].val[jtag]), sum0 + pj[itag].val[jtag]);
#else///defined(KAHAN_SUM_CORRECTION) && defined(ACCURATE_ACCUMULATION)
    atomicAdd(&(iacc[idx].val[jtag]), pj[itag].val[jtag] + pj[itag + 1].val[jtag]);    jtag ^= 2;
    atomicAdd(&(iacc[idx].val[jtag]), pj[itag].val[jtag] + pj[itag + 1].val[jtag]);
#endif//defined(KAHAN_SUM_CORRECTION) && defined(ACCURATE_ACCUMULATION)
#endif//NWARP == 2
#   if  NWARP == 4
#   if  defined(KAHAN_SUM_CORRECTION) && defined(ACCURATE_ACCUMULATION)
    /* T := S, S := S + R; T := S - T; R := R - T */
    real sum = pj[itag].val[jtag] + pj[itag + 1].val[jtag] + pj[itag + 2].val[jtag] + pj[itag + 3].val[jtag];
#if 1
    real tmp = atomicAdd(&(iacc[idx].val[jtag]), sum);    tmp = (tmp + sum) - tmp;    sum -= tmp;
#else
    real tmp = atomicAdd(&(iacc[idx].val[jtag]), sum);    tmp = CAST_D2R(CAST_R2D(tmp + sum) - CAST_R2D(tmp));    sum = CAST_D2R(CAST_R2D(sum) - CAST_R2D(tmp));
#endif
    pj[hp + lane].ai = res;
    atomicAdd(&(ires[idx].val[jtag]), sum + pj[itag].val[jtag]);
#else///defined(KAHAN_SUM_CORRECTION) && defined(ACCURATE_ACCUMULATION)
#if 0
    if( jtag == 0 )
      printf("%e\t%e\t%e\t%e\t%e\n", pi.x,
	     pj[itag].ai.x   + pj[itag + 1].ai.x   + pj[itag + 2].ai.x   + pj[itag + 3].ai.x,
	     pj[itag].ai.y   + pj[itag + 1].ai.y   + pj[itag + 2].ai.y   + pj[itag + 3].ai.y,
	     pj[itag].ai.z   + pj[itag + 1].ai.z   + pj[itag + 2].ai.z   + pj[itag + 3].ai.z,
	     pj[itag].ai.pot + pj[itag + 1].ai.pot + pj[itag + 2].ai.pot + pj[itag + 3].ai.pot);
#endif
    atomicAdd(&(iacc[idx].val[jtag]), pj[itag].val[jtag] + pj[itag + 1].val[jtag] + pj[itag + 2].val[jtag] + pj[itag + 3].val[jtag]);
#endif//defined(KAHAN_SUM_CORRECTION) && defined(ACCURATE_ACCUMULATION)
#endif//NWARP == 4
#   if  NWARP >= 8
    pj[itag].val[jtag] += pj[itag + 1].val[jtag] + pj[itag + 2].val[jtag] + pj[itag + 3].val[jtag];
#   if  NWARP == 32
    if( gtag < 4 )
      pj[itag].val[jtag] += pj[itag + 16].val[jtag];
#endif//NWARP == 32
#   if  NWARP >= 16
    if( gtag < 2 )
      pj[itag].val[jtag] += pj[itag + 8].val[jtag];
#endif//NWARP >= 16
    if( gtag == 0 ){
#   if  defined(KAHAN_SUM_CORRECTION) && defined(ACCURATE_ACCUMULATION)
      /* T := S, S := S + R; T := S - T; R := R - T */
      real sum = pj[itag].val[jtag] + pj[itag + 4].val[jtag];
      real tmp = atomicAdd(&(iacc[idx].val[jtag]), sum);      tmp = (tmp + sum) - tmp;      sum -= tmp;
      pj[hp + lane].ai = res;
      atomicAdd(&(ires[idx].val[jtag]), sum + pj[itag].val[jtag]);
#else///defined(KAHAN_SUM_CORRECTION) && defined(ACCURATE_ACCUMULATION)
      atomicAdd(&(iacc[idx].val[jtag]), pj[itag].val[jtag] + pj[itag + 4].val[jtag]);
#endif//defined(KAHAN_SUM_CORRECTION) && defined(ACCURATE_ACCUMULATION)
    }/* if( gtag == 0 ){ */
#endif//NWARP >= 8
#else///NWARP > 1
#   if  defined(KAHAN_SUM_CORRECTION) && defined(ACCURATE_ACCUMULATION)
    /* /\* R := R + x_i *\/ */
    /* res.x += ai.x;    res.y += ai.y;    res.z += ai.z;    res.pot += ai.pot; */
    /* T := S, S := S + R; T := S - T; R := R - T */
    acceleration tmp;
    tmp.x   = atomicAdd(&(iacc[idx].ai.x  ), ai.x  );    tmp.x   = (tmp.x   + ai.x  ) - tmp.x  ;    ai.x   -= tmp.x  ;    atomicAdd(&(ires[idx].ai.x  ), ai.x   + res.x  );
    tmp.y   = atomicAdd(&(iacc[idx].ai.y  ), ai.y  );    tmp.y   = (tmp.y   + ai.y  ) - tmp.y  ;    ai.y   -= tmp.y  ;    atomicAdd(&(ires[idx].ai.y  ), ai.y   + res.y  );
    tmp.z   = atomicAdd(&(iacc[idx].ai.z  ), ai.z  );    tmp.z   = (tmp.z   + ai.z  ) - tmp.z  ;    ai.z   -= tmp.z  ;    atomicAdd(&(ires[idx].ai.z  ), ai.z   + res.z  );
    tmp.pot = atomicAdd(&(iacc[idx].ai.pot), ai.pot);    tmp.pot = (tmp.pot + ai.pot) - tmp.pot;    ai.pot -= tmp.pot;    atomicAdd(&(ires[idx].ai.pot), ai.pot + res.pot);
    /* acceleration tmp; */
    /* tmp.x   = atomicAdd(&(iacc[idx].ai.x  ), ai.x  );    tmp.x   = iacc[idx].ai.x   - tmp.x  ;    ai.x   -= tmp.x  ;    atomicAdd(&(ires[idx].ai.x  ), ai.x   + res.x  ); */
    /* tmp.y   = atomicAdd(&(iacc[idx].ai.y  ), ai.y  );    tmp.y   = iacc[idx].ai.y   - tmp.y  ;    ai.y   -= tmp.y  ;    atomicAdd(&(ires[idx].ai.y  ), ai.y   + res.y  ); */
    /* tmp.z   = atomicAdd(&(iacc[idx].ai.z  ), ai.z  );    tmp.z   = iacc[idx].ai.z   - tmp.z  ;    ai.z   -= tmp.z  ;    atomicAdd(&(ires[idx].ai.z  ), ai.z   + res.z  ); */
    /* tmp.pot = atomicAdd(&(iacc[idx].ai.pot), ai.pot);    tmp.pot = iacc[idx].ai.pot - tmp.pot;    ai.pot -= tmp.pot;    atomicAdd(&(ires[idx].ai.pot), ai.pot + res.pot); */
#else///defined(KAHAN_SUM_CORRECTION) && defined(ACCURATE_ACCUMULATION)
#ifndef DPADD_FOR_ACC
    atomicAdd(&(iacc[idx].ai.x  ), ai.x  );
    atomicAdd(&(iacc[idx].ai.y  ), ai.y  );
    atomicAdd(&(iacc[idx].ai.z  ), ai.z  );
    atomicAdd(&(iacc[idx].ai.pot), ai.pot);
#else///DPADD_FOR_ACC
    atomicAdd(&(dacc[idx].x  ), CAST_R2D(ai.x  ) + CAST_R2D(res.x  ));
    atomicAdd(&(dacc[idx].y  ), CAST_R2D(ai.y  ) + CAST_R2D(res.y  ));
    atomicAdd(&(dacc[idx].z  ), CAST_R2D(ai.z  ) + CAST_R2D(res.z  ));
    atomicAdd(&(dacc[idx].pot), CAST_R2D(ai.pot) + CAST_R2D(res.pot));
#endif//DPADD_FOR_ACC
#endif//defined(KAHAN_SUM_CORRECTION) && defined(ACCURATE_ACCUMULATION)
#endif//NWARP > 1
#endif//defined(SERIALIZED_EXECUTION) && (NWARP == 1)
    //---------------------------------------------------------------------
#ifdef  COUNT_INTERACTIONS
    atomicAdd(&(stockNj  [idx]), Nj_tot);
    atomicAdd(&(stockNbuf[idx]), Nb_max);
#endif//COUNT_INTERACTIONS
    //---------------------------------------------------------------------
    if( tidx == head )
      atomicAdd(overflow, fail);
    //---------------------------------------------------------------------
#ifdef  DBG_TREE_WALK
    if( tidx == head )
      printf("mjtot is %e for %d-th thread\n", mjtot, tidx);
#endif//DBG_TREE_WALK
    //---------------------------------------------------------------------
  }/* if( !skip ){ */
  //-----------------------------------------------------------------------
#endif//PRINT_PSEUDO_PARTICLE_INFO
  //-----------------------------------------------------------------------


  //-----------------------------------------------------------------------
#ifdef  USE_SMID_TO_GET_BUFID
  releaseBuffer(tidx, freeLst, (uint)bufIdx, bufTarget);
#else///USE_SMID_TO_GET_BUFID
#ifdef  TRY_MODE_ABOUT_BUFFER
  releaseBuffer(tidx, freeLst, (uint)bufIdx, bufTarget);
#else///TRY_MODE_ABOUT_BUFFER
  releaseBuffer(tidx, freeNum, freeLst, bufIdx, active);
#endif//TRY_MODE_ABOUT_BUFFER
#endif//USE_SMID_TO_GET_BUFID
  //-----------------------------------------------------------------------
#   if  !defined(USE_CUDA_EVENT) && !defined(SERIALIZED_EXECUTION) && !defined(PRINT_PSEUDO_PARTICLE_INFO)
  long long int exitCycle = clock64();
  if( tidx == 0 ){
    unsigned long long int elapsed = (unsigned long long int)(exitCycle - initCycle);
    atomicAdd(cycles, elapsed);
  }/* if( tidx == 0 ){ */
#endif//!defined(USE_CUDA_EVENT) && !defined(SERIALIZED_EXECUTION) && !defined(PRINT_PSEUDO_PARTICLE_INFO)
  //-----------------------------------------------------------------------
}
//-------------------------------------------------------------------------


//-------------------------------------------------------------------------
#if 0
//-------------------------------------------------------------------------
__global__ void printTreeNode_kernel(const int num, READ_ONLY jparticle * RESTRICT pj, READ_ONLY jmass * RESTRICT mj)
{
  for(int ii = 0; ii < num; ii++){
    const jparticle jpos = pj[ii];
    const jmass     mass = mj[ii];
    printf("%e\t%e\t%e\t%e\n", mass, jpos.x, jpos.y, jpos.z);
  }/* for(int ii = 0; ii < num; ii++){ */
}
//-------------------------------------------------------------------------
void printTreeNode_dev(const int num, const soaTreeNode node, MPIinfo mpi)
{
  for(int ii = 0; ii < mpi.size; ii++){
    if( ii == mpi.rank )
      printTreeNode_kernel<<<1, 1>>>(num, node.jpos, node.mj);
    checkCudaErrors(hipDeviceSynchronize());
    fflush(stdout);
    MPI_Barrier(mpi.comm);
    if( ii == mpi.rank )
      printf("# rank %d / %d\n", mpi.rank, mpi.size);
    fflush(stdout);
    MPI_Barrier(mpi.comm);
  }
  MPI_Finalize();
  exit(0);
}
//-------------------------------------------------------------------------
void printFullTree_dev(int n0, int n1, int h1, const soaTreeNode node, MPIinfo mpi)
{
  for(int ii = 0; ii < mpi.size; ii++){
    if( ii == mpi.rank ){
      printTreeNode_kernel<<<1, 1>>>(n0,   node.jpos     ,   node.mj     );
      printTreeNode_kernel<<<1, 1>>>(n1, &(node.jpos[h1]), &(node.mj[h1]));
    }
    checkCudaErrors(hipDeviceSynchronize());
    fflush(stdout);
    MPI_Barrier(mpi.comm);
    if( ii == mpi.rank )
      printf("# rank %d / %d\n", mpi.rank, mpi.size);
    fflush(stdout);
    MPI_Barrier(mpi.comm);
  }
  MPI_Finalize();
  exit(0);
}
//-------------------------------------------------------------------------
__global__ void printTreeLink_kernel(READ_ONLY uint * RESTRICT more, READ_ONLY jparticle * RESTRICT pj, READ_ONLY jmass * RESTRICT mj, uint * RESTRICT buffer)
{
  buffer[0] = 0;
  int rem = 1;
  int head = rem;
  int ii = 0;

  while( true ){
    if( rem == 0 )
      break;

    uint jj = buffer[ii];
    /* printf("%e\t%e\t%e\t%e\t%e\n", mj[jj], pj[jj].x, pj[jj].y, pj[jj].z, pj[jj].w); */
    printf("%e\t%e\t%e\t%e\n", mj[jj], pj[jj].x, pj[jj].y, pj[jj].z);
    rem--;

    uint jcell = more[jj];
    int num = 1 + (jcell >> IDXBITS);
    jcell &= IDXMASK;
    if( jcell == jj )
      num = 0;
    for(int kk = 0; kk < num; kk++)
      buffer[head + kk] = jcell + kk;

    head += num;
    rem += num;
    ii++;
  }
}
//-------------------------------------------------------------------------
 void printTreeLink_dev(int head, const soaTreeNode node, uint * buf, MPIinfo mpi)
{
  checkCudaErrors(hipDeviceSynchronize());
  MPI_Barrier(mpi.comm);
  for(int ii = 0; ii < mpi.size; ii++){
    if( ii == mpi.rank ){
      printTreeLink_kernel<<<1, 1>>>(  node.more       ,   node.jpos     ,     node.mj       , buf);
      printTreeLink_kernel<<<1, 1>>>(&(node.more[head]), &(node.jpos[head]), &(node.mj[head]), buf);
    }
    checkCudaErrors(hipDeviceSynchronize());
    fflush(stdout);
    MPI_Barrier(mpi.comm);
    if( ii == mpi.rank )
      printf("# rank %d / %d\n", mpi.rank, mpi.size);
    fflush(stdout);
    MPI_Barrier(mpi.comm);
  }
  MPI_Finalize();
  exit(0);
}
//-------------------------------------------------------------------------
#endif
//-------------------------------------------------------------------------


//-------------------------------------------------------------------------
/* calculate gravitational acceleration and potential */
//-------------------------------------------------------------------------
/* Ni     :: input          :: total number of N-body particles stored in this process */
/* ipos   :: input          :: position and mass of N-body particles */
/* iacc   ::         output :: acceleration and potential of N-body particles */
/* more   :: input          :: head index and number of child particles of the corresponding j-particle */
/* jpos   :: input          :: position and squared radius of pseudo N-body particle as j-particles */
/* mj     :: input          :: mass of pseudo N-body particle as j-particles */
/* buffer ::                :: tentative memory space to store tree cells which does not fit within the limited space of the shared memory */
//-------------------------------------------------------------------------
static inline void callCalcGravityFunc
(const dim3 blck, const dim3 thrd, kernelStream *sinfo, int *sidx,
 laneinfo * RESTRICT laneInfo, const iparticle pi, const int rootIdx, const soaTreeNode tree
#ifndef SERIALIZED_EXECUTION
 , const int grpNum, const int jhead
#endif//SERIALIZED_EXECUTION
#   if  !defined(SERIALIZED_EXECUTION) || defined(PRINT_PSEUDO_PARTICLE_INFO)
#ifdef  USE_CUDA_EVENT
 , int *Nwalk, hipEvent_t *iniEvent, hipEvent_t *finEvent
#else///USE_CUDA_EVENT
 , unsigned long long int * RESTRICT cycles
#endif//USE_CUDA_EVENT
#endif//!defined(SERIALIZED_EXECUTION) || defined(PRINT_PSEUDO_PARTICLE_INFO)
 , const soaTreeWalkBuf buf
#ifdef  COUNT_INTERACTIONS
 , iparticle_treeinfo treeInfo
#endif//COUNT_INTERACTIONS
 )
{
  //-----------------------------------------------------------------------
  __NOTE__("%s (grpNum = %d, jhead = %d)\n", "start", grpNum, jhead);
  //-----------------------------------------------------------------------
#ifdef  DEBUG_PRINT_FOR_PARTICLE_ACCELERATION
  checkCudaErrors(hipDeviceSynchronize());
  chkMPIerr(MPI_Barrier(MPI_COMM_WORLD));
#endif//DEBUG_PRINT_FOR_PARTICLE_ACCELERATION
  //-----------------------------------------------------------------------
#if 0
  int deviceID;
  checkCudaErrors(hipGetDevice(&deviceID));
  fprintf(stdout, "jhead = %d on device %d\n", jhead, deviceID);
  fflush(stdout);
#endif
  //-----------------------------------------------------------------------
#   if  defined(USE_CUDA_EVENT) && (!defined(SERIALIZED_EXECUTION) || defined(PRINT_PSEUDO_PARTICLE_INFO))
  checkCudaErrors(hipEventRecord(iniEvent[*Nwalk], 0));
#endif//defined(USE_CUDA_EVENT) && (!defined(SERIALIZED_EXECUTION) || defined(PRINT_PSEUDO_PARTICLE_INFO))
  //-----------------------------------------------------------------------
#   if  defined(BLOCK_TIME_STEP) && !defined(SERIALIZED_EXECUTION)
  if( grpNum != 0 ){
#endif//defined(BLOCK_TIME_STEP) && !defined(SERIALIZED_EXECUTION)
    //---------------------------------------------------------------------
    if( blck.x <= MAX_BLOCKS_PER_GRID ){
      //-------------------------------------------------------------------
#if 1
      calcAcc_kernel<<<blck, thrd, SMEM_SIZE, sinfo->stream[*sidx]>>>
#else
      calcAcc_kernel<<<blck, thrd>>>
#endif
	(laneInfo,
#ifdef  BLOCK_TIME_STEP
	 pi.jpos,
#else///BLOCK_TIME_STEP
	 pi.pos,
#endif//BLOCK_TIME_STEP
	 (jnode *)pi.acc,
#ifdef  GADGET_MAC
	 pi.acc_old,
#endif//GADGET_MAC
	 rootIdx,
#ifdef  SERIALIZED_EXECUTION
	 tree.more, tree.jpos, tree.mj,
#else///SERIALIZED_EXECUTION
	 &(tree.more[jhead]), &(tree.jpos[jhead]), &(tree.mj[jhead]),
#endif//SERIALIZED_EXECUTION
#ifdef  DPADD_FOR_ACC
	 pi.tmp,
#endif//DPADD_FOR_ACC
#   if  defined(KAHAN_SUM_CORRECTION) && defined(ACCURATE_ACCUMULATION) && (!defined(SERIALIZED_EXECUTION) || (NWARP > 1))
	 (jnode *)pi.res,
#endif//defined(KAHAN_SUM_CORRECTION) && defined(ACCURATE_ACCUMULATION) && (!defined(SERIALIZED_EXECUTION) || (NWARP > 1))
#ifndef USE_SMID_TO_GET_BUFID
#ifndef TRY_MODE_ABOUT_BUFFER
	 buf.active,
#endif//TRY_MODE_ABOUT_BUFFER
	 buf.freeNum,
#endif//USE_SMID_TO_GET_BUFID
	 buf.freeLst, buf.buffer, buf.bufSize, buf.fail
#   if  !defined(USE_CUDA_EVENT) && (!defined(SERIALIZED_EXECUTION) || defined(PRINT_PSEUDO_PARTICLE_INFO))
	 , cycles
#endif//!defined(USE_CUDA_EVENT) && (!defined(SERIALIZED_EXECUTION) || defined(PRINT_PSEUDO_PARTICLE_INFO))
#ifdef  COUNT_INTERACTIONS
	 , treeInfo.Nj, treeInfo.Nbuf
#endif//COUNT_INTERACTIONS
	 );
      //-------------------------------------------------------------------
      *sidx ^= 1;
      //-------------------------------------------------------------------
    }/* if( blck.x <= MAX_BLOCKS_PER_GRID ){ */
    //---------------------------------------------------------------------
    else{
      //-------------------------------------------------------------------
      int Nrem = blck.x;
      const int Niter = BLOCKSIZE(Nrem, MAX_BLOCKS_PER_GRID);
      int hidx = 0;
      //-------------------------------------------------------------------
      for(int iter = 0; iter < Niter; iter++){
	//-----------------------------------------------------------------
	int Nblck = MAX_BLOCKS_PER_GRID;
	if( Nblck > Nrem )	  Nblck = Nrem;
	//-----------------------------------------------------------------
	int Nsub = Nblck * NGROUPS;
	calcAcc_kernel<<<Nblck, thrd.x, SMEM_SIZE, sinfo->stream[*sidx]>>>
	  (&laneInfo[hidx],
#ifdef  BLOCK_TIME_STEP
	   pi.jpos,
#else///BLOCK_TIME_STEP
	   pi.pos,
#endif//BLOCK_TIME_STEP
	   (jnode *)pi.acc,
#ifdef  GADGET_MAC
	   pi.acc_old,
#endif//GADGET_MAC
	   rootIdx,
#ifdef  SERIALIZED_EXECUTION
	   tree.more, tree.jpos, tree.mj,
#else///SERIALIZED_EXECUTION
	   &(tree.more[jhead]), &(tree.jpos[jhead]), &(tree.mj[jhead]),
#endif//SERIALIZED_EXECUTION
#ifdef  DPADD_FOR_ACC
	   pi.tmp,
#endif//DPADD_FOR_ACC
#   if  defined(KAHAN_SUM_CORRECTION) && defined(ACCURATE_ACCUMULATION) && (!defined(SERIALIZED_EXECUTION) || (NWARP > 1))
	   (jnode *)pi.res,
#endif//defined(KAHAN_SUM_CORRECTION) && defined(ACCURATE_ACCUMULATION) && (!defined(SERIALIZED_EXECUTION) || (NWARP > 1))
#ifndef USE_SMID_TO_GET_BUFID
#ifndef TRY_MODE_ABOUT_BUFFER
	   buf.active,
#endif//TRY_MODE_ABOUT_BUFFER
	   buf.freeNum,
#endif//USE_SMID_TO_GET_BUFID
	   buf.freeLst, buf.buffer, buf.bufSize, buf.fail
#   if  !defined(USE_CUDA_EVENT) && (!defined(SERIALIZED_EXECUTION) || defined(PRINT_PSEUDO_PARTICLE_INFO))
	   , cycles
#endif//!defined(USE_CUDA_EVENT) && (!defined(SERIALIZED_EXECUTION) || defined(PRINT_PSEUDO_PARTICLE_INFO))
#ifdef  COUNT_INTERACTIONS
	   , treeInfo.Nj, treeInfo.Nbuf
#endif//COUNT_INTERACTIONS
	   );
	//-----------------------------------------------------------------
	hidx += Nsub;
	Nrem -= Nblck;
	//-----------------------------------------------------------------
	*sidx ^= 1;
	//-----------------------------------------------------------------
      }/* for(int iter = 0; iter < Niter; iter++){ */
      //-------------------------------------------------------------------
    }/* else{ */
    //---------------------------------------------------------------------
/* #ifndef SERIALIZED_EXECUTION */
/*     /\* evaluate GPU time based on clock cycle counter *\/ */
/*     checkCudaErrors(hipMemcpyAsync(&clockCycles, *cycles, sizeof(unsigned long long int), hipMemcpyDeviceToHost, sinfo->stream[*sidx])); */
/* #endif//SERIALIZED_EXECUTION */
    //---------------------------------------------------------------------
#   if  defined(BLOCK_TIME_STEP) && !defined(SERIALIZED_EXECUTION)
  }/* if( grpNum != 0 ){ */
#endif//defined(BLOCK_TIME_STEP) && !defined(SERIALIZED_EXECUTION)
  //-----------------------------------------------------------------------
#   if  defined(USE_CUDA_EVENT) && (!defined(SERIALIZED_EXECUTION) || defined(PRINT_PSEUDO_PARTICLE_INFO))
  checkCudaErrors(hipEventRecord(finEvent[*Nwalk], 0));
  *Nwalk += 1;
#endif//defined(USE_CUDA_EVENT) && (!defined(SERIALIZED_EXECUTION) || defined(PRINT_PSEUDO_PARTICLE_INFO))
  //-----------------------------------------------------------------------
#ifdef  DEBUG_PRINT_FOR_PARTICLE_ACCELERATION
  checkCudaErrors(hipDeviceSynchronize());
  chkMPIerr(MPI_Barrier(MPI_COMM_WORLD));
  fflush(NULL);
  printf("calcAcc finish\n");
  fflush(NULL);
  chkMPIerr(MPI_Barrier(MPI_COMM_WORLD));
#endif//DEBUG_PRINT_FOR_PARTICLE_ACCELERATION
  //-----------------------------------------------------------------------
  __NOTE__("%s\n", "end");
  //-----------------------------------------------------------------------
}
//-------------------------------------------------------------------------
extern "C"
void calcGravity_dev
(const int grpNum
#ifdef  BLOCK_TIME_STEP
 , double *reduce, const int totNum
#endif//BLOCK_TIME_STEP
 , laneinfo * RESTRICT laneInfo, const int Ni, const iparticle pi, const soaTreeNode tree, const soaTreeWalkBuf buf
 , kernelStream *sinfo, deviceProp devProp, double *time
#ifdef  PRINT_PSEUDO_PARTICLE_INFO
 , char *file
#endif//PRINT_PSEUDO_PARTICLE_INFO
#   if  !defined(SERIALIZED_EXECUTION) || defined(PRINT_PSEUDO_PARTICLE_INFO)
#ifdef  USE_CUDA_EVENT
 , hipEvent_t *iniCalcAcc, hipEvent_t *finCalcAcc
#else///USE_CUDA_EVENT
 , unsigned long long int *cycles_hst, unsigned long long int *cycles_dev
#endif//USE_CUDA_EVENT
#endif//!defined(SERIALIZED_EXECUTION) || defined(PRINT_PSEUDO_PARTICLE_INFO)
#ifndef SERIALIZED_EXECUTION
 , measuredTime *measured, const int pjNum
#ifdef  LET_COMMUNICATION_VIA_HOST
 , const soaTreeNode tree_hst
#endif//LET_COMMUNICATION_VIA_HOST
 , const int Nlet, domainInfo *let, const int Nstream_let, hipStream_t stream_let[], MPIcfg_tree mpi
#ifdef  MONITOR_LETGEN_TIME
#ifdef  USE_CUDA_EVENT
 , hipEvent_t *iniMakeLET, hipEvent_t *finMakeLET
#else///USE_CUDA_EVENT
 , unsigned long long int *cycles_let_hst, unsigned long long int *cycles_let_dev
#endif//USE_CUDA_EVENT
#endif//MONITOR_LETGEN_TIME
#endif//SERIALIZED_EXECUTION
#ifdef  COUNT_INTERACTIONS
 , iparticle_treeinfo treeInfo
#endif//COUNT_INTERACTIONS
#ifdef  EXEC_BENCHMARK
 , wall_clock_time *elapsed
#endif//EXEC_BENCHMARK
#ifdef  COMPARE_WITH_DIRECT_SOLVER
 , const bool approxGravity
#ifdef  INDIVIDUAL_GRAVITATIONAL_SOFTENING
 , const real eps2
#endif//INDIVIDUAL_GRAVITATIONAL_SOFTENING
#endif//COMPARE_WITH_DIRECT_SOLVER
 )
{
  //-----------------------------------------------------------------------
  __NOTE__("%s\n", "start");
  //-----------------------------------------------------------------------
  int Nrem;
  //-----------------------------------------------------------------------
#ifdef  COUNT_INTERACTIONS
  /* initialize count of Nj and Nbuf */
  initCounter_kernel<<<BLOCKSIZE(Ni, NTHREADS), NTHREADS>>>(treeInfo.Nj, treeInfo.Nbuf);
  getLastCudaError("initCounter_kernel");
#endif//COUNT_INTERACTIONS
  //-----------------------------------------------------------------------
#if 0
  fprintf(stdout, "rank %d: grpNum = %d\n", mpi.rank, grpNum);
  fflush(stdout);
#endif
  //-----------------------------------------------------------------------
#if 0
  jparticle *pj_hst;
  mycudaMallocHost((void **)&pj_hst, pjNum * sizeof(jparticle));
  checkCudaErrors(hipMemcpy(pj_hst, tree.jpos, pjNum * sizeof(jparticle), hipMemcpyDeviceToHost));
  jmass *mj_hst;
  mycudaMallocHost((void **)&mj_hst, pjNum * sizeof(jmass));
  checkCudaErrors(hipMemcpy(mj_hst, tree.mj, pjNum * sizeof(jmass), hipMemcpyDeviceToHost));

  FILE *fp;
  char filename[128];
  sprintf(filename, "%s/%s.%d_%d.txt", "dat", "jpos", mpi.rank, mpi.size);
  fp = fopen(filename, "w");
  for(int ii = 0; ii < pjNum; ii++)
    fprintf(fp, "%e\t%e\t%e\t%e\n", pj_hst[ii].x, pj_hst[ii].y, pj_hst[ii].z, mj_hst[ii]);

  fclose(fp);
  mycudaFreeHost(pj_hst);
  mycudaFreeHost(mj_hst);
  MPI_Finalize();
  exit(0);
#endif
  //-----------------------------------------------------------------------


  //-----------------------------------------------------------------------
  /* set thread-block configuration */
  static dim3 thrd, blck;
  thrd.x =                           NTHREADS;  thrd.y = 1;  thrd.z = 1;
  /* blck.x = BLOCKSIZE(grpNum * NWARP, NGROUPS);  blck.y = 1;  blck.z = 1; */
  blck.x = BLOCKSIZE(grpNum, NGROUPS);  blck.y = 1;  blck.z = 1;
  //-----------------------------------------------------------------------
#if 0
  fprintf(stdout, "rank %d: grpNum = %d with Nsm is %d (%d loops); amin = %e, pos = (%e, %e, %e), r = %e; bufSize is %d\n",
	  mpi.rank, grpNum, devProp.numSM, BLOCKSIZE(blck.x, NBLOCKS_PER_SM * devProp.numSM),
	  let[0].amin, let[0].icom.x, let[0].icom.y, let[0].icom.z, SQRT(let[0].icom.m),
	  buf.bufSize);
  fflush(stdout);
#endif
  //-----------------------------------------------------------------------
  /* initialize measurement counters */
#ifdef  USE_CUDA_EVENT
#   if  !defined(SERIALIZED_EXECUTION) || defined(PRINT_PSEUDO_PARTICLE_INFO)
  int Nwalk = 0;
#endif//!defined(SERIALIZED_EXECUTION) || defined(PRINT_PSEUDO_PARTICLE_INFO)
#   if  !defined(SERIALIZED_EXECUTION) && defined(MONITOR_LETGEN_TIME)
  int Nmake = 0;
#endif//!defined(SERIALIZED_EXECUTION) && defined(MONITOR_LETGEN_TIME)
#else///USE_CUDA_EVENT
#   if  !defined(SERIALIZED_EXECUTION) || defined(PRINT_PSEUDO_PARTICLE_INFO)
  *cycles_hst = 0;
  checkCudaErrors(hipMemcpy(cycles_dev, cycles_hst, sizeof(unsigned long long int), hipMemcpyHostToDevice));
#endif//!defined(SERIALIZED_EXECUTION) || defined(PRINT_PSEUDO_PARTICLE_INFO)
#   if  !defined(SERIALIZED_EXECUTION) && defined(MONITOR_LETGEN_TIME)
  *cycles_let_hst = 0;
  checkCudaErrors(hipMemcpy(cycles_let_dev, cycles_let_hst, sizeof(unsigned long long int), hipMemcpyHostToDevice));
#endif//!defined(SERIALIZED_EXECUTION) && defined(MONITOR_LETGEN_TIME)
#endif//USE_CUDA_EVENT
  //-----------------------------------------------------------------------
#ifdef  USE_MEASURED_CLOCK_FREQ
  uint clockWalk;/* in MHz */
#endif//USE_MEASURED_CLOCK_FREQ
  //-----------------------------------------------------------------------
#   if  defined(SERIALIZED_EXECUTION) || defined(EXEC_BENCHMARK)
  static struct timeval start;
  checkCudaErrors(hipDeviceSynchronize());
  gettimeofday(&start, NULL);
#endif//defined(SERIALIZED_EXECUTION) || defined(EXEC_BENCHMARK)
  //-----------------------------------------------------------------------


  //-----------------------------------------------------------------------
  /* initialize acceleration and potential */
  //-----------------------------------------------------------------------
#ifdef  BLOCK_TIME_STEP
  Nrem = BLOCKSIZE(grpNum, NWARP * NGROUPS);
#else///BLOCK_TIME_STEP
  Nrem = BLOCKSIZE(Ni, NTHREADS);
#endif//BLOCK_TIME_STEP
  //-----------------------------------------------------------------------
  /* when grid splitting is not required... */
  if( Nrem <= MAX_BLOCKS_PER_GRID ){
#ifdef  BLOCK_TIME_STEP
#ifndef SERIALIZED_EXECUTION
    if( grpNum != 0 )
#endif//SERIALIZED_EXECUTION
      initAcc_kernel<<<Nrem, thrd>>>
	(pi.acc, BLOCKSIZE(grpNum, NGROUPS) * NGROUPS, laneInfo
#ifdef  GADGET_MAC
	 , pi.acc_old
#endif//GADGET_MAC
#ifdef  DPADD_FOR_ACC
	 , pi.tmp
#endif//DPADD_FOR_ACC
#   if  defined(KAHAN_SUM_CORRECTION) && defined(ACCURATE_ACCUMULATION) && (!defined(SERIALIZED_EXECUTION) || (NWARP > 1))
	 , pi.res
#endif//defined(KAHAN_SUM_CORRECTION) && defined(ACCURATE_ACCUMULATION) && (!defined(SERIALIZED_EXECUTION) || (NWARP > 1))
	 );
#else///BLOCK_TIME_STEP
    initAcc_kernel<<<Nrem, NTHREADS>>>
      (pi.acc
#ifdef  GADGET_MAC
       , pi.acc_old
#endif//GADGET_MAC
#ifdef  DPADD_FOR_ACC
       , pi.tmp
#endif//DPADD_FOR_ACC
#   if  defined(KAHAN_SUM_CORRECTION) && defined(ACCURATE_ACCUMULATION) && (!defined(SERIALIZED_EXECUTION) || (NWARP > 1))
       , pi.res
#endif//defined(KAHAN_SUM_CORRECTION) && defined(ACCURATE_ACCUMULATION) && (!defined(SERIALIZED_EXECUTION) || (NWARP > 1))
       );
#endif//BLOCK_TIME_STEP
  }/* if( Nrem <= MAX_BLOCKS_PER_GRID ){ */
  //-----------------------------------------------------------------------
  /* when grid splitting is required... */
  else{
    //---------------------------------------------------------------------
    const int Niter = BLOCKSIZE(Nrem, MAX_BLOCKS_PER_GRID);
    int hidx = 0;
    //---------------------------------------------------------------------
    for(int iter = 0; iter < Niter; iter++){
      //-------------------------------------------------------------------
      int Nblck = MAX_BLOCKS_PER_GRID;
      if( Nblck > Nrem )	Nblck = Nrem;
      //-------------------------------------------------------------------
#ifdef  BLOCK_TIME_STEP
      int Nsub = Nblck * NWARP * NGROUPS;
      initAcc_kernel<<<Nblck, thrd.x>>>
	(pi.acc, BLOCKSIZE(Nsub, NGROUPS) * NGROUPS, &laneInfo[hidx]
#ifdef  GADGET_MAC
	 , pi.acc_old
#endif//GADGET_MAC
#ifdef  DPADD_FOR_ACC
	 , pi.tmp
#endif//DPADD_FOR_ACC
#   if  defined(KAHAN_SUM_CORRECTION) && defined(ACCURATE_ACCUMULATION) && (!defined(SERIALIZED_EXECUTION) || (NWARP > 1))
	 , pi.res
#endif//defined(KAHAN_SUM_CORRECTION) && defined(ACCURATE_ACCUMULATION) && (!defined(SERIALIZED_EXECUTION) || (NWARP > 1))
	 );
#else///BLOCK_TIME_STEP
      int Nsub = Nblck * NTHREADS;
      initAcc_kernel<<<Nblck, NTHREADS>>>
	(&pi.acc[hidx]
#ifdef  GADGET_MAC
	 , &pi.acc_old[hidx]
#endif//GADGET_MAC
#ifdef  DPADD_FOR_ACC
	 , &pi.tmp[hidx]
#endif//DPADD_FOR_ACC
#   if  defined(KAHAN_SUM_CORRECTION) && defined(ACCURATE_ACCUMULATION) && (!defined(SERIALIZED_EXECUTION) || (NWARP > 1))
	 , &pi.res[hidx]
#endif//defined(KAHAN_SUM_CORRECTION) && defined(ACCURATE_ACCUMULATION) && (!defined(SERIALIZED_EXECUTION) || (NWARP > 1))
	 );
#endif//BLOCK_TIME_STEP
      //-------------------------------------------------------------------
      hidx += Nsub;
      Nrem -= Nblck;
      //-------------------------------------------------------------------
    }/* for(int iter = 0; iter < Niter; iter++){ */
    //---------------------------------------------------------------------
  }/* else{ */
  //-----------------------------------------------------------------------
#if 0
  checkCudaErrors(hipDeviceSynchronize());
#endif
  getLastCudaError("initAcc_kernel");
  //-----------------------------------------------------------------------
  /* calculate gravitational acceleration based on the width-first tree traversal */
  //-----------------------------------------------------------------------
#ifdef  COMPARE_WITH_DIRECT_SOLVER
  if( approxGravity )
#endif//COMPARE_WITH_DIRECT_SOLVER
    {
      //-------------------------------------------------------------------
      /* gravity from j-particles within local process */
      //-------------------------------------------------------------------
      /* set CUDA streams */
      int sidx = sinfo->idx;
      //-------------------------------------------------------------------
      callCalcGravityFunc(blck, thrd, sinfo, &sidx, laneInfo, pi, 0, tree
#ifndef SERIALIZED_EXECUTION
			  , grpNum, 0
#endif//SERIALIZED_EXECUTION
#   if  !defined(SERIALIZED_EXECUTION) || defined(PRINT_PSEUDO_PARTICLE_INFO)
#ifdef  USE_CUDA_EVENT
			  , &Nwalk, iniCalcAcc, finCalcAcc
#else///USE_CUDA_EVENT
			  , cycles_dev
#endif//USE_CUDA_EVENT
#endif//!defined(SERIALIZED_EXECUTION) || defined(PRINT_PSEUDO_PARTICLE_INFO)
			  , buf
#ifdef  COUNT_INTERACTIONS
			  , treeInfo
#endif//COUNT_INTERACTIONS
			  );
      //-------------------------------------------------------------------
      /* estimate performance indicator of block time step */
#ifdef  BLOCK_TIME_STEP
      const double block = (double)BLOCKSIZE(BLOCKSIZE(grpNum, NGROUPS), NBLOCKS_PER_SM * devProp.numSM);
      const double share = (double)BLOCKSIZE(BLOCKSIZE(totNum, NGROUPS), NBLOCKS_PER_SM * devProp.numSM);
#ifdef  WALK_TREE_COMBINED_MODEL
      *reduce = share / block;
#else///WALK_TREE_COMBINED_MODEL
      *reduce = block / share;
#endif//WALK_TREE_COMBINED_MODEL
#endif//BLOCK_TIME_STEP
      //-------------------------------------------------------------------
#ifdef  USE_MEASURED_CLOCK_FREQ
      /* measure clock frequency as a reference value */
      nvmlDeviceGetClock(deviceHandler, NVML_CLOCK_SM, NVML_CLOCK_ID_CURRENT, &clockWalk);
#endif//USE_MEASURED_CLOCK_FREQ
      //-------------------------------------------------------------------


      //-------------------------------------------------------------------
#ifndef SERIALIZED_EXECUTION
      //-------------------------------------------------------------------
      /* gravity from j-particles within other process(es) */
      //-------------------------------------------------------------------
      /* rewrite from MPI_Isend/MPI_Irecv to MPI_Put may accelerate the simulation */
      //-------------------------------------------------------------------
      int idxProcs = 0;
      int remProcs = Nlet - 1;
#ifdef  DOUBLE_BUFFER_FOR_LET
      static int headLETsend[2], headLETrecv[2], sizeLETbuf[2], sizeLETsend[2], sizeLETrecv[2];
      /* 1st half */
      headLETsend[0] = ALIGN_BUF_FOR_LET(pjNum);
      sizeLETbuf [0] = ((int)ceilf(EXTEND_NUM_TREE_NODE * (float)NUM_ALLOC_TREE_NODE) - headLETsend[0]) >> 1;
      headLETrecv[0] = ALIGN_BUF_FOR_LET(headLETsend[0] + (sizeLETbuf[0] >> 1));
      sizeLETsend[0] = headLETrecv[0] - headLETsend[0];
      sizeLETrecv[0] = sizeLETbuf [0] - sizeLETsend[0];
      /* 2nd half */
      headLETsend[1] = ALIGN_BUF_FOR_LET(headLETrecv[0] + sizeLETrecv[0]);
      sizeLETbuf [1] = (int)ceilf(EXTEND_NUM_TREE_NODE * (float)NUM_ALLOC_TREE_NODE) - headLETsend[1];
      headLETrecv[1] = ALIGN_BUF_FOR_LET(headLETsend[1] + (sizeLETbuf[1] >> 1));
      sizeLETsend[1] = headLETrecv[1] - headLETsend[1];
      sizeLETrecv[1] = sizeLETbuf [1] - sizeLETsend[1];
#else///DOUBLE_BUFFER_FOR_LET
      int LETsteps = 0;
      const int headLETsend = ALIGN_BUF_FOR_LET(pjNum);
      const int  remLETbuf  = (int)ceilf(EXTEND_NUM_TREE_NODE * (float)NUM_ALLOC_TREE_NODE) - headLETsend;
      const int  remLETsend = ALIGN_BUF_FOR_LET(remLETbuf >> 1);
      const int  remLETrecv = remLETbuf - remLETsend;
      const int headLETrecv = headLETsend + remLETsend;
#endif//DOUBLE_BUFFER_FOR_LET
#   if  defined(USE_CUDA_EVENT) && defined(MONITOR_LETGEN_TIME)
      int prevLETstreams = 0;
#endif//defined(USE_CUDA_EVENT) && defined(MONITOR_LETGEN_TIME)
      while( true ){
	//-----------------------------------------------------------------
	/* get maximum number of processes which possible to communicate by limitation of memory capacity */
	//-----------------------------------------------------------------
#ifdef  DOUBLE_BUFFER_FOR_LET
	int remSend = sizeLETsend[sidx];
	int remRecv = sizeLETrecv[sidx];
#else///DOUBLE_BUFFER_FOR_LET
	int remSend = remLETsend;
	int remRecv = remLETrecv;
#endif//DOUBLE_BUFFER_FOR_LET
	int numProcs = remProcs;
	for(int ii = 0; ii < remProcs; ii++){
	  //---------------------------------------------------------------
	  remSend -= let[idxProcs + ii].maxSend;
	  remRecv -= let[idxProcs + ii].maxRecv;
	  //---------------------------------------------------------------
	  if( (remSend < 0) || (remRecv < 0) ){
	    numProcs = ii - 1;
	    break;
	  }/* if( (remSend < 0) || (remRecv < 0) ){ */
	  //---------------------------------------------------------------
	}/* for(int ii = 0; ii < remProcs; ii++){ */
	//-----------------------------------------------------------------
	if( (numProcs < 1) && (mpi.size > 1) ){
	  __KILL__(stderr, "ERROR: numProcs is %d, due to lack of sizeLETsend(%d) or sizeLETrecv(%d) while 0-th target requires numSend(%d) and numRecv(%d) @ rank %d.\n\tIncrease EXTEND_NUM_TREE_NODE(%f) defined in src/tree/let.h and/or TREE_SAFETY_VAL(%f) defined in src/tree/make.h.\n", numProcs,
#ifdef  DOUBLE_BUFFER_FOR_LET
		   sizeLETsend[sidx], sizeLETrecv[sidx],
#else///DOUBLE_BUFFER_FOR_LET
		   remLETsend, remLETrecv,
#endif//DOUBLE_BUFFER_FOR_LET
		   let[idxProcs].maxSend, let[idxProcs].maxRecv, mpi.rank, EXTEND_NUM_TREE_NODE, TREE_SAFETY_VAL);
	}/* if( (numProcs < 1) && (mpi.size > 1) ){ */
	chkMPIerr(MPI_Allreduce(MPI_IN_PLACE, &numProcs, 1, MPI_INT, MPI_MIN, mpi.comm));
	//-----------------------------------------------------------------
	/* set send buffer for LET on device */
#ifdef  DOUBLE_BUFFER_FOR_LET
	let[idxProcs].headSend = headLETsend[sidx];
#else///DOUBLE_BUFFER_FOR_LET
	let[idxProcs].headSend = headLETsend;
#endif//DOUBLE_BUFFER_FOR_LET
	for(int ii = 0; ii < numProcs - 1; ii++)
	  let[idxProcs + ii + 1].headSend = let[idxProcs + ii].headSend + ALIGN_BUF_FOR_LET(let[idxProcs + ii].maxSend);
	//-----------------------------------------------------------------


	//-----------------------------------------------------------------
	/* FUTURE UPDATE: divide below procedure to overlap communication and calculation */
	//-----------------------------------------------------------------
	/* generate numProcs LET(s) */
	for(int ii = idxProcs; ii < idxProcs + numProcs; ii++){
	  //---------------------------------------------------------------
	  const int streamIdxLET = ii % Nstream_let;
	  //---------------------------------------------------------------
	  callGenLET(stream_let[streamIdxLET], &let[ii], mpi, tree, buf
#ifdef  MONITOR_LETGEN_TIME
#ifdef  USE_CUDA_EVENT
		     , iniMakeLET[Nmake], finMakeLET[Nmake]
#else///USE_CUDA_EVENT
		     , cycles_let_dev
#endif//USE_CUDA_EVENT
#endif//MONITOR_LETGEN_TIME
		     );
#   if  defined(USE_CUDA_EVENT) && defined(MONITOR_LETGEN_TIME)
	  Nmake++;
#endif//defined(USE_CUDA_EVENT) && defined(MONITOR_LETGEN_TIME)
	  //---------------------------------------------------------------
	  checkCudaErrors(hipMemcpyAsync(let[ii].numSend_hst, let[ii].numSend_dev, sizeof(int), hipMemcpyDeviceToHost, stream_let[streamIdxLET]));
	  //---------------------------------------------------------------
#if 0
	  printf("rank %d: grpNum = %d, Nlet = %d\n", mpi.rank, grpNum, *(let[ii].numSend_hst));
	  fflush(stdout);
#endif
	  //---------------------------------------------------------------
	}/* for(int ii = idxProcs; ii < idxProcs + numProcs; ii++){ */
	//-----------------------------------------------------------------
	/* share # of LET nodes */
	for(int ii = idxProcs; ii < idxProcs + numProcs; ii++){
	  //---------------------------------------------------------------
	  const int streamIdxLET = ii % Nstream_let;
	  //---------------------------------------------------------------
	  checkCudaErrors(hipStreamSynchronize(stream_let[streamIdxLET]));
	  let[ii].numSend = *(let[ii].numSend_hst);
	  if( let[ii].numSend > let[ii].maxSend ){
	    __KILL__(stderr, "ERROR: predicted size of send buffer (%d) is not sufficient for true size of that (%d) @ rank %d for rand %d.\n\tsuggestion: consider increasing \"LETSIZE_REDUCE_FACTOR\" defined in src/tree/let.h (current value is %f) to at least %f.\n", let[ii].maxSend, let[ii].numSend, mpi.rank, let[ii].rank, LETSIZE_REDUCE_FACTOR, LETSIZE_REDUCE_FACTOR * (float)let[ii].numSend / (float)let[ii].maxSend);
	  }/* if( let[ii].numSend > let[ii].maxSend ){ */
	  __NOTE__("numSend = %d, numFull = %d @ rank %d\n", let[ii].numSend, let[ii].numFull, mpi.rank);
	  //---------------------------------------------------------------
	  /* send # of LET nodes */
	  chkMPIerr(MPI_Isend(&(let[ii].numSend), 1, MPI_INT, let[ii].rank,  mpi.rank, mpi.comm, &(let[ii].reqSendInfo)));
	  let[ii].numRecvGuess = let[ii].maxRecv;
	  chkMPIerr(MPI_Irecv(&(let[ii].numRecv), 1, MPI_INT, let[ii].rank, let[ii].rank, mpi.comm, &(let[ii].reqRecvInfo)));
#ifdef  DBG_LETGEN_ON_GPU
	  fprintf(stdout, "rank = %d: ii = %d: LET(target is %d): numSend = %d out of %d nodes\n", mpi.rank, ii, let[ii].rank, let[ii].numSend, pjNum);
	  fflush(stdout);
#endif//DBG_LETGEN_ON_GPU
	  //---------------------------------------------------------------
	  /* copy LET nodes from device to host */
#ifdef  LET_COMMUNICATION_VIA_HOST
	  checkCudaErrors(hipMemcpyAsync(&(tree_hst.more[let[ii].headSend]), &(tree.more[let[ii].headSend]), sizeof(     uint) * let[ii].numSend, hipMemcpyDeviceToHost, stream_let[streamIdxLET]));
	  checkCudaErrors(hipMemcpyAsync(&(tree_hst.jpos[let[ii].headSend]), &(tree.jpos[let[ii].headSend]), sizeof(jparticle) * let[ii].numSend, hipMemcpyDeviceToHost, stream_let[streamIdxLET]));
	  checkCudaErrors(hipMemcpyAsync(&(tree_hst.mj  [let[ii].headSend]), &(tree.mj  [let[ii].headSend]), sizeof(    jmass) * let[ii].numSend, hipMemcpyDeviceToHost, stream_let[streamIdxLET]));
#endif//LET_COMMUNICATION_VIA_HOST
	  //---------------------------------------------------------------
	}/* for(int ii = idxProcs; ii < idxProcs + numProcs; ii++){ */
	//-----------------------------------------------------------------
	/* send numProcs LET(s) to other process(es) */
	for(int ii = idxProcs; ii < idxProcs + numProcs; ii++){
	  //---------------------------------------------------------------
	  const int streamIdxLET = ii % Nstream_let;
	  //---------------------------------------------------------------
	  /* send LET nodes using MPI_Isend */
#ifdef  LET_COMMUNICATION_VIA_HOST
	  checkCudaErrors(hipStreamSynchronize(stream_let[streamIdxLET]));
	  chkMPIerr(MPI_Isend(&(tree_hst.more[let[ii].headSend]), let[ii].numSend, mpi.more, let[ii].rank, mpi.rank, mpi.comm, &(let[ii].reqSendMore)));
	  chkMPIerr(MPI_Isend(&(tree_hst.jpos[let[ii].headSend]), let[ii].numSend, mpi.jpos, let[ii].rank, mpi.rank, mpi.comm, &(let[ii].reqSendJpos)));
	  chkMPIerr(MPI_Isend(&(tree_hst.mj  [let[ii].headSend]), let[ii].numSend, mpi.mass, let[ii].rank, mpi.rank, mpi.comm, &(let[ii].reqSendMass)));
#else///LET_COMMUNICATION_VIA_HOST
	  chkMPIerr(MPI_Isend(&(tree.more[let[ii].headSend]), let[ii].numSend, mpi.more, let[ii].rank, mpi.rank, mpi.comm, &(let[ii].reqSendMore)));
	  chkMPIerr(MPI_Isend(&(tree.jpos[let[ii].headSend]), let[ii].numSend, mpi.jpos, let[ii].rank, mpi.rank, mpi.comm, &(let[ii].reqSendJpos)));
	  chkMPIerr(MPI_Isend(&(tree.mj  [let[ii].headSend]), let[ii].numSend, mpi.mass, let[ii].rank, mpi.rank, mpi.comm, &(let[ii].reqSendMass)));
#endif//LET_COMMUNICATION_VIA_HOST
	  //---------------------------------------------------------------
	}/* for(int ii = idxProcs; ii < idxProcs + numProcs; ii++){ */
	//-----------------------------------------------------------------
	/* chkMPIerr(MPI_Barrier(mpi.comm)); */
	//-----------------------------------------------------------------

	//-----------------------------------------------------------------
	/* receive # of LET nodes */
	//-----------------------------------------------------------------
	for(int ii = idxProcs; ii < idxProcs + numProcs; ii++){
	  //---------------------------------------------------------------
	  MPI_Status status;
	  chkMPIerr(MPI_Wait(&(let[ii].reqRecvInfo), &status));
#ifdef  DBG_LETGEN_ON_GPU
	  fprintf(stdout, "rank = %d: ii = %d: LET(origin is %d): numRecv = %d\n",
		  mpi.rank, ii, let[ii].rank, let[ii].numRecv);
	  fflush(stdout);
#endif//DBG_LETGEN_ON_GPU
	  //---------------------------------------------------------------
	}/* for(int ii = idxProcs; ii < idxProcs + numProcs; ii++){ */
	//-----------------------------------------------------------------
	/* chkMPIerr(MPI_Barrier(mpi.comm)); */
	//-----------------------------------------------------------------
	/* set receive buffer for LET on device */
#ifdef  DOUBLE_BUFFER_FOR_LET
	let[idxProcs].headRecv = headLETrecv[sidx];
#else///DOUBLE_BUFFER_FOR_LET
	let[idxProcs].headRecv = headLETrecv;
#endif//DOUBLE_BUFFER_FOR_LET
	int numRecv = 0;
	for(int ii = 0; ii < numProcs - 1; ii++){
	  const int numRecvBuf = ALIGN_BUF_FOR_LET(let[idxProcs + ii].numRecv);
	  let[idxProcs + ii + 1].headRecv = numRecvBuf + let[idxProcs + ii].headRecv;
	  numRecv                        += numRecvBuf;
	}/* for(int ii = 0; ii < numProcs - 1; ii++){ */
	numRecv += ALIGN_BUF_FOR_LET(let[idxProcs + numProcs - 1].numRecv);
	//-----------------------------------------------------------------
#ifdef  DOUBLE_BUFFER_FOR_LET
	if( numRecv > sizeLETrecv[sidx] )
#else///DOUBLE_BUFFER_FOR_LET
	if( numRecv > remLETrecv )
#endif//DOUBLE_BUFFER_FOR_LET
	  {
	    __KILL__(stderr, "ERROR: lack of remLETrecv(%d) to store numRecv(%d) LET nodes.\n\tIncrease EXTEND_NUM_TREE_NODE(%f) defined in src/tree/let.h and/or TREE_SAFETY_VAL(%f) defined in src/tree/make.h.\n",
#ifdef  DOUBLE_BUFFER_FOR_LET
		     sizeLETrecv[sidx],
#else///DOUBLE_BUFFER_FOR_LET
		     remLETrecv,
#endif//DOUBLE_BUFFER_FOR_LET
		     numRecv, EXTEND_NUM_TREE_NODE, TREE_SAFETY_VAL);
	  }
	//-----------------------------------------------------------------

	//-----------------------------------------------------------------
	/* receive LET nodes */
	//-----------------------------------------------------------------
	/* before receiving LET nodes, gravity calculation using LET nodes stored in the receive buffer in the previous loop must be finished */
#ifndef DOUBLE_BUFFER_FOR_LET
	if( LETsteps > 0 ){
#   if  defined(BLOCK_TIME_STEP) && !defined(SERIALIZED_EXECUTION)
	  if( grpNum != 0 ){
#endif//defined(BLOCK_TIME_STEP) && !defined(SERIALIZED_EXECUTION)
	    checkCudaErrors(hipStreamSynchronize(sinfo->stream[sidx ^ 1]));
	    if( blck.x > MAX_BLOCKS_PER_GRID )
	      checkCudaErrors(hipStreamSynchronize(sinfo->stream[sidx ]));
#   if  defined(BLOCK_TIME_STEP) && !defined(SERIALIZED_EXECUTION)
	  }/* if( grpNum != 0 ){ */
#endif//defined(BLOCK_TIME_STEP) && !defined(SERIALIZED_EXECUTION)
	}/* if( LETsteps > 0 ){ */
#endif//DOUBLE_BUFFER_FOR_LET
	//-----------------------------------------------------------------
	for(int ii = idxProcs; ii < idxProcs + numProcs; ii++){
	  //---------------------------------------------------------------
#ifdef  LET_COMMUNICATION_VIA_HOST
	  chkMPIerr(MPI_Irecv(&(tree_hst.more[let[ii].headRecv]), let[ii].numRecv, mpi.more, let[ii].rank, let[ii].rank, mpi.comm, &(let[ii].reqRecvMore)));
	  chkMPIerr(MPI_Irecv(&(tree_hst.jpos[let[ii].headRecv]), let[ii].numRecv, mpi.jpos, let[ii].rank, let[ii].rank, mpi.comm, &(let[ii].reqRecvJpos)));
	  chkMPIerr(MPI_Irecv(&(tree_hst.mj  [let[ii].headRecv]), let[ii].numRecv, mpi.mass, let[ii].rank, let[ii].rank, mpi.comm, &(let[ii].reqRecvMass)));
#else///LET_COMMUNICATION_VIA_HOST
	  chkMPIerr(MPI_Irecv(&(tree    .more[let[ii].headRecv]), let[ii].numRecv, mpi.more, let[ii].rank, let[ii].rank, mpi.comm, &(let[ii].reqRecvMore)));
	  chkMPIerr(MPI_Irecv(&(tree    .jpos[let[ii].headRecv]), let[ii].numRecv, mpi.jpos, let[ii].rank, let[ii].rank, mpi.comm, &(let[ii].reqRecvJpos)));
	  chkMPIerr(MPI_Irecv(&(tree    .mj  [let[ii].headRecv]), let[ii].numRecv, mpi.mass, let[ii].rank, let[ii].rank, mpi.comm, &(let[ii].reqRecvMass)));
#endif//LET_COMMUNICATION_VIA_HOST
	  //---------------------------------------------------------------
	}/* for(int ii = idxProcs; ii < idxProcs + numProcs; ii++){ */
	//-----------------------------------------------------------------

	//-----------------------------------------------------------------
	/* receive numProcs LET(s) and calculate gravity from them */
	//-----------------------------------------------------------------
	for(int ii = idxProcs; ii < idxProcs + numProcs; ii++){
	  //---------------------------------------------------------------
	  /* copy LET nodes from host to device */
	  //---------------------------------------------------------------
	  MPI_Status statusMore;	  chkMPIerr(MPI_Wait(&(let[ii].reqRecvJpos), &statusMore));
#ifdef  LET_COMMUNICATION_VIA_HOST
	  checkCudaErrors(hipMemcpyAsync(&(tree.jpos[let[ii].headRecv]), &(tree_hst.jpos[let[ii].headRecv]), sizeof(jparticle) * let[ii].numRecv, hipMemcpyHostToDevice, sinfo->stream[sidx]));
#endif//LET_COMMUNICATION_VIA_HOST
	  MPI_Status statusMass;	  chkMPIerr(MPI_Wait(&(let[ii].reqRecvMass), &statusMass));
#ifdef  LET_COMMUNICATION_VIA_HOST
	  checkCudaErrors(hipMemcpyAsync(&(tree.mj  [let[ii].headRecv]), &(tree_hst.mj  [let[ii].headRecv]), sizeof(    jmass) * let[ii].numRecv, hipMemcpyHostToDevice, sinfo->stream[sidx]));
#endif//LET_COMMUNICATION_VIA_HOST
	  MPI_Status statusJpos;	  chkMPIerr(MPI_Wait(&(let[ii].reqRecvMore), &statusJpos));
#ifdef  LET_COMMUNICATION_VIA_HOST
	  checkCudaErrors(hipMemcpyAsync(&(tree.more[let[ii].headRecv]), &(tree_hst.more[let[ii].headRecv]), sizeof(     uint) * let[ii].numRecv, hipMemcpyHostToDevice, sinfo->stream[sidx]));
#endif//LET_COMMUNICATION_VIA_HOST
	  //---------------------------------------------------------------
#ifdef  DBG_LETGEN_ON_GPU
	  fprintf(stdout, "received LET from rank %d (%d-th partner)\n", let[ii].rank, ii);
	  fflush(stdout);
	  /* MPI_Finalize(); */
	  /* exit(0); */
#endif//DBG_LETGEN_ON_GPU
	  //---------------------------------------------------------------

	  //---------------------------------------------------------------
	  /* calculate gravity from LET */
	  //---------------------------------------------------------------
#ifdef  DEBUG_PRINT_FOR_PARTICLE_ACCELERATION
	  printf("# %d-th LET\n", ii);
	  checkCudaErrors(hipDeviceSynchronize());
	  chkMPIerr(MPI_Barrier(MPI_COMM_WORLD));
	  fflush(NULL);
#endif//DEBUG_PRINT_FOR_PARTICLE_ACCELERATION
	  callCalcGravityFunc(blck, thrd, sinfo, &sidx, laneInfo, pi, 0, tree, grpNum, let[ii].headRecv
#ifdef  USE_CUDA_EVENT
			      , &Nwalk, iniCalcAcc, finCalcAcc
#else///USE_CUDA_EVENT
			      , cycles_dev
#endif//USE_CUDA_EVENT
			      , buf
#ifdef  COUNT_INTERACTIONS
			      , treeInfo
#endif//COUNT_INTERACTIONS
			      );
	  //---------------------------------------------------------------
	}/* for(int ii = idxProcs; ii < idxProcs + numProcs; ii++){ */
	//-----------------------------------------------------------------
	for(int ii = 0; ii < numProcs; ii++){
	  //---------------------------------------------------------------
	  MPI_Status statusInfo;	  chkMPIerr(MPI_Wait(&(let[ii].reqSendInfo), &statusInfo));
	  MPI_Status statusMore;	  chkMPIerr(MPI_Wait(&(let[ii].reqSendMore), &statusMore));
	  MPI_Status statusJpos;	  chkMPIerr(MPI_Wait(&(let[ii].reqSendJpos), &statusJpos));
	  MPI_Status statusMass;	  chkMPIerr(MPI_Wait(&(let[ii].reqSendMass), &statusMass));
	  //---------------------------------------------------------------
	}/* for(int ii = 0; ii < numProcs; ii++){ */
	//-----------------------------------------------------------------
	idxProcs += numProcs;
	remProcs -= numProcs;
#ifndef DOUBLE_BUFFER_FOR_LET
	LETsteps++;
#endif//DOUBLE_BUFFER_FOR_LET
	//-----------------------------------------------------------------

	//-----------------------------------------------------------------
	if( remProcs <= 0 )	  break;
	//-----------------------------------------------------------------
      }/* while( true ){ */
      //-------------------------------------------------------------------
      /* preparation for communication in the next step */
#ifdef  BLOCK_TIME_STEP
      const float letsize_scaler = (float)(share / block);
#else///BLOCK_TIME_STEP
      const float letsize_scaler = UNITY;
#endif//BLOCK_TIME_STEP
      for(int ii = 0; ii < Nlet - 1; ii++){
	//-----------------------------------------------------------------
	/* /\* guess the minimum size of the buffer *\/ */
	/* int minSend = (int)ceilf(letsize_scaler * (float)let[ii].numSend);	minSend += 32 - (minSend & 31); */
	/* int minRecv = (int)ceilf(letsize_scaler * (float)let[ii].numRecv);	minRecv += 32 - (minRecv & 31); */
	//-----------------------------------------------------------------
	if( ceilf(letsize_scaler * (float)let[ii].numSend) < (LETSIZE_REDUCE_CRITERION * (float)let[ii].maxSend) )	  let[ii].overEstimateSend++;
	if( ceilf(letsize_scaler * (float)let[ii].numRecv) < (LETSIZE_REDUCE_CRITERION * (float)let[ii].maxRecv) )	  let[ii].overEstimateRecv++;
	//-----------------------------------------------------------------
	if( let[ii].overEstimateSend >= LETSIZE_OVERESTIMATION_STEPS ){
	  let[ii].maxSend = (int)ceilf(LETSIZE_REDUCE_FACTOR * (float)let[ii].maxSend);	  let[ii].maxSend += 32 - (let[ii].maxSend & 31);
	  let[ii].overEstimateSend = 0;
	}/* if( let[ii].overEstimateSend >= LETSIZE_OVERESTIMATION_STEPS ){ */
	//-----------------------------------------------------------------
	if( let[ii].overEstimateRecv >= LETSIZE_OVERESTIMATION_STEPS ){
	  let[ii].maxRecv = (int)ceilf(LETSIZE_REDUCE_FACTOR * (float)let[ii].maxRecv);	  let[ii].maxRecv += 32 - (let[ii].maxRecv & 31);
	  let[ii].overEstimateRecv = 0;
	}/* if( let[ii].overEstimateRecv >= LETSIZE_OVERESTIMATION_STEPS ){ */
	//-----------------------------------------------------------------
	/* let[ii].maxSend = (int)ceilf(fminf((float)let[ii].maxSend, letsize_scaler * (float)let[ii].numSend));	let[ii].maxSend += 32 - (let[ii].maxSend & 31); */
	/* let[ii].maxRecv = (int)ceilf(fminf((float)let[ii].maxRecv, letsize_scaler * (float)let[ii].numRecv));	let[ii].maxRecv += 32 - (let[ii].maxRecv & 31); */
	//-----------------------------------------------------------------
      }/* for(int ii = 0; ii < Nlet - 1; ii++){ */
      setLETpartition(Nlet, let);
#if 0
      fprintf(stderr, "maxSend = %d while pjNum = %d @ rank %d\n", let[0].maxSend, pjNum, mpi.rank);
      fflush(stderr);
#endif
      //-------------------------------------------------------------------
#endif//SERIALIZED_EXECUTION
      //-------------------------------------------------------------------


      //-------------------------------------------------------------------
      sinfo->idx = sidx;
#if 0
      checkCudaErrors(hipDeviceSynchronize());
      /* checkCudaErrors(hipStreamSynchronize(sinfo->stream[sidx    ])); */
      /* checkCudaErrors(hipStreamSynchronize(sinfo->stream[sidx ^ 1])); */
#endif
      //-------------------------------------------------------------------
#ifdef  DBG_LETGEN_ON_GPU
      fprintf(stdout, "force calculation finished on rank %d\n", mpi.rank);
      fflush(stdout);
#endif//DBG_LETGEN_ON_GPU
      //-------------------------------------------------------------------
      int fail_hst;
      checkCudaErrors(hipMemcpy(&fail_hst, buf.fail, sizeof(int), hipMemcpyDeviceToHost));
      if( fail_hst != 0 ){
#ifdef  SERIALIZED_EXECUTION
	__KILL__(stderr, "ERROR: bufUsed exceeds bufSize of %d at least %d times.\nPLEASE re-simulate after decreasing NUM_BODY_MAX(%d) or GLOBAL_MEMORY_SYSBUF(%zu) defined in src/misc/structure.h or TREE_SAFETY_VAL(%f) defined in src/tree/make.h.\n", buf.bufSize, fail_hst, NUM_BODY_MAX, (size_t)GLOBAL_MEMORY_SYSBUF, TREE_SAFETY_VAL);
#else///SERIALIZED_EXECUTION
	__KILL__(stderr, "ERROR: bufUsed exceeds bufSize of %d at least %d times.\nPLEASE re-simulate after decreasing NUM_BODY_MAX(%d) or GLOBAL_MEMORY_SYSBUF(%zu) defined in src/misc/structure.h or TREE_SAFETY_VAL(%f) defined in src/tree/make.h, or EXTEND_NUM_TREE_NODE(%f) defined in src/tree/let.h.\n", buf.bufSize, fail_hst, NUM_BODY_MAX, (size_t)GLOBAL_MEMORY_SYSBUF, TREE_SAFETY_VAL, EXTEND_NUM_TREE_NODE);
#endif//SERIALIZED_EXECUTION
      }/* if( fail_hst != 0 ){ */
      //-------------------------------------------------------------------
/* #   if  !defined(SERIALIZED_EXECUTION) && defined(USE_CUDA_EVENT) && defined(MONITOR_LETGEN_TIME) */
/*       for(int ii = 0; ii < prevLETstreams; ii++){ */
/* 	checkCudaErrors(hipEventSynchronize(finMakeLET[ii])); */
/* 	checkCudaErrors(hipEventElapsedTime(&makeLET_ms, iniCalcAcc[ii], finCalcAcc[ii])); */
/* 	makeLET += (double)makeLET_ms * 1.0e-3; */
/*       }/\* for(int jj = 0; jj < prevLETstreams; jj++){ *\/ */
/* #endif//!defined(SERIALIZED_EXECUTION) && defined(USE_CUDA_EVENT) && defined(MONITOR_LETGEN_TIME) */
      //-------------------------------------------------------------------
/* #   if  defined(USE_CUDA_EVENT) && (!defined(SERIALIZED_EXECUTION) || defined(PRINT_PSEUDO_PARTICLE_INFO)) */
/*       checkCudaErrors(hipEventSynchronize(finCalcAcc[sidx ^ 1])); */
/*       checkCudaErrors(hipEventElapsedTime(&calcAcc_ms, iniCalcAcc[sidx ^ 1], finCalcAcc[sidx ^ 1])); */
/*       calcAcc += (double)calcAcc_ms * 1.0e-3; */
/* #endif//defined(USE_CUDA_EVENT) && (!defined(SERIALIZED_EXECUTION) || defined(PRINT_PSEUDO_PARTICLE_INFO)) */
      //-------------------------------------------------------------------
    }
  //-----------------------------------------------------------------------
#ifdef  COMPARE_WITH_DIRECT_SOLVER
  else{
    //---------------------------------------------------------------------
    Nrem = BLOCKSIZE(Ni, NTHREADS);
    if( Nrem <= MAX_BLOCKS_PER_GRID )
      calcAccDirect_kernel<<<Nrem, NTHREADS>>>
	(pi.pos, pi.acc, pi.pos, Ni
#ifdef  INDIVIDUAL_GRAVITATIONAL_SOFTENING
	 , eps2
#endif//INDIVIDUAL_GRAVITATIONAL_SOFTENING
	 );
    //---------------------------------------------------------------------
    else{
      //-------------------------------------------------------------------
      const int Niter = BLOCKSIZE(Nrem, MAX_BLOCKS_PER_GRID);
      int hidx = 0;
      //-------------------------------------------------------------------
      for(int iter = 0; iter < Niter; iter++){
	//-----------------------------------------------------------------
	int Nblck = MAX_BLOCKS_PER_GRID;
	if( Nblck > Nrem )	  Nblck = Nrem;
	//-----------------------------------------------------------------
	int Nsub = Nblck * NTHREADS;
	calcAccDirect_kernel<<<Nblck, NTHREADS>>>
	  (&pi.pos[hidx], &pi.acc[hidx], &pi.pos[hidx], Nsub
#ifdef  INDIVIDUAL_GRAVITATIONAL_SOFTENING
	   , &eps2[hidx]
#endif//INDIVIDUAL_GRAVITATIONAL_SOFTENING
	   );
	//-----------------------------------------------------------------
	hidx += Nsub;
	Nrem -= Nblck;
	//-----------------------------------------------------------------
      }/* for(int iter = 0; iter < Niter; iter++){ */
      //-------------------------------------------------------------------
    }/* else{ */
    //---------------------------------------------------------------------
    getLastCudaError("calcAccDirect");
    //---------------------------------------------------------------------
  }
#endif//COMPARE_WITH_DIRECT_SOLVER
  //-----------------------------------------------------------------------
#ifdef  PRINT_PSEUDO_PARTICLE_INFO
  checkCudaErrors(hipDeviceSynchronize());
  /* get total clock cycles to compute enclosing ball */
  checkCudaErrors(hipMemcpy(cycles_hst, cycles_dev, sizeof(unsigned long long int), hipMemcpyDeviceToHost));
  /* get information on enclosing ball */
  acceleration *seb;
  const int Nseb = blck.x * NGROUPS;
  mycudaMallocHost((void **)&seb, (size_t)Nseb * sizeof(acceleration));
  checkCudaErrors(hipMemcpy(seb, pi.acc, (size_t)Nseb * sizeof(acceleration), hipMemcpyDeviceToHost));
  /* set file tag */
  FILE *fp;
#ifndef ADOPT_ENCLOSING_BALL
  char sebfile[] = "com";
#else///ADOPT_ENCLOSING_BALL
#ifdef  ADOPT_SMALLEST_ENCLOSING_BALL
  char sebfile[] = "fischer03";
#endif//ADOPT_SMALLEST_ENCLOSING_BALL
#ifdef  ADOPT_APPROXIMATED_ENCLOSING_BALL
  char sebfile[] = "ritter90";
#endif//ADOPT_APPROXIMATED_ENCLOSING_BALL
#ifdef  COMPARE_ENCLOSING_BALLS
  char sebfile[] = "smaller";
#endif//COMPARE_ENCLOSING_BALLS
#   if  !defined(ADOPT_SMALLEST_ENCLOSING_BALL) && !defined(ADOPT_APPROXIMATED_ENCLOSING_BALL) && !defined(COMPARE_ENCLOSING_BALLS)
  char sebfile[] = "cartesian";
#endif//!defined(ADOPT_SMALLEST_ENCLOSING_BALL) && !defined(ADOPT_APPROXIMATED_ENCLOSING_BALL) && !defined(COMPARE_ENCLOSING_BALLS)
#endif//ADOPT_ENCLOSING_BALL
  char filename[128];
  /* output computing cost of enclosing ball */
  sprintf(filename, "%s/%s.ball.clock.%s.txt", LOGFOLDER, file, sebfile);
  fp = fopen(filename, "a");
  if( fp == NULL ){    __KILL__(stderr, "ERROR: failure to open \"%s\"\n", filename);  }
  char date[64], hostname[64];
  getPresentDateInStrings(date);
  gethostname(hostname, sizeof(hostname));
  fprintf(fp, "\twith %s equipped on %s\n", devProp.name, devProp.host);
  fprintf(fp, "\tmeasured on %s", date);
  fprintf(fp, "Nseb = %d, Ntot = %d, enclosing ball of continuous %d particles\n", Nseb, Ni, DIV_NWARP(TSUB));
  fprintf(fp, "%Lu cycles @ Ttot = %d, Tsub = %d, Nb_sm = %d, Nsm = %d\n", *cycles_hst, NTHREADS, TSUB, NBLOCKS_PER_SM, devProp.numSM);
  *cycles_hst /= (unsigned long long int)((NTHREADS >> 5) * (devProp.numSM * NBLOCKS_PER_SM));/* divide by product of (# of warps within a thread) and (# of concurrent blocks) */
  fprintf(fp, "%Lu cycles after divided by the product of (# of warps within a thread) and (# of concurrent blocks)\n", *cycles_hst);
  fprintf(fp, "%le seconds @ %lf GHz\n", (double)(*cycles_hst) / (devProp.coreClk * 1.0e+9), devProp.coreClk);
  fclose(fp);
  /* output properties of enclosing ball */
  sprintf(filename, "%s/%s.ball.%s.dat", DATAFOLDER, file, sebfile);
  fp = fopen(filename, "wb");
  if( fp == NULL ){    __KILL__(stderr, "ERROR: failure to open \"%s\"\n", filename);  }
  fwrite(seb, sizeof(acceleration), Nseb, fp);
  fclose(fp);
  /* output summary of enclosing ball for future analysis */
  sprintf(date, "%s/%s.ball.info.txt", LOGFOLDER, file);
  fp = fopen(date, "a");
  if( fp == NULL ){    __KILL__(stderr, "ERROR: failure to open \"%s\"\n", date);  }
  fprintf(fp, "%s\t%d\n", filename, Nseb);
  fclose(fp);
  /* finalize the computation */
  mycudaFreeHost(seb);
  exit(0);
#endif//PRINT_PSEUDO_PARTICLE_INFO
  //-----------------------------------------------------------------------


  //-----------------------------------------------------------------------
  /* mutiply Gravitational constant and subtract self-interaction for potential */
  //-----------------------------------------------------------------------
#ifdef  BLOCK_TIME_STEP
  Nrem = BLOCKSIZE(grpNum, NWARP * NGROUPS);
#else///BLOCK_TIME_STEP
  Nrem = BLOCKSIZE(Ni, NTHREADS);
#endif//BLOCK_TIME_STEP
  //-----------------------------------------------------------------------
  /* when grid splitting is not required... */
  if( Nrem <= MAX_BLOCKS_PER_GRID ){
#ifdef  BLOCK_TIME_STEP
#ifndef SERIALIZED_EXECUTION
    if( grpNum != 0 )
#endif//SERIALIZED_EXECUTION
      trimAcc_kernel<<<Nrem, thrd>>>
	(pi.acc, pi.pos, BLOCKSIZE(grpNum, NGROUPS) * NGROUPS, laneInfo
#ifdef  DPADD_FOR_ACC
	 , pi.tmp
#endif//DPADD_FOR_ACC
#   if  defined(KAHAN_SUM_CORRECTION) && defined(ACCURATE_ACCUMULATION) && (!defined(SERIALIZED_EXECUTION) || (NWARP > 1))
	 , pi.res
#endif//defined(KAHAN_SUM_CORRECTION) && defined(ACCURATE_ACCUMULATION) && (!defined(SERIALIZED_EXECUTION) || (NWARP > 1))
	 );
#else///BLOCK_TIME_STEP
    trimAcc_kernel<<<Nrem, NTHREADS>>>
      (pi.acc, pi.pos
#ifdef  DPADD_FOR_ACC
       , pi.tmp
#endif//DPADD_FOR_ACC
#   if  defined(KAHAN_SUM_CORRECTION) && defined(ACCURATE_ACCUMULATION) && (!defined(SERIALIZED_EXECUTION) || (NWARP > 1))
       , pi.res
#endif//defined(KAHAN_SUM_CORRECTION) && defined(ACCURATE_ACCUMULATION) && (!defined(SERIALIZED_EXECUTION) || (NWARP > 1))
       );
#endif//BLOCK_TIME_STEP
  }/* if( Nrem <= MAX_BLOCKS_PER_GRID ){ */
  //-----------------------------------------------------------------------
  /* when grid splitting is required... */
  else{
    //---------------------------------------------------------------------
    const int Niter = BLOCKSIZE(Nrem, MAX_BLOCKS_PER_GRID);
    int hidx = 0;
    //---------------------------------------------------------------------
    for(int iter = 0; iter < Niter; iter++){
      //-------------------------------------------------------------------
      int Nblck = MAX_BLOCKS_PER_GRID;
      if( Nblck > Nrem )	Nblck = Nrem;
      //-------------------------------------------------------------------
#ifdef  BLOCK_TIME_STEP
      int Nsub = Nblck * NWARP * NGROUPS;
      trimAcc_kernel<<<Nblck, thrd.x>>>
	(pi.acc, pi.pos, BLOCKSIZE(Nsub, NGROUPS) * NGROUPS, &laneInfo[hidx]
#ifdef  DPADD_FOR_ACC
	 , pi.tmp
#endif//DPADD_FOR_ACC
#   if  defined(KAHAN_SUM_CORRECTION) && defined(ACCURATE_ACCUMULATION) && (!defined(SERIALIZED_EXECUTION) || (NWARP > 1))
	 , pi.res
#endif//defined(KAHAN_SUM_CORRECTION) && defined(ACCURATE_ACCUMULATION) && (!defined(SERIALIZED_EXECUTION) || (NWARP > 1))
	 );
#else///BLOCK_TIME_STEP
      int Nsub = Nblck * NTHREADS;
      trimAcc_kernel<<<Nblck, NTHREADS>>>
	(&pi.acc[hidx], &pi.pos[hidx]
#ifdef  DPADD_FOR_ACC
	 , &pi.tmp[hidx]
#endif//DPADD_FOR_ACC
#   if  defined(KAHAN_SUM_CORRECTION) && defined(ACCURATE_ACCUMULATION) && (!defined(SERIALIZED_EXECUTION) || (NWARP > 1))
	 , &pi.res[hidx]
#endif//defined(KAHAN_SUM_CORRECTION) && defined(ACCURATE_ACCUMULATION) && (!defined(SERIALIZED_EXECUTION) || (NWARP > 1))
	 );
#endif//BLOCK_TIME_STEP
      //-------------------------------------------------------------------
      hidx += Nsub;
      Nrem -= Nblck;
      //-------------------------------------------------------------------
    }/* for(int iter = 0; iter < Niter; iter++){ */
    //---------------------------------------------------------------------
  }/* else{ */
  //-----------------------------------------------------------------------
  getLastCudaError("trimAcc_kernel");
  //-----------------------------------------------------------------------
#if 0
  checkCudaErrors(hipDeviceSynchronize());
  MPI_Finalize();
  exit(0);
#endif
  //-----------------------------------------------------------------------


  //-----------------------------------------------------------------------
#   if  defined(SERIALIZED_EXECUTION) || defined(EXEC_BENCHMARK)
  static struct timeval finish;
  checkCudaErrors(hipDeviceSynchronize());
  gettimeofday(&finish, NULL);
  *time = calcElapsedTimeInSec(start, finish);
#ifdef  EXEC_BENCHMARK
  elapsed->calcGravity_dev += *time;
#endif//EXEC_BENCHMARK
#endif//defined(SERIALIZED_EXECUTION) || defined(EXEC_BENCHMARK)
  //-----------------------------------------------------------------------
  /* evaluate GPU time */
#ifndef SERIALIZED_EXECUTION
  checkCudaErrors(hipDeviceSynchronize());
#ifdef  USE_CUDA_EVENT
  double calcAcc = 0.0;
  for(int ii = 0; ii < Nwalk; ii++){
    float tmp_ms;
    checkCudaErrors(hipEventElapsedTime(&tmp_ms, iniCalcAcc[ii], finCalcAcc[ii]));
    calcAcc += (double)tmp_ms;
  }/* for(int ii = 0; ii < Nwalk; ii++){ */
  calcAcc *= 1.0e-3;
#else///USE_CUDA_EVENT
  checkCudaErrors(hipMemcpy(cycles_hst, cycles_dev, sizeof(unsigned long long int), hipMemcpyDeviceToHost));
  /* # of launched blocks for tree traversal = # of blocks per kernel function * (# of local tree + # of LETs) = # of blocks per kernel function * # of GPUs */
#ifdef  USE_MEASURED_CLOCK_FREQ
  double devClock = (double)clockWalk * 1.0e+6;
#if 0
  printf("%e Hz on rank %d\n", devClock, mpi.rank);
  MPI_Finalize();
  exit(0);
#endif
#endif//USE_MEASURED_CLOCK_FREQ
#ifdef  USE_GPU_BASE_CLOCK_FREQ
  const double devClock = devProp.coreClk * 1.0e+9;
#endif//USE_GPU_BASE_CLOCK_FREQ
  /* const double calcAcc = ((double)(*cycles_hst) / (devClock * (double)(blck.x * mpi.size))) * (double)BLOCKSIZE(blck.x * mpi.size, devProp.numSM * NBLOCKS_PER_SM); */
  const double calcAcc = ((double)(*cycles_hst) / (devClock * (double)(blck.x * mpi.size))) * (double)BLOCKSIZE(blck.x * mpi.size, devProp.numSM);
#endif//USE_CUDA_EVENT
  measured->sum_excg    += calcAcc;
  measured->sum_rebuild += calcAcc;
  *time   = calcAcc;
#ifdef  MONITOR_LETGEN_TIME
#ifdef  USE_CUDA_EVENT
  double makeLET = 0.0;
  for(int ii = 0; ii < Nmake; ii++){
    float tmp_ms;
    checkCudaErrors(hipEventElapsedTime(&tmp_ms, iniMakeLET[ii], finMakeLET[ii]));
    makeLET += (double)tmp_ms;
  }/* for(int ii = 0; ii < Nwalk; ii++){ */
  makeLET *= 1.0e-3;
#else///USE_CUDA_EVENT
  checkCudaErrors(hipMemcpy(cycles_let_hst, cycles_let_dev, sizeof(unsigned long long int), hipMemcpyDeviceToHost));
  /* # of launched blocks for LET generator = # of LETs = # of GPUs - 1 = # of MPI processes - 1 */
  /* const double makeLET = ((double)(*cycles_let_hst) / (devClock * (double)(mpi.size - 1))) * BLOCKSIZE(mpi.size - 1, devProp.numSM * NBLOCKS_PER_SM); */
  const double makeLET = ((double)(*cycles_let_hst) / (devClock * (double)(mpi.size - 1))) * BLOCKSIZE(mpi.size - 1, devProp.numSM);
#endif//USE_CUDA_EVENT
  measured->sum_excg    += makeLET;
  measured->sum_rebuild += makeLET;
#if 0
  static struct timeval finish;
  checkCudaErrors(hipDeviceSynchronize());
  gettimeofday(&finish, NULL);
  fprintf(stdout, "rank %d: %e + %e | %e\n", mpi.rank, calcAcc, makeLET, calcElapsedTimeInSec(start, finish));
  fflush(stdout);
#endif
#endif//MONITOR_LETGEN_TIME
#endif//SERIALIZED_EXECUTION
  //-----------------------------------------------------------------------
  __NOTE__("%s\n", "end");
  //-----------------------------------------------------------------------
#ifdef  DEBUG_PRINT_FOR_PARTICLE_ACCELERATION
  MPI_Finalize();
  exit(0);
#endif//DEBUG_PRINT_FOR_PARTICLE_ACCELERATION
  //-----------------------------------------------------------------------
#if 0
  MPIinfo mpi_tmp;
  mpi_tmp.rank = mpi.rank;
  mpi_tmp.size = mpi.size;
  mpi_tmp.comm = mpi.comm;
  printTreeNode_dev(pjNum, tree, mpi_tmp);
#endif
  //-----------------------------------------------------------------------
}
//-------------------------------------------------------------------------


//-------------------------------------------------------------------------
extern "C"
void setGlobalConstants_walk_dev_cu
(const real newton_hst, const real eps2_hst
#ifndef WS93_MAC
 , const real theta2_hst
#endif//WS93_MAC
)
{
  //-----------------------------------------------------------------------
  __NOTE__("%s\n", "start");
  //-----------------------------------------------------------------------

  //-----------------------------------------------------------------------
  const real epsinv_hst = RSQRT(eps2_hst);
  //-----------------------------------------------------------------------
  jnode jnode0_hst;
#pragma unroll
  for(int ii = 0; ii < NSTOCK; ii++)
    jnode0_hst.idx[ii] = 0;
  //-----------------------------------------------------------------------
#   if  CUDART_VERSION >= 5000
  hipMemcpyToSymbol(HIP_SYMBOL( newton ), &newton_hst, sizeof( real), 0, hipMemcpyHostToDevice);
#ifndef INDIVIDUAL_GRAVITATIONAL_SOFTENING
  hipMemcpyToSymbol(HIP_SYMBOL( eps2   ), &  eps2_hst, sizeof( real), 0, hipMemcpyHostToDevice);
#endif//INDIVIDUAL_GRAVITATIONAL_SOFTENING
  hipMemcpyToSymbol(HIP_SYMBOL( epsinv ), &epsinv_hst, sizeof( real), 0, hipMemcpyHostToDevice);
#ifndef WS93_MAC
  hipMemcpyToSymbol(HIP_SYMBOL( theta2 ), &theta2_hst, sizeof( real), 0, hipMemcpyHostToDevice);
#endif//WS93_MAC
  hipMemcpyToSymbol(HIP_SYMBOL( jnode0 ), &jnode0_hst, sizeof(jnode), 0, hipMemcpyHostToDevice);
#else//CUDART_VERSION >= 5000
  hipMemcpyToSymbol(HIP_SYMBOL("newton"), &newton_hst, sizeof( real), 0, hipMemcpyHostToDevice);
#ifndef INDIVIDUAL_GRAVITATIONAL_SOFTENING
  hipMemcpyToSymbol(HIP_SYMBOL("eps2"  ), &  eps2_hst, sizeof( real), 0, hipMemcpyHostToDevice);
#endif//INDIVIDUAL_GRAVITATIONAL_SOFTENING
  hipMemcpyToSymbol(HIP_SYMBOL("epsinv"), &epsinv_hst, sizeof( real), 0, hipMemcpyHostToDevice);
#ifndef WS93_MAC
  hipMemcpyToSymbol(HIP_SYMBOL("theta2"), &theta2_hst, sizeof( real), 0, hipMemcpyHostToDevice);
#endif//WS93_MAC
  hipMemcpyToSymbol(HIP_SYMBOL("jnode0"), &jnode0_hst, sizeof(jnode), 0, hipMemcpyHostToDevice);
#endif//CUDART_VERSION >= 5000
  //-----------------------------------------------------------------------
#   if  SMPREF == 1
  checkCudaErrors(hipFuncSetCacheConfig(reinterpret_cast<const void*>(calcAcc_kernel), hipFuncCachePreferShared));
#endif//SMPREF == 1
#   if  WIDEBANK == 1
  checkCudaErrors(hipDeviceSetSharedMemConfig(hipSharedMemBankSizeFourByte));
#endif//WIDEBANK == 1
  //-----------------------------------------------------------------------

  //-----------------------------------------------------------------------
  /* error checking before running the kernel */
  //-----------------------------------------------------------------------
  struct hipFuncAttributes funcAttr;
  checkCudaErrors(hipFuncGetAttributes(&funcAttr, reinterpret_cast<const void*>(calcAcc_kernel)));
  int regLimit = MAX_REGISTERS_PER_SM / (funcAttr.numRegs * NTHREADS);
  int memLimit = ((SMPREF == 1) ? (48 * 1024) : (16 * 1024)) / funcAttr.sharedSizeBytes;
  int Nblck = (regLimit <= memLimit) ? regLimit : memLimit;
  if( Nblck != NBLOCKS_PER_SM ){
    //---------------------------------------------------------------------
    __KILL__(stderr, "ERROR: # of blocks per SM for calcAcc_kernel is mispredicted (%d).\n\tThe limits come from register and shared memory are %d and %d, respectively.\n\tHowever, the expected value of NBLOCKS_PER_SM defined in src/tree/walk_dev.cu is %d\n", Nblck, regLimit, memLimit, NBLOCKS_PER_SM);
    //---------------------------------------------------------------------
  }/* if( Nblck != NBLOCKS_PER_SM ){ */
  //-----------------------------------------------------------------------

  //-----------------------------------------------------------------------
  __NOTE__("%s\n", "end");
  //-----------------------------------------------------------------------
}
//-------------------------------------------------------------------------
