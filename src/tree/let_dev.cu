#include "hip/hip_runtime.h"
/*************************************************************************\
 *                                                                       *
                  last updated on 2016/07/22(Fri) 10:48:19
 *                                                                       *
 *    Octree N-body calculation for collisionless systems on NVIDIA GPUs *
 *                                                                       *
 *                                                                       *
 *                                                                       *
 *                                             written by Yohei MIKI     *
 *                                                                       *
\*************************************************************************/
//-------------------------------------------------------------------------
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <math.h>
#include <mpi.h>
#include <hip/hip_runtime_api.h>
#include <sys/time.h>
//-------------------------------------------------------------------------
#include <macro.h>
#include <cudalib.h>
#include <timer.h>
#include <mpilib.h>
//-------------------------------------------------------------------------
#include "../misc/benchmark.h"
#include "../misc/structure.h"
#include "../misc/device.h"
//-------------------------------------------------------------------------
#include "macutil.h"
#include "make.h"
#include "buf_inc.h"
//-------------------------------------------------------------------------
#include "../para/mpicfg.h"
#include "let.h"
#include "walk_dev.h"
#include "let_dev.h"
//-------------------------------------------------------------------------
#include "buf_inc.cu"
//-------------------------------------------------------------------------
#   if  !defined(GADGET_MAC) && !defined(WS93_MAC)
__constant__  real theta2;
#endif//!defined(GADGET_MAC) && !defined(WS93_MAC)
//-------------------------------------------------------------------------


//-------------------------------------------------------------------------
muse configLETtopology(domainInfo **info, position **ipos,
#ifdef  GADGET_MAC
		       real **amin,
#endif//GADGET_MAC
#ifdef  BUILD_LET_ON_DEVICE
		       int **numSend_hst, int **numSend_dev,
#endif//BUILD_LET_ON_DEVICE
		       hipStream_t **stream, int *Nstream, const deviceProp gpu, MPIcfg_tree mpi)
{
  //-----------------------------------------------------------------------
  __NOTE__("%s\n", "start");
  //-----------------------------------------------------------------------
  muse alloc = {0, 0};
  //-----------------------------------------------------------------------

  //-----------------------------------------------------------------------
  *info = (domainInfo *)malloc(mpi.size * sizeof(domainInfo));  if( *info == NULL ){    __KILL__(stderr, "ERROR: failure to allocate info\n");  }
  alloc.host                += mpi.size * sizeof(domainInfo);
  *ipos = (position   *)malloc(mpi.size * sizeof(position  ));  if( *ipos == NULL ){    __KILL__(stderr, "ERROR: failure to allocate ipos\n");  }
  alloc.host                += mpi.size * sizeof(position  );
#ifdef  GADGET_MAC
  *amin = (real       *)malloc(mpi.size * sizeof(real      ));  if( *amin == NULL ){    __KILL__(stderr, "ERROR: failure to allocate amin\n");  }
  alloc.host                += mpi.size * sizeof(real      );
#endif//GADGET_MAC
  //-----------------------------------------------------------------------
  for(int ii = 0; ii < mpi.size - 1; ii++)
    (*info)[ii].rank = mpi.rank ^ (1 + ii);
  //-----------------------------------------------------------------------
#ifdef  BUILD_LET_ON_DEVICE
  mycudaMalloc    ((void **)numSend_dev, mpi.size * sizeof(int));  alloc.device += mpi.size * sizeof(int);
  mycudaMallocHost((void **)numSend_hst, mpi.size * sizeof(int));  alloc.host   += mpi.size * sizeof(int);
  for(int ii = 0; ii < mpi.size; ii++){
    (*info)[ii].numSend_hst = &((*numSend_hst)[ii]);
    (*info)[ii].numSend_dev = &((*numSend_dev)[ii]);
  }/* for(int ii = 0; ii < mpi.size; ii++){ */
#endif//BUILD_LET_ON_DEVICE
  //-----------------------------------------------------------------------
  *Nstream = NBLOCKS_PER_SM * gpu.numSM;
  *stream = (hipStream_t *)malloc((*Nstream) * sizeof(hipStream_t));
  alloc.host +=                    (*Nstream) * sizeof(hipStream_t);
  if( *stream == NULL ){    __KILL__(stderr, "ERROR: failure to allocate stream");  }
#pragma unroll
  for(int ii = 0; ii < *Nstream; ii++)
    checkCudaErrors(hipStreamCreate(&((*stream)[ii])));
  //-----------------------------------------------------------------------

  //-----------------------------------------------------------------------
  __NOTE__("%s\n", "end");
  //-----------------------------------------------------------------------
  return (alloc);
  //-----------------------------------------------------------------------
}
//-------------------------------------------------------------------------
void releaseLETtopology(domainInfo  *info, position  *ipos,
#ifdef  GADGET_MAC
			real  *amin,
#endif//GADGET_MAC
#ifdef  BUILD_LET_ON_DEVICE
			int  *numSend_hst, int  *numSend_dev,
#endif//BUILD_LET_ON_DEVICE
			hipStream_t  *stream, int  Nstream
			)
{
  //-----------------------------------------------------------------------
  __NOTE__("%s\n", "start");
  //-----------------------------------------------------------------------
  free(info);
  free(ipos);
#ifdef  GADGET_MAC
  free(amin);
#endif//GADGET_MAC
#ifdef  BUILD_LET_ON_DEVICE
  mycudaFree    (numSend_dev);
  mycudaFreeHost(numSend_hst);
#endif//BUILD_LET_ON_DEVICE
  //-----------------------------------------------------------------------
  for(int ii = 0; ii < Nstream; ii++)
    mycudaStreamDestroy(stream[ii]);
  free(stream);
  //-----------------------------------------------------------------------
  __NOTE__("%s\n", "end");
  //-----------------------------------------------------------------------
}
//-------------------------------------------------------------------------


//-------------------------------------------------------------------------
/* parallel prefix sum within a block */
/* type of prefix sum is inclusive */
/* NOTE: implicit synchronization within 32 threads (a warp) is assumed */
//-------------------------------------------------------------------------
__device__ __forceinline__ int prefixSum(int val, const int tidx, const int lane, volatile int * smem)
{
  //-----------------------------------------------------------------------
  /* 1. prefix sum within a warp */
  //-----------------------------------------------------------------------
#ifdef  USE_WARP_SHUFFLE_FUNC_MAKE_LET
  //-----------------------------------------------------------------------
  /* load index */
  int tmp;
  /* calculate inclusive prefix sum */
  tmp = __shfl_up(val,  1, warpSize);  if( lane >=  1 )    val += tmp;
  tmp = __shfl_up(val,  2, warpSize);  if( lane >=  2 )    val += tmp;
  tmp = __shfl_up(val,  4, warpSize);  if( lane >=  4 )    val += tmp;
  tmp = __shfl_up(val,  8, warpSize);  if( lane >=  8 )    val += tmp;
  tmp = __shfl_up(val, 16, warpSize);  if( lane >= 16 )    val += tmp;
  /* return calculated inclusive prefix sum */
  smem[tidx] = val;
  //-----------------------------------------------------------------------
#else///USE_WARP_SHUFFLE_FUNC_MAKE_LET
  //-----------------------------------------------------------------------
  smem[tidx] = val;
  if( lane >=  1 ){    val += smem[tidx -  1];    smem[tidx] = val;  }
  if( lane >=  2 ){    val += smem[tidx -  2];    smem[tidx] = val;  }
  if( lane >=  4 ){    val += smem[tidx -  4];    smem[tidx] = val;  }
  if( lane >=  8 ){    val += smem[tidx -  8];    smem[tidx] = val;  }
  if( lane >= 16 ){    val += smem[tidx - 16];    smem[tidx] = val;  }
  //-----------------------------------------------------------------------
#endif//USE_WARP_SHUFFLE_FUNC_MAKE_LET
  //-----------------------------------------------------------------------

  //-----------------------------------------------------------------------
#   if  NTHREADS_MAKE_LET >= 64
  //-----------------------------------------------------------------------
  /* 2. prefix sum about the tail of each warp */
  //-----------------------------------------------------------------------
  int scan = val;
  __syncthreads();
  /* warpSize = 32 = 2^5 */
  if( tidx < (NTHREADS_MAKE_LET >> 5) ){
    //---------------------------------------------------------------------
    val = smem[tidx * warpSize + warpSize - 1];
#ifdef  USE_WARP_SHUFFLE_FUNC_MAKE_LET
#   if  NTHREADS_MAKE_LET >=   64
    const int groupSize = NTHREADS_MAKE_LET >> 5;
    tmp = __shfl_up(val,  1, groupSize);    if( lane >=  1 )      val += tmp;
#   if  NTHREADS_MAKE_LET >=  128
    tmp = __shfl_up(val,  2, groupSize);    if( lane >=  2 )      val += tmp;
#   if  NTHREADS_MAKE_LET >=  256
    tmp = __shfl_up(val,  4, groupSize);    if( lane >=  4 )      val += tmp;
#   if  NTHREADS_MAKE_LET >=  512
    tmp = __shfl_up(val,  8, groupSize);    if( lane >=  8 )      val += tmp;
#   if  NTHREADS_MAKE_LET == 1024
    tmp = __shfl_up(val, 16, groupSize);    if( lane >= 16 )      val += tmp;
#endif//NTHREADS_MAKE_LET == 1024
#endif//NTHREADS_MAKE_LET >=  512
#endif//NTHREADS_MAKE_LET >=  256
#endif//NTHREADS_MAKE_LET >=  128
#endif//NTHREADS_MAKE_LET >=   64
    smem[tidx] = val;
#else///USE_WARP_SHUFFLE_FUNC_MAKE_LET
    smem[tidx] = val;
#   if  NTHREADS_MAKE_LET >=   64
    if( lane >=  1 )      smem[tidx] += smem[tidx -  1];
#   if  NTHREADS_MAKE_LET >=  128
    if( lane >=  2 )      smem[tidx] += smem[tidx -  2];
#   if  NTHREADS_MAKE_LET >=  256
    if( lane >=  4 )      smem[tidx] += smem[tidx -  4];
#   if  NTHREADS_MAKE_LET >=  512
    if( lane >=  8 )      smem[tidx] += smem[tidx -  8];
#   if  NTHREADS_MAKE_LET == 1024
    if( lane >= 16 )      smem[tidx] += smem[tidx - 16];
#endif//NTHREADS_MAKE_LET == 1024
#endif//NTHREADS_MAKE_LET >=  512
#endif//NTHREADS_MAKE_LET >=  256
#endif//NTHREADS_MAKE_LET >=  128
#endif//NTHREADS_MAKE_LET >=   64
#endif//USE_WARP_SHUFFLE_FUNC_MAKE_LET
    //---------------------------------------------------------------------
  }/* if( tidx < (NTHREADS_MAKE_LET >> 5) ){ */
  __syncthreads();
  //-----------------------------------------------------------------------

  //-----------------------------------------------------------------------
  /* 3. prefix sum within a block */
  //-----------------------------------------------------------------------
  /* warpSize = 32 = 2^5 */
  if( tidx >= warpSize )
    scan += smem[(tidx >> 5) - 1];
  __syncthreads();
  //-----------------------------------------------------------------------

  //-----------------------------------------------------------------------
  /* 4. upload calculate prefix sum */
  //-----------------------------------------------------------------------
  smem[tidx] = scan;
  val = scan;
  __syncthreads();
  //-----------------------------------------------------------------------
#endif//NTHREADS_MAKE_LET >= 64
  //-----------------------------------------------------------------------
  return (val);
  //-----------------------------------------------------------------------
}
//-------------------------------------------------------------------------


//-------------------------------------------------------------------------
__device__ __forceinline__ void copyData_s2s
(volatile uint * src, int sidx,
 volatile uint * dst, int didx, const int num, const int tidx)
{
  //-----------------------------------------------------------------------
  const int iter = DIV_NTHREADS_MAKE_LET(num);
  const int frac = num & (NTHREADS_MAKE_LET - 1);/* := Nload % NTHREADS_MAKE_LET */
  //-----------------------------------------------------------------------
  union {uint4 i; uint a[4];} tmp;
  for(int kk = 0; kk < (iter >> 2); kk++){
    /* load */
    tmp.i.x = src[sidx                        ];
    tmp.i.y = src[sidx +     NTHREADS_MAKE_LET];
    tmp.i.z = src[sidx + 2 * NTHREADS_MAKE_LET];
    tmp.i.w = src[sidx + 3 * NTHREADS_MAKE_LET];
    sidx += 4 * NTHREADS_MAKE_LET;
    __syncthreads();
    /* store; */
    dst[didx                        ] = tmp.i.x;
    dst[didx +     NTHREADS_MAKE_LET] = tmp.i.y;
    dst[didx + 2 * NTHREADS_MAKE_LET] = tmp.i.z;
    dst[didx + 3 * NTHREADS_MAKE_LET] = tmp.i.w;
    didx += 4 * NTHREADS_MAKE_LET;
  }/* for(int kk = 0; kk < (iter >> 2); kk++){ */
  //-----------------------------------------------------------------------
  const int loop = iter & 3;
  /* load */
#pragma unroll
  for(int ii = 0; ii < loop; ii++){
    tmp.a[ii] = src[sidx];
    sidx += NTHREADS_MAKE_LET;
  }/* for(int ii = 0; ii < loop; ii++){ */
  if( loop != 0 )
    __syncthreads();
  /* store; */
#pragma unroll
  for(int ii = 0; ii < loop; ii++){
    dst[didx] = tmp.a[ii];
    didx += NTHREADS_MAKE_LET;
  }/* for(int ii = 0; ii < loop; ii++){ */
  //-----------------------------------------------------------------------
  if( frac > 0 ){
    if( tidx < frac )      tmp.i.x = src[sidx];
    __syncthreads();
    if( tidx < frac )      dst[didx] = tmp.i.x;
  }/* if( frac > 0 ){ */
  //-----------------------------------------------------------------------
  __syncthreads();
  //-----------------------------------------------------------------------
}
//-------------------------------------------------------------------------
__device__ __forceinline__ void copyData_g2s
(uint * RESTRICT gbuf, int srcHead, uint * RESTRICT sbuf, int dstHead, int numCopy, const int tidx)
{
  //-----------------------------------------------------------------------
  /* fraction processing at loading from the head of destination array */
  //-----------------------------------------------------------------------
  const int numTemp = NTHREADS_MAKE_LET - (srcHead & (NTHREADS_MAKE_LET - 1));/* := NTHREADS_MAKE_LET - (srcHead % NTHREADS_MAKE_LET) */
  const int numHead = (numTemp < numCopy) ? numTemp : numCopy;
  if( tidx < numHead )
    sbuf[dstHead + tidx] = gbuf[srcHead + tidx];
  dstHead += numHead;
  srcHead += numHead;
  numCopy -= numHead;
  //-----------------------------------------------------------------------

  //-----------------------------------------------------------------------
  /* sequential load from source on the global memory and store to destination on the shared memory */
  //-----------------------------------------------------------------------
  for(int ii = tidx; ii < numCopy; ii += NTHREADS_MAKE_LET)
    sbuf[dstHead + ii] = gbuf[srcHead + ii];
  //-----------------------------------------------------------------------
  __syncthreads();
  //-----------------------------------------------------------------------
}
//-------------------------------------------------------------------------
__device__ __forceinline__ void copyData_s2g
(uint * RESTRICT sbuf, int srcHead, uint * RESTRICT gbuf, int dstHead, int numCopy, const int tidx)
{
  //-----------------------------------------------------------------------
  /* fraction processing at storing to the head of destination array */
  //-----------------------------------------------------------------------
  const int numTemp = NTHREADS_MAKE_LET - (dstHead & (NTHREADS_MAKE_LET - 1));/* := NTHREADS_MAKE_LET - (dstHead % NTHREADS_MAKE_LET) */
  const int numHead = (numTemp < numCopy) ? numTemp : numCopy;
  if( tidx < numHead )
    gbuf[dstHead + tidx] = sbuf[srcHead + tidx];
  dstHead += numHead;
  srcHead += numHead;
  numCopy -= numHead;
  //-----------------------------------------------------------------------

  //-----------------------------------------------------------------------
  /* sequential load from source on the shared memory and store to destination on the global memory */
  //-----------------------------------------------------------------------
  for(int ii = tidx; ii < numCopy; ii += NTHREADS_MAKE_LET)
    gbuf[dstHead + ii] = sbuf[srcHead + ii];
  //-----------------------------------------------------------------------
  __syncthreads();
  //-----------------------------------------------------------------------
}
//-------------------------------------------------------------------------
__device__ __forceinline__ void copyData_g2g
(uint * gbuf, int srcHead, int dstHead, int Ncopy, const int Ndisp, const int tidx)
{
  //-----------------------------------------------------------------------
  /* configure the settings */
  //-----------------------------------------------------------------------
  const int Nfirst = Ndisp & (NTHREADS_MAKE_LET - 1);/* := Ndisp % NTHREADS_MAKE_LET */
  /* ldIdx is Nfirst, Nfirst + 1, ..., NTHREADS_MAKE_LET - 1, 0, 1, ..., Nfirst - 1 for tidx of 0, 1, 2, ..., NTHREADS_MAKE_LET - 1 */
  const int  ldIdx = (tidx + Nfirst) & (NTHREADS_MAKE_LET - 1);/* := (tidx + Nfirst) % NTHREADS_MAKE_LET */
  const int grpIdx = (ldIdx < Nfirst) ? 0 : 1;
  //-----------------------------------------------------------------------
  srcHead += Ndisp - Nfirst;/* hereafter, srcHead is NTHREADS_MAKE_LET elements aligned */
  //-----------------------------------------------------------------------


  //-----------------------------------------------------------------------
  /* fraction processing at loading from the head of source array */
  //-----------------------------------------------------------------------
  uint temp = gbuf[srcHead + ldIdx];
  srcHead += NTHREADS_MAKE_LET;
  //-----------------------------------------------------------------------

  //-----------------------------------------------------------------------
  /* sequential load and store from source to destination on the global memory */
  //-----------------------------------------------------------------------
  const int Niter = BLOCKSIZE(Ncopy, NTHREADS_MAKE_LET);
  /* for(int iter = 0; iter < Niter; iter++){ */
  for(int iter = 0; iter < Niter; iter += 4){
    //---------------------------------------------------------------------
    /* const int Nmove = (Ncopy > NTHREADS_MAKE_LET) ? (NTHREADS_MAKE_LET) : (Ncopy); */
    const int Nmove = (Ncopy > (4 * NTHREADS_MAKE_LET)) ? (4 * NTHREADS_MAKE_LET) : (Ncopy);
    //---------------------------------------------------------------------

    //---------------------------------------------------------------------
    /* load from the source array on the global memory */
    //---------------------------------------------------------------------
    /* load from temp (fraction processing) as initialization */
    /* uint local = temp; */
    union {uint4 i; uint a[4];} local;
    //---------------------------------------------------------------------
    /* load from global memory, store to shared memory or temp (fraction processing) */
    /* temp = gbuf[srcHead + ldIdx]; */
    /* if( !grpIdx ) */
    /*   local = temp; */
    const int Nloop = BLOCKSIZE(Nmove, NTHREADS_MAKE_LET);
#pragma unroll
    for(int ii = 0; ii < Nloop; ii++){
      if(  grpIdx )      	local.a[ii] = temp;
      temp = gbuf[srcHead + ldIdx + ii * NTHREADS_MAKE_LET];
      if( !grpIdx )      	local.a[ii] = temp;
    }/* for(int ii = 0; ii < Nloop; ii++){ */
    //---------------------------------------------------------------------
    __syncthreads();
    //---------------------------------------------------------------------

    //---------------------------------------------------------------------
    /* store to the destination array on the global memory */
    //---------------------------------------------------------------------
    /* gbuf[dstHead + tidx] = local; */
#pragma unroll
    for(int ii = 0; ii < Nloop; ii++)
      gbuf[dstHead + tidx + ii * NTHREADS_MAKE_LET] = local.a[ii];
    //---------------------------------------------------------------------
    Ncopy   -= Nmove;
    srcHead += Nmove;
    dstHead += Nmove;
    //---------------------------------------------------------------------
  }/* for(int iter = 0; iter < Niter; iter += 4){ */
  //-----------------------------------------------------------------------
  __syncthreads();
  //-----------------------------------------------------------------------
}
//-------------------------------------------------------------------------


//-------------------------------------------------------------------------
/* uint smem[NTHREADS_MAKE_LET]; */
/* uint node[NTHREADS_MAKE_LET]; */
/* sizes of smem and node are the same, do not used at the same time ==>> use smem as node */
//-------------------------------------------------------------------------
__device__ __forceinline__ void enqueueChildNodes
(const int tidx, const int lane, int * RESTRICT smem, const int leaf, const uint subNode,
 uint * RESTRICT smbuf,                  int *rem_sm, int *num_sm,
 uint * RESTRICT gmbuf, const size_t hb, int *rem_gm, int *num_gm, int *head_gm, int *tail_gm
)
{
  //-----------------------------------------------------------------------
  /* 1. compact the given sparse tree nodes */
  //-----------------------------------------------------------------------
  int add = prefixSum(leaf, tidx, lane, smem) - leaf;/* exclusive prefix sum of leaf */
  //-----------------------------------------------------------------------
  __syncthreads();
  int Ntot = smem[NTHREADS_MAKE_LET - 1];
  /* node[(leaf) ? (add) : (Ntot + tidx - add)] = (leaf) ? subNode : NULL_NODE; */
  smem[(leaf) ? (add) : (Ntot + tidx - add)] = (leaf) ? subNode : NULL_NODE;
  //-----------------------------------------------------------------------

  //-----------------------------------------------------------------------
  /* 2. copy tree nodes to the shared memory */
  //-----------------------------------------------------------------------
  const int Nsm = (Ntot < *rem_sm) ? (Ntot) : (*rem_sm);
  /* copyData_s2s(node, tidx, smbuf, tidx + (*num_sm), Nsm, tidx); */
  copyData_s2s((uint *)smem, tidx, smbuf, tidx + (*num_sm), Nsm, tidx);
  //-----------------------------------------------------------------------
  *num_sm += Nsm;
  *rem_sm -= Nsm;
  Ntot    -= Nsm;
  //-----------------------------------------------------------------------

  //-----------------------------------------------------------------------
  /* 3. move tree nodes on the global memory, if necessary */
  //-----------------------------------------------------------------------
  if( Ntot > *rem_gm ){
    //---------------------------------------------------------------------
    copyData_g2g(gmbuf, hb, hb, *num_gm, *head_gm, tidx);
/* #pragma unroll */
/*     for(int ii = tidx; ii < *num_gm; ii += NTHREADS_MAKE_LET) */
/*       gmbuf[hb + (size_t)ii] = gmbuf[hb + (size_t)((*head_gm) + ii)]; */
/*       /\* gmbuf[hb + (size_t)((*head_gm) + ii)] = gmbuf[hb + (size_t)ii]; *\/ */
    //---------------------------------------------------------------------
    * rem_gm += *head_gm;
    *tail_gm -= *head_gm;
    *head_gm  = 0;
    //---------------------------------------------------------------------
  }/* if( Ntot > *rem_gm ){ */
  //-----------------------------------------------------------------------

  //-----------------------------------------------------------------------
  /* 4. copy tree nodes to the global memory */
  //-----------------------------------------------------------------------
#if 0
  memcpy((void *)&gmbuf[hb + (size_t)(*tail_gm)], (const void *)&smem[Nsm], sizeof(uint) * Ntot);
#else
  /* copyData_s2g(node, Nsm, gmbuf, hb + (size_t)(*tail_gm), Ntot, tidx); */
  copyData_s2g((uint *)smem, Nsm, gmbuf, hb + (size_t)(*tail_gm), Ntot, tidx);
#endif
  //-----------------------------------------------------------------------
  * rem_gm -= Ntot;
  * num_gm += Ntot;
  *tail_gm += Ntot;
  //-----------------------------------------------------------------------
}
//-------------------------------------------------------------------------


//-------------------------------------------------------------------------
/* make width-first LET (Locally Essential Tree) */
//-------------------------------------------------------------------------
/* icom       :: input          :: position and squared radius of a pseudo i-particle corresponding to N-body particles in a different domain */
/* numLETnode ::         output :: the total number of LET nodes */
/* more_org   :: input          :: head index and number of child particles of the corresponding j-particle (full tree data; i.e., local data) */
/* jpos_org   :: input          :: position and squared radius of pseudo N-body particle as j-particles (full tree data; i.e., local data) */
/*   mj_org   :: input          :: mass of pseudo N-body particle as j-particles (full tree data; i.e., local data) */
/* more_let   ::         output :: head index and number of child particles of the corresponding j-particle (subtracted tree data; i.e., LET) */
/* jpos_let   ::         output :: position and squared radius of pseudo N-body particle as j-particles (subtracted tree data; i.e., LET) */
/*   mj_let   ::         output :: mass of pseudo N-body particle as j-particles (subtracted tree data; i.e., LET) */
/* active     ::                :: a shared value to lock the shared quantities (freeNum, freeLst) to control usage of buffer */
/* freeNum    ::                :: an unsigned integer represents # of unused bufferes */
/* freeLst    ::                :: a list of unused bufferes */
/* buffer     ::                :: tentative memory space to store tree cells which does not fit within the limited space of the shared memory */
/* bufSize    :: input          :: size of the buffer */
/* overflow   ::         output :: a variable to detect buffer overflow */
//-------------------------------------------------------------------------
__global__ void makeLET_kernel
(READ_ONLY position icom,
#ifdef  GADGET_MAC
 READ_ONLY real amin,
#endif//GADGET_MAC
 int * RESTRICT numLETnode,
 READ_ONLY uint * RESTRICT more_org, READ_ONLY jparticle * RESTRICT jpos_org, READ_ONLY real * RESTRICT mj_org,
           uint * RESTRICT more_let,           jparticle * RESTRICT jpos_let,           real * RESTRICT mj_let,
#   if  !defined(USE_SMID_TO_GET_BUFID) && !defined(TRY_MODE_ABOUT_BUFFER)
 int * RESTRICT active, uint * RESTRICT freeNum,
#endif//!defined(USE_SMID_TO_GET_BUFID) && !defined(TRY_MODE_ABOUT_BUFFER)
#   if  !defined(USE_SMID_TO_GET_BUFID) &&  defined(TRY_MODE_ABOUT_BUFFER)
 const int freeNum,
#endif//!defined(USE_SMID_TO_GET_BUFID) &&  defined(TRY_MODE_ABOUT_BUFFER)
 uint * RESTRICT freeLst, uint * RESTRICT buffer, const int bufSize, int * RESTRICT overflow
#ifdef  MONITOR_LETGEN_TIME
 , unsigned long long int * RESTRICT cycles
#endif//MONITOR_LETGEN_TIME
)
{
  //-----------------------------------------------------------------------
  /* start stop watch */
  //-----------------------------------------------------------------------
#ifdef  MONITOR_LETGEN_TIME
  const long long int initCycle = clock64();
#endif//MONITOR_LETGEN_TIME
  //-----------------------------------------------------------------------

  //-----------------------------------------------------------------------
  /* identify thread properties */
  //-----------------------------------------------------------------------
  const int tidx = THREADIDX_X1D;
  const int lane = tidx & (warpSize - 1);/* index of the thread within a thread group */
  //-----------------------------------------------------------------------
  /* const int head = tidx - lane; */
  /* const int tail = head + (warpSize - 1); */
  //-----------------------------------------------------------------------


  //-----------------------------------------------------------------------
  /* shared values within the threads */
  //-----------------------------------------------------------------------
  __shared__ uint queue[NTHREADS_MAKE_LET * NQUEUE_LET];
  __shared__  int  smem[NTHREADS_MAKE_LET];
  //-----------------------------------------------------------------------


  //-----------------------------------------------------------------------
#ifdef  USE_SMID_TO_GET_BUFID
  const int target = occupyBuffer(tidx, freeLst, queue);
#else///USE_SMID_TO_GET_BUFID
#ifdef  TRY_MODE_ABOUT_BUFFER
  const int target = occupyBuffer(tidx, BLOCKIDX_X1D, freeNum, freeLst, queue);
#else///TRY_MODE_ABOUT_BUFFER
  occupyBuffer(tidx, freeNum, freeLst, queue, active);
#endif//TRY_MODE_ABOUT_BUFFER
#endif//USE_SMID_TO_GET_BUFID
  const int bufIdx = (int)queue[0];
  __syncthreads();
  size_t buf0Head = (size_t)bufIdx * (size_t)bufSize;
  //-----------------------------------------------------------------------


  //-----------------------------------------------------------------------
  /* sweep all tree nodes by executing tree-traversal */
  //-----------------------------------------------------------------------
  /* initialize queue for tree nodes */
#pragma unroll
  for(int jj = 0; jj < NQUEUE_LET; jj++)
    queue[tidx + NTHREADS_MAKE_LET * jj] = NULL_NODE;/* size >= NTHREADS_MAKE_LET * NQUEUE_LET */
  //-----------------------------------------------------------------------
  /* initialize queue for j-cells and interaction list by a representative thread */
  int sendNum = 0;/* # of LET nodes already stored in the global memory */
  int letTail = 0;/* the tail index of child cells for LET nodes already stored in the global memory */
  int bufHead = 0;
  int bufTail = 0;
  int bufOpen = bufSize;
  int bufUsed = 0;
  /* set child j-cells in queue on the shared memory */
  const int root = 0;
  uint jcell = more_org[root];
  int rem = 1 + (int)(jcell >> IDXBITS);
  jcell &= IDXMASK;
  //-----------------------------------------------------------------------
  if( rem > NTHREADS_MAKE_LET ){
    //---------------------------------------------------------------------
    /* if rem exceeds NTHREADS_MAKE_LET, then number of child j-cells must be shrunk */
    //---------------------------------------------------------------------
    queue[tidx] = jcell + (uint)tidx;/* size >= NTHREADS_MAKE_LET */
    //---------------------------------------------------------------------
    if( tidx == (NTHREADS_MAKE_LET - 1) )
      queue[tidx] += (uint)((rem - NTHREADS_MAKE_LET) << IDXBITS);/* size >= NTHREADS_MAKE_LET */
    //---------------------------------------------------------------------
    rem = NTHREADS_MAKE_LET;
    //---------------------------------------------------------------------
  }/* if( rem > NTHREADS_MAKE_LET ){ */
  else{
    //---------------------------------------------------------------------
    /* upload rem (<= NTHREADS_MAKE_LET) child j-cells to the shared memory */
    //---------------------------------------------------------------------
    if( tidx < rem )
      queue[tidx] = more_org[jcell + tidx];/* size >= NTHREADS_MAKE_LET */
    //---------------------------------------------------------------------
  }/* else{ */
  //-----------------------------------------------------------------------


  //-----------------------------------------------------------------------
  /* tree traversal in a width-first manner */
  //-----------------------------------------------------------------------
  int fail = 0;
  //-----------------------------------------------------------------------
  while( true ){
    //---------------------------------------------------------------------
    /* if the queue becomes empty, then exit the while loop */
    //---------------------------------------------------------------------
    __syncthreads();
    if( rem == 0 )
      break;
    //---------------------------------------------------------------------

    //---------------------------------------------------------------------
    /* pick up a queue from stack */
    //---------------------------------------------------------------------
    /* tentative load from the stack */
    int cnum = 0;
    jcell = NULL_NODE;
    if( tidx < rem ){
      jcell = queue[tidx];
      cnum = 1 + (int)(jcell >> IDXBITS);
    }/* if( lane < rem ){ */
    jcell &= IDXMASK;
    //---------------------------------------------------------------------
    /* predict the head index on the shared memory by parallel prefix sum */
    int hidx = prefixSum(cnum, tidx, lane, smem) - cnum;/* exclusive prefix sum of cnum */
    //---------------------------------------------------------------------
    smem[tidx] = NULL_NODE;
    __syncthreads();
    //---------------------------------------------------------------------
    int remove = 0;
    if( (cnum != 0) && (hidx < NTHREADS_MAKE_LET) ){
      //-------------------------------------------------------------------
      /* local data can be uploaded to the shared memory */
      int unum = NTHREADS_MAKE_LET - hidx;
      if( cnum < unum )	  unum = cnum;
      //-------------------------------------------------------------------
      /* upload local data */
      for(int jj = 0; jj < unum; jj++){
	/* list[hidx & (NTHREADS_MAKE_LET - 1)] = jcell; */
	smem[hidx] = (int)jcell;/* because hidx < NTHREADS_MAKE_LET */
	hidx++;
	jcell++;
      }/* for(int jj = 0; jj < unum; jj++){ */
      //-------------------------------------------------------------------
      /* eliminate stocked j-cells from the queue */
      if( unum == cnum )
	remove = 1;
      else{
	jcell += ((uint)(cnum - unum - 1) << IDXBITS);
	queue[tidx] = jcell;
      }/* else{ */
      //-------------------------------------------------------------------
    }/* if( (cnum != 0) && (hidx < NTHREADS_MAKE_LET) ){ */
    //---------------------------------------------------------------------
    /* set an index of j-cell */
    __syncthreads();
    const int target = smem[tidx];
    //---------------------------------------------------------------------
    /* remove scanned j-cells if possible */
    prefixSum(remove, tidx, lane, smem);
    remove = smem[NTHREADS_MAKE_LET - 1];
    //---------------------------------------------------------------------
    if( remove != 0 ){
      rem -= remove;
      copyData_s2s(queue, tidx + remove, queue, tidx, rem, tidx);
    }/* if( remove != 0 ){ */
    //---------------------------------------------------------------------


    //---------------------------------------------------------------------
    /* pick up pseudo particles */
    //---------------------------------------------------------------------
    /* prefixSum to submit an LET node */
    int returnLET = (target != NULL_NODE) ? 1 : 0;
    hidx = sendNum + prefixSum(returnLET, tidx, lane, smem) - returnLET;/* index of the corresponding LET node, which is based on exclusive prefix sum of calc */
    sendNum += smem[NTHREADS_MAKE_LET - 1];
    //---------------------------------------------------------------------

    //---------------------------------------------------------------------
    /* only the active threads pick up a j-cell from the global memory */
    //---------------------------------------------------------------------
    jparticle jpos_tmp;
    uint      more_tmp;
    int childNum = 0;
    int hasChild = 0;
    //---------------------------------------------------------------------
    if( returnLET ){
      //-------------------------------------------------------------------
      jpos_tmp     = jpos_org[target];      /* get position of pseudo j-particle */
      mj_let[hidx] =   mj_org[target];      /* send mj of an LET node */
      //-------------------------------------------------------------------
      /* set a pseudo i-particle */
      const real rx = jpos_tmp.x - icom.x;
      const real ry = jpos_tmp.y - icom.y;
      const real rz = jpos_tmp.z - icom.z;
      const real r2 = 1.0e-30f + rx * rx + ry * ry + rz * rz;
#if 1
      real lambda = FMAX(UNITY - SQRTRATIO(icom.m, r2), ZERO);
#else
      real lambda = UNITY - SQRTRATIO(icom.m, r2);
      if( lambda < EPSILON )	lambda = ZERO;
#endif
      /* calculate distance between the pseudo i-particle and the candidate j-particle */
      //-------------------------------------------------------------------
#ifdef  GADGET_MAC
      /* alpha * |a| * r^4 > G * M * l^2 */
#if 1
      lambda *= lambda * r2;
      if(   jpos_tmp.w < lambda               * lambda               * amin )
#else
	if( jpos_tmp.w < lambda * lambda * r2 * lambda * lambda * r2 * amin )
#endif
#else///GADGET_MAC
#ifdef  WS93_MAC
	  if(   jpos_tmp.w < lambda * lambda * r2 )
#else///WS93_MAC
	    /* (l / r) < theta */
	    if( jpos_tmp.w < lambda * lambda * r2 * theta2 )
#endif//WS93_MAC
#endif//GADGET_MAC
	      {
		//---------------------------------------------------------
		/* distant node ==>> child cells are not included in the LET */
		//---------------------------------------------------------
		more_tmp = hidx;
		jpos_tmp.w = -UNITY;/* squared size for the distant node is set to be negative */
		//---------------------------------------------------------
	      }
	    else
	      {
		//---------------------------------------------------------
		/* near node ==> child cells are included in the LET */
		//---------------------------------------------------------
		/* add child-cells of near tree-cells to the tentative stack */
		more_tmp = more_org[target];
		childNum = 1 + (int)(more_tmp >> IDXBITS);
		hasChild = 1;
		//---------------------------------------------------------
	      }
      //-------------------------------------------------------------------
    }/* if( returnLET ){ */
    //---------------------------------------------------------------------


    //---------------------------------------------------------------------
    /* if the shared memory has open space and some tree cells are stored on the global memory, then load tree-cells from the global memory to the shared memory */
    //---------------------------------------------------------------------
    /* evaluate available size of the queue on the shared memory */
    int Nsm_rem = NQUEUE_LET * NTHREADS_MAKE_LET - rem;
    //---------------------------------------------------------------------
    if( (bufUsed != 0) && (Nsm_rem > 0) ){
      //-------------------------------------------------------------------
      /* hq is tidx */
      const int Nload = (Nsm_rem < bufUsed) ? (Nsm_rem) : (bufUsed);
#if 0
      memcpy((void *)&queue[rem], (const void *)&buffer[buf0Head + bufHead], sizeof(uint) * Nload);
#else
      copyData_g2s(buffer, buf0Head + bufHead, queue, rem, Nload, tidx);
#endif
      //-------------------------------------------------------------------
      rem     += Nload;
      Nsm_rem -= Nload;
      bufUsed -= Nload;
      bufHead += Nload;
      //-------------------------------------------------------------------
      if( bufUsed == 0 ){
	bufHead = 0;
	bufTail = 0;
	bufOpen = bufSize;
      }/* if( bufUsed == 0 ){ */
      //-------------------------------------------------------------------
    }/* if( (bufUsed != 0) && (Nsm_rem > 0) ){ */
    //---------------------------------------------------------------------


    //---------------------------------------------------------------------
    /* copy child-cells of near tree-cells stored in the tentative stack to the stack on the shared memory and/or the global memory */
    //---------------------------------------------------------------------
    enqueueChildNodes(tidx, lane, smem, hasChild, more_tmp, queue, &Nsm_rem, &rem, buffer, buf0Head, &bufOpen, &bufUsed, &bufHead, &bufTail);
    fail += (bufOpen < 0);
    //---------------------------------------------------------------------


    //---------------------------------------------------------------------
    /* if current node has child nodes in LET, then head index of more_tmp must be rewritten */
    //---------------------------------------------------------------------
    /* prefixSum to extend LET */
    int leafHead = prefixSum(childNum, tidx, lane, smem) - childNum;/* exclusive prefix sum of nchild */
    //---------------------------------------------------------------------
    /* modify more pointer using leafHead */
    if( childNum > 0 )
      more_tmp = ((uint)(childNum - 1) << IDXBITS) + (uint)(letTail + leafHead);
    letTail += smem[NTHREADS_MAKE_LET - 1];
    //---------------------------------------------------------------------


    //---------------------------------------------------------------------
    /* add tree nodes to LET (mj_tmp is already stored) */
    //---------------------------------------------------------------------
    if( returnLET ){
      jpos_let[hidx] = jpos_tmp;
      more_let[hidx] = more_tmp;
    }/* if( returnLET ){ */
    //---------------------------------------------------------------------
  }/* while( true ){ */
  //-----------------------------------------------------------------------


  //-----------------------------------------------------------------------
  /* finalizing LET generator */
  //-----------------------------------------------------------------------
  if( tidx == 0 ){
    *numLETnode = sendNum;
    atomicAdd(overflow, fail);
  }/* if( tidx == 0 ){ */
  //-----------------------------------------------------------------------
#ifdef  USE_SMID_TO_GET_BUFID
  releaseBuffer(tidx, freeLst, (uint)bufIdx, target);
#else///USE_SMID_TO_GET_BUFID
#ifdef  TRY_MODE_ABOUT_BUFFER
  releaseBuffer(tidx, freeLst, (uint)bufIdx, target);
#else///TRY_MODE_ABOUT_BUFFER
  releaseBuffer(tidx, freeNum, freeLst, bufIdx, active);
#endif//TRY_MODE_ABOUT_BUFFER
#endif//USE_SMID_TO_GET_BUFID
  //-----------------------------------------------------------------------
#ifdef  MONITOR_LETGEN_TIME
  long long int exitCycle = clock64();
  if( tidx == 0 ){
    unsigned long long int elapsed = (unsigned long long int)(exitCycle - initCycle);
    atomicAdd(cycles, elapsed);
  }/* if( tidx == 0 ){ */
#endif//MONITOR_LETGEN_TIME
  //-----------------------------------------------------------------------
}
//-------------------------------------------------------------------------


//-------------------------------------------------------------------------
 /* generate LET (Locally Essential Tree) */
//-------------------------------------------------------------------------
extern "C"
void callGenLET
  (const hipStream_t stream, domainInfo *let, MPIcfg_tree mpi,
   const soaTreeNode tree, const int numSendGuess, const soaTreeWalkBuf buf
#ifdef  MONITOR_LETGEN_TIME
   , unsigned long long int * RESTRICT cycles
#endif//MONITOR_LETGEN_TIME
   )
{
  //-----------------------------------------------------------------------
  __NOTE__("%s\n", "start");
  //-----------------------------------------------------------------------
  /* checkCudaErrors(hipStreamSynchronize(stream)); */
  /* makeLET_kernel<<<1, NTHREADS_MAKE_LET>>> */
  makeLET_kernel<<<1, NTHREADS_MAKE_LET, SMEM_SIZE, stream>>>
    ((*let).icom,
#ifdef  GADGET_MAC
     (*let).amin,
#endif//GADGET_MAC
     (*let).numSend_dev,
     tree.more, tree.jpos, tree.mj,
     &(tree.more[(*let).headSend]), &(tree.jpos[(*let).headSend]), &(tree.mj[(*let).headSend]),
#ifndef USE_SMID_TO_GET_BUFID
#ifndef TRY_MODE_ABOUT_BUFFER
     buf.active,
#endif//TRY_MODE_ABOUT_BUFFER
     buf.freeNum,
#endif//USE_SMID_TO_GET_BUFID
     buf.freeLst, buf.buffer, NGROUPS * buf.bufSize, buf.fail
#ifdef  MONITOR_LETGEN_TIME
     , cycles
#endif//MONITOR_LETGEN_TIME
     );
  //-----------------------------------------------------------------------
  checkCudaErrors(hipMemcpy((*let).numSend_hst, (*let).numSend_dev, sizeof(int), hipMemcpyDeviceToHost));
  /* checkCudaErrors(hipMemcpyAsync((*let).numSend_hst, (*let).numSend_dev, sizeof(int), hipMemcpyDeviceToHost, stream)); */
  /* checkCudaErrors(hipStreamSynchronize(stream)); */
  let->numSend = *((*let).numSend_hst);
  if( let->numSend > numSendGuess ){
    __KILL__(stderr, "ERROR: predicted size of send buffer(%d) is not sufficient for true size of that(%d) @ rank %d for rand %d.\n\tsuggestion: consider increasing \"LETSIZE_OVERESTIMATION_FACTOR\" defined in src/tree/let.h (current value is %f).\n", numSendGuess, let->numSend, mpi.rank, let->rank, LETSIZE_OVERESTIMATION_FACTOR);
  }/* if( *numSend_hst > numSendGuess ){ */
  //-----------------------------------------------------------------------
#ifdef  DBG_LETGEN_ON_GPU
  fprintf(stdout, "numSend = %d @ rank %d\n", (*let).numSend, mpi.rank);
  /* checkCudaErrors(hipDeviceSynchronize()); */
  /* MPI_Finalize(); */
  /* exit(0); */
#endif//DBG_LETGEN_ON_GPU
  //-----------------------------------------------------------------------
  __NOTE__("%s\n", "end");
  //-----------------------------------------------------------------------
}
//-------------------------------------------------------------------------


//-------------------------------------------------------------------------
extern "C"
void setGlobalConstants_let_dev_cu
(
#   if  !defined(GADGET_MAC) && !defined(WS93_MAC)
 const real theta2_hst
#else///endif//!defined(GADGET_MAC) && !defined(WS93_MAC)
 void
#endif//!defined(GADGET_MAC) && !defined(WS93_MAC)
   )
{
  //-----------------------------------------------------------------------
  __NOTE__("%s\n", "start");
  //-----------------------------------------------------------------------

  //-----------------------------------------------------------------------
#   if  !defined(GADGET_MAC) && !defined(WS93_MAC)
#   if  CUDART_VERSION >= 5000
  hipMemcpyToSymbol(HIP_SYMBOL( theta2 ), &theta2_hst, sizeof( real), 0, hipMemcpyHostToDevice);
#else//CUDART_VERSION >= 5000
  hipMemcpyToSymbol(HIP_SYMBOL("theta2"), &theta2_hst, sizeof( real), 0, hipMemcpyHostToDevice);
#endif//CUDART_VERSION >= 5000
#endif//!defined(GADGET_MAC) && !defined(WS93_MAC)
  //-----------------------------------------------------------------------
#   if  SMPREF_LET == 1
  checkCudaErrors(hipFuncSetCacheConfig(reinterpret_cast<const void*>(makeLET_kernel), hipFuncCachePreferShared));
#endif//SMPREF_LET == 1
  //-----------------------------------------------------------------------

  //-----------------------------------------------------------------------
  __NOTE__("%s\n", "end");
  //-----------------------------------------------------------------------
}
//-------------------------------------------------------------------------
